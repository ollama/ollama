#include "hip/hip_runtime.h"
#include "common.cuh"
#include "wkv.cuh"

template <int block_size>
static __global__ void rwkv_wkv_f32(const int B, const int T, const int C, const int H, const float * k, const float * v, const float * r, const float * tf, const float * td, const float * s, float * dst) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int head_size = block_size;
    const int batch_i = bid / H;
    const int head_i = bid % H;
    const int state_size = C * head_size;
    const int n_seq_tokens = T / B;

    float state[head_size];
    __shared__ float _k[head_size], _r[head_size], _tf[head_size], _td[head_size];

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        state[i] = s[batch_i * state_size + head_i * head_size * head_size + i * head_size + tid];
    }

    __syncthreads();
    _tf[tid] = tf[head_i * head_size + tid];
    __syncthreads();

    for (int t = batch_i * n_seq_tokens * C + head_i * head_size + tid; t < (batch_i + 1) * n_seq_tokens * C + head_i * head_size + tid; t += C) {
        __syncthreads();
        _k[tid] = k[t];
        _r[tid] = r[t];
        _td[tid] = td[t];
        __syncthreads();

        const float _v = v[t];
        float y = 0;
        for (int j = 0; j < head_size; j += 4) {
            const float4& k = (float4&)(_k[j]);
            const float4& r = (float4&)(_r[j]);
            const float4& tf = (float4&)(_tf[j]);
            const float4& td = (float4&)(_td[j]);
            float4& s = (float4&)(state[j]);
            float4 kv;

            kv.x = k.x * _v;
            kv.y = k.y * _v;
            kv.z = k.z * _v;
            kv.w = k.w * _v;

            y += r.x * (tf.x * kv.x + s.x);
            y += r.y * (tf.y * kv.y + s.y);
            y += r.z * (tf.z * kv.z + s.z);
            y += r.w * (tf.w * kv.w + s.w);

            s.x = s.x * td.x + kv.x;
            s.y = s.y * td.y + kv.y;
            s.z = s.z * td.z + kv.z;
            s.w = s.w * td.w + kv.w;
        }
        dst[t] = y;
    }

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        dst[T * C + batch_i * state_size + head_i * head_size * head_size + i * head_size + tid] = state[i];
    }
}

template <int block_size>
static __global__ void rwkv_wkv7_f32(const int B, const int T, const int C, const int H, const float * r, const float * w, const float * k, const float * v, const float * a, const float * b, const float * s, float * dst) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int head_size = block_size;
    const int batch_i = bid / H;
    const int head_i = bid % H;
    const int state_size = C * head_size;
    const int n_seq_tokens = T / B;

    float state[head_size];
    __shared__ float _r[head_size], _w[head_size], _k[head_size], _a[head_size], _b[head_size];

#ifndef GGML_USE_MUSA
    #pragma unroll
#endif
    for (int i = 0; i < head_size; i++) {
        state[i] = s[batch_i * state_size + head_i * head_size * head_size + tid * head_size + i];
    }

    for (int t = batch_i * n_seq_tokens * C + head_i * head_size + tid; t < (batch_i + 1) * n_seq_tokens * C + head_i * head_size + tid; t += C) {
        __syncthreads();
        _r[tid] = r[t];
        _w[tid] = w[t];
        _k[tid] = k[t];
        _a[tid] = a[t];
        _b[tid] = b[t];
        __syncthreads();

        float sa = 0;
        #pragma unroll
        for (int j = 0; j < head_size; j += 4)
        {
            const float4& a = (float4&)(_a[j]);
            const float4& s = (float4&)(state[j]);
            sa += a.x * s.x;
            sa += a.y * s.y;
            sa += a.z * s.z;
            sa += a.w * s.w;
        }

        const float _v = v[t];
        float y = 0;
        for (int j = 0; j < head_size; j += 4) {
            const float4& r = (float4&)(_r[j]);
            const float4& w = (float4&)(_w[j]);
            const float4& k = (float4&)(_k[j]);
            const float4& b = (float4&)(_b[j]);
            float4& s = (float4&)(state[j]);
            float4 kv;

            kv.x = k.x * _v;
            kv.y = k.y * _v;
            kv.z = k.z * _v;
            kv.w = k.w * _v;

            s.x = s.x * w.x + kv.x + sa * b.x;
            s.y = s.y * w.y + kv.y + sa * b.y;
            s.z = s.z * w.z + kv.z + sa * b.z;
            s.w = s.w * w.w + kv.w + sa * b.w;

            y += s.x * r.x;
            y += s.y * r.y;
            y += s.z * r.z;
            y += s.w * r.w;
        }
        dst[t] = y;
    }

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        dst[T * C + batch_i * state_size + head_i * head_size * head_size + tid * head_size + i] = state[i];
    }
}

void ggml_cuda_op_rwkv_wkv6(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const float * k_d  = (const float *)dst->src[0]->data;
    const float * v_d  = (const float *)dst->src[1]->data;
    const float * r_d  = (const float *)dst->src[2]->data;
    const float * tf_d = (const float *)dst->src[3]->data;
    const float * td_d = (const float *)dst->src[4]->data;
    const float * s_d  = (const float *)dst->src[5]->data;

    const int64_t B = dst->src[5]->ne[1];
    const int64_t T = dst->src[0]->ne[2];
    const int64_t C = dst->ne[0];
    const int64_t H = dst->src[0]->ne[1];

    float * dst_d = (float *)dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(dst->src[5]->type == GGML_TYPE_F32);
    GGML_ASSERT(C % H == 0);
    GGML_ASSERT(C / H == CUDA_WKV_BLOCK_SIZE || C / H == CUDA_WKV_BLOCK_SIZE * 2);

    if (C / H == CUDA_WKV_BLOCK_SIZE) {
        rwkv_wkv_f32<CUDA_WKV_BLOCK_SIZE><<<B * H, C / H, 0, stream>>>(B, T, C, H, k_d, v_d, r_d, tf_d, td_d, s_d, dst_d);
    } else {
        rwkv_wkv_f32<CUDA_WKV_BLOCK_SIZE * 2><<<B * H, C / H, 0, stream>>>(B, T, C, H, k_d, v_d, r_d, tf_d, td_d, s_d, dst_d);
    }
}

void ggml_cuda_op_rwkv_wkv7(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const float * r_d = (const float *)dst->src[0]->data;
    const float * w_d = (const float *)dst->src[1]->data;
    const float * k_d = (const float *)dst->src[2]->data;
    const float * v_d = (const float *)dst->src[3]->data;
    const float * a_d = (const float *)dst->src[4]->data;
    const float * b_d = (const float *)dst->src[5]->data;
    const float * s_d = (const float *)dst->src[6]->data;

    const int64_t B = dst->src[6]->ne[1];
    const int64_t T = dst->src[0]->ne[2];
    const int64_t C = dst->ne[0];
    const int64_t H = dst->src[0]->ne[1];

    float * dst_d = (float *)dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(dst->src[6]->type == GGML_TYPE_F32);
    GGML_ASSERT(C % H == 0);
    GGML_ASSERT(C / H == CUDA_WKV_BLOCK_SIZE || C / H == CUDA_WKV_BLOCK_SIZE * 2);

    if (C / H == CUDA_WKV_BLOCK_SIZE) {
        rwkv_wkv7_f32<CUDA_WKV_BLOCK_SIZE><<<B * H, C / H, 0, stream>>>(B, T, C, H, r_d, w_d, k_d, v_d, a_d, b_d, s_d, dst_d);
    } else {
        rwkv_wkv7_f32<CUDA_WKV_BLOCK_SIZE * 2><<<B * H, C / H, 0, stream>>>(B, T, C, H, r_d, w_d, k_d, v_d, a_d, b_d, s_d, dst_d);
    }
}

#include "hip/hip_runtime.h"
// Old and deprecated WMMA FlashAttention implementation.
// It is still needed for Volta since the memory layout of NVIDIA tensor cores changed with Turing.
// Long-term the WMMA code should be replaced with a dedicated Volta implementation.

#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-wmma-f16.cuh"

#ifdef FP16_MMA_AVAILABLE
#if !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))
#include <mma.h>
namespace wmma = nvcuda::wmma;
#elif defined(GGML_HIP_ROCWMMA_FATTN) && defined(FP16_MMA_AVAILABLE)
#undef HIP_ENABLE_WARP_SYNC_BUILTINS // conflicts with rocWMMA headers
#include <rocwmma/rocwmma.hpp>
namespace wmma = rocwmma;
#endif // !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))
#endif // FP16_MMA_AVAILABLE

// D == head size, VKQ_stride == num VKQ rows calculated in parallel:
template<int D, int ncols, int nwarps, int VKQ_stride, typename KQ_acc_t, bool use_logit_softcap>
__launch_bounds__(nwarps*ggml_cuda_get_physical_warp_size(), 1)
static __global__ void flash_attn_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const float logit_softcap,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int nb21,
        const int nb22,
        const int nb23,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if defined(FLASH_ATTN_AVAILABLE) && (__CUDA_ARCH__ == GGML_CUDA_CC_VOLTA || (defined(GGML_HIP_ROCWMMA_FATTN) && defined(FP16_MMA_AVAILABLE)))
    // Skip unused kernel variants for faster compilation:
    if (use_logit_softcap && !(D == 128 || D == 256)) {
        NO_DEVICE_CODE;
        return;
    }

    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    constexpr int warp_size = ggml_cuda_get_physical_warp_size();

    const int ic0 = ncols*blockIdx.x; // Index of the first Q/QKV column to work on.

    static_assert(D <= FATTN_KQ_STRIDE, "D must be <= FATTN_KQ_STRIDE.");
    static_assert(ncols == 8 || ncols % 16 == 0, "ncols must be 8 or a multiple of 16.");
    constexpr int frag_m = ncols == 8 ? 32 : 16;
    constexpr int frag_n = ncols == 8 ?  8 : 16;
    static_assert(D % frag_m == 0, "If ncols == 8 then D % frag_m must be 0.");
    typedef wmma::fragment<wmma::matrix_a,    frag_m, frag_n, 16, half, wmma::row_major> frag_a_K;
    typedef wmma::fragment<wmma::matrix_a,    frag_m, frag_n, 16, half, wmma::col_major> frag_a_V;
    typedef wmma::fragment<wmma::matrix_b,    frag_m, frag_n, 16, half, wmma::col_major> frag_b;
    typedef wmma::fragment<wmma::accumulator, frag_m, frag_n, 16, KQ_acc_t>                      frag_c_KQ;
    typedef wmma::fragment<wmma::accumulator, frag_m, frag_n, 16, half>                          frag_c_VKQ;

    constexpr int KQ_stride_tc  = nwarps*frag_m; // Number of KQ rows calculated in parallel.
    constexpr int VKQ_ratio = KQ_stride_tc/VKQ_stride; // Number of parallel VKQ accumulators needed to keep all warps busy.
    static_assert(VKQ_ratio <= nwarps, "VKQ_ratio must be <= nwarps.");

    // Pad internal representation of KQ, KQV to reduce shared memory bank conflicts:
    constexpr int D_padded = D + 8;
    constexpr int kqs_padded = FATTN_KQ_STRIDE + 8;
    constexpr int kqar = sizeof(KQ_acc_t)/sizeof(half);

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float * Q_f   = (const float *) (Q + nb02* blockIdx.z              + nb01*ic0);
    const half  * K_h   = (const half  *) (K + nb12*(blockIdx.z / gqa_ratio));
    const half  * V_h   = (const half  *) (V + nb12*(blockIdx.z / gqa_ratio)); // K and V have same shape
    const half  * maskh = (const half  *)  mask + (nb31/sizeof(half))* ic0;
    const half2 * mask2 = (const half2 *)  mask + (nb31/sizeof(half))*(ic0/2);

    const int stride_Q  = nb01 / sizeof(float);
    const int stride_KV = nb11 / sizeof(half);

    const float slopef = get_alibi_slope(max_bias, blockIdx.z, n_head_log2, m0, m1);
    const half  slopeh = __float2half(slopef);
    const half2 slope2 = make_half2(slopef, slopef);

    const half2 logit_softcap_2 = make_half2(logit_softcap, logit_softcap);

    frag_b Q_b[D/16][ncols/frag_n];

    // A single buffer for temporarily holding tiles of KQ and VKQ parts:
    constexpr int mem_KQ = ncols*kqs_padded*kqar;
    constexpr int mem_VKQ_parts = VKQ_ratio*ncols*D_padded;
    __shared__ half KQ[mem_KQ >= mem_VKQ_parts ? mem_KQ : mem_VKQ_parts];
    float * KQ_f = (float *) KQ;
    half2 * KQ2 = (half2 *) KQ;

    float    KQ_rowsum_f[ncols/nwarps] = {0.0f};
    float       KQ_max_f[ncols/nwarps];
    float KQ_max_scale_f[ncols/nwarps] = {0.0f};

#pragma unroll
    for (int j = 0; j < ncols/nwarps; ++j) {
        KQ_max_f[j] = -FLT_MAX/2.0f;
    }

    half2    KQ_rowsum_h2[ncols/nwarps] = {{0.0f, 0.0f}};
    half2       KQ_max_h2[ncols/nwarps];
    half2 KQ_max_scale_h2[ncols/nwarps] = {{0.0f, 0.0f}};

#pragma unroll
    for (int j = 0; j < ncols/nwarps; ++j) {
        KQ_max_h2[j] = make_half2(-HALF_MAX_HALF, -HALF_MAX_HALF);
    }

    __shared__ half VKQ[ncols*D_padded]; // Accumulator for final VKQ slice.
    half2 * VKQ2 = (half2 *) VKQ;
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j = j0 + threadIdx.y;
#pragma unroll
        for (int i0 = 0; i0 < D/2; i0 += warp_size) {
            const int i = i0 + threadIdx.x;
            if (i0 + warp_size > D/2 && i >= D/2) {
                break;
            }
            VKQ2[j*(D_padded/2) + i] = make_half2(0.0f, 0.0f);
        }
    }

    // Convert Q to half and apply scale, temporarily store in KQ:
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j = j0 + threadIdx.y;
#pragma unroll
        for (int i0 = 0; i0 < D; i0 += warp_size) {
            const int i = i0 + threadIdx.x;
            if (i0 + warp_size > D && i >= D) {
                break;
            }
            KQ[j*D_padded + i] = ic0 + j < ne01 ? Q_f[j*stride_Q + i] * scale : 0.0f;
        }
    }

    __syncthreads();

    // Load Q into tensor core fragments/registers since it will be used frequently:
#pragma unroll
    for (int i0 = 0; i0 < D; i0 += 16) {
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += frag_n) {
            wmma::load_matrix_sync(Q_b[i0/16][j0/frag_n], KQ + j0*D_padded + i0, D_padded);
        }
    }

    __syncthreads();

    // Iterate over ne11 == previous tokens:
    for (int k_VKQ_0 = blockIdx.y*FATTN_KQ_STRIDE; k_VKQ_0 < ne11; k_VKQ_0 += gridDim.y*FATTN_KQ_STRIDE) {
        // Calculate tile of KQ:
#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE; i_KQ_0 += KQ_stride_tc) {
            frag_c_KQ KQ_c[ncols/frag_n];
#pragma unroll
            for (int j = 0; j < ncols/frag_n; ++j) {
                wmma::fill_fragment(KQ_c[j], static_cast<KQ_acc_t>(0.0f));
            }
#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D; k_KQ_0 += 16) {
                frag_a_K K_a;
                wmma::load_matrix_sync(K_a, K_h + (k_VKQ_0 + i_KQ_0 + frag_m*threadIdx.y)*stride_KV + k_KQ_0, stride_KV);
#pragma unroll
                for (int j = 0; j < ncols/frag_n; ++j) {
                    wmma::mma_sync(KQ_c[j], K_a, Q_b[k_KQ_0/16][j], KQ_c[j]);
                }
            }
#pragma unroll
            for (int j0 = 0; j0 < ncols; j0 += frag_n) {
                wmma::store_matrix_sync((KQ_acc_t *) KQ + j0*kqs_padded + i_KQ_0 + frag_m*threadIdx.y, KQ_c[j0/frag_n], kqs_padded, wmma::mem_col_major);
            }
        }

        __syncthreads();

        // Calculate softmax for each KQ column using the current max. value.
        // The divisor is stored in KQ_rowsum and will be applied at the end.
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            if (std::is_same<KQ_acc_t, float>::value) {
                float KQ_f_tmp[FATTN_KQ_STRIDE / warp_size];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += warp_size) {
                    const int k = k0 + threadIdx.x;

                    KQ_f_tmp[k0/warp_size] = KQ_f[j*kqs_padded + k];

                    if (use_logit_softcap) {
                        KQ_f_tmp[k0/warp_size] = logit_softcap*tanhf(KQ_f_tmp[k0/warp_size]);
                    }
                }

                float KQ_max_new = KQ_max_f[j0/nwarps];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += warp_size) {
                    const int k = k0 + threadIdx.x;

                    KQ_f_tmp[k0/warp_size] += mask ? __half2float(slopeh*maskh[j*(nb31/sizeof(half)) + k_VKQ_0 + k]) : 0.0f;
                    KQ_max_new = max(KQ_max_new, KQ_f_tmp[k0/warp_size]);
                }
                KQ_max_new = warp_reduce_max<warp_size>(KQ_max_new);

                const float diff = KQ_max_f[j0/nwarps] - KQ_max_new;
                KQ_max_scale_f[j0/nwarps] = expf(diff);
                if (diff <= SOFTMAX_FTZ_THRESHOLD) {
                    KQ_max_scale_f[j0/nwarps] = 0.0f;
                }
                KQ_max_f[j0/nwarps] = KQ_max_new;

                float KQ_rowsum_add = 0.0f;
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += warp_size) {
                    const int k = k0 + threadIdx.x;

                    const float diff = KQ_f_tmp[k0/warp_size] - KQ_max_f[j0/nwarps];
                    KQ_f_tmp[k0/warp_size] = expf(diff);
                    if (diff <= SOFTMAX_FTZ_THRESHOLD) {
                        KQ_f_tmp[k0/warp_size] = 0.0f;
                    }
                    KQ_rowsum_add += KQ_f_tmp[k0/warp_size];
                    KQ[j*(kqar*kqs_padded) + k] = KQ_f_tmp[k0/warp_size];
                }
                KQ_rowsum_add = warp_reduce_sum<warp_size>(KQ_rowsum_add);

                // Scale previous KQ_rowsum to account for a potential increase in KQ_max:
                KQ_rowsum_f[j0/nwarps] = KQ_max_scale_f[j0/nwarps]*KQ_rowsum_f[j0/nwarps] + KQ_rowsum_add;
            } else {
                half2 KQ2_tmp[FATTN_KQ_STRIDE/(2*warp_size)];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE/2; k0 += warp_size) {
                    const int k = k0 + threadIdx.x;

                    KQ2_tmp[k0/warp_size] = KQ2[j*(kqs_padded/2) + k];

                    if (use_logit_softcap) {
                        // There is no dedicated tangens hyperbolicus function for half2.
                        KQ2_tmp[k0/warp_size] = h2exp(KQ2_tmp[k0/warp_size]*make_half2(2.0f, 2.0f));
                        KQ2_tmp[k0/warp_size] = (KQ2_tmp[k0/warp_size] - make_half2(1.0f, 1.0f))
                                               /(KQ2_tmp[k0/warp_size] + make_half2(1.0f, 1.0f));

                        KQ2_tmp[k0/warp_size] *= logit_softcap_2;
                    }
                }

                half2 KQ_max_new = KQ_max_h2[j0/nwarps];
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE/2; k0 += warp_size) {
                    const int k = k0 + threadIdx.x;

                    KQ2_tmp[k0/warp_size] += mask ? slope2*mask2[(j*ne11 + k_VKQ_0)/2 + k] : make_half2(0.0f, 0.0f);
                    KQ_max_new = ggml_cuda_hmax2(KQ_max_new, KQ2_tmp[k0/warp_size]);
                }
                KQ_max_new = __half2half2(warp_reduce_max<warp_size>(ggml_cuda_hmax(__low2half(KQ_max_new), __high2half(KQ_max_new))));
                const half2 diff = KQ_max_h2[j0/nwarps] - KQ_max_new;
                KQ_max_scale_h2[j0/nwarps] = h2exp(diff);
                const uint32_t ftz_mask = __hgt2_mask(diff, make_half2(SOFTMAX_FTZ_THRESHOLD, SOFTMAX_FTZ_THRESHOLD));
                *((uint32_t *) &KQ_max_scale_h2[j0/nwarps]) &= ftz_mask;
                KQ_max_h2[j0/nwarps] = KQ_max_new;

                half2 KQ_rowsum_add = make_half2(0.0f, 0.0f);
#pragma unroll
                for (int k0 = 0; k0 < FATTN_KQ_STRIDE/2; k0 += warp_size) {
                    const int k = k0 + threadIdx.x;

                    const half2 diff = KQ2_tmp[k0/warp_size] - KQ_max_h2[j0/nwarps];
                    KQ2_tmp[k0/warp_size] = h2exp(diff);
                    const uint32_t ftz_mask = __hgt2_mask(diff, make_half2(SOFTMAX_FTZ_THRESHOLD, SOFTMAX_FTZ_THRESHOLD));
                    *((uint32_t *) &KQ2_tmp[k0/warp_size]) &= ftz_mask;
                    KQ_rowsum_add += KQ2_tmp[k0/warp_size];
                    KQ2[j*(kqs_padded/2) + k] = KQ2_tmp[k0/warp_size];
                }
                KQ_rowsum_add = warp_reduce_sum<warp_size>(KQ_rowsum_add);

                // Scale previous KQ_rowsum to account for a potential increase in KQ_max:
                KQ_rowsum_h2[j0/nwarps] = KQ_max_scale_h2[j0/nwarps]*KQ_rowsum_h2[j0/nwarps] + KQ_rowsum_add;
            }
        }

        __syncthreads();

        frag_b KQ_b[FATTN_KQ_STRIDE/(VKQ_ratio*16)][ncols/frag_n];
#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += frag_n) {
#pragma unroll
            for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += VKQ_ratio*16) {
                const int k = k0 + (threadIdx.y % VKQ_ratio)*16;
                wmma::load_matrix_sync(
                    KQ_b[k0/(VKQ_ratio*16)][j0/frag_n],
                    KQ + j0*(kqar*kqs_padded) + k,
                    kqar*kqs_padded);
            }
        }

        frag_c_VKQ VKQ_c[D/VKQ_stride][ncols/frag_n];
#pragma unroll
        for (int i_VKQ_0 = 0; i_VKQ_0 < D; i_VKQ_0 += VKQ_stride) {
#pragma unroll
            for (int j = 0; j < ncols/frag_n; ++j) {
                wmma::fill_fragment(VKQ_c[i_VKQ_0/VKQ_stride][j], static_cast<half>(0.0f));
            }

#pragma unroll
            for (int k0 = 0; k0 < FATTN_KQ_STRIDE; k0 += VKQ_ratio*16) {
                const int k = k0 + (threadIdx.y % VKQ_ratio)*16;

                frag_a_V v_a;
                wmma::load_matrix_sync(v_a, V_h + (k_VKQ_0 + k)*stride_KV + i_VKQ_0 + frag_m*(threadIdx.y/VKQ_ratio), stride_KV);
#pragma unroll
                for (int j = 0; j < ncols/frag_n; ++j) {
                    wmma::mma_sync(VKQ_c[i_VKQ_0/VKQ_stride][j], v_a, KQ_b[k0/(VKQ_ratio*16)][j], VKQ_c[i_VKQ_0/VKQ_stride][j]);
                }
            }
        }

        __syncthreads();

        const int offset_k = (threadIdx.y % VKQ_ratio) * (ncols*D_padded);
#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < D; i_KQ_0 += VKQ_stride) {
#pragma unroll
            for (int j0 = 0; j0 < ncols; j0 += frag_n) {
                wmma::store_matrix_sync(
                    KQ + offset_k + j0*D_padded + i_KQ_0 + frag_m*(threadIdx.y/VKQ_ratio),
                    VKQ_c[i_KQ_0/VKQ_stride][j0/frag_n],
                    D_padded, wmma::mem_col_major);
            }
        }

        __syncthreads();

#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            half2 VKQ_scale;
            if (std::is_same<KQ_acc_t, float>::value) {
                VKQ_scale = make_half2(KQ_max_scale_f[j0/nwarps], KQ_max_scale_f[j0/nwarps]);
            } else {
                VKQ_scale = KQ_max_scale_h2[j0/nwarps];
            }

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += warp_size) {
                const int i = i0 + threadIdx.x;
                if (i0 + warp_size > D/2 && i >= D/2) {
                    break;
                }

                half2 VKQ_add = make_half2(0.0f, 0.0f);
#pragma unroll
                for (int l = 0; l < VKQ_ratio; ++l) {
                    VKQ_add += KQ2[l*(ncols*D_padded/2) + j*(D_padded/2) + i];
                }
                VKQ2[j*(D_padded/2) + i] = VKQ_scale*VKQ2[j*(D_padded/2) + i] + VKQ_add;
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j_VKQ = j0 + threadIdx.y;
        if (ic0 + j_VKQ >= ne01) {
            return;
        }
        const int j_dst = (ic0 + j_VKQ)*gridDim.y + blockIdx.y;

        float KQ_rowsum_j;
        if (std::is_same<KQ_acc_t, float>::value) {
            KQ_rowsum_j = KQ_rowsum_f[j0/nwarps];
        } else {
            KQ_rowsum_j = __low2float(KQ_rowsum_h2[j0/nwarps]) + __high2float(KQ_rowsum_h2[j0/nwarps]);
        }

#pragma unroll
        for (int i0 = 0; i0 < D; i0 += warp_size) {
            const int i = i0 + threadIdx.x;
            if (i0 + warp_size > D && i >= D) {
                break;
            }
            float dst_val = VKQ[j_VKQ*D_padded + i];
            if (gridDim.y == 1) {
                dst_val /= KQ_rowsum_j;
            }
            dst[j_dst*gridDim.z*D + blockIdx.z*D + i] = dst_val;
        }

        if (gridDim.y == 1 || threadIdx.x != 0) {
            continue;
        }

        float2 dst_meta_val;
        if (std::is_same<KQ_acc_t, float>::value) {
            dst_meta_val.x = KQ_max_f[j0/nwarps];
        } else {
            dst_meta_val.x = __low2float(KQ_max_h2[j0/nwarps]);
        }
        dst_meta_val.y = KQ_rowsum_j;
        dst_meta[((ic0 + j_VKQ)*gridDim.z + blockIdx.z) * gridDim.y + blockIdx.y] = dst_meta_val;
    }
#else
    GGML_UNUSED(Q); GGML_UNUSED(K); GGML_UNUSED(V); GGML_UNUSED(mask);
    GGML_UNUSED(dst); GGML_UNUSED(dst_meta); GGML_UNUSED(scale);
    GGML_UNUSED(max_bias); GGML_UNUSED(m0); GGML_UNUSED(m1);
    GGML_UNUSED(n_head_log2); GGML_UNUSED(logit_softcap);
    GGML_UNUSED(ne00); GGML_UNUSED(ne01); GGML_UNUSED(ne02); GGML_UNUSED(ne03);
    GGML_UNUSED(ne10); GGML_UNUSED(ne11); GGML_UNUSED(ne12); GGML_UNUSED(ne13);
    GGML_UNUSED(ne31); GGML_UNUSED(nb31); GGML_UNUSED(nb01); GGML_UNUSED(nb02);
    GGML_UNUSED(nb03); GGML_UNUSED(nb11); GGML_UNUSED(nb12); GGML_UNUSED(nb13);
    GGML_UNUSED(nb21); GGML_UNUSED(nb22); GGML_UNUSED(nb23);
    GGML_UNUSED(ne0); GGML_UNUSED(ne1); GGML_UNUSED(ne2); GGML_UNUSED(ne3);
    NO_DEVICE_CODE;
#endif // defined(FLASH_ATTN_AVAILABLE) && (__CUDA_ARCH__ == GGML_CUDA_CC_VOLTA || (defined(GGML_HIP_ROCWMMA_FATTN) && defined(FP16_MMA_AVAILABLE)))
}

constexpr int get_max_power_of_2(int x) {
    return x % 2 == 0 ? 2*get_max_power_of_2(x/2) : 1;
}

static_assert(get_max_power_of_2(1) == 1, "Test failed.");
static_assert(get_max_power_of_2(2) == 2, "Test failed.");
static_assert(get_max_power_of_2(4) == 4, "Test failed.");
static_assert(get_max_power_of_2(6) == 2, "Test failed.");

// Number of VKQ rows calculated in parallel:
constexpr int get_VKQ_stride(int D, int nwarps, int frag_m) {
    return (get_max_power_of_2(D/frag_m) < nwarps ? get_max_power_of_2(D/frag_m) : nwarps)*frag_m;
}

static_assert(get_VKQ_stride(128, 1, 32) ==  32, "Test failed.");
static_assert(get_VKQ_stride(128, 2, 32) ==  64, "Test failed.");
static_assert(get_VKQ_stride(128, 4, 32) == 128, "Test failed.");
static_assert(get_VKQ_stride( 64, 1, 32) ==  32, "Test failed.");
static_assert(get_VKQ_stride( 64, 2, 32) ==  64, "Test failed.");
static_assert(get_VKQ_stride( 64, 4, 32) ==  64, "Test failed.");
static_assert(get_VKQ_stride( 80, 1, 16) ==  16, "Test failed.");
static_assert(get_VKQ_stride( 80, 2, 16) ==  16, "Test failed.");
static_assert(get_VKQ_stride( 80, 4, 16) ==  16, "Test failed.");

template <int D, int cols_per_block, typename KQ_acc_t>
void ggml_cuda_flash_attn_ext_wmma_f16_case(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV = dst;

    constexpr int nwarps = 4;

    constexpr int frag_m = cols_per_block == 8 && D % 32 == 0 ? 32 : 16;
    const int warp_size = ggml_cuda_info().devices[ggml_cuda_get_device()].warp_size;

    float logit_softcap;
    memcpy(&logit_softcap, (const float *) KQV->op_params + 2, sizeof(float));

    fattn_kernel_t fattn_kernel;
    if (logit_softcap == 0.0f) {
        constexpr bool use_logit_softcap = false;
        fattn_kernel = flash_attn_ext_f16<
            D, cols_per_block, nwarps, get_VKQ_stride(D, nwarps, frag_m), KQ_acc_t, use_logit_softcap>;
    } else {
        constexpr bool use_logit_softcap = true;
        fattn_kernel = flash_attn_ext_f16<
            D, cols_per_block, nwarps, get_VKQ_stride(D, nwarps, frag_m), KQ_acc_t, use_logit_softcap>;
    }
    launch_fattn<D, cols_per_block, 1, -1>(ctx, dst, fattn_kernel, nwarps, 0, FATTN_KQ_STRIDE, true, true, false, warp_size);
}

void ggml_cuda_flash_attn_ext_wmma_f16(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV = dst;
    const ggml_tensor * Q   = dst->src[0];

    const enum ggml_prec prec = ggml_flash_attn_ext_get_prec(KQV);
    const int warp_size = ggml_cuda_info().devices[ctx.device].warp_size;

    if (prec != GGML_PREC_DEFAULT) {
        if (Q->ne[1] <= 32 || Q->ne[0] > 128) {
            constexpr int cols_per_block = 16;
            switch (Q->ne[0]) {
                case 64:
                    ggml_cuda_flash_attn_ext_wmma_f16_case< 64, cols_per_block, float>(ctx, dst);
                    break;
                case 80:
                    ggml_cuda_flash_attn_ext_wmma_f16_case< 80, cols_per_block, float>(ctx, dst);
                    break;
                case 96:
                    ggml_cuda_flash_attn_ext_wmma_f16_case< 96, cols_per_block, float>(ctx, dst);
                    break;
                case 112:
                    ggml_cuda_flash_attn_ext_wmma_f16_case<112, cols_per_block, float>(ctx, dst);
                    break;
                case 128:
                    ggml_cuda_flash_attn_ext_wmma_f16_case<128, cols_per_block, float>(ctx, dst);
                    break;
                case 256:
                    ggml_cuda_flash_attn_ext_wmma_f16_case<256, cols_per_block, float>(ctx, dst);
                    break;
                default:
                    GGML_ABORT("fatal error");
                    break;
            }
        } else {
            constexpr int cols_per_block = 32;
            switch (Q->ne[0]) {
                case 64:
                    ggml_cuda_flash_attn_ext_wmma_f16_case< 64, cols_per_block, float>(ctx, dst);
                    break;
                case 80:
                    ggml_cuda_flash_attn_ext_wmma_f16_case< 80, cols_per_block, float>(ctx, dst);
                    break;
                case 96:
                    ggml_cuda_flash_attn_ext_wmma_f16_case< 96, cols_per_block, float>(ctx, dst);
                    break;
                case 112:
                    ggml_cuda_flash_attn_ext_wmma_f16_case<112, cols_per_block, float>(ctx, dst);
                    break;
                case 128:
                    ggml_cuda_flash_attn_ext_wmma_f16_case<128, cols_per_block, float>(ctx, dst);
                    break;
                // case 256:
                //     ggml_cuda_flash_attn_ext_wmma_f16_case<256, cols_per_block, float>(ctx, dst);
                //     break;
                default:
                    GGML_ABORT("fatal error");
                    break;
            }
        }
        return;
    }

#if !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))
    if (Q->ne[1] <= 8 && Q->ne[0] % warp_size == 0) {
        constexpr int cols_per_block = 8;
        switch (Q->ne[0]) {
            case 64:
                ggml_cuda_flash_attn_ext_wmma_f16_case< 64, cols_per_block, half>(ctx, dst);
                break;
            case 96:
                ggml_cuda_flash_attn_ext_wmma_f16_case< 96, cols_per_block, half>(ctx, dst);
                break;
            case 128:
                ggml_cuda_flash_attn_ext_wmma_f16_case<128, cols_per_block, half>(ctx, dst);
                break;
            case 256:
                ggml_cuda_flash_attn_ext_wmma_f16_case<256, cols_per_block, half>(ctx, dst);
                break;
            default:
                GGML_ABORT("fatal error");
                break;
        }
        return;
    }
#endif // !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))

    if (Q->ne[1] <= 32) {
        constexpr int cols_per_block = 16;
        switch (Q->ne[0]) {
            case 64:
                ggml_cuda_flash_attn_ext_wmma_f16_case< 64, cols_per_block, half>(ctx, dst);
                break;
            case 80:
                ggml_cuda_flash_attn_ext_wmma_f16_case< 80, cols_per_block, half>(ctx, dst);
                break;
            case 96:
                ggml_cuda_flash_attn_ext_wmma_f16_case< 96, cols_per_block, half>(ctx, dst);
                break;
            case 112:
                ggml_cuda_flash_attn_ext_wmma_f16_case<112, cols_per_block, half>(ctx, dst);
                break;
            case 128:
                ggml_cuda_flash_attn_ext_wmma_f16_case<128, cols_per_block, half>(ctx, dst);
                break;
            case 256:
                ggml_cuda_flash_attn_ext_wmma_f16_case<256, cols_per_block, half>(ctx, dst);
                break;
            default:
                GGML_ABORT("fatal error");
                break;
        }
        return;
    }

    constexpr int cols_per_block = 32;
    switch (Q->ne[0]) {
        case 64:
            ggml_cuda_flash_attn_ext_wmma_f16_case< 64, cols_per_block, half>(ctx, dst);
            break;
        case 80:
            ggml_cuda_flash_attn_ext_wmma_f16_case< 80, cols_per_block, half>(ctx, dst);
            break;
        case 96:
            ggml_cuda_flash_attn_ext_wmma_f16_case< 96, cols_per_block, half>(ctx, dst);
            break;
        case 112:
            ggml_cuda_flash_attn_ext_wmma_f16_case<112, cols_per_block, half>(ctx, dst);
            break;
        case 128:
            ggml_cuda_flash_attn_ext_wmma_f16_case<128, cols_per_block, half>(ctx, dst);
            break;
        case 256:
            ggml_cuda_flash_attn_ext_wmma_f16_case<256, cols_per_block, half>(ctx, dst);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

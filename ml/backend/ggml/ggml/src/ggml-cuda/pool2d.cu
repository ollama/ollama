#include "hip/hip_runtime.h"
#include "pool2d.cuh"

template <typename Ti, typename To>
static  __global__ void pool2d_nchw_kernel(
        const int ih, const int iw, const int oh, const int ow,
        const int kh, const int kw, const int sh, const int sw,
        const int ph, const int pw, const int parallel_elements,
        const Ti* src, To* dst, const enum ggml_op_pool op) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= parallel_elements) {
        return;
    }

    const int I_HW = ih * iw;
    const int O_HW = oh * ow;
    const int nc = idx / O_HW;
    const int cur_oh = idx % O_HW / ow;
    const int cur_ow = idx % O_HW % ow;
    const Ti* i_ptr = src + nc * I_HW;
    To* o_ptr = dst + nc * O_HW;
    const int start_h = cur_oh * sh - ph;
    const int bh = max(0, start_h);
    const int eh = min(ih, start_h + kh);
    const int start_w = cur_ow * sw - pw;
    const int bw = max(0, start_w);
    const int ew = min(iw, start_w + kw);
    const To scale = 1. / (kh * kw);
    To res = 0;

    switch (op) {
        case GGML_OP_POOL_AVG: res = 0; break;
        case GGML_OP_POOL_MAX: res = -FLT_MAX; break;
        default: assert(false);
    }

    for (int i = bh; i < eh; i += 1) {
        for (int j = bw; j < ew; j += 1) {
#if __CUDA_ARCH__ >= 350
            Ti cur = __ldg(i_ptr + i * iw + j);
#else
            Ti cur = i_ptr[i * iw + j];
#endif
            switch (op) {
                case GGML_OP_POOL_AVG: res += cur * scale; break;
                case GGML_OP_POOL_MAX: res = max(res, (To)cur); break;
                default: assert(false);
            }
        }
    }
    o_ptr[cur_oh * ow + cur_ow] = res;
}

static void pool2d_nchw_kernel_f32_f32_cuda(
        const int ih, const int iw, const int oh, const int ow,
        const int kh, const int kw, const int sh, const int sw,
        const int ph, const int pw, const int parallel_elements,
        const float * src, float * dst, const enum ggml_op_pool op,
        hipStream_t stream) {

    const int num_blocks = (parallel_elements + CUDA_POOL2D_BLOCK_SIZE - 1) / CUDA_POOL2D_BLOCK_SIZE;
    dim3 block_nums(num_blocks);
    pool2d_nchw_kernel<<<block_nums, CUDA_POOL2D_BLOCK_SIZE, 0, stream>>>(ih, iw, oh, ow, kh, kw, sh, sw, ph, pw, parallel_elements, src, dst, op);
}

void ggml_cuda_op_pool2d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int32_t * opts = (const int32_t *)dst->op_params;
    enum ggml_op_pool op = static_cast<ggml_op_pool>(opts[0]);
    const int k0 = opts[1];
    const int k1 = opts[2];
    const int s0 = opts[3];
    const int s1 = opts[4];
    const int p0 = opts[5];
    const int p1 = opts[6];

    const int64_t IH = src0->ne[1];
    const int64_t IW = src0->ne[0];

    const int64_t N = dst->ne[3];
    const int64_t OC = dst->ne[2];
    const int64_t OH = dst->ne[1];
    const int64_t OW = dst->ne[0];

    const int parallel_elements = N * OC * OH * OW;

    pool2d_nchw_kernel_f32_f32_cuda(IH, IW, OH, OW, k1, k0, s1, s0, p1, p0, parallel_elements, src0_d, dst_d, op, stream);
}

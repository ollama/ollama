#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-mma-f16.cuh"
#include "fattn-tile.cuh"
#include "fattn-vec.cuh"
#include "fattn-wmma-f16.cuh"
#include "fattn.cuh"

template <int DKQ, int DV, int ncols2>
static void ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const int cc = ggml_cuda_info().devices[ggml_cuda_get_device()].cc;
    const ggml_tensor * Q = dst->src[0];

    if constexpr (ncols2 <= 8) {
        if (Q->ne[1] <= 8/ncols2) {
            ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 8/ncols2, ncols2>(ctx, dst);
            return;
        }
    }

    if (Q->ne[1] <= 16/ncols2) {
        ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 16/ncols2, ncols2>(ctx, dst);
        return;
    }

    if (ggml_cuda_highest_compiled_arch(cc) == GGML_CUDA_CC_TURING || Q->ne[1] <= 32/ncols2) {
        ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 32/ncols2, ncols2>(ctx, dst);
        return;
    }

    ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 64/ncols2, ncols2>(ctx, dst);
}

template <int DKQ, int DV>
static void ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV  = dst;
    const ggml_tensor * Q    = dst->src[0];
    const ggml_tensor * K    = dst->src[1];
    const ggml_tensor * mask = dst->src[3];

    float max_bias = 0.0f;
    memcpy(&max_bias, (const float *) KQV->op_params + 1, sizeof(float));

    const bool use_gqa_opt = mask && max_bias == 0.0f;

    GGML_ASSERT(Q->ne[2] % K->ne[2] == 0);
    const int gqa_ratio = Q->ne[2] / K->ne[2];

    if (use_gqa_opt && gqa_ratio % 8 == 0) {
        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 8>(ctx, dst);
        return;
    }

    if (use_gqa_opt && gqa_ratio % 4 == 0) {
        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 4>(ctx, dst);
        return;
    }

    if (use_gqa_opt && gqa_ratio % 2 == 0) {
        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 2>(ctx, dst);
        return;
    }

    ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 1>(ctx, dst);
}

static void ggml_cuda_flash_attn_ext_mma_f16(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV  = dst;
    const ggml_tensor * Q    = dst->src[0];
    const ggml_tensor * K    = dst->src[1];
    const ggml_tensor * V    = dst->src[2];
    const ggml_tensor * mask = dst->src[3];

    switch (Q->ne[0]) {
        case 64:
            GGML_ASSERT(V->ne[0] == 64);
            ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2< 64,  64>(ctx, dst);
            break;
        case 80:
            GGML_ASSERT(V->ne[0] == 80);
            ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2< 80,  80>(ctx, dst);
            break;
        case 96:
            GGML_ASSERT(V->ne[0] == 96);
            ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2< 96,  96>(ctx, dst);
            break;
        case 112:
            GGML_ASSERT(V->ne[0] == 112);
            ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2<112, 112>(ctx, dst);
            break;
        case 128:
            GGML_ASSERT(V->ne[0] == 128);
            ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2<128, 128>(ctx, dst);
            break;
        case 256:
            GGML_ASSERT(V->ne[0] == 256);
            ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2<256, 256>(ctx, dst);
            break;
        case 576: {
            // For Deepseek, go straight to the ncols1 switch to avoid compiling unnecessary kernels.
            GGML_ASSERT(V->ne[0] == 512);
            float max_bias = 0.0f;
            memcpy(&max_bias, (const float *) KQV->op_params + 1, sizeof(float));

            const bool use_gqa_opt = mask && max_bias == 0.0f;
            GGML_ASSERT(use_gqa_opt);

            GGML_ASSERT(Q->ne[2] % K->ne[2] == 0);
            const int gqa_ratio = Q->ne[2] / K->ne[2];
            GGML_ASSERT(gqa_ratio % 16 == 0);
            ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<576, 512, 16>(ctx, dst);
        } break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

#define FATTN_VEC_CASE(D, type_K, type_V)                                                                        \
    {                                                                                                            \
        const bool type_K_okay = K->type == (type_K) || (K->type == GGML_TYPE_F32 && (type_K) == GGML_TYPE_F16); \
        const bool type_V_okay = V->type == (type_V) || (V->type == GGML_TYPE_F32 && (type_V) == GGML_TYPE_F16); \
        if (Q->ne[0] == (D) && type_K_okay && type_V_okay) {                                                     \
            ggml_cuda_flash_attn_ext_vec_case<D, type_K, type_V>(ctx, dst);                                      \
            return;                                                                                              \
        }                                                                                                        \
    }                                                                                                            \

#define FATTN_VEC_CASES_ALL_D(type_K, type_V) \
    FATTN_VEC_CASE( 64, type_K, type_V)       \
    FATTN_VEC_CASE(128, type_K, type_V)       \
    FATTN_VEC_CASE(256, type_K, type_V)       \

static void ggml_cuda_flash_attn_ext_vec(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_tensor * Q = dst->src[0];
    ggml_tensor * K = dst->src[1];
    ggml_tensor * V = dst->src[2];

#ifdef GGML_CUDA_FA_ALL_QUANTS
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_F16,  GGML_TYPE_F16)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_0, GGML_TYPE_F16)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_1, GGML_TYPE_F16)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_0, GGML_TYPE_F16)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_1, GGML_TYPE_F16)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q8_0, GGML_TYPE_F16)

    FATTN_VEC_CASES_ALL_D(GGML_TYPE_F16,  GGML_TYPE_Q4_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_0, GGML_TYPE_Q4_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_1, GGML_TYPE_Q4_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_0, GGML_TYPE_Q4_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_1, GGML_TYPE_Q4_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q8_0, GGML_TYPE_Q4_0)

    FATTN_VEC_CASES_ALL_D(GGML_TYPE_F16,  GGML_TYPE_Q4_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_0, GGML_TYPE_Q4_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_1, GGML_TYPE_Q4_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_0, GGML_TYPE_Q4_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_1, GGML_TYPE_Q4_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q8_0, GGML_TYPE_Q4_1)

    FATTN_VEC_CASES_ALL_D(GGML_TYPE_F16,  GGML_TYPE_Q5_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_0, GGML_TYPE_Q5_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_1, GGML_TYPE_Q5_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_0, GGML_TYPE_Q5_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_1, GGML_TYPE_Q5_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q8_0, GGML_TYPE_Q5_0)

    FATTN_VEC_CASES_ALL_D(GGML_TYPE_F16,  GGML_TYPE_Q5_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_0, GGML_TYPE_Q5_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_1, GGML_TYPE_Q5_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_0, GGML_TYPE_Q5_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_1, GGML_TYPE_Q5_1)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q8_0, GGML_TYPE_Q5_1)

    FATTN_VEC_CASES_ALL_D(GGML_TYPE_F16,  GGML_TYPE_Q8_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_0, GGML_TYPE_Q8_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_1, GGML_TYPE_Q8_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_0, GGML_TYPE_Q8_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q5_1, GGML_TYPE_Q8_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q8_0, GGML_TYPE_Q8_0)
#else
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_F16,  GGML_TYPE_F16)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q4_0, GGML_TYPE_Q4_0)
    FATTN_VEC_CASES_ALL_D(GGML_TYPE_Q8_0, GGML_TYPE_Q8_0)
#endif // GGML_CUDA_FA_ALL_QUANTS

    GGML_ABORT("fatal error");
}

// Best FlashAttention kernel for a specific GPU:
enum best_fattn_kernel {
    BEST_FATTN_KERNEL_NONE     =   0,
    BEST_FATTN_KERNEL_TILE     = 200,
    BEST_FATTN_KERNEL_VEC      = 100,
    BEST_FATTN_KERNEL_WMMA_F16 = 300,
    BEST_FATTN_KERNEL_MMA_F16  = 400,
};

static best_fattn_kernel ggml_cuda_get_best_fattn_kernel(const int device, const ggml_tensor * dst) {
#ifndef FLASH_ATTN_AVAILABLE
    GGML_UNUSED(device); GGML_UNUSED(dst);
    return BEST_FATTN_KERNEL_NONE;
#endif// FLASH_ATTN_AVAILABLE

    const ggml_tensor * KQV   = dst;
    const ggml_tensor * Q     = dst->src[0];
    const ggml_tensor * K     = dst->src[1];
    const ggml_tensor * V     = dst->src[2];
    const ggml_tensor * mask  = dst->src[3];

    const int gqa_ratio = Q->ne[2] / K->ne[2];
    GGML_ASSERT(Q->ne[2] % K->ne[2] == 0);

    float max_bias = 0.0f;
    memcpy(&max_bias, (const float *) KQV->op_params + 1, sizeof(float));

    // The effective batch size for the kernel can be increased by gqa_ratio.
    // The kernel versions without this optimization are also used for ALiBi, if there is no mask, or if the KV cache is not padded,
    const bool gqa_opt_applies = gqa_ratio % 2 == 0 && mask && max_bias == 0.0f && K->ne[1] % FATTN_KQ_STRIDE == 0;

    const int cc = ggml_cuda_info().devices[device].cc;

    switch (K->ne[0]) {
        case  40:
        case  64:
        case  80:
        case  96:
        case 128:
        case 112:
        case 256:
            if (V->ne[0] != K->ne[0]) {
                return BEST_FATTN_KERNEL_NONE;
            }
            break;
        case 576:
            if (V->ne[0] != 512) {
                return BEST_FATTN_KERNEL_NONE;
            }
            if (!gqa_opt_applies || gqa_ratio % 16 != 0) {
                return BEST_FATTN_KERNEL_NONE;
            }
            break;
        default:
            return BEST_FATTN_KERNEL_NONE;
    }

#ifndef GGML_CUDA_FA_ALL_QUANTS
    if (K->type != V->type) {
        return BEST_FATTN_KERNEL_NONE;
    }
#endif // GGML_CUDA_FA_ALL_QUANTS

    switch (K->type) {
        case GGML_TYPE_F32:
        case GGML_TYPE_F16:
            break;
        case GGML_TYPE_Q4_1:
        case GGML_TYPE_Q5_0:
        case GGML_TYPE_Q5_1:
#ifndef GGML_CUDA_FA_ALL_QUANTS
            return BEST_FATTN_KERNEL_NONE;
#endif // GGML_CUDA_FA_ALL_QUANTS
        case GGML_TYPE_Q4_0:
        case GGML_TYPE_Q8_0:
            break;
        default:
            return BEST_FATTN_KERNEL_NONE;
    }

    if (mask && mask->ne[2] != 1) {
        return BEST_FATTN_KERNEL_NONE;
    }

    // For small batch sizes the vector kernel may be preferable over the kernels optimized for large batch sizes:
    const bool can_use_vector_kernel = Q->ne[0] <= 256 && Q->ne[0] % 64 == 0 && K->ne[1] % FATTN_KQ_STRIDE == 0;

    // If Turing tensor cores available, use them:
    if (turing_mma_available(cc) && K->ne[1] % FATTN_KQ_STRIDE == 0 && Q->ne[0] != 40) {
        if (can_use_vector_kernel) {
            if (!ggml_is_quantized(K->type) && !ggml_is_quantized(V->type)) {
                if (cc >= GGML_CUDA_CC_ADA_LOVELACE && Q->ne[1] == 1 && Q->ne[3] == 1 && !(gqa_ratio > 4 && K->ne[1] >= 8192)) {
                    return BEST_FATTN_KERNEL_VEC;
                }
            } else {
                if (cc >= GGML_CUDA_CC_ADA_LOVELACE) {
                    if (Q->ne[1] <= 2) {
                        return BEST_FATTN_KERNEL_VEC;
                    }
                } else {
                    if (Q->ne[1] == 1) {
                        return BEST_FATTN_KERNEL_VEC;
                    }
                }
            }
            if (!gqa_opt_applies && Q->ne[1] == 1) {
                return BEST_FATTN_KERNEL_VEC;
            }
        }

        return BEST_FATTN_KERNEL_MMA_F16;
    }

    // Use the WMMA kernel if possible:
    if (ggml_cuda_should_use_wmma_fattn(cc) && K->ne[1] % FATTN_KQ_STRIDE == 0 && Q->ne[0] != 40 && Q->ne[0] != 576) {
        if (can_use_vector_kernel && Q->ne[1] <= 2) {
            return BEST_FATTN_KERNEL_VEC;
        }
        return BEST_FATTN_KERNEL_WMMA_F16;
    }

    // If there are no tensor cores available, use the generic tile kernel:
    if (can_use_vector_kernel) {
        if (!ggml_is_quantized(K->type) && !ggml_is_quantized(V->type)) {
            if (Q->ne[1] == 1) {
                if (!gqa_opt_applies) {
                    return BEST_FATTN_KERNEL_VEC;
                }
            }
        } else {
            if (Q->ne[1] <= 2) {
                return BEST_FATTN_KERNEL_VEC;
            }
        }
    }
    return BEST_FATTN_KERNEL_TILE;
}

void ggml_cuda_flash_attn_ext(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_set_device(ctx.device);
    switch (ggml_cuda_get_best_fattn_kernel(ggml_cuda_get_device(), dst)) {
        case BEST_FATTN_KERNEL_NONE:
            GGML_ABORT("fatal error");
        case BEST_FATTN_KERNEL_TILE:
            ggml_cuda_flash_attn_ext_tile(ctx, dst);
            break;
        case BEST_FATTN_KERNEL_VEC:
            ggml_cuda_flash_attn_ext_vec(ctx, dst);
            break;
        case BEST_FATTN_KERNEL_WMMA_F16:
            ggml_cuda_flash_attn_ext_wmma_f16(ctx, dst);
            break;
        case BEST_FATTN_KERNEL_MMA_F16:
            ggml_cuda_flash_attn_ext_mma_f16(ctx, dst);
            break;
    }
}

bool ggml_cuda_flash_attn_ext_supported(int device, const ggml_tensor * dst) {
    return ggml_cuda_get_best_fattn_kernel(device, dst) != BEST_FATTN_KERNEL_NONE;
}

#include "hip/hip_runtime.h"
#include "common.cuh"
#include "wkv6.cuh"

static __global__ void rwkv_wkv_f32(const int B, const int T, const int C, const int H, const float * k, const float * v, const float * r, const float * tf, const float * td, const float * s, float * dst) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int head_size = CUDA_WKV_BLOCK_SIZE;
    const int batch_i = bid / H;
    const int head_i = bid % H;
    const int state_size = C * head_size;
    const int n_seq_tokens = T / B;

    float state[head_size];
    __shared__ float _k[head_size], _r[head_size], _tf[head_size], _td[head_size];

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        state[i] = s[batch_i * state_size + head_i * head_size * head_size + i * head_size + tid];
    }

    __syncthreads();
    _tf[tid] = tf[head_i * head_size + tid];
    __syncthreads();

    for (int t = batch_i * n_seq_tokens * C + head_i * head_size + tid; t < (batch_i + 1) * n_seq_tokens * C + head_i * head_size + tid; t += C) {
        __syncthreads();
        _k[tid] = k[t];
        _r[tid] = r[t];
        _td[tid] = td[t];
        __syncthreads();

        const float _v = v[t];
        float y = 0;
        for (int j = 0; j < head_size; j += 4) {
            const float4& k = (float4&)(_k[j]);
            const float4& r = (float4&)(_r[j]);
            const float4& tf = (float4&)(_tf[j]);
            const float4& td = (float4&)(_td[j]);
            float4& s = (float4&)(state[j]);
            float4 kv;

            kv.x = k.x * _v;
            kv.y = k.y * _v;
            kv.z = k.z * _v;
            kv.w = k.w * _v;

            y += r.x * (tf.x * kv.x + s.x);
            y += r.y * (tf.y * kv.y + s.y);
            y += r.z * (tf.z * kv.z + s.z);
            y += r.w * (tf.w * kv.w + s.w);

            s.x = s.x * td.x + kv.x;
            s.y = s.y * td.y + kv.y;
            s.z = s.z * td.z + kv.z;
            s.w = s.w * td.w + kv.w;
        }
        dst[t] = y;
    }

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        dst[T * C + batch_i * state_size + head_i * head_size * head_size + i * head_size + tid] = state[i];
    }
}

void ggml_cuda_op_rwkv_wkv6(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const float * k_d  = (const float *)dst->src[0]->data;
    const float * v_d  = (const float *)dst->src[1]->data;
    const float * r_d  = (const float *)dst->src[2]->data;
    const float * tf_d = (const float *)dst->src[3]->data;
    const float * td_d = (const float *)dst->src[4]->data;
    const float * s_d  = (const float *)dst->src[5]->data;

    const int64_t B = dst->src[5]->ne[1];
    const int64_t T = dst->src[0]->ne[3];
    const int64_t C = dst->ne[0];
    const int64_t H = dst->src[0]->ne[2];

    float * dst_d = (float *)dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(dst->src[5]->type == GGML_TYPE_F32);
    GGML_ASSERT(C % H == 0);
    GGML_ASSERT(C / H == CUDA_WKV_BLOCK_SIZE); // The current cuda kernel is designed for RWKV6, HEAD_SIZE == 64

    rwkv_wkv_f32<<<B * H, C / H, 0, stream>>>(B, T, C, H, k_d, v_d, r_d, tf_d, td_d, s_d, dst_d);
}

#include "hip/hip_runtime.h"
#include "common.cuh"
#include "cross-entropy-loss.cuh"
#include "sum.cuh"

#include <cmath>
#include <cstdint>

static __global__ void cross_entropy_loss_f32(const float * logits, const float * labels, float * dst, const int nclasses, const int k) {
    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int i0 = blockDim.x*blockIdx.x + warp_id*WARP_SIZE;

    const int ne_tmp = WARP_SIZE*nclasses;

    extern __shared__ float tmp_all[];
    float * tmp_logits = tmp_all + (2*warp_id + 0)*ne_tmp;
    float * tmp_labels = tmp_all + (2*warp_id + 1)*ne_tmp;

    // Each warp first loads ne_tmp logits/labels into shared memory:
    for (int i = lane_id; i < ne_tmp; i += WARP_SIZE) {
        const int ig = i0*nclasses + i; // ig == i global

        tmp_logits[i] = ig < k*nclasses ? logits[ig] : 0.0f;
        tmp_labels[i] = ig < k*nclasses ? labels[ig] : 0.0f;
    }

    // Each thread in the warp then calculates the cross entropy loss for a single row.
    // TODO: pad in order to avoid shared memory bank conflicts.

    // Find maximum for softmax:
    float max = -INFINITY;
    for (int i = 0; i < nclasses; ++i) {
        max = fmaxf(max, tmp_logits[lane_id*nclasses + i]);
    }

    // Calculate log(softmax(logits)) which is just logits - max:
    float sum = 0.0f;
    for (int i = 0; i < nclasses; ++i) {
        float val = tmp_logits[lane_id*nclasses + i] - max;
        sum += expf(val);
        tmp_logits[lane_id*nclasses + i] = val;
    }
    sum = logf(sum);

    // log(exp(logits - max) / sum) = (logits - max) - log(sum)
    float loss = 0.0f;
    for (int i = 0; i < nclasses; ++i) {
        loss += (tmp_logits[lane_id*nclasses + i] - sum) * tmp_labels[lane_id*nclasses + i];
    }
    loss = -warp_reduce_sum(loss) / (float)k;

    __syncthreads();

    if (lane_id == 0) {
        tmp_all[warp_id] = loss;
    }

    __syncthreads();

    if (warp_id != 0) {
        return;
    }

    loss = lane_id < CUDA_CROSS_ENTROPY_LOSS_BLOCK_SIZE/WARP_SIZE ? tmp_all[lane_id] : 0.0f;
    loss = warp_reduce_sum(loss);

    if (lane_id != 0) {
        return;
    }

    dst[blockIdx.x] = loss;
}

static __global__ void cross_entropy_loss_back_f32(const float * logits, const float * labels, const float * loss, float * dst, const int nclasses) {
    extern __shared__ float tmp[];

    float maxval = -INFINITY;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        const float val = logits[blockIdx.x*nclasses + i];
        maxval = fmaxf(maxval, val);
        tmp[i] = val;
    }
    maxval = warp_reduce_max(maxval);

    float sum = 0.0f;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        const float val = expf(tmp[i] - maxval);
        sum += val;
        tmp[i] = val;
    }
    sum = warp_reduce_sum(sum);
    const float sm_scale = 1.0f/sum;

    const float d_by_nrows = *loss/gridDim.x;
    for (int i = threadIdx.x; i < nclasses; i += WARP_SIZE) {
        dst[blockIdx.x*nclasses + i] = (tmp[i]*sm_scale - labels[blockIdx.x*nclasses + i])*d_by_nrows;
    }
}

void ggml_cuda_cross_entropy_loss(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(dst));

    const int64_t ne00  = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    ggml_cuda_pool & pool = ctx.pool();
    hipStream_t stream = ctx.stream();

    const dim3 blocks_dim(CUDA_CROSS_ENTROPY_LOSS_BLOCK_SIZE, 1, 1);
    const dim3 blocks_num((nrows + CUDA_CROSS_ENTROPY_LOSS_BLOCK_SIZE - 1) / CUDA_CROSS_ENTROPY_LOSS_BLOCK_SIZE, 1, 1);
    const int shmem = 2*CUDA_CROSS_ENTROPY_LOSS_BLOCK_SIZE*ne00*sizeof(float);

    ggml_cuda_pool_alloc<float> dst_tmp(pool, blocks_num.x);

    cross_entropy_loss_f32<<<blocks_num, blocks_dim, shmem, stream>>>(src0_d, src1_d, dst_tmp.ptr, ne00, nrows);

    // Combine results from individual blocks:
    sum_f32_cuda(pool, dst_tmp.ptr, dst_d, blocks_num.x, stream);
}

void ggml_cuda_cross_entropy_loss_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const ggml_tensor * opt0 = dst->src[2];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(opt0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(opt0));
    GGML_ASSERT(ggml_is_contiguous(dst));
    GGML_ASSERT(ggml_are_same_shape(src0, src1));
    GGML_ASSERT(ggml_are_same_shape(src0, dst));

    const int64_t ne00  = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    const float * opt0_d = (const float *) opt0->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    const dim3 blocks_dim(WARP_SIZE, 1, 1);
    const dim3 blocks_num(nrows, 1, 1);
    const int shmem = ne00*sizeof(float);

    cross_entropy_loss_back_f32<<<blocks_num, blocks_dim, shmem, stream>>>(src0_d, src1_d, opt0_d, dst_d, ne00);
}

#include "hip/hip_runtime.h"
#include "ggml-impl.h"
#include "opt-step-sgd.cuh"

#include <cstdint>

static __global__ void opt_step_sgd_f32(
    float * __restrict__ x, const float * __restrict__ g,
    const float * __restrict__ pars, const int64_t k) {

    const int64_t i = (int64_t) blockIdx.x*blockDim.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    x[i] = x[i] * (1.0f - pars[0] * pars[1]) - pars[0] * g[i];
}

static void opt_step_sgd_f32_cuda(
    float * x, const float * g, const float * __restrict__ pars, const int64_t k, hipStream_t stream) {

    const dim3 block_dims(CUDA_OPT_STEP_SGD_BLOCK_SIZE, 1, 1);
    const dim3 block_nums((k + CUDA_OPT_STEP_SGD_BLOCK_SIZE - 1) / CUDA_OPT_STEP_SGD_BLOCK_SIZE, 1, 1);
    opt_step_sgd_f32<<<block_nums, block_dims, 0, stream>>>(x, g, pars, k);
}

void ggml_cuda_opt_step_sgd(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0      = dst->src[0];
    const ggml_tensor * src0_grad = dst->src[1];
    const ggml_tensor * params    = dst->src[2];

    GGML_ASSERT(src0->type      == GGML_TYPE_F32);
    GGML_ASSERT(src0_grad->type == GGML_TYPE_F32);
    GGML_ASSERT(params->type    == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src0_grad));
    GGML_ASSERT(ggml_is_contiguous(params));
    GGML_ASSERT(ggml_are_same_shape(src0, src0_grad));
    GGML_ASSERT(ggml_nelements(params) == 2);

    float       * src0_d      = (float       *) src0->data;
    const float * src0_grad_d = (const float *) src0_grad->data;
    const float * params_d    = (const float *) params->data;

    hipStream_t stream = ctx.stream();

    const int64_t ne = ggml_nelements(src0);

    opt_step_sgd_f32_cuda(src0_d, src0_grad_d, params_d, ne, stream);
}

#include "hip/hip_runtime.h"
#include "acc.cuh"

static __global__ void acc_f32(const float * x, const float * y, float * dst, const int ne,
    const int ne10, const int ne11, const int ne12,
    const int nb1, const int nb2, int offset) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= ne) {
        return;
    }
    int src1_idx = i - offset;
    int oz = src1_idx / nb2;
    int oy = (src1_idx - (oz * nb2)) / nb1;
    int ox = src1_idx % nb1;
    if (src1_idx >= 0 && ox < ne10 && oy < ne11 && oz < ne12) {
        dst[i] = x[i] + y[ox + oy * ne10 + oz * ne10 * ne11];
    } else {
        dst[i] = x[i];
    }
}

static void acc_f32_cuda(const float * x, const float * y, float * dst, const int n_elements,
    const int ne10, const int ne11, const int ne12,
    const int nb1, const int nb2, const int offset, hipStream_t stream) {
    int num_blocks = (n_elements + CUDA_ACC_BLOCK_SIZE - 1) / CUDA_ACC_BLOCK_SIZE;
    acc_f32<<<num_blocks, CUDA_ACC_BLOCK_SIZE, 0, stream>>>(x, y, dst, n_elements, ne10, ne11, ne12, nb1, nb2, offset);
}

void ggml_cuda_op_acc(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->ne[3] == 1); // just 3D tensors supported

    int nb1 = dst->op_params[0] / 4; // 4 bytes of float32
    int nb2 = dst->op_params[1] / 4; // 4 bytes of float32
    // int nb3 = dst->op_params[2] / 4; // 4 bytes of float32 - unused
    int offset = dst->op_params[3] / 4; // offset in bytes

    acc_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), src1->ne[0], src1->ne[1], src1->ne[2], nb1, nb2, offset, stream);
}

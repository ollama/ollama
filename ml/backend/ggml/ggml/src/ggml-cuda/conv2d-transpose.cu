#include "hip/hip_runtime.h"
#include <algorithm>

#include "conv2d-transpose.cuh"
#include "ggml.h"

__global__ void conv2d_transpose_kernel(const float * __restrict__ input, const half * __restrict__ kernel,
                                        float * __restrict__ output, const int in_w, const int in_h, const int out_w,
                                        const int out_h, const int kernel_w, const int kernel_h, const int stride,
                                        const int c_in, const int c_out, const int batches) {
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    const int total_elements = out_w * out_h * c_out * batches;

    if (global_idx >= total_elements) {
        return;
    }

    const int out_x_idx = global_idx % out_w;
    const int out_y_idx = (global_idx / out_w) % out_h;
    const int c_idx     = (global_idx / (out_w * out_h)) % c_out;
    const int n_idx     = global_idx / (out_w * out_h * c_out);

    float accumulator = 0;
    // For each output idx, find the inputs that contribute to it by checking stride alignment and bounds

    for (int c_in_idx = 0; c_in_idx < c_in; c_in_idx++) {
        for (int kh = 0; kh < kernel_h; ++kh) {
            int in_y = out_y_idx - kh;
            if (in_y < 0 || in_y % stride) continue;
            in_y /= stride;
            if (in_y >= in_h) continue;

            for (int kw = 0; kw < kernel_w; ++kw) {
                int in_x = out_x_idx - kw;
                if (in_x < 0 || in_x % stride) continue;
                in_x /= stride;
                if (in_x >= in_w) continue;

                const int input_idx = (in_w * in_h * c_in) * n_idx + (in_w * in_h) * c_in_idx + (in_w) *in_y + in_x;
                const int kernel_idx =
                    (kernel_h * kernel_w * c_out) * c_in_idx + (kernel_h * kernel_w) * c_idx + (kernel_w) *kh + kw;

                float input_val = input[input_idx];
                half  kern_val  = kernel[kernel_idx];

                accumulator += input_val * (float) kern_val;
            }
        }
    }

    output[(out_w * out_h * c_out) * n_idx + (out_w * out_h) * c_idx + (out_w) *out_y_idx + out_x_idx] = accumulator;
}

//input is (W, H, C_in, N), Kernel is (W, H, C_out, C_in)
void ggml_cuda_conv_2d_transpose_p0(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * kernel = dst->src[0];
    const ggml_tensor * input  = dst->src[1];

    GGML_ASSERT(kernel->type == GGML_TYPE_F16 && input->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);

    const float * input_data  = (const float *) input->data;
    float *       output_data = (float *) dst->data;
    const half * kernel_data = (const half *) kernel->data;

    const int input_w      = input->ne[0];
    const int input_h      = input->ne[1];
    const int output_w     = dst->ne[0];
    const int output_h     = dst->ne[1];
    const int channels_in  = input->ne[2];
    const int channels_out = kernel->ne[2];
    const int kernel_w     = kernel->ne[0];
    const int kernel_h     = kernel->ne[1];
    const int stride       = dst->op_params[0];
    const int batches      = input->ne[3];

    GGML_ASSERT(channels_in == kernel->ne[3]);
    GGML_ASSERT(stride > 0);

    hipStream_t st = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(input));
    GGML_ASSERT(ggml_is_contiguous(kernel));
    GGML_ASSERT(ggml_is_contiguous(dst));

    const int total  = (output_w * output_h * channels_out * batches);
    const int blocks = (total + CUDA_CONV2D_TRANSPOSE_BLOCK_SIZE - 1) / CUDA_CONV2D_TRANSPOSE_BLOCK_SIZE;

    conv2d_transpose_kernel<<<blocks, CUDA_CONV2D_TRANSPOSE_BLOCK_SIZE, 0, st>>>(
        input_data, kernel_data, output_data, input_w, input_h, output_w, output_h, kernel_w, kernel_h, stride,
        channels_in, channels_out, batches);
}

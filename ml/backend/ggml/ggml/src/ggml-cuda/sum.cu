#if !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11700
#define USE_CUB
#endif // !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11700

#ifdef USE_CUB
#include <hipcub/hipcub.hpp>
using namespace hipcub;
#endif // USE_CUB

#include "sumrows.cuh"
#include "sum.cuh"

#include <cstdint>

void sum_f32_cuda(ggml_cuda_pool & pool, const float * x, float * dst, const int64_t ne, hipStream_t stream) {
#ifdef USE_CUB
    size_t tmp_size = 0;
    DeviceReduce::Sum(nullptr,       tmp_size, x, dst, ne, stream);
    ggml_cuda_pool_alloc<uint8_t> tmp_alloc(pool, tmp_size);
    DeviceReduce::Sum(tmp_alloc.ptr, tmp_size, x, dst, ne, stream);
#else
    // Use (inefficient) sum_rows implementation as a fallback.
    // For AMD there is rocPRIM which could be used as a drop-in replacement via hipcub but this would require C++11 -> C++14.
    sum_rows_f32_cuda(x, dst, ne, 1, stream);
    GGML_UNUSED(pool);
#endif // USE_CUB
}

void ggml_cuda_op_sum(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const float * src0_d = (const float *) src0->data;
    float * dst_d = (float *) dst->data;

    const int64_t ne = ggml_nelements(src0);

    ggml_cuda_pool & pool = ctx.pool();
    hipStream_t stream = ctx.stream();

    sum_f32_cuda(pool, src0_d, dst_d, ne, stream);
}

#include "hip/hip_runtime.h"
#include "softcap.cuh"

static __global__ void softcap_f32(const float * x, float * dst, const float scale, const float softcap, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = tanhf(scale * x[i]) * softcap;
}

static void softcap_f32_cuda(const float * x, float * dst, const float scale, const float softcap, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SOFTCAP_BLOCK_SIZE - 1) / CUDA_SOFTCAP_BLOCK_SIZE;
    softcap_f32<<<num_blocks, CUDA_SOFTCAP_BLOCK_SIZE, 0, stream>>>(x, dst, scale, softcap, k);
}

// fused GGML_OP_SCALE + GGML_UNARY_OP_TANH + GGML_OP_SCALE
void ggml_cuda_op_softcap(ggml_backend_cuda_context & ctx, ggml_tensor * dst, ggml_tensor * src) {
    const ggml_tensor * src0 = src->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float scale;
    float softcap;
    memcpy(&scale,   (float *) src->op_params + 0, sizeof(float));
    memcpy(&softcap, (float *) dst->op_params + 0, sizeof(float));

    softcap_f32_cuda(src0_d, dst_d, scale, softcap, ggml_nelements(src0), stream);
}

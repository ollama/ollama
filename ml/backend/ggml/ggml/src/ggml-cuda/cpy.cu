#include "hip/hip_runtime.h"
#include "cpy.cuh"

typedef void (*cpy_kernel_t)(const char * cx, char * cdst);

static __device__ void cpy_1_f32_f32(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    float * dsti = (float *) cdsti;

    *dsti = *xi;
}

static __device__ void cpy_1_f32_f16(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    half * dsti = (half *) cdsti;

    *dsti = __float2half(*xi);
}

static __device__ void cpy_1_f16_f16(const char * cxi, char * cdsti) {
    const half * xi = (const half *) cxi;
    half * dsti = (half *) cdsti;

    *dsti = *xi;
}

static __device__ void cpy_1_f16_f32(const char * cxi, char * cdsti) {
    const half * xi = (const half *) cxi;
    float * dsti = (float *) cdsti;

    *dsti = *xi;
}

template <cpy_kernel_t cpy_1>
static __global__ void cpy_f32_f16(const char * cx, char * cdst, const int ne,
                                   const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                                   const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                                   const int nb12, const int nb13) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    // determine indices i03/i13, i02/i12, i01/i11, i00/i10 as a function of index i of flattened tensor
    // then combine those indices with the corresponding byte offsets to get the total offsets
    const int64_t i03 = i/(ne00 * ne01 * ne02);
    const int64_t i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int64_t i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int64_t i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int64_t x_offset = i00*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int64_t i13 = i/(ne10 * ne11 * ne12);
    const int64_t i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int64_t i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int64_t i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int64_t dst_offset = i10*nb10 + i11*nb11 + i12*nb12 + i13 * nb13;

    cpy_1(cx + x_offset, cdst + dst_offset);
}

static __device__ void cpy_blck_f32_q8_0(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q8_0 * dsti = (block_q8_0 *) cdsti;

    float amax = 0.0f; // absolute max

    for (int j = 0; j < QK8_0; j++) {
        const float v = xi[j];
        amax = fmaxf(amax, fabsf(v));
    }

    const float d = amax / ((1 << 7) - 1);
    const float id = d ? 1.0f/d : 0.0f;

    dsti->d = d;

    for (int j = 0; j < QK8_0; ++j) {
        const float x0 = xi[j]*id;

        dsti->qs[j] = roundf(x0);
    }
}

static __device__ void cpy_blck_q8_0_f32(const char * cxi, char * cdsti) {
    const block_q8_0 * xi = (const block_q8_0 *) cxi;
    float * dsti = (float *) cdsti;

    const float d = (float)xi->d;

    for (int j = 0; j < QK8_0; j++) {
       dsti[j] = xi->qs[j] * d;
    }
}

static __device__ void cpy_blck_f32_q4_0(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q4_0 * dsti = (block_q4_0 *) cdsti;

    float amax = 0.0f;
    float vmax = 0.0f;

    for (int j = 0; j < QK4_0; ++j) {
        const float v = xi[j];
        if (amax < fabsf(v)) {
            amax = fabsf(v);
            vmax = v;
        }
    }

    const float d  = vmax / -8;
    const float id = d ? 1.0f/d : 0.0f;

    dsti->d = d;

    for (int j = 0; j < QK4_0/2; ++j) {
        const float x0 = xi[0       + j]*id;
        const float x1 = xi[QK4_0/2 + j]*id;

        const uint8_t xi0 = min(15, (int8_t)(x0 + 8.5f));
        const uint8_t xi1 = min(15, (int8_t)(x1 + 8.5f));

        dsti->qs[j]  = xi0;
        dsti->qs[j] |= xi1 << 4;
    }
}

static __device__ void cpy_blck_f32_q4_1(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q4_1 * dsti = (block_q4_1 *) cdsti;

    float vmin = FLT_MAX;
    float vmax = -FLT_MAX;

    for (int j = 0; j < QK4_1; ++j) {
        const float v = xi[j];

        if (v < vmin) vmin = v;
        if (v > vmax) vmax = v;
    }

    const float d  = (vmax - vmin) / ((1 << 4) - 1);
    const float id = d ? 1.0f/d : 0.0f;

    dsti->dm.x = d;
    dsti->dm.y = vmin;

    for (int j = 0; j < QK4_1/2; ++j) {
        const float x0 = (xi[0       + j] - vmin)*id;
        const float x1 = (xi[QK4_1/2 + j] - vmin)*id;

        const uint8_t xi0 = min(15, (int8_t)(x0 + 0.5f));
        const uint8_t xi1 = min(15, (int8_t)(x1 + 0.5f));

        dsti->qs[j]  = xi0;
        dsti->qs[j] |= xi1 << 4;
    }
}

static __device__ void cpy_blck_f32_q5_0(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q5_0 * dsti = (block_q5_0 *) cdsti;

    float amax = 0.0f;
    float vmax = 0.0f;

    for (int j = 0; j < QK5_0; ++j) {
        const float v = xi[j];
        if (amax < fabsf(v)) {
            amax = fabsf(v);
            vmax = v;
        }
    }

    const float d  = vmax / -16;
    const float id = d ? 1.0f/d : 0.0f;

    dsti->d = d;

    uint32_t qh = 0;
    for (int j = 0; j < QK5_0/2; ++j) {
        const float x0 = xi[0       + j]*id;
        const float x1 = xi[QK5_0/2 + j]*id;

        const uint8_t xi0 = min(31, (int8_t)(x0 + 16.5f));
        const uint8_t xi1 = min(31, (int8_t)(x1 + 16.5f));

        dsti->qs[j]  = (xi0 & 0xf) | ((xi1 & 0xf) << 4);
        qh |= ((xi0 & 0x10u) >> 4) << (j + 0);
        qh |= ((xi1 & 0x10u) >> 4) << (j + QK5_0/2);
    }
    memcpy(dsti->qh, &qh, sizeof(qh));
}

static __device__ void cpy_blck_f32_q5_1(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q5_1 * dsti = (block_q5_1 *) cdsti;

    float min = xi[0];
    float max = xi[0];

    for (int j = 1; j < QK5_1; ++j) {
        const float v = xi[j];
        min = v < min ? v : min;
        max = v > max ? v : max;
    }

    const float d  = (max - min) / 31;
    const float id = d ? 1.0f/d : 0.0f;

    dsti->dm.x = d;
    dsti->dm.y = min;

    uint32_t qh = 0;
    for (int j = 0; j < QK5_1/2; ++j) {
        const float x0 = (xi[0       + j] - min)*id;
        const float x1 = (xi[QK5_1/2 + j] - min)*id;

        const uint8_t xi0 = (uint8_t)(x0 + 0.5f);
        const uint8_t xi1 = (uint8_t)(x1 + 0.5f);

        dsti->qs[j]  = (xi0 & 0xf) | ((xi1 & 0xf) << 4);
        qh |= ((xi0 & 0x10u) >> 4) << (j + 0);
        qh |= ((xi1 & 0x10u) >> 4) << (j + QK5_1/2);
    }
    memcpy(dsti->qh, &qh, sizeof(qh));
}


static __device__ __forceinline__ int best_index_int8(int n, const int8_t * val, float x) {
    if (x <= val[0]) return 0;
    if (x >= val[n-1]) return n-1;
    int ml = 0, mu = n-1;
    while (mu-ml > 1) {
        int mav = (ml+mu)/2;
        if (x < val[mav]) mu = mav; else ml = mav;
    }
    return x - val[mu-1] < val[mu] - x ? mu-1 : mu;
}

static __device__ void cpy_blck_f32_iq4_nl(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_iq4_nl * dsti = (block_iq4_nl *) cdsti;

    float amax = 0.0f;
    float vmax = 0.0f;

    for (int j = 0; j < QK4_NL; ++j) {
        const float v = xi[j];
        if (amax < fabsf(v)) {
            amax = fabsf(v);
            vmax = v;
        }
    }

    float d = vmax / kvalues_iq4nl[0];
    const float id = d ? 1.0f/d : 0.0f;

    float sumqx = 0, sumq2 = 0;
    for (int j = 0; j < QK4_NL/2; ++j) {
        const float x0 = xi[0        + j]*id;
        const float x1 = xi[QK4_NL/2 + j]*id;
        const uint8_t xi0 = best_index_int8(16, kvalues_iq4nl, x0);
        const uint8_t xi1 = best_index_int8(16, kvalues_iq4nl, x1);
        dsti->qs[j] = xi0 | (xi1 << 4);
        const float v0 = kvalues_iq4nl[xi0];
        const float v1 = kvalues_iq4nl[xi1];
        const float w0 = xi[0        + j]*xi[0        + j];
        const float w1 = xi[QK4_NL/2 + j]*xi[QK4_NL/2 + j];
        sumqx += w0*v0*xi[j] + w1*v1*xi[QK4_NL/2 + j];
        sumq2 += w0*v0*v0 + w1*v1*v1;
    }

    dsti->d = sumq2 > 0 ? sumqx/sumq2 : d;
}

template <cpy_kernel_t cpy_blck, int qk>
static __global__ void cpy_f32_q(const char * cx, char * cdst, const int ne,
                                 const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                                 const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                                 const int nb12, const int nb13) {
    const int i = (blockDim.x*blockIdx.x + threadIdx.x)*qk;

    if (i >= ne) {
        return;
    }

    const int i03 = i/(ne00 * ne01 * ne02);
    const int i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int x_offset = i00*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int i13 = i/(ne10 * ne11 * ne12);
    const int i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int dst_offset = (i10/qk)*nb10 + i11*nb11 + i12*nb12 + i13*nb13;

    cpy_blck(cx + x_offset, cdst + dst_offset);
}

template <cpy_kernel_t cpy_blck, int qk>
static __global__ void cpy_q_f32(const char * cx, char * cdst, const int ne,
                                 const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                                 const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                                 const int nb12, const int nb13) {
    const int i = (blockDim.x*blockIdx.x + threadIdx.x)*qk;

    if (i >= ne) {
        return;
    }

    const int i03 = i/(ne00 * ne01 * ne02);
    const int i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int x_offset = (i00/qk)*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int i13 = i/(ne10 * ne11 * ne12);
    const int i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int dst_offset = i10*nb10 + i11*nb11 + i12*nb12 + i13*nb13;

    cpy_blck(cx + x_offset, cdst + dst_offset);
}

static void ggml_cpy_f16_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f16_f32><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f32><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_f16_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f16><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q8_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK8_0 == 0);
    const int num_blocks = ne / QK8_0;
    cpy_f32_q<cpy_blck_f32_q8_0, QK8_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_q8_0_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = ne;
    cpy_q_f32<cpy_blck_q8_0_f32, QK8_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q4_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK4_0 == 0);
    const int num_blocks = ne / QK4_0;
    cpy_f32_q<cpy_blck_f32_q4_0, QK4_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q4_1_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK4_1 == 0);
    const int num_blocks = ne / QK4_1;
    cpy_f32_q<cpy_blck_f32_q4_1, QK4_1><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q5_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK5_0 == 0);
    const int num_blocks = ne / QK5_0;
    cpy_f32_q<cpy_blck_f32_q5_0, QK5_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q5_1_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK5_1 == 0);
    const int num_blocks = ne / QK5_1;
    cpy_f32_q<cpy_blck_f32_q5_1, QK5_1><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_iq4_nl_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK4_NL == 0);
    const int num_blocks = ne / QK4_NL;
    cpy_f32_q<cpy_blck_f32_iq4_nl, QK4_NL><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f16_f16_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f16_f16><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

void ggml_cuda_cpy(ggml_backend_cuda_context & ctx, const ggml_tensor * src0, ggml_tensor * src1) {
    const int64_t ne = ggml_nelements(src0);
    GGML_ASSERT(ne == ggml_nelements(src1));

    GGML_ASSERT(ggml_nbytes(src0) <= INT_MAX);
    GGML_ASSERT(ggml_nbytes(src1) <= INT_MAX);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    //GGML_ASSERT(src0->ne[3] == 1);

    const int64_t nb00 = src0->nb[0];
    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];
    const int64_t nb03 = src0->nb[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    const int64_t ne12 = src1->ne[2];

    //GGML_ASSERT(src1->ne[3] == 1);

    const int64_t nb10 = src1->nb[0];
    const int64_t nb11 = src1->nb[1];
    const int64_t nb12 = src1->nb[2];
    const int64_t nb13 = src1->nb[3];

    hipStream_t main_stream = ctx.stream();

    char * src0_ddc = (char *) src0->data;
    char * src1_ddc = (char *) src1->data;

    if (src0->type == src1->type && ggml_is_contiguous(src0) && ggml_is_contiguous(src1)) {
        GGML_ASSERT(ggml_nbytes(src0) == ggml_nbytes(src1));
        CUDA_CHECK(hipMemcpyAsync(src1_ddc, src0_ddc, ggml_nbytes(src0), hipMemcpyDeviceToDevice, main_stream));
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_f32_f32_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_f32_f16_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q8_0) {
        ggml_cpy_f32_q8_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_Q8_0 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_q8_0_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_0) {
        ggml_cpy_f32_q4_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_1) {
        ggml_cpy_f32_q4_1_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_0) {
        ggml_cpy_f32_q5_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_IQ4_NL) {
        ggml_cpy_f32_iq4_nl_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_1) {
        ggml_cpy_f32_q5_1_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_f16_f16_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_f16_f32_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else {
        GGML_ABORT("%s: unsupported type combination (%s to %s)\n", __func__,
                ggml_type_name(src0->type), ggml_type_name(src1->type));
    }
}

void ggml_cuda_dup(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    ggml_cuda_cpy(ctx, src0, dst);
}

void* ggml_cuda_cpy_fn(const ggml_tensor * src0, ggml_tensor * src1) {
    if (src0->type == src1->type && ggml_is_contiguous(src0) && ggml_is_contiguous(src1)) {
        return nullptr;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_f32_f16<cpy_1_f32_f32>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        return (void*) cpy_f32_f16<cpy_1_f32_f16>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q8_0) {
        return (void*) cpy_f32_q<cpy_blck_f32_q8_0, QK8_0>;
    } else if (src0->type == GGML_TYPE_Q8_0 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_q_f32<cpy_blck_q8_0_f32, QK8_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_0) {
        return (void*) cpy_f32_q<cpy_blck_f32_q4_0, QK4_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_1) {
        return (void*) cpy_f32_q<cpy_blck_f32_q4_1, QK4_1>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_0) {
        return (void*) cpy_f32_q<cpy_blck_f32_q5_0, QK5_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_IQ4_NL) {
        return (void*) cpy_f32_q<cpy_blck_f32_iq4_nl, QK4_NL>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_1) {
        return (void*) cpy_f32_q<cpy_blck_f32_q5_1, QK5_1>;
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16) {
        return (void*) cpy_f32_f16<cpy_1_f32_f16>;
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_f32_f16<cpy_1_f16_f32>;
    } else {
        GGML_ABORT("%s: unsupported type combination (%s to %s)\n", __func__,
                ggml_type_name(src0->type), ggml_type_name(src1->type));
    }
}

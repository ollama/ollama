#include "hip/hip_runtime.h"
#include "cpy.cuh"
#include "dequantize.cuh"
#include "cpy-utils.cuh"
#if defined(GGML_USE_MUSA) && defined(GGML_MUSA_MUDNN_COPY)
#include "ggml-musa/mudnn.cuh"
#endif // GGML_USE_MUSA && GGML_MUSA_MUDNN_COPY

typedef void (*cpy_kernel_t)(const char * cx, char * cdst);

template <cpy_kernel_t cpy_1>
static __global__ void cpy_flt(const char * cx, char * cdst_direct, const int ne,
                               const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                               const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                               const int nb12, const int nb13, char ** cdst_indirect, int graph_cpynode_index) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    char * cdst = (cdst_indirect != nullptr) ? cdst_indirect[graph_cpynode_index]: cdst_direct;

    // determine indices i03/i13, i02/i12, i01/i11, i00/i10 as a function of index i of flattened tensor
    // then combine those indices with the corresponding byte offsets to get the total offsets
    const int64_t i03 = i/(ne00 * ne01 * ne02);
    const int64_t i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int64_t i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int64_t i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int64_t x_offset = i00*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int64_t i13 = i/(ne10 * ne11 * ne12);
    const int64_t i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int64_t i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int64_t i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int64_t dst_offset = i10*nb10 + i11*nb11 + i12*nb12 + i13 * nb13;

    cpy_1(cx + x_offset, cdst + dst_offset);
}

static __device__ void cpy_blck_q8_0_f32(const char * cxi, char * cdsti) {
    float * cdstf = (float *)(cdsti);

#pragma unroll
    for (int j = 0; j < QK8_0; j += 2) {
        dfloat2 dq;
        dequantize_q8_0(cxi, 0, j, dq);
        *(cdstf + j) = dq.x;
        *(cdstf + j + 1) = dq.y;
    }
}

template<dequantize_kernel_t dequant, int qk>
static __device__ void cpy_blck_q_f32(const char * cxi, char * cdsti) {
    float * cdstf = (float *)(cdsti);

#pragma unroll
    for (int j = 0; j < qk/2; j++) {
        dfloat2 dq;
        dequant(cxi, 0, j, dq);
        *(cdstf + j) = dq.x;
        *(cdstf + j + qk/2) = dq.y;
    }
}

template <cpy_kernel_t cpy_blck, int qk>
static __global__ void cpy_f32_q(const char * cx, char * cdst_direct, const int ne,
                                 const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                                 const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                                 const int nb12, const int nb13, char ** cdst_indirect, int graph_cpynode_index) {
    const int i = (blockDim.x*blockIdx.x + threadIdx.x)*qk;

    if (i >= ne) {
        return;
    }

    char * cdst = (cdst_indirect != nullptr) ? cdst_indirect[graph_cpynode_index]: cdst_direct;

    const int i03 = i/(ne00 * ne01 * ne02);
    const int i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int x_offset = i00*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int i13 = i/(ne10 * ne11 * ne12);
    const int i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int dst_offset = (i10/qk)*nb10 + i11*nb11 + i12*nb12 + i13*nb13;

    cpy_blck(cx + x_offset, cdst + dst_offset);
}

template <cpy_kernel_t cpy_blck, int qk>
static __global__ void cpy_q_f32(const char * cx, char * cdst_direct, const int ne,
                                 const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                                 const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                                 const int nb12, const int nb13, char ** cdst_indirect, int graph_cpynode_index) {
    const int i = (blockDim.x*blockIdx.x + threadIdx.x)*qk;

    if (i >= ne) {
        return;
    }

    char * cdst = (cdst_indirect != nullptr) ? cdst_indirect[graph_cpynode_index]: cdst_direct;

    const int i03 = i/(ne00 * ne01 * ne02);
    const int i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int x_offset = (i00/qk)*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int i13 = i/(ne10 * ne11 * ne12);
    const int i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int dst_offset = i10*nb10 + i11*nb11 + i12*nb12 + i13*nb13;

    cpy_blck(cx + x_offset, cdst + dst_offset);
}

// Copy destination pointers to GPU to be available when pointer indirection is in use

void ggml_cuda_cpy_dest_ptrs_copy(ggml_cuda_graph * cuda_graph, char ** host_dest_ptrs, const int host_dest_ptrs_size, hipStream_t stream) {
#if defined(GGML_CUDA_USE_GRAPHS) || defined(GGML_HIP_GRAPHS) || defined(GGML_MUSA_GRAPHS)
    if (cuda_graph->dest_ptrs_size < host_dest_ptrs_size) { // (re-)allocate GPU memory for destination pointers
        CUDA_CHECK(hipStreamSynchronize(stream));
        if (cuda_graph->dest_ptrs_d != nullptr) {
            CUDA_CHECK(hipFree(cuda_graph->dest_ptrs_d));
        }
        CUDA_CHECK(hipMalloc(&cuda_graph->dest_ptrs_d, host_dest_ptrs_size*sizeof(char *)));
        cuda_graph->dest_ptrs_size = host_dest_ptrs_size;
    }
    // copy destination pointers to GPU
    CUDA_CHECK(hipMemcpyAsync(cuda_graph->dest_ptrs_d, host_dest_ptrs, host_dest_ptrs_size*sizeof(char *), hipMemcpyHostToDevice, stream));
    cuda_graph->graph_cpynode_index = 0; // reset index
#else
    GGML_UNUSED(cuda_graph); GGML_UNUSED(host_dest_ptrs);
    GGML_UNUSED(host_dest_ptrs_size); GGML_UNUSED(stream);
#endif
}

template<typename src_t, typename dst_t>
static void ggml_cpy_flt_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_flt<cpy_1_flt<src_t, dst_t>><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_f32_q8_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    GGML_ASSERT(ne % QK8_0 == 0);
    const int num_blocks = ne / QK8_0;
    cpy_f32_q<cpy_blck_f32_q8_0, QK8_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_q8_0_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    const int num_blocks = ne;
    cpy_q_f32<cpy_blck_q8_0_f32, QK8_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_f32_q4_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    GGML_ASSERT(ne % QK4_0 == 0);
    const int num_blocks = ne / QK4_0;
    cpy_f32_q<cpy_blck_f32_q4_0, QK4_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_q4_0_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02,
    const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12,
    const int nb10, const int nb11, const int nb12, const int nb13,
    hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {
    const int num_blocks = ne;
    cpy_q_f32<cpy_blck_q_f32<dequantize_q4_0, QK4_0>, QK4_0><<<num_blocks, 1, 0, stream>>>(
        cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03,
         ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_f32_q4_1_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    GGML_ASSERT(ne % QK4_1 == 0);
    const int num_blocks = ne / QK4_1;
    cpy_f32_q<cpy_blck_f32_q4_1, QK4_1><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_q4_1_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02,
    const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12,
    const int nb10, const int nb11, const int nb12, const int nb13,
    hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {
    const int num_blocks = ne;
    cpy_q_f32<cpy_blck_q_f32<dequantize_q4_1, QK4_1>, QK4_1><<<num_blocks, 1, 0, stream>>>(
        cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03,
         ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_f32_q5_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    GGML_ASSERT(ne % QK5_0 == 0);
    const int num_blocks = ne / QK5_0;
    cpy_f32_q<cpy_blck_f32_q5_0, QK5_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_q5_0_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02,
    const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12,
    const int nb10, const int nb11, const int nb12, const int nb13,
    hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {
    const int num_blocks = ne;
    cpy_q_f32<cpy_blck_q_f32<dequantize_q5_0, QK5_0>, QK5_0><<<num_blocks, 1, 0, stream>>>(
        cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03,
        ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_f32_q5_1_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    GGML_ASSERT(ne % QK5_1 == 0);
    const int num_blocks = ne / QK5_1;
    cpy_f32_q<cpy_blck_f32_q5_1, QK5_1><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_q5_1_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02,
    const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12,
    const int nb10, const int nb11, const int nb12, const int nb13,
    hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {
    const int num_blocks = ne;
    cpy_q_f32<cpy_blck_q_f32<dequantize_q5_1, QK5_1>, QK5_1><<<num_blocks, 1, 0, stream>>>(
        cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03,
        ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

static void ggml_cpy_f32_iq4_nl_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    GGML_ASSERT(ne % QK4_NL == 0);
    const int num_blocks = ne / QK4_NL;
    cpy_f32_q<cpy_blck_f32_iq4_nl, QK4_NL><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, cdst_indirect, graph_cpynode_index++);
}

template <cpy_kernel_t cpy_1>
static __global__ void cpy_i32_i32(
    const char *cx, char *cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02, const int nb03,
    const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13,
    hipStream_t stream, char ** cdst_indirect, int & graph_cpynode_index) {

    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    const int64_t i03 = i / (ne00 * ne01 * ne02);
    const int64_t i02 = (i - i03 * ne00 * ne01 * ne02) / (ne00 * ne01);
    const int64_t i01 = (i - i03 * ne00 * ne01 * ne02 - i02 * ne01 * ne00) / ne00;
    const int64_t i00 = i - i03 * ne00 * ne01 * ne02 - i02 * ne01 * ne00 - i01 * ne00;
    const int64_t x_offset = i00 * nb00 + i01 * nb01 + i02 * nb02 + i03 * nb03;

    const int64_t i13 = i / (ne10 * ne11 * ne12);
    const int64_t i12 = (i - i13 * ne10 * ne11 * ne12) / (ne10 * ne11);
    const int64_t i11 = (i - i13 * ne10 * ne11 * ne12 - i12 * ne10 * ne11) / ne10;
    const int64_t i10 = i - i13 * ne10 * ne11 * ne12 - i12 * ne10 * ne11 - i11 * ne10;
    const int64_t dst_offset = i10 * nb10 + i11 * nb11 + i12 * nb12 + i13 * nb13;

    char * cdst_ptr = (cdst_indirect != nullptr) ? cdst_indirect[graph_cpynode_index] : cdst;
    cpy_1(cx + x_offset, cdst_ptr + dst_offset);
}


static void ggml_cpy_i32_i32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02, const int nb03,
    const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13,
    hipStream_t stream, char ** cdst_indirect, int graph_cpynode_index) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_i32_i32<cpy_1_i32_i32><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, stream, cdst_indirect, graph_cpynode_index);
}

void ggml_cuda_cpy(ggml_backend_cuda_context & ctx, const ggml_tensor * src0, ggml_tensor * src1, bool disable_indirection_for_this_node) {
    const int64_t ne = ggml_nelements(src0);
    GGML_ASSERT(ne == ggml_nelements(src1));

    GGML_ASSERT(ggml_nbytes(src0) <= INT_MAX);
    GGML_ASSERT(ggml_nbytes(src1) <= INT_MAX);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    //GGML_ASSERT(src0->ne[3] == 1);

    const int64_t nb00 = src0->nb[0];
    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];
    const int64_t nb03 = src0->nb[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    const int64_t ne12 = src1->ne[2];

    //GGML_ASSERT(src1->ne[3] == 1);

    const int64_t nb10 = src1->nb[0];
    const int64_t nb11 = src1->nb[1];
    const int64_t nb12 = src1->nb[2];
    const int64_t nb13 = src1->nb[3];

    hipStream_t main_stream = ctx.stream();

    char * src0_ddc = (char *) src0->data;
    char * src1_ddc = (char *) src1->data;

    char ** dest_ptrs_d = nullptr;
    int graph_cpynode_index = -1;
#if defined(GGML_CUDA_USE_GRAPHS) || defined(GGML_HIP_GRAPHS) || defined(GGML_MUSA_GRAPHS)
    if(ctx.cuda_graph->use_cpy_indirection && !disable_indirection_for_this_node) {
        dest_ptrs_d = ctx.cuda_graph->dest_ptrs_d;
        graph_cpynode_index = ctx.cuda_graph->graph_cpynode_index;
    }
#else
    GGML_UNUSED(disable_indirection_for_this_node);
#endif
    if (src0->type == src1->type && ggml_is_contiguous(src0) && ggml_is_contiguous(src1)) {
        GGML_ASSERT(ggml_nbytes(src0) == ggml_nbytes(src1));
#if defined(GGML_USE_MUSA) && defined(GGML_MUSA_MUDNN_COPY)
        if (src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16) {
            CUDA_CHECK(mudnnMemcpyAsync(ctx, src1, src0));
        } else
#endif // GGML_USE_MUSA && GGML_MUSA_MUDNN_COPY
        {
            CUDA_CHECK(hipMemcpyAsync(src1_ddc, src0_ddc, ggml_nbytes(src0), hipMemcpyDeviceToDevice, main_stream));
        }
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_flt_cuda<float, float> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_BF16) {
        ggml_cpy_flt_cuda<float, hip_bfloat16> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_flt_cuda<float, half> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q8_0) {
        ggml_cpy_f32_q8_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_Q8_0 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_q8_0_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_0) {
        ggml_cpy_f32_q4_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_Q4_0 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_q4_0_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02,
            nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_1) {
        ggml_cpy_f32_q4_1_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_Q4_1 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_q4_1_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02,
            nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_0) {
        ggml_cpy_f32_q5_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_Q5_0 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_q5_0_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02,
            nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_IQ4_NL) {
        ggml_cpy_f32_iq4_nl_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_1) {
        ggml_cpy_f32_q5_1_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_Q5_1 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_q5_1_f32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_flt_cuda<half, half> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_BF16) {
        ggml_cpy_flt_cuda<half, hip_bfloat16> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_flt_cuda<half, float> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_I32 && src1->type == GGML_TYPE_I32) {
        ggml_cpy_i32_i32_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_BF16 && src1->type == GGML_TYPE_BF16) {
        ggml_cpy_flt_cuda<hip_bfloat16, hip_bfloat16> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_BF16 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_flt_cuda<hip_bfloat16, half> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else if (src0->type == GGML_TYPE_BF16 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_flt_cuda<hip_bfloat16, float> (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream, dest_ptrs_d, graph_cpynode_index);
    } else {
        GGML_ABORT("%s: unsupported type combination (%s to %s)\n", __func__,
                ggml_type_name(src0->type), ggml_type_name(src1->type));
    }
#if defined(GGML_CUDA_USE_GRAPHS) || defined(GGML_HIP_GRAPHS) || defined(GGML_MUSA_GRAPHS)
    if(ctx.cuda_graph->use_cpy_indirection && !disable_indirection_for_this_node) {
        ctx.cuda_graph->graph_cpynode_index = graph_cpynode_index;
    }
#else
    GGML_UNUSED(disable_indirection_for_this_node);
#endif

}

void ggml_cuda_dup(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    bool disable_indirection = true;
    ggml_cuda_cpy(ctx, src0, dst, disable_indirection);
}

void* ggml_cuda_cpy_fn(const ggml_tensor * src0, ggml_tensor * src1) {
    if (src0->type == src1->type && ggml_is_contiguous(src0) && ggml_is_contiguous(src1)) {
        return nullptr;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_flt<cpy_1_flt<float, float>>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_BF16) {
        return (void*) cpy_flt<cpy_1_flt<float, hip_bfloat16>>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        return (void*) cpy_flt<cpy_1_flt<float, half>>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q8_0) {
        return (void*) cpy_f32_q<cpy_blck_f32_q8_0, QK8_0>;
    } else if (src0->type == GGML_TYPE_Q8_0 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_q_f32<cpy_blck_q8_0_f32, QK8_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_0) {
        return (void*) cpy_f32_q<cpy_blck_f32_q4_0, QK4_0>;
    } else if (src0->type == GGML_TYPE_Q4_0 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_q_f32<cpy_blck_q_f32<dequantize_q4_0, QK4_0>, QK4_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_1) {
        return (void*) cpy_f32_q<cpy_blck_f32_q4_1, QK4_1>;
    } else if (src0->type == GGML_TYPE_Q4_1 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_q_f32<cpy_blck_q_f32<dequantize_q4_1, QK4_1>, QK4_1>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_0) {
        return (void*) cpy_f32_q<cpy_blck_f32_q5_0, QK5_0>;
    } else if (src0->type == GGML_TYPE_Q5_0 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_q_f32<cpy_blck_q_f32<dequantize_q5_0, QK5_0>, QK5_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_IQ4_NL) {
        return (void*) cpy_f32_q<cpy_blck_f32_iq4_nl, QK4_NL>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_1) {
        return (void*) cpy_f32_q<cpy_blck_f32_q5_1, QK5_1>;
    } else if (src0->type == GGML_TYPE_Q5_1 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_q_f32<cpy_blck_q_f32<dequantize_q5_1, QK5_1>, QK5_1>;
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16) {
        return (void*) cpy_flt<cpy_1_flt<half, half>>;
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_BF16) {
        return (void*) cpy_flt<cpy_1_flt<half, hip_bfloat16>>;
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_flt<cpy_1_flt<half, float>>;
    } else if (src0->type == GGML_TYPE_BF16 && src1->type == GGML_TYPE_F16) {
        return (void*) cpy_flt<cpy_1_flt<hip_bfloat16, half>>;
    } else if (src0->type == GGML_TYPE_BF16 && src1->type == GGML_TYPE_BF16) {
        return (void*) cpy_flt<cpy_1_flt<hip_bfloat16, hip_bfloat16>>;
    } else if (src0->type == GGML_TYPE_BF16 && src1->type == GGML_TYPE_F32) {
        return (void*) cpy_flt<cpy_1_flt<hip_bfloat16, float>>;
    } else {
        GGML_ABORT("%s: unsupported type combination (%s to %s)\n", __func__,
                ggml_type_name(src0->type), ggml_type_name(src1->type));
    }
}

#include "hip/hip_runtime.h"
#include "common.cuh"
#include "ggml.h"
#include "softmax.cuh"
#include <cstdint>
#include <utility>

template <typename T>
static __device__ __forceinline__ float t2f32(T val) {
    return (float) val;
}

template <>
__device__ float __forceinline__ t2f32<half>(half val) {
    return __half2float(val);
}

struct soft_max_params {

    int64_t nheads;
    uint32_t n_head_log2;
    int64_t ncols;
    int64_t nrows_x;
    int64_t nrows_y;
    int64_t ne00;
    int64_t ne01;
    int64_t ne02;
    int64_t ne03;
    int64_t nb11;
    int64_t nb12;
    int64_t nb13;

    int64_t ne12;
    int64_t ne13;
    float scale;
    float max_bias;
    float m0;
    float m1;
};

// When ncols_template == 0 the bounds for the loops in this function are not known and can't be unrolled.
// As we want to keep pragma unroll for all other cases we supress the clang transformation warning here.
#ifdef __clang__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wpass-failed"
#endif // __clang__
template <bool use_shared, int ncols_template, int block_size_template, typename T>
static __global__ void soft_max_f32(
        const float * x, const T * mask, float * dst, const soft_max_params p) {
    const int ncols = ncols_template == 0 ? p.ncols : ncols_template;

    const int tid  = threadIdx.x;

    const int64_t i03 = blockIdx.z;
    const int64_t i02 = blockIdx.y;
    const int64_t i01 = blockIdx.x;

    //TODO: noncontigous inputs/outputs
    const int rowx = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

    const int64_t i11 = i01;
    const int64_t i12 = i02 % p.ne12;
    const int64_t i13 = i03 % p.ne13;

    x    += int64_t(rowx)*ncols;
    mask += (i11*p.nb11 + i12*p.nb12 + i13*p.nb13) / sizeof(T) * (mask != nullptr);
    dst  += int64_t(rowx)*ncols;

    const int block_size = block_size_template == 0 ? blockDim.x : block_size_template;

    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    const float slope = get_alibi_slope(p.max_bias, i02, p.n_head_log2, p.m0, p.m1);

    extern __shared__ float data_soft_max_f32[];
    float * buf_iw = data_soft_max_f32; // shared memory buffer for inter-warp communication
    // shared memory buffer to cache values between iterations:
    float * vals = use_shared ? buf_iw + WARP_SIZE : dst;

    float max_val = -INFINITY;

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            break;
        }

        const float val = x[col]*p.scale + (mask ? slope*t2f32(mask[col]) : 0.0f);

        vals[col] = val;
        max_val = max(max_val, val);
    }

    // find the max value in the block
    max_val = warp_reduce_max(max_val);
    if (block_size > WARP_SIZE) {
        if (warp_id == 0) {
            buf_iw[lane_id] = -INFINITY;
        }
        __syncthreads();

        if (lane_id == 0) {
            buf_iw[warp_id] = max_val;
        }
        __syncthreads();

        max_val = buf_iw[lane_id];
        max_val = warp_reduce_max(max_val);
    }

    float tmp = 0.0f; // partial sum

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            break;
        }

        const float val = expf(vals[col] - max_val);
        tmp += val;
        vals[col] = val;
    }

    // find the sum of exps in the block
    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __syncthreads();
        if (warp_id == 0) {
            buf_iw[lane_id] = 0.0f;
        }
        __syncthreads();

        if (lane_id == 0) {
            buf_iw[warp_id] = tmp;
        }
        __syncthreads();

        tmp = buf_iw[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float inv_sum = 1.0f / tmp;

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            return;
        }

        dst[col] = vals[col] * inv_sum;
    }
}
#ifdef __clang__
#pragma clang diagnostic pop
#endif // __clang__

static __global__ void soft_max_back_f32(
        const float * grad, const float * dstf, float * dst, const int ncols, const float scale) {
    const int tid  = threadIdx.x;
    const int rowx = blockIdx.x;

    grad += int64_t(rowx)*ncols;
    dstf += int64_t(rowx)*ncols;
    dst  += int64_t(rowx)*ncols;

    float dgf_dot = 0.0f; // dot product of dst from forward pass and gradients

    for (int col = tid; col < ncols; col += WARP_SIZE) {
        dgf_dot += dstf[col]*grad[col];
    }

    dgf_dot = warp_reduce_sum(dgf_dot);

    for (int col = tid; col < ncols; col += WARP_SIZE) {
        dst[col] = scale * (grad[col] - dgf_dot) * dstf[col];
    }
}

template<int... Ns, typename T>
static void launch_soft_max_kernels(const float * x, const T * mask, float * dst,
                             const soft_max_params & p, hipStream_t stream, dim3 block_dims, dim3 block_nums, size_t nbytes_shared)
{
    const int id       = ggml_cuda_get_device();
    const size_t smpbo = ggml_cuda_info().devices[id].smpbo;

    auto launch_kernel = [=](auto I) -> bool {
        constexpr int ncols = decltype(I)::value;
        constexpr int block = (ncols > 1024 ? 1024 : ncols);

        if (p.ncols == ncols) {
            CUDA_SET_SHARED_MEMORY_LIMIT((soft_max_f32<true, ncols, block, T>), smpbo);
            soft_max_f32<true, ncols, block><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, mask, dst, p);
            return true;
        }
        return false;
    };

    // unary fold over launch_kernel
    if ((launch_kernel(std::integral_constant<int, Ns>{}) || ...)) {
        return;
    }

    //default case
    CUDA_SET_SHARED_MEMORY_LIMIT((soft_max_f32<true, 0, 0, T>), smpbo);
    soft_max_f32<true, 0, 0><<<block_nums, block_dims, nbytes_shared, stream>>>(x, mask, dst, p);
}


template<typename T>
static void soft_max_f32_cuda(const float * x, const T * mask, float * dst, const soft_max_params & params, hipStream_t stream) {
    int nth = WARP_SIZE;
    const int64_t ncols_x = params.ncols;

    while (nth < ncols_x && nth < CUDA_SOFT_MAX_BLOCK_SIZE) nth *= 2;
    const dim3 block_dims(nth,     1, 1);
    const dim3 block_nums(params.ne01, params.ne02, params.ne03);
    const size_t nbytes_shared = (GGML_PAD(ncols_x, WARP_SIZE) + WARP_SIZE)*sizeof(float);
    static_assert(CUDA_SOFT_MAX_BLOCK_SIZE == 1024, "These values need to be adjusted.");


    const int id       = ggml_cuda_get_device();
    const size_t smpbo = ggml_cuda_info().devices[id].smpbo;


    if (nbytes_shared <= smpbo) {
        launch_soft_max_kernels<32, 64, 128, 256, 512, 1024, 2048, 4096>(x, mask, dst, params, stream, block_dims, block_nums, nbytes_shared);
    } else {
        const size_t nbytes_shared_low = WARP_SIZE*sizeof(float);
        soft_max_f32<false, 0, 0><<<block_nums, block_dims, nbytes_shared_low, stream>>>(x, mask, dst, params);
    }
}

static void soft_max_back_f32_cuda(
        const float * grad, const float * dstf, float * dst,
        const int ncols, const int nrows, const float scale, hipStream_t stream) {
    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums(nrows,     1, 1);

    soft_max_back_f32<<<block_nums, block_dims, 0, stream>>>(grad, dstf, dst, ncols, scale);
}

void ggml_cuda_op_soft_max(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    const float * src0_d = (const float *) src0->data;
    const void  * src1_d = src1 ? (const void *) src1->data : nullptr;
    float       *  dst_d = (float *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(!src1 || src1->type == GGML_TYPE_F16 || src1->type == GGML_TYPE_F32); // src1 contains mask and it is optional

    const int64_t nrows_x = ggml_nrows(src0);
    const int64_t nrows_y = src0->ne[1];

    const int64_t ne00 = src0->ne[0];

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (const float *) dst->op_params + 0, sizeof(float));
    memcpy(&max_bias, (const float *) dst->op_params + 1, sizeof(float));

    const bool use_f16 = (src1 && src1->type == GGML_TYPE_F16);

    const int64_t nb11 = src1 ? src1->nb[1] : 1;
    const int64_t nb12 = src1 ? src1->nb[2] : 1;
    const int64_t nb13 = src1 ? src1->nb[3] : 1;

    const int64_t ne12 = src1 ? src1->ne[2] : 1;
    const int64_t ne13 = src1 ? src1->ne[3] : 1;

    const uint32_t n_head      = src0->ne[2];
    const uint32_t n_head_log2 = 1u << (uint32_t) floorf(log2f((float) n_head));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);


    soft_max_params params = {};
    params.nheads = src0->ne[2];
    params.n_head_log2 = n_head_log2;
    params.ncols = ne00;
    params.nrows_x = nrows_x;
    params.nrows_y = nrows_y;
    params.ne00 = src0->ne[0];
    params.ne01 = src0->ne[1];
    params.ne02 = src0->ne[2];
    params.ne03 = src0->ne[3];
    params.nb11 = nb11;
    params.nb12 = nb12;
    params.nb13 = nb13;
    params.ne12 = ne12;
    params.ne13 = ne13;
    params.scale = scale;
    params.max_bias = max_bias;
    params.m0 = m0;
    params.m1 = m1;

    if (use_f16) {
        soft_max_f32_cuda(src0_d, (const half  *) src1_d, dst_d, params, stream);
    } else {
        soft_max_f32_cuda(src0_d, (const float *) src1_d, dst_d, params, stream);
    }
}

void ggml_cuda_op_soft_max_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // grad
    const ggml_tensor * src1 = dst->src[1]; // forward pass output

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (const float *) dst->op_params + 0, sizeof(float));
    memcpy(&max_bias, (const float *) dst->op_params + 1, sizeof(float));

    GGML_ASSERT(max_bias == 0.0f);

    soft_max_back_f32_cuda(src0_d, src1_d, dst_d, ncols, nrows, scale, stream);
}

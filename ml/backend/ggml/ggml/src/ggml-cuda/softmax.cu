#include "hip/hip_runtime.h"
#include "common.cuh"
#include "softmax.cuh"

template <typename T>
static __device__ __forceinline__ float t2f32(T val) {
    return (float) val;
}

template <>
__device__ float __forceinline__ t2f32<half>(half val) {
    return __half2float(val);
}

template <bool vals_smem, int ncols_template, int block_size_template, typename T>
static __global__ void soft_max_f32(const float * x, const T * mask, float * dst, const int ncols_par, const int nrows_y, const float scale, const float max_bias, const float m0, const float m1, uint32_t n_head_log2) {
    const int ncols = ncols_template == 0 ? ncols_par : ncols_template;

    const int tid  = threadIdx.x;
    const int rowx = blockIdx.x;
    const int rowy = rowx % nrows_y; // broadcast the mask in the row dimension

    const int block_size = block_size_template == 0 ? blockDim.x : block_size_template;

    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    const float slope = get_alibi_slope(max_bias, rowx/nrows_y, n_head_log2, m0, m1);

    extern __shared__ float data_soft_max_f32[];
    float * buf_iw = data_soft_max_f32; // shared memory buffer for inter-warp communication
    // shared memory buffer to cache values between iterations:
    float * vals = vals_smem ? buf_iw + WARP_SIZE : dst + (int64_t)rowx*ncols;

    float max_val = -INFINITY;

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            break;
        }

        const int64_t ix = (int64_t)rowx*ncols + col;
        const int64_t iy = (int64_t)rowy*ncols + col;

        const float val = x[ix]*scale + (mask ? slope*t2f32(mask[iy]) : 0.0f);

        vals[col] = val;
        max_val = max(max_val, val);
    }

    // find the max value in the block
    max_val = warp_reduce_max(max_val);
    if (block_size > WARP_SIZE) {
        if (warp_id == 0) {
            buf_iw[lane_id] = -INFINITY;
        }
        __syncthreads();

        if (lane_id == 0) {
            buf_iw[warp_id] = max_val;
        }
        __syncthreads();

        max_val = buf_iw[lane_id];
        max_val = warp_reduce_max(max_val);
    }

    float tmp = 0.0f; // partial sum

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            break;
        }

        const float val = expf(vals[col] - max_val);
        tmp += val;
        vals[col] = val;
    }

    // find the sum of exps in the block
    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __syncthreads();
        if (warp_id == 0) {
            buf_iw[lane_id] = 0.0f;
        }
        __syncthreads();

        if (lane_id == 0) {
            buf_iw[warp_id] = tmp;
        }
        __syncthreads();

        tmp = buf_iw[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float inv_sum = 1.0f / tmp;

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            return;
        }

        const int64_t idst = (int64_t)rowx*ncols + col;
        dst[idst] = vals[col] * inv_sum;
    }
}

template<typename T>
static void soft_max_f32_cuda(const float * x, const T * mask, float * dst, const int ncols_x, const int nrows_x, const int nrows_y, const float scale, const float max_bias, hipStream_t stream) {
    int nth = WARP_SIZE;
    while (nth < ncols_x && nth < CUDA_SOFT_MAX_BLOCK_SIZE) nth *= 2;
    const dim3 block_dims(nth,     1, 1);
    const dim3 block_nums(nrows_x, 1, 1);
    const size_t shmem = (GGML_PAD(ncols_x, WARP_SIZE) + WARP_SIZE)*sizeof(float);
    static_assert(CUDA_SOFT_MAX_BLOCK_SIZE == 1024, "These values need to be adjusted.");

    const uint32_t n_head      = nrows_x/nrows_y;
    const uint32_t n_head_log2 = 1u << (uint32_t) floorf(log2f((float) n_head));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);

    // FIXME: this limit could be raised by ~2-4x on Ampere or newer
    if (shmem < ggml_cuda_info().devices[ggml_cuda_get_device()].smpb) {
        switch (ncols_x) {
            case 32:
                soft_max_f32<true, 32, 32><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 64:
                soft_max_f32<true, 64, 64><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 128:
                soft_max_f32<true, 128, 128><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 256:
                soft_max_f32<true, 256, 256><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 512:
                soft_max_f32<true, 512, 512><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 1024:
                soft_max_f32<true, 1024, 1024><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 2048:
                soft_max_f32<true, 2048, 1024><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 4096:
                soft_max_f32<true, 4096, 1024><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            default:
                soft_max_f32<true, 0, 0><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
        }
    } else {
        const size_t shmem_low = WARP_SIZE*sizeof(float);
        soft_max_f32<false, 0, 0><<<block_nums, block_dims, shmem_low, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
    }
}

void ggml_cuda_op_soft_max(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    const float * src0_d = (const float *)src0->data;
    const void  * src1_d = src1 ? (const void *)src1->data : nullptr;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(!src1 || src1->type == GGML_TYPE_F16 || src1->type == GGML_TYPE_F32); // src1 contains mask and it is optional

    const int64_t ne00    = src0->ne[0];
    const int64_t nrows_x = ggml_nrows(src0);
    const int64_t nrows_y = src0->ne[1];

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (float *) dst->op_params + 0, sizeof(float));
    memcpy(&max_bias, (float *) dst->op_params + 1, sizeof(float));

    const bool use_f16 = (src1 && src1->type == GGML_TYPE_F16);

    if (use_f16) {
        const half * src1_dd = (const half *)src1_d;

        soft_max_f32_cuda(src0_d, src1_dd, dst_d, ne00, nrows_x, nrows_y, scale, max_bias, stream);
    } else {
        const float * src1_dd = (const float *)src1_d;

        soft_max_f32_cuda(src0_d, src1_dd, dst_d, ne00, nrows_x, nrows_y, scale, max_bias, stream);
    }
}

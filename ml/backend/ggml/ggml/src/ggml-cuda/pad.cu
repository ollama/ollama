#include "hip/hip_runtime.h"
#include "pad.cuh"

static __global__ void pad_f32(const float * x, float * dst, const int ne0, const int ne00, const int ne01, const int ne02, const int ne03) {
    // blockIdx.z: idx of ne2*ne3, aka ne02*ne03
    // blockIdx.y: idx of ne1
    // blockIDx.x: idx of ne0 / BLOCK_SIZE
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    // operation
    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;
    if (nidx < ne00 && blockIdx.y < ne01 && blockIdx.z < ne02*ne03) {
        int offset_src =
            nidx +
            blockIdx.y * ne00 +
            blockIdx.z * ne00 * ne01;
        dst[offset_dst] = x[offset_src];
    } else {
        dst[offset_dst] = 0.0f;
    }
}

static void pad_f32_cuda(const float * x, float * dst,
    const int ne00, const int ne01, const int ne02, const int ne03,
    const int ne0, const int ne1, const int ne2, const int ne3, hipStream_t stream) {
    int num_blocks = (ne0 + CUDA_PAD_BLOCK_SIZE - 1) / CUDA_PAD_BLOCK_SIZE;
    dim3 gridDim(num_blocks, ne1, ne2*ne3);
    pad_f32<<<gridDim, CUDA_PAD_BLOCK_SIZE, 0, stream>>>(x, dst, ne0, ne00, ne01, ne02, ne03);
}

void ggml_cuda_op_pad(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(src0->ne[3] == 1 && dst->ne[3] == 1); // just 3D tensors

    pad_f32_cuda(src0_d, dst_d,
        src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3],
        dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3], stream);
}

static __global__ void unpad_f32(const float * x, float * dst, const int ne0, const int ne00, const int ne01, const int ne02, const int ne03) {
    // blockIdx.z: idx of ne2*ne3, aka ne02*ne03
    // blockIdx.y: idx of ne1
    // blockIDx.x: idx of ne0 / BLOCK_SIZE
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    // operation
    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;
    if (nidx < ne00 && blockIdx.y < ne01 && blockIdx.z < ne02*ne03) {
        int offset_src =
            nidx +
            blockIdx.y * ne00 +
            blockIdx.z * ne00 * ne01;
        dst[offset_dst] = x[offset_src];
    }
}

static void unpad_f32_cuda(const float * x, float * dst,
    const int ne00, const int ne01, const int ne02, const int ne03,
    const int ne0, const int ne1, const int ne2, const int ne3, hipStream_t stream) {
    int num_blocks = (ne0 + CUDA_PAD_BLOCK_SIZE - 1) / CUDA_PAD_BLOCK_SIZE;
    dim3 gridDim(num_blocks, ne1, ne2*ne3);
    unpad_f32<<<gridDim, CUDA_PAD_BLOCK_SIZE, 0, stream>>>(x, dst, ne0, ne00, ne01, ne02, ne03);
}

void ggml_cuda_op_unpad(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(src0->ne[3] == 1 && dst->ne[3] == 1); // just 3D tensors

    unpad_f32_cuda(src0_d, dst_d,
        src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3],
        dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3], stream);
}

#include "device.cuh"
#include <mutex>

static hipDeviceProp_t  deviceProp;
static std::once_flag  devicePropInitFlag;

static void InitDeviceProp() {
    int dev = 0;
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) {
        // fallback: assume device 0
        dev = 0;
        hipGetDevice(&dev);
    }
    // query and stash all of its properties
    hipGetDeviceProperties(&deviceProp, dev);
}

const hipDeviceProp_t & getCachedDeviceProperties() {
    std::call_once(devicePropInitFlag, InitDeviceProp);
    return deviceProp;
}

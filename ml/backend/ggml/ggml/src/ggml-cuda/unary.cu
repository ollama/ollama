#include "hip/hip_runtime.h"
#include "unary.cuh"

static __device__ __forceinline__ float op_abs(float x) {
    return fabsf(x);
}

static __device__ __forceinline__ float op_sgn(float x) {
    return (x > 0.f ? 1.f : ((x < 0.f ? -1.f : 0.f)));
}

static __device__ __forceinline__ float op_neg(float x) {
    return -x;
}

static __device__ __forceinline__ float op_step(float x) {
    return x > 0.0f;
}

static __device__ __forceinline__ float op_gelu(float x) {
    const float GELU_COEF_A    = 0.044715f;
    const float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;

    return 0.5f*x*(1.0f + tanhf(SQRT_2_OVER_PI*x*(1.0f + GELU_COEF_A*x*x)));
}

static __device__ __forceinline__ float op_gelu_erf(float x) {
    const float SQRT_2_INV = 0.70710678118654752440084436210484f;

    return 0.5f*x*(1.0f + erff(x*SQRT_2_INV));
}

static __device__ __forceinline__ float op_gelu_quick(float x) {
    const float GELU_QUICK_COEF = -1.702f;

    return x * (1.0f / (1.0f + expf(GELU_QUICK_COEF * x)));
}

static __device__ __forceinline__ float op_silu(float x) {
    return x / (1.0f + expf(-x));
}

static __device__ __forceinline__ float op_tanh(float x) {
    return tanhf(x);
}

static __device__ __forceinline__ float op_relu(float x) {
    return fmaxf(x, 0);
}

static __device__ __forceinline__ float op_sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

static __device__ __forceinline__ float op_hardsigmoid(float x) {
    return fminf(1.0f, fmaxf(0.0f, (x + 3.0f) / 6.0f));
}

static __device__ __forceinline__ float op_hardswish(float x) {
    return x * fminf(1.0f, fmaxf(0.0f, (x + 3.0f) / 6.0f));
}

static __device__ __forceinline__ float op_exp(float x) {
    return expf(x);
}

static __device__ __forceinline__ float op_sqr(float x) {
    return x * x;
}

static __device__ __forceinline__ float op_sqrt(float x) {
    return sqrtf(x);
}

static __device__ __forceinline__ float op_sin(float x) {
    return sinf(x);
}

static __device__ __forceinline__ float op_cos(float x) {
    return cosf(x);
}

static __device__ __forceinline__ float op_log(float x) {
    return logf(x);
}

static __device__ __forceinline__ float op_elu(float x) {
    return (x > 0.f) ? x : expm1f(x);
}

template <float (*op)(float), typename T>
static __global__ void unary_op_kernel(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op((float)x[i]);
}

template <float (*op)(float), typename T>
static void unary_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    unary_op_kernel<op><<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <float (*op)(float)>
void ggml_cuda_op_unary(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        unary_cuda<op>((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        unary_cuda<op>((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_abs(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_abs>(ctx, dst);
}

void ggml_cuda_op_sgn(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sgn>(ctx, dst);
}

void ggml_cuda_op_neg(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_neg>(ctx, dst);
}

void ggml_cuda_op_step(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_step>(ctx, dst);
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_gelu>(ctx, dst);
}

void ggml_cuda_op_gelu_erf(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_gelu_erf>(ctx, dst);
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_gelu_quick>(ctx, dst);
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_silu>(ctx, dst);
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_tanh>(ctx, dst);
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_relu>(ctx, dst);
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sigmoid>(ctx, dst);
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_hardsigmoid>(ctx, dst);
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_hardswish>(ctx, dst);
}

void ggml_cuda_op_exp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_exp>(ctx, dst);
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sqr>(ctx, dst);
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sqrt>(ctx, dst);
}

void ggml_cuda_op_sin(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sin>(ctx, dst);
}

void ggml_cuda_op_cos(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_cos>(ctx, dst);
}

void ggml_cuda_op_log(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_log>(ctx, dst);
}

void ggml_cuda_op_elu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_elu>(ctx, dst);
}
/* gated ops */

template <float (*op)(float), typename T>
static __global__ void unary_gated_op_kernel(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1) {
    const int64_t i = int64_t(blockDim.x)*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    // perform base op and multiply with gate (either offset in same tensor or a separate one)
    const int64_t j0 = (i / n) * o0 + (i % n);
    const int64_t j1 = o0 == o1 ? j0 : (i / n) * o1 + (i % n);

    dst[i] = (T)(op((float)x[j0]) * (float)g[j1]);
}

template <float (*op)(float), typename T>
static void unary_gated_cuda(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1, hipStream_t stream) {
    const int64_t num_blocks = (k + CUDA_GLU_BLOCK_SIZE - 1) / CUDA_GLU_BLOCK_SIZE;
    unary_gated_op_kernel<op><<<num_blocks, CUDA_GLU_BLOCK_SIZE, 0, stream>>>(x, g, dst, k, n, o0, o1);
}

template <float (*op)(float)>
void ggml_cuda_op_unary_gated(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    void * src0_d = src0->data;
    void * src1_d = src1 ? src1->data : src0->data;
    const int64_t src0_o = src0->nb[1];
    const int64_t src1_o = src1 ? src1->nb[1] : src0->nb[1];
    void * dst_d = dst->data;
    const int64_t nc = src1 ? src0->ne[0] : src0->ne[0] / 2;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous_1(src0));
    GGML_ASSERT(src0->nb[0] == ggml_element_size(src0));
    GGML_ASSERT(ggml_is_contiguous(dst));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);
    GGML_ASSERT(dst->ne[0] == nc);
    GGML_ASSERT(ggml_nrows(dst) == ggml_nrows(src0));

    if (src1) {
        GGML_ASSERT(ggml_is_contiguous_1(src1));
        GGML_ASSERT(src1->nb[0] == ggml_element_size(src1));
        GGML_ASSERT(src1->ne[0] == nc);
        GGML_ASSERT(src0->type == src1->type);
    }

    const int32_t swapped = ((const int32_t *) dst->op_params)[1];

    if (src0->type == GGML_TYPE_F16) {
        half * src0_p = (half *) src0_d;
        half * src1_p = (half *) src1_d;

        if (!src1) {
            src0_p += swapped ? nc : 0;
            src1_p += swapped ? 0 : nc;
        }

        unary_gated_cuda<op>(src0_p, src1_p, (half *)dst_d, ggml_nelements(dst), nc, src0_o / sizeof(half), src1_o / sizeof(half), stream);
    } else {
        float * src0_p = (float *) src0_d;
        float * src1_p = (float *) src1_d;

        if (!src1) {
            src0_p += swapped ? nc : 0;
            src1_p += swapped ? 0 : nc;
        }

        unary_gated_cuda<op>(src0_p, src1_p, (float *)dst_d, ggml_nelements(dst), nc, src0_o / sizeof(float), src1_o / sizeof(float), stream);
    }
}

void ggml_cuda_op_reglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_relu>(ctx, dst);
}

void ggml_cuda_op_geglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_gelu>(ctx, dst);
}

void ggml_cuda_op_swiglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_silu>(ctx, dst);
}

void ggml_cuda_op_geglu_erf(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_gelu_erf>(ctx, dst);
}

void ggml_cuda_op_geglu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_gelu_quick>(ctx, dst);
}

/* silu_back */

static __device__ __forceinline__ float op_silu_back(float grad, float x) {
    const float s = 1.0f / (1.0f + expf(-x));
    return grad * s * (1.0f + x * (1.0f - s));
}

template <class T>
static __global__ void silu_back_kernel(const T * grad, const T * xf, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op_silu_back((float)grad[i], (float)xf[i]);
}

template <class T>
static void silu_back_cuda(const T * grad, const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BACK_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_back_kernel<<<num_blocks, CUDA_SILU_BACK_BLOCK_SIZE, 0, stream>>>(grad, x, dst, k);
}

void ggml_cuda_op_silu_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // input from forward pass
    const ggml_tensor * src1 = dst->src[1]; // grads of forward pass output

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_back_cuda((const half *)src0_d, (const half *)src1_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_back_cuda((const float*)src0_d, (const float*)src1_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

/* leaky relu */

static __device__ __forceinline__ float op_leaky_relu(float x, const float negative_slope) {
    return fmaxf(x, 0) + fminf(x, 0.0f) * negative_slope;
}

template <class T>
static __global__ void leaky_relu_kernel(const T * x, T * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op_leaky_relu((float)x[i], negative_slope);
}

template <class T>
static void leaky_relu_cuda(const T * x, T * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    leaky_relu_kernel<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    if (src0->type == GGML_TYPE_F16) {
        leaky_relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), negative_slope, stream);
    } else {
        leaky_relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), negative_slope, stream);
    }
}

#include "hip/hip_runtime.h"
#include "unary.cuh"

static __device__ __forceinline__ float op_abs(float x) {
    return fabsf(x);
}

static __device__ __forceinline__ float op_sgn(float x) {
    return (x > 0.f ? 1.f : ((x < 0.f ? -1.f : 0.f)));
}

static __device__ __forceinline__ float op_neg(float x) {
    return -x;
}

static __device__ __forceinline__ float op_step(float x) {
    return x > 0.0f;
}

static __device__ __forceinline__ float op_gelu(float x) {
    const float GELU_COEF_A    = 0.044715f;
    const float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;

    return 0.5f*x*(1.0f + tanhf(SQRT_2_OVER_PI*x*(1.0f + GELU_COEF_A*x*x)));
}

static __device__ __forceinline__ float op_gelu_quick(float x) {
    const float GELU_QUICK_COEF = -1.702f;

    return x * (1.0f / (1.0f + expf(GELU_QUICK_COEF * x)));
}

static __device__ __forceinline__ float op_silu(float x) {
    return x / (1.0f + expf(-x));
}

static __device__ __forceinline__ float op_tanh(float x) {
    return tanhf(x);
}

static __device__ __forceinline__ float op_relu(float x) {
    return fmaxf(x, 0);
}

static __device__ __forceinline__ float op_sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

static __device__ __forceinline__ float op_hardsigmoid(float x) {
    return fminf(1.0f, fmaxf(0.0f, (x + 3.0f) / 6.0f));
}

static __device__ __forceinline__ float op_hardswish(float x) {
    return x * fminf(1.0f, fmaxf(0.0f, (x + 3.0f) / 6.0f));
}

static __device__ __forceinline__ float op_exp(float x) {
    return expf(x);
}

static __device__ __forceinline__ float op_sqr(float x) {
    return x * x;
}

static __device__ __forceinline__ float op_sqrt(float x) {
    return sqrtf(x);
}

static __device__ __forceinline__ float op_sin(float x) {
    return sinf(x);
}

static __device__ __forceinline__ float op_cos(float x) {
    return cosf(x);
}

static __device__ __forceinline__ float op_log(float x) {
    return logf(x);
}

template <float (*op)(float), typename T>
static __global__ void unary_op_kernel(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op((float)x[i]);
}

template <float (*op)(float), typename T>
static void unary_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    unary_op_kernel<op><<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <float (*op)(float)>
void ggml_cuda_op_unary(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        unary_cuda<op>((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        unary_cuda<op>((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_abs(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_abs>(ctx, dst);
}

void ggml_cuda_op_sgn(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sgn>(ctx, dst);
}

void ggml_cuda_op_neg(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_neg>(ctx, dst);
}

void ggml_cuda_op_step(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_step>(ctx, dst);
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_gelu>(ctx, dst);
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_gelu_quick>(ctx, dst);
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_silu>(ctx, dst);
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_tanh>(ctx, dst);
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_relu>(ctx, dst);
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sigmoid>(ctx, dst);
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_hardsigmoid>(ctx, dst);
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_hardswish>(ctx, dst);
}

void ggml_cuda_op_exp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_exp>(ctx, dst);
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sqr>(ctx, dst);
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sqrt>(ctx, dst);
}

void ggml_cuda_op_sin(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sin>(ctx, dst);
}

void ggml_cuda_op_cos(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_cos>(ctx, dst);
}

void ggml_cuda_op_log(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_log>(ctx, dst);
}

/* silu_back */

static __device__ __forceinline__ float op_silu_back(float grad, float x) {
    const float s = 1.0f / (1.0f + expf(-x));
    return grad * s * (1.0f + x * (1.0f - s));
}

template <class T>
static __global__ void silu_back_kernel(const T * grad, const T * xf, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op_silu_back((float)grad[i], (float)xf[i]);
}

template <class T>
static void silu_back_cuda(const T * grad, const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BACK_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_back_kernel<<<num_blocks, CUDA_SILU_BACK_BLOCK_SIZE, 0, stream>>>(grad, x, dst, k);
}

void ggml_cuda_op_silu_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // input from forward pass
    const ggml_tensor * src1 = dst->src[1]; // grads of forward pass output

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_back_cuda((const half *)src0_d, (const half *)src1_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_back_cuda((const float*)src0_d, (const float*)src1_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

/* leaky relu */

static __device__ __forceinline__ float op_leaky_relu(float x, const float negative_slope) {
    return fmaxf(x, 0) + fminf(x, 0.0f) * negative_slope;
}

template <class T>
static __global__ void leaky_relu_kernel(const T * x, T * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op_leaky_relu((float)x[i], negative_slope);
}

template <class T>
static void leaky_relu_cuda(const T * x, T * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    leaky_relu_kernel<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    if (src0->type == GGML_TYPE_F16) {
        leaky_relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), negative_slope, stream);
    } else {
        leaky_relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), negative_slope, stream);
    }
}

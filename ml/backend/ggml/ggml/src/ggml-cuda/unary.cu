#include "hip/hip_runtime.h"
#include "unary.cuh"

static __global__ void neg_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = -x[i];
}

static __global__ void step_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = x[i] > 0.0f;
}

static __global__ void gelu_f32(const float * x, float * dst, const int k) {
    const float GELU_COEF_A    = 0.044715f;
    const float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    float xi = x[i];
    dst[i] = 0.5f*xi*(1.0f + tanhf(SQRT_2_OVER_PI*xi*(1.0f + GELU_COEF_A*xi*xi)));
}

static __global__ void gelu_quick_f32(const float * x, float * dst, int k) {
    const float GELU_QUICK_COEF = -1.702f;
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = x[i] * (1.0f / (1.0f + expf(GELU_QUICK_COEF * x[i])));
}

static __global__ void silu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / (1.0f + expf(-x[i]));
}

static __global__ void tanh_f32(const float * x, float * dst, int k) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = tanhf(x[i]);
}

static __global__ void relu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0);
}

static __global__ void sigmoid_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = 1.0f / (1.0f + expf(-x[i]));
}

static __global__ void hardsigmoid_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fminf(1.0f, fmaxf(0.0f, (x[i] + 3.0f) / 6.0f));
}

static __global__ void hardswish_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * fminf(1.0f, fmaxf(0.0f, (x[i] + 3.0f) / 6.0f));
}

static __global__ void exp_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = expf(x[i]);
}

static __global__ void leaky_relu_f32(const float * x, float * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0) + fminf(x[i], 0.0f) * negative_slope;
}

static __global__ void sqr_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * x[i];
}

static __global__ void sqrt_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sqrtf(x[i]);
}

static __global__ void sin_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sinf(x[i]);
}

static __global__ void cos_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = cosf(x[i]);
}

static void neg_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    neg_f32<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void step_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_STEP_BLOCK_SIZE - 1) / CUDA_STEP_BLOCK_SIZE;
    step_f32<<<num_blocks, CUDA_STEP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void gelu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_f32<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void gelu_quick_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_quick_f32<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void silu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void tanh_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_TANH_BLOCK_SIZE - 1) / CUDA_TANH_BLOCK_SIZE;
    tanh_f32<<<num_blocks, CUDA_TANH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void relu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    relu_f32<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void sigmoid_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIGMOID_BLOCK_SIZE - 1) / CUDA_SIGMOID_BLOCK_SIZE;
    sigmoid_f32<<<num_blocks, CUDA_SIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void hardsigmoid_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSIGMOID_BLOCK_SIZE - 1) / CUDA_HARDSIGMOID_BLOCK_SIZE;
    hardsigmoid_f32<<<num_blocks, CUDA_HARDSIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void hardswish_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSWISH_BLOCK_SIZE - 1) / CUDA_HARDSWISH_BLOCK_SIZE;
    hardswish_f32<<<num_blocks, CUDA_HARDSWISH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void exp_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_EXP_BLOCK_SIZE - 1) / CUDA_EXP_BLOCK_SIZE;
    exp_f32<<<num_blocks, CUDA_EXP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void leaky_relu_f32_cuda(const float * x, float * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    leaky_relu_f32<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

static void sqr_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQR_BLOCK_SIZE - 1) / CUDA_SQR_BLOCK_SIZE;
    sqr_f32<<<num_blocks, CUDA_SQR_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void sqrt_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQRT_BLOCK_SIZE - 1) / CUDA_SQRT_BLOCK_SIZE;
    sqrt_f32<<<num_blocks, CUDA_SQRT_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void sin_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIN_BLOCK_SIZE - 1) / CUDA_SIN_BLOCK_SIZE;
    sin_f32<<<num_blocks, CUDA_SIN_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void cos_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_COS_BLOCK_SIZE - 1) / CUDA_COS_BLOCK_SIZE;
    cos_f32<<<num_blocks, CUDA_COS_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

void ggml_cuda_op_neg(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    neg_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_step(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    step_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    gelu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    silu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    gelu_quick_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    tanh_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    relu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sigmoid_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    hardsigmoid_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    hardswish_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_exp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    exp_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    leaky_relu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), negative_slope, stream);
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sqr_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sqrt_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_sin(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sin_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_cos(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    cos_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

#include "hip/hip_runtime.h"
#include "sumrows.cuh"

static __global__ void k_sum_rows_f32(const float * x, float * dst, const int ncols) {
    const int row = blockIdx.x;
    const int col = threadIdx.x;

    float sum = 0.0f;
    for (int i = col; i < ncols; i += blockDim.x) {
        sum += x[row * ncols + i];
    }

    sum = warp_reduce_sum(sum);

    if (col == 0) {
        dst[row] = sum;
    }
}

void sum_rows_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums(nrows, 1, 1);
    k_sum_rows_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols);
}

void ggml_cuda_op_sum_rows(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    sum_rows_f32_cuda(src0_d, dst_d, ncols, nrows, stream);
}

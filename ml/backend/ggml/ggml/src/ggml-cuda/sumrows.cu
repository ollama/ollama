#include "hip/hip_runtime.h"
#include "reduce_rows.cuh"
#include "sumrows.cuh"

void sum_rows_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    const int  id  = ggml_cuda_get_device();
    const int  nsm = ggml_cuda_info().devices[id].nsm;
    const dim3 block_nums(nrows, 1, 1);
    if ((nrows / nsm) < 2) {
        const dim3 block_dims(512, 1, 1);
        reduce_rows_f32</*norm=*/false><<<block_nums, block_dims, 0, stream>>>(x, dst, ncols);
    } else {
        const dim3 block_dims(ncols < 1024 ? 32 : 128, 1, 1);
        reduce_rows_f32</*norm=*/false><<<block_nums, block_dims, 0, stream>>>(x, dst, ncols);
    }
}

void ggml_cuda_op_sum_rows(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const dim3 block_nums(nrows, 1, 1);

    const int id  = ggml_cuda_get_device();
    const int nsm = ggml_cuda_info().devices[id].nsm;
    if ((nrows / nsm) < 2) {
        // Increase num threads to 512 for small nrows to better hide the latency
        const dim3 block_dims(512, 1, 1);
        reduce_rows_f32</*norm=*/false><<<block_nums, block_dims, 0, stream>>>(src0_d, dst_d, ncols);
    } else {
        // Enough active SMs to hide latency, use smaller blocks to allow better scheduling
        const dim3 block_dims(ncols < 1024 ? 32 : 128, 1, 1);
        reduce_rows_f32</*norm=*/false><<<block_nums, block_dims, 0, stream>>>(src0_d, dst_d, ncols);
    }
}

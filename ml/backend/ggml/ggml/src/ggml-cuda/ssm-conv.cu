#include "hip/hip_runtime.h"
#include "ssm-conv.cuh"

template <size_t split_d_inner, size_t d_conv>
static __global__ void ssm_conv_f32(const float * __restrict__ src0, const float * __restrict__ src1,
                                    const int src0_nb0, const int src0_nb1, const int src0_nb2, const int src1_nb1,
                                    float * __restrict__ dst, const int dst_nb0, const int dst_nb1, const int dst_nb2,
                                    const int64_t n_t) {
    GGML_UNUSED(src0_nb0);
    const int tid  = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;

    const float * x_block = (const float *) ((const char *) src0 + bidx * src0_nb2 + bidy * split_d_inner * src0_nb1);
    const float * w_block = (const float *) ((const char *) src1 + bidy * split_d_inner * src1_nb1);
    float *       y_block = (float *) ((char *) dst + bidx * dst_nb2 + bidy * split_d_inner * dst_nb0);

    const int stride_x = src0_nb1 / sizeof(float);
    const int stride_w = src1_nb1 / sizeof(float);
    const int stride_y = dst_nb1 / sizeof(float);

    float x[d_conv] = { 0.0f };
    float w[d_conv] = { 0.0f };

#pragma unroll
    for (size_t j = 0; j < d_conv; j++) {
        w[j] = w_block[tid * stride_w + j];
    }

    for (int64_t i = 0; i < n_t; i++) {
        float sumf = 0.0f;

        if (i == 0) {
            for (size_t j = 0; j < d_conv; j++) {
                x[j] = x_block[tid * stride_x + j];
            }
        } else {
            x[(i - 1) % d_conv] = x_block[tid * stride_x + i + d_conv - 1];
        }

#pragma unroll
        for (size_t j = 0; j < d_conv; j++) {
            sumf += x[(i + j) % d_conv] * w[j];
        }
        y_block[i * stride_y + tid] = sumf;
    }
}

template <size_t split_d_inner, size_t d_conv, int64_t split_n_t>
static __global__ void ssm_conv_long_token_f32(const float * __restrict__ src0, const float * __restrict__ src1,
                                               const int src0_nb0, const int src0_nb1, const int src0_nb2,
                                               const int src1_nb1, float * __restrict__ dst, const int dst_nb0,
                                               const int dst_nb1, const int dst_nb2, const int64_t n_t) {
    const int tid  = threadIdx.x;
    const int bidx = blockIdx.x;
    const int bidy = blockIdx.y;
    const int bidz = blockIdx.z;

    const float * x_block = (const float *) ((const char *) src0 + bidx * src0_nb2 + bidy * split_d_inner * src0_nb1 +
                                             bidz * split_n_t * src0_nb0);
    const float * w_block = (const float *) ((const char *) src1 + bidy * split_d_inner * src1_nb1);
    float *       y_block =
        (float *) ((char *) dst + bidx * dst_nb2 + bidz * split_n_t * dst_nb1 + bidy * split_d_inner * dst_nb0);

    const int stride_x = src0_nb1 / sizeof(float);
    const int stride_w = src1_nb1 / sizeof(float);
    const int stride_y = dst_nb1 / sizeof(float);

    float x[d_conv] = { 0.0f };
    float w[d_conv] = { 0.0f };

#pragma unroll
    for (size_t j = 0; j < d_conv; j++) {
        w[j] = w_block[tid * stride_w + j];
    }

#pragma unroll
    for (int64_t i = 0; i < split_n_t; i++) {
        if (bidz * split_n_t + i < n_t) {
            float sumf = 0.0f;

            if (i == 0) {
                for (size_t j = 0; j < d_conv; j++) {
                    x[j] = x_block[tid * stride_x + j];
                }
            } else {
                x[(i - 1) % d_conv] = x_block[tid * stride_x + i + d_conv - 1];
            }

#pragma unroll
            for (size_t j = 0; j < d_conv; j++) {
                sumf += x[(i + j) % d_conv] * w[j];
            }
            y_block[i * stride_y + tid] = sumf;
        }
    }
}

static void ssm_conv_f32_cuda(const float * src0, const float * src1, const int src0_nb0, const int src0_nb1,
                              const int src0_nb2, const int src1_nb1, float * dst, const int dst_nb0, const int dst_nb1,
                              const int dst_nb2, const int64_t nc, const int64_t nr, const int64_t n_t,
                              const int64_t n_s, hipStream_t stream) {
    const int threads = 128;
    GGML_ASSERT(nr % threads == 0);

    if (n_t <= 32) {
        const dim3 blocks(n_s, (nr + threads - 1) / threads, 1);
        if (nc == 4) {
            ssm_conv_f32<threads, 4><<<blocks, threads, 0, stream>>>(src0, src1, src0_nb0, src0_nb1, src0_nb2, src1_nb1,
                                                                     dst, dst_nb0, dst_nb1, dst_nb2, n_t);
        } else {
            GGML_ABORT("Only support kernel size = 4  now.");
        }
    } else {
        if (nc == 4) {
            const int64_t split_n_t = 32;
            dim3          blocks(n_s, (nr + threads - 1) / threads, (n_t + split_n_t - 1) / split_n_t);
            ssm_conv_long_token_f32<threads, 4, split_n_t><<<blocks, threads, 0, stream>>>(
                src0, src1, src0_nb0, src0_nb1, src0_nb2, src1_nb1, dst, dst_nb0, dst_nb1, dst_nb2, n_t);
        } else {
            GGML_ABORT("Only support kernel size = 4 right now.");
        }
    }
}

void ggml_cuda_op_ssm_conv(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0];  // conv_x
    const struct ggml_tensor * src1 = dst->src[1];  // conv1d.weight

    const int64_t nc  = src1->ne[0];                // d_conv
    const int64_t nr  = src0->ne[1];                // d_inner
    const int64_t n_t = dst->ne[1];                 // tokens per sequence
    const int64_t n_s = dst->ne[2];                 // number of sequences in the batch

    GGML_ASSERT(dst->ne[0] == nr);
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src0->nb[1] == src0->ne[0] * sizeof(float));

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float *       dst_d  = (float *) dst->data;
    hipStream_t  stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    ssm_conv_f32_cuda(src0_d, src1_d, src0->nb[0], src0->nb[1], src0->nb[2], src1->nb[1], dst_d, dst->nb[0], dst->nb[1],
                      dst->nb[2], nc, nr, n_t, n_s, stream);
}

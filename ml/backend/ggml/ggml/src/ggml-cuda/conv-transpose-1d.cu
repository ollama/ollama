#include "hip/hip_runtime.h"
#include "conv-transpose-1d.cuh"

static  __global__ void conv_transpose_1d_kernel(
        const int s0, const int p0, const int d0, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst) {
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index >= output_size) {
        return;
    }

    int out_index = global_index / dst_ne0;

    float accumulator = 0;

    for (int c = 0; c < src0_ne2; c++) {
        int idx = global_index % dst_ne0;

        int kernel_offset = (src0_ne0 * src0_ne1 * c) + (out_index * src0_ne0);
        int input_offset = src1_ne0 * c;

        for (int i = 0; i < src1_ne0; i++) {
            if (!(idx >= i*s0 && idx < i*s0 + src0_ne0)) {
                continue;
            }
            int weight_idx = idx - i*s0;

            float kernel_weight = src0[kernel_offset + weight_idx];
            float input_value =  src1[input_offset+i];

            accumulator += kernel_weight * input_value;
        }
    }
    dst[global_index] = accumulator;
}

static void conv_transpose_1d_f32_f32_cuda(
        const int s0, const int p0, const int d0, const int output_size,
        const int src0_ne0, const int src0_ne1, const int src0_ne2, const int src0_ne3,
        const int src1_ne0, const int src1_ne1, const int src1_ne2, const int src1_ne3,
        const int dst_ne0, const int dst_ne1, const int dst_ne2, const int dst_ne3,
        const float * src0, const float * src1,  float * dst,
        hipStream_t stream) {

    const int num_blocks = (output_size + CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE - 1) / CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE;
    conv_transpose_1d_kernel<<<num_blocks,CUDA_CONV_TRANPOSE_1D_BLOCK_SIZE, 0, stream>>>(
        s0,p0,d0,output_size,
        src0_ne0, src0_ne1,  src0_ne2, src0_ne3,
        src1_ne0, src1_ne1,  src1_ne2, src1_ne3,
        dst_ne0,  dst_ne1,   dst_ne2,  dst_ne3,
        src0,src1, dst);
}

void ggml_cuda_op_conv_transpose_1d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;

    const ggml_tensor * src1 = dst->src[1];
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));

    const int32_t * opts = (const int32_t *)dst->op_params;

    const int s0 = opts[0];
    const int p0 = 0;//opts[3];
    const int d0 = 1;//opts[4];

    const int64_t kernel_size = ggml_nelements(src0);
    const int64_t input_size = ggml_nelements(src1);
    const int64_t output_size = ggml_nelements(dst);

    conv_transpose_1d_f32_f32_cuda(s0, p0, d0, output_size,
        src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3],
        src1->ne[0], src1->ne[1], src1->ne[2], src1->ne[3],
        dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3],
        src0_d, src1_d, dst_d, stream);
}

#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>

#include "argmax.cuh"
#include "common.cuh"
#include "sum.cuh"

static __global__ void argmax_f32(const float * __restrict__ x, int32_t * __restrict__ dst, const int64_t ncols) {
    const int64_t row = blockIdx.x;

    float maxval = -FLT_MAX;
    int   argmax = -1;
    const float * rowx = x + row * ncols;

    for (int32_t col = threadIdx.x; col < ncols; col += blockDim.x) {
        const float val = rowx[col];
        if (val > maxval) {
            maxval = val;
            argmax = col;
        }
    }

#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        const float val = __shfl_xor_sync(0xFFFFFFFF, maxval, offset, WARP_SIZE);
        const int   col = __shfl_xor_sync(0xFFFFFFFF, argmax, offset, WARP_SIZE);
        if (val > maxval) {
            maxval = val;
            argmax = col;
        }
    }

    const int n_warps = blockDim.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warp_id = threadIdx.x / WARP_SIZE;
    if (n_warps > 1) {
        constexpr int    max_warps = 1024 / WARP_SIZE;
        __shared__ float shared_maxval[max_warps];
        __shared__ int   shared_argmax[max_warps];
        if (lane_id == 0) {
            shared_maxval[warp_id] = maxval;
            shared_argmax[warp_id] = argmax;
        }

        __syncthreads();

        if (warp_id == 0) {
            if (lane_id < n_warps) {
                maxval = shared_maxval[lane_id];
                argmax = shared_argmax[lane_id];
            }
#pragma unroll
            for (int offset = 16; offset > 0; offset >>= 1) {
                const float val = __shfl_xor_sync(0xFFFFFFFF, maxval, offset, WARP_SIZE);
                const int   col = __shfl_xor_sync(0xFFFFFFFF, argmax, offset, WARP_SIZE);
                if (val > maxval) {
                    maxval = val;
                    argmax = col;
                }
            }
        }
    }

    if (warp_id == 0 && lane_id == 0) {
        dst[row] = argmax;
    }
}

void ggml_cuda_argmax(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);

    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ne00  = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const float * src0_d = (const float *) src0->data;
    int32_t     * dst_d  = (int32_t     *) dst->data;

    hipStream_t stream = ctx.stream();

    const int64_t num_blocks = nrows;
    const int64_t num_threads = std::min<int64_t>(1024, (ne00 + WARP_SIZE - 1) / WARP_SIZE * WARP_SIZE);
    const dim3 blocks_dim(num_threads, 1, 1);
    const dim3 blocks_num(num_blocks, 1, 1);

    argmax_f32<<<blocks_num, blocks_dim, 0, stream>>>(src0_d, dst_d, ne00);
}

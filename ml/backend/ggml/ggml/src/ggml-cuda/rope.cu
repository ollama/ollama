#include "hip/hip_runtime.h"
#include "rope.cuh"

struct rope_corr_dims {
    float v[2];
};


struct mrope_sections {
    int v[4];
};

static __device__ float rope_yarn_ramp(const float low, const float high, const int i0) {
    const float y = (i0 / 2 - low) / max(0.001f, high - low);
    return 1.0f - min(1.0f, max(0.0f, y));
}

// YaRN algorithm based on LlamaYaRNScaledRotaryEmbedding.py from https://github.com/jquesnelle/yarn
// MIT licensed. Copyright (c) 2023 Jeffrey Quesnelle and Bowen Peng.
template<bool forward>
static __device__ void rope_yarn(
        const float theta_extrap, const float freq_scale, const rope_corr_dims corr_dims, const int64_t i0, const float ext_factor,
        float mscale, float & cos_theta, float & sin_theta) {
    // Get n-d rotational scaling corrected for extrapolation
    float theta_interp = freq_scale * theta_extrap;
    float theta = theta_interp;
    if (ext_factor != 0.0f) {
        float ramp_mix = rope_yarn_ramp(corr_dims.v[0], corr_dims.v[1], i0) * ext_factor;
        theta = theta_interp * (1 - ramp_mix) + theta_extrap * ramp_mix;

        // Get n-d magnitude scaling corrected for interpolation
        mscale *= 1.0f + 0.1f * logf(1.0f / freq_scale);
    }
    cos_theta = cosf(theta) * mscale;
    sin_theta = sinf(theta) * mscale;
    if (!forward) {
        sin_theta *= -1.0f;
    }
}

template<bool forward, bool has_ff, typename T>
static __global__ void rope_norm(
        const T * x, T * dst, const int ne0, const int ne1, const int s1, const int s2, const int n_dims,
        const int32_t * pos, const float freq_scale, const float ext_factor, const float attn_factor,
        const rope_corr_dims corr_dims, const float theta_scale, const float * freq_factors) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row_dst = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row_dst*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int row_x     = row_dst % ne1;
    const int channel_x = row_dst / ne1;

    const int idst = row_dst*ne0 + i0;
    const int ix   = channel_x*s2 + row_x*s1 + i0;

    const float theta_base = pos[channel_x]*powf(theta_scale, i0/2.0f);

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn<forward>(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, cos_theta, sin_theta);

    const float x0 = x[ix + 0];
    const float x1 = x[ix + 1];

    dst[idst + 0] = x0*cos_theta - x1*sin_theta;
    dst[idst + 1] = x0*sin_theta + x1*cos_theta;
}

template<bool forward, bool has_ff, typename T>
static __global__ void rope_neox(
        const T * x, T * dst, const int ne0, const int ne1, const int s1, const int s2, const int n_dims,
        const int32_t * pos, const float freq_scale, const float ext_factor, const float attn_factor,
        const rope_corr_dims corr_dims, const float theta_scale, const float * freq_factors) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row_dst = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row_dst*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int row_x     = row_dst % ne1;
    const int channel_x = row_dst / ne1;

    const int idst = row_dst*ne0 + i0/2;
    const int ix   = channel_x*s2 + row_x*s1 + i0/2;

    const float theta_base = pos[channel_x]*powf(theta_scale, i0/2.0f);

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn<forward>(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, cos_theta, sin_theta);

    const float x0 = x[ix + 0];
    const float x1 = x[ix + n_dims/2];

    dst[idst + 0]        = x0*cos_theta - x1*sin_theta;
    dst[idst + n_dims/2] = x0*sin_theta + x1*cos_theta;
}

template<bool forward, bool has_ff, typename T>
static __global__ void rope_multi(
        const T * x, T * dst, const int ne0, const int ne1, const int ne2, const int s1, const int s2,
        const int n_dims, const int32_t * pos, const float freq_scale, const float ext_factor, const float attn_factor,
        const rope_corr_dims corr_dims, const float theta_scale, const float * freq_factors, const mrope_sections sections) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row_dst = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row_dst*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int row_x     = row_dst % ne1;
    const int channel_x = row_dst / ne1;

    const int idst = row_dst*ne0 + i0/2;
    const int ix   = channel_x*s2 + row_x*s1 + i0/2;

    const int sect_dims = sections.v[0] + sections.v[1] + sections.v[2] + sections.v[3];
    const int sec_w = sections.v[1] + sections.v[0];
    const int sector = (i0 / 2) % sect_dims;

    float theta_base = 0.0;
    if (sector < sections.v[0]) {
        theta_base = pos[channel_x]*powf(theta_scale, i0/2.0f);
    }
    else if (sector >= sections.v[0] && sector < sec_w) {
        theta_base = pos[channel_x + ne2 * 1]*powf(theta_scale, i0/2.0f);
    }
    else if (sector >= sec_w && sector < sec_w + sections.v[2]) {
        theta_base = pos[channel_x + ne2 * 2]*powf(theta_scale, i0/2.0f);
    }
    else if (sector >= sec_w + sections.v[2]) {
        theta_base = pos[channel_x + ne2 * 3]*powf(theta_scale, i0/2.0f);
    }

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn<forward>(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, cos_theta, sin_theta);

    const float x0 = x[ix + 0];
    const float x1 = x[ix + n_dims/2];

    dst[idst + 0]        = x0*cos_theta - x1*sin_theta;
    dst[idst + n_dims/2] = x0*sin_theta + x1*cos_theta;
}

template<bool forward, bool has_ff, typename T>
static __global__ void rope_vision(
        const T * x, T * dst, const int ne0, const int ne1, const int ne2, const int s1, const int s2, const int n_dims,
        const int32_t * pos, const float freq_scale, const float ext_factor, const float attn_factor, const rope_corr_dims corr_dims,
        const float theta_scale, const float * freq_factors, const mrope_sections sections) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row_dst = blockDim.x*blockIdx.x + threadIdx.x;

    const int row_x     = row_dst % ne1;
    const int channel_x = row_dst / ne1;

    const int idst = row_dst*ne0 + i0/2;
    const int ix   = channel_x*s2 + row_x*s1 + i0/2;

    const int sect_dims = sections.v[0] + sections.v[1];
    const int sec_w = sections.v[1] + sections.v[0];
    const int sector = (i0 / 2) % sect_dims;

    float theta_base = 0.0;
    if (sector < sections.v[0]) {
        const int p = sector;
        theta_base = pos[channel_x]*powf(theta_scale, p);
    }
    else if (sector >= sections.v[0] && sector < sec_w) {
        const int p = sector - sections.v[0];
        theta_base = pos[channel_x + ne2]*powf(theta_scale, p);
    }

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn<forward>(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, cos_theta, sin_theta);

    const float x0 = x[ix + 0];
    const float x1 = x[ix + n_dims];

    dst[idst + 0]      = x0*cos_theta - x1*sin_theta;
    dst[idst + n_dims] = x0*sin_theta + x1*cos_theta;
}

template<bool forward, typename T>
static void rope_norm_cuda(
        const T * x, T * dst, const int ne0, const int ne1, const int s1, const int s2, const int n_dims, const int nr,
        const int32_t * pos, const float freq_scale, const float freq_base, const float ext_factor, const float attn_factor,
        const rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_norm<forward, false><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors);
    } else {
        rope_norm<forward, true><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors);
    }
}

template<bool forward, typename T>
static void rope_neox_cuda(
        const T * x, T * dst, const int ne0, const int ne1, const int s1, const int s2, const int n_dims, const int nr,
        const int32_t * pos, const float freq_scale, const float freq_base, const float ext_factor, const float attn_factor,
        const rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_neox<forward, false, T><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors);
    } else {
        rope_neox<forward, true, T><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors);
    }
}

template<bool forward, typename T>
static void rope_multi_cuda(
        const T * x, T * dst, const int ne0, const int ne1, const int ne2, const int s1, const int s2, const int n_dims, const int nr,
        const int32_t * pos, const float freq_scale, const float freq_base, const float ext_factor, const float attn_factor,
        const rope_corr_dims corr_dims, const float * freq_factors, const mrope_sections sections, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_multi<forward, false, T><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, ne2, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors, sections);
    } else {
        rope_multi<forward, true, T><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, ne2, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors, sections);
    }
}

template<bool forward, typename T>
static void rope_vision_cuda(
        const T * x, T * dst, const int ne0, const int ne1, const int ne2, const int s1, const int s2, const int n_dims, const int nr,
        const int32_t * pos, const float freq_scale, const float freq_base, const float ext_factor, const float attn_factor,
        const rope_corr_dims corr_dims, const float * freq_factors, const mrope_sections sections, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);
    // break down (head_dim, heads, seq) into (CUDA_ROPE_BLOCK_SIZE, x, heads * seq)
    // where x ~= ceil(head_dim / CUDA_ROPE_BLOCK_SIZE);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_vision<forward, false, T><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, ne2, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors, sections);
    } else {
        rope_vision<forward, true, T><<<block_nums, block_dims, 0, stream>>>(
            x, dst, ne0, ne1, ne2, s1, s2, n_dims, pos, freq_scale, ext_factor,
            attn_factor, corr_dims, theta_scale, freq_factors, sections);
    }
}

template <bool forward>
void ggml_cuda_op_rope_impl(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const ggml_tensor * src2 = dst->src[2];

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    const int64_t ne00 = src0->ne[0]; // head dims
    const int64_t ne01 = src0->ne[1]; // num heads
    const int64_t ne02 = src0->ne[2]; // num heads
    const int64_t nr = ggml_nrows(src0);

    const size_t s01 = src0->nb[1] / ggml_type_size(src0->type);
    const size_t s02 = src0->nb[2] / ggml_type_size(src0->type);

    //const int n_past     = ((int32_t *) dst->op_params)[0];
    const int n_dims     = ((int32_t *) dst->op_params)[1];
    const int mode       = ((int32_t *) dst->op_params)[2];
    //const int n_ctx      = ((int32_t *) dst->op_params)[3];
    const int n_ctx_orig = ((int32_t *) dst->op_params)[4];
    mrope_sections sections;

    // RoPE alteration for extended context
    float freq_base;
    float freq_scale;
    float ext_factor;
    float attn_factor;
    float beta_fast;
    float beta_slow;

    memcpy(&freq_base,   (int32_t *) dst->op_params +  5, sizeof(float));
    memcpy(&freq_scale,  (int32_t *) dst->op_params +  6, sizeof(float));
    memcpy(&ext_factor,  (int32_t *) dst->op_params +  7, sizeof(float));
    memcpy(&attn_factor, (int32_t *) dst->op_params +  8, sizeof(float));
    memcpy(&beta_fast,   (int32_t *) dst->op_params +  9, sizeof(float));
    memcpy(&beta_slow,   (int32_t *) dst->op_params + 10, sizeof(float));
    memcpy(&sections.v,  (int32_t *) dst->op_params + 11, sizeof(int)*4);

    const bool is_neox = mode & GGML_ROPE_TYPE_NEOX;
    const bool is_mrope = mode & GGML_ROPE_TYPE_MROPE;
    const bool is_vision = mode == GGML_ROPE_TYPE_VISION;

    if (is_mrope) {
        GGML_ASSERT(sections.v[0] > 0 || sections.v[1] > 0 || sections.v[2] > 0);
    }

    if (is_vision) {
        GGML_ASSERT(n_dims == ne00/2);
    }

    const int32_t * pos = (const int32_t *) src1_d;

    const float * freq_factors = nullptr;
    if (src2 != nullptr) {
        freq_factors = (const float *) src2->data;
    }

    rope_corr_dims corr_dims;
    ggml_rope_yarn_corr_dims(n_dims, n_ctx_orig, freq_base, beta_fast, beta_slow, corr_dims.v);

    // compute
    if (is_neox) {
        if (src0->type == GGML_TYPE_F32) {
            rope_neox_cuda<forward>(
                (const float *) src0_d, (float *) dst_d, ne00, ne01, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
        } else if (src0->type == GGML_TYPE_F16) {
            rope_neox_cuda<forward>(
                (const half *) src0_d, (half *) dst_d, ne00, ne01, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
        } else {
            GGML_ABORT("fatal error");
        }
    } else if (is_mrope && !is_vision) {
        if (src0->type == GGML_TYPE_F32) {
            rope_multi_cuda<forward>(
                (const float *) src0_d, (float *) dst_d, ne00, ne01, ne02, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
        } else if (src0->type == GGML_TYPE_F16) {
            rope_multi_cuda<forward>(
                (const half *) src0_d, (half *) dst_d, ne00, ne01, ne02, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
        } else {
            GGML_ABORT("fatal error");
        }
    } else if (is_vision) {
        if (src0->type == GGML_TYPE_F32) {
            rope_vision_cuda<forward>(
                (const float *) src0_d, (float *) dst_d, ne00, ne01, ne02, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
        } else if (src0->type == GGML_TYPE_F16) {
            rope_vision_cuda<forward>(
                (const half *) src0_d, (half *) dst_d, ne00, ne01, ne02, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
        } else {
            GGML_ABORT("fatal error");
        }
    } else {
        if (src0->type == GGML_TYPE_F32) {
            rope_norm_cuda<forward>(
                (const float *) src0_d, (float *) dst_d, ne00, ne01, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
        } else if (src0->type == GGML_TYPE_F16) {
            rope_norm_cuda<forward>(
                (const half *) src0_d, (half *) dst_d, ne00, ne01, s01, s02, n_dims, nr, pos, freq_scale,
                freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
        } else {
            GGML_ABORT("fatal error");
        }
    }
}

void ggml_cuda_op_rope(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_rope_impl<true>(ctx, dst);
}

void ggml_cuda_op_rope_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_rope_impl<false>(ctx, dst);
}

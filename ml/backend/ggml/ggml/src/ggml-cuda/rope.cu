#include "hip/hip_runtime.h"
#include "rope.cuh"

struct rope_corr_dims {
    float v[2];
};


struct mrope_sections {
    int v[4];
};

static __device__ float rope_yarn_ramp(const float low, const float high, const int i0) {
    const float y = (i0 / 2 - low) / max(0.001f, high - low);
    return 1.0f - min(1.0f, max(0.0f, y));
}

// YaRN algorithm based on LlamaYaRNScaledRotaryEmbedding.py from https://github.com/jquesnelle/yarn
// MIT licensed. Copyright (c) 2023 Jeffrey Quesnelle and Bowen Peng.
static __device__ void rope_yarn(
    float theta_extrap, float freq_scale, rope_corr_dims corr_dims, int64_t i0, float ext_factor, float mscale,
    float * cos_theta, float * sin_theta) {
    // Get n-d rotational scaling corrected for extrapolation
    float theta_interp = freq_scale * theta_extrap;
    float theta = theta_interp;
    if (ext_factor != 0.0f) {
        float ramp_mix = rope_yarn_ramp(corr_dims.v[0], corr_dims.v[1], i0) * ext_factor;
        theta = theta_interp * (1 - ramp_mix) + theta_extrap * ramp_mix;

        // Get n-d magnitude scaling corrected for interpolation
        mscale *= 1.0f + 0.1f * logf(1.0f / freq_scale);
    }
    *cos_theta = cosf(theta) * mscale;
    *sin_theta = sinf(theta) * mscale;
}

template<typename T, bool has_ff>
static __global__ void rope_norm(
    const T * x, T * dst, int ne0, int n_dims, const int32_t * pos, float freq_scale, int p_delta_rows,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims, float theta_scale, const float * freq_factors) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int i  = row*ne0 + i0;
    const int i2 = row/p_delta_rows;

    const float theta_base = pos[i2]*powf(theta_scale, i0/2.0f);

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + 1];

    dst[i + 0] = x0*cos_theta - x1*sin_theta;
    dst[i + 1] = x0*sin_theta + x1*cos_theta;
}

template<typename T, bool has_ff>
static __global__ void rope_neox(
    const T * x, T * dst, int ne0, int n_dims, const int32_t * pos, float freq_scale, int p_delta_rows,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims, float theta_scale, const float * freq_factors) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int i  = row*ne0 + i0/2;
    const int i2 = row/p_delta_rows;

    const float theta_base = pos[i2]*powf(theta_scale, i0/2.0f);

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + n_dims/2];

    dst[i + 0]        = x0*cos_theta - x1*sin_theta;
    dst[i + n_dims/2] = x0*sin_theta + x1*cos_theta;
}

template<typename T, bool has_ff>
static __global__ void rope_multi(
    const T * x, T * dst, int ne0, int ne2, int n_dims, const int32_t * pos, float freq_scale, int p_delta_rows,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims, float theta_scale, const float * freq_factors, mrope_sections sections) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int i  = row*ne0 + i0/2;
    const int i2 = row/p_delta_rows;

    int sect_dims = sections.v[0] + sections.v[1] + sections.v[2] + sections.v[3];
    int sec_w = sections.v[1] + sections.v[0];
    int sector = (i0 / 2) % sect_dims;

    float theta_base = 0.0;
    if (sector < sections.v[0]) {
        theta_base = pos[i2]*powf(theta_scale, i0/2.0f);
    }
    else if (sector >= sections.v[0] && sector < sec_w) {
        theta_base = pos[i2 + ne2 * 1]*powf(theta_scale, i0/2.0f);
    }
    else if (sector >= sec_w && sector < sec_w + sections.v[2]) {
        theta_base = pos[i2 + ne2 * 2]*powf(theta_scale, i0/2.0f);
    }
    else if (sector >= sec_w + sections.v[2]) {
        theta_base = pos[i2 + ne2 * 3]*powf(theta_scale, i0/2.0f);
    }

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + n_dims/2];

    dst[i + 0]        = x0*cos_theta - x1*sin_theta;
    dst[i + n_dims/2] = x0*sin_theta + x1*cos_theta;
}

template<typename T, bool has_ff>
static __global__ void rope_vision(
    const T * x, T * dst, int ne0, int ne2, int n_dims, const int32_t * pos, float freq_scale, int p_delta_rows,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims, float theta_scale, const float * freq_factors, mrope_sections sections) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    const int i  = row*ne0 + i0/2;
    const int i2 = row/p_delta_rows; // i2-th tokens

    int sect_dims = sections.v[0] + sections.v[1];
    int sec_w = sections.v[1] + sections.v[0];
    int sector = (i0 / 2) % sect_dims;

    float theta_base = 0.0;
    if (sector < sections.v[0]) {
        const int p = sector;
        theta_base = pos[i2]*powf(theta_scale, p);
    }
    else if (sector >= sections.v[0] && sector < sec_w) {
        const int p = sector - sections.v[0];
        theta_base = pos[i2 + ne2]*powf(theta_scale, p);
    }

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + n_dims];

    dst[i + 0]      = x0*cos_theta - x1*sin_theta;
    dst[i + n_dims] = x0*sin_theta + x1*cos_theta;
}

template<typename T>
static void rope_norm_cuda(
    const T * x, T * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_norm<T, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    } else {
        rope_norm<T, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    }
}

template<typename T>
static void rope_neox_cuda(
    const T * x, T * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_neox<T, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    } else {
        rope_neox<T, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    }
}

template<typename T>
static void rope_multi_cuda(
    const T * x, T * dst, int ne0, int ne2, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, mrope_sections sections, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_multi<T, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, ne2, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors, sections
                );
    } else {
        rope_multi<T, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, ne2, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors, sections
                );
    }
}

template<typename T>
static void rope_vision_cuda(
    const T * x, T * dst, int ne0, int ne2, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, mrope_sections sections, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);
    // break down (head_dim, heads, seq) into (CUDA_ROPE_BLOCK_SIZE, x, heads * seq)
    // where x ~= ceil(head_dim / CUDA_ROPE_BLOCK_SIZE);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_vision<T, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, ne2, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors, sections
                );
    } else {
        rope_vision<T, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, ne2, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors, sections
                );
    }
}

static void rope_norm_cuda_f16(
    const half * x, half * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {

    rope_norm_cuda<half>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_norm_cuda_f32(
    const float * x, float * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {

    rope_norm_cuda<float>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_neox_cuda_f16(
    const half * x, half * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {

    rope_neox_cuda<half>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_neox_cuda_f32(
    const float * x, float * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream
) {

    rope_neox_cuda<float>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_multi_cuda_f16(
    const half * x, half * dst, int ne0, int ne2, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, mrope_sections sections, hipStream_t stream
) {

    rope_multi_cuda<half>(x, dst, ne0, ne2, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
}

static void rope_multi_cuda_f32(
    const float * x, float * dst, int ne0, int ne2, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, mrope_sections sections, hipStream_t stream
) {

    rope_multi_cuda<float>(x, dst, ne0, ne2, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
}

static void rope_vision_cuda_f16(
    const half * x, half * dst, int ne0, int ne2, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, mrope_sections sections, hipStream_t stream
) {

    rope_vision_cuda<half>(x, dst, ne0, ne2, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
}

static void rope_vision_cuda_f32(
    const float * x, float * dst, int ne0, int ne2, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, mrope_sections sections, hipStream_t stream
) {

    rope_vision_cuda<float>(x, dst, ne0, ne2, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, sections, stream);
}

void ggml_cuda_op_rope(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const ggml_tensor * src2 = dst->src[2];

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    const int64_t ne00 = src0->ne[0]; // head dims
    const int64_t ne01 = src0->ne[1]; // num heads
    const int64_t ne02 = src0->ne[2]; // num heads
    const int64_t nr = ggml_nrows(src0);

    //const int n_past     = ((int32_t *) dst->op_params)[0];
    const int n_dims     = ((int32_t *) dst->op_params)[1];
    const int mode       = ((int32_t *) dst->op_params)[2];
    //const int n_ctx      = ((int32_t *) dst->op_params)[3];
    const int n_ctx_orig = ((int32_t *) dst->op_params)[4];
    mrope_sections sections;

    // RoPE alteration for extended context
    float freq_base;
    float freq_scale;
    float ext_factor;
    float attn_factor;
    float beta_fast;
    float beta_slow;

    memcpy(&freq_base,   (int32_t *) dst->op_params +  5, sizeof(float));
    memcpy(&freq_scale,  (int32_t *) dst->op_params +  6, sizeof(float));
    memcpy(&ext_factor,  (int32_t *) dst->op_params +  7, sizeof(float));
    memcpy(&attn_factor, (int32_t *) dst->op_params +  8, sizeof(float));
    memcpy(&beta_fast,   (int32_t *) dst->op_params +  9, sizeof(float));
    memcpy(&beta_slow,   (int32_t *) dst->op_params + 10, sizeof(float));
    memcpy(&sections.v,  (int32_t *) dst->op_params + 11, sizeof(int)*4);

    const bool is_neox = mode & GGML_ROPE_TYPE_NEOX;
    const bool is_mrope = mode & GGML_ROPE_TYPE_MROPE;
    const bool is_vision = mode == GGML_ROPE_TYPE_VISION;

    if (is_mrope) {
        GGML_ASSERT(sections.v[0] > 0 || sections.v[1] > 0 || sections.v[2] > 0);
    }

    if (is_vision) {
        GGML_ASSERT(n_dims == ne00/2);
    }

    const int32_t * pos = (const int32_t *) src1_d;

    const float * freq_factors = nullptr;
    if (src2 != nullptr) {
        freq_factors = (const float *) src2->data;
    }

    rope_corr_dims corr_dims;
    ggml_rope_yarn_corr_dims(n_dims, n_ctx_orig, freq_base, beta_fast, beta_slow, corr_dims.v);

    // compute
    if (is_neox) {
        if (src0->type == GGML_TYPE_F32) {
            rope_neox_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_neox_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else {
            GGML_ABORT("fatal error");
        }
    } else if (is_mrope && !is_vision) {
        if (src0->type == GGML_TYPE_F32) {
            rope_multi_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, ne02, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, sections, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_multi_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, ne02, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, sections, stream
            );
        } else {
            GGML_ABORT("fatal error");
        }
    } else if (is_vision) {
        if (src0->type == GGML_TYPE_F32) {
            rope_vision_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, ne02, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, sections, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_vision_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, ne02, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, sections, stream
            );
        } else {
            GGML_ABORT("fatal error");
        }
    } else {
        if (src0->type == GGML_TYPE_F32) {
            rope_norm_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_norm_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else {
            GGML_ABORT("fatal error");
        }
    }
}

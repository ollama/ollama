#include "hip/hip_runtime.h"
#include "ggml-cuda/common.cuh"
#include "roll.cuh"

static __forceinline__ __device__ int64_t wrap_index(const int64_t idx, const int64_t ne) {
    if (idx < 0) {
        return idx + ne;
    }
    if (idx >= ne) {
        return idx - ne;
    }
    return idx;
}

static __global__ void roll_f32_cuda(const float * __restrict__ src,
                                     float * __restrict__ dst,
                                     const int64_t ne00,
                                     const int64_t ne01,
                                     const int64_t ne02,
                                     const int64_t ne03,
                                     const int     s0,
                                     const int     s1,
                                     const int     s2,
                                     const int     s3) {
    const int64_t idx        = int64_t(blockDim.x) * blockIdx.x + threadIdx.x;
    const int64_t n_elements = ne00 * ne01 * ne02 * ne03;

    if (idx >= n_elements) {
        return;
    }

    const int64_t i0 = idx % ne00;
    const int64_t i1 = (idx / ne00) % ne01;
    const int64_t i2 = (idx / (ne00 * ne01)) % ne02;
    const int64_t i3 = (idx / (ne00 * ne01 * ne02)) % ne03;

    const int64_t d0 = wrap_index(i0 - s0, ne00);
    const int64_t d1 = wrap_index(i1 - s1, ne01);
    const int64_t d2 = wrap_index(i2 - s2, ne02);
    const int64_t d3 = wrap_index(i3 - s3, ne03);

    dst[i3 * (ne00 * ne01 * ne02) + i2 * (ne01 * ne00) + i1 * ne00 + i0] =
        src[d3 * (ne00 * ne01 * ne02) + d2 * (ne01 * ne00) + d1 * ne00 + d0];
}

void ggml_cuda_op_roll(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    int s0 = dst->op_params[0];
    int s1 = dst->op_params[1];
    int s2 = dst->op_params[2];
    int s3 = dst->op_params[3];

    const ggml_tensor * src0   = dst->src[0];
    const float *       src0_d = (const float *) dst->src[0]->data;
    float *             dst_d  = (float *) dst->data;

    GGML_TENSOR_UNARY_OP_LOCALS;

    GGML_ASSERT(dst->src[0]->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_are_same_shape(dst->src[0], dst));

    hipStream_t stream = ctx.stream();

    int64_t sz         = (ne00 * ne01 * ne02 * ne03);
    int64_t num_blocks = (sz + CUDA_ROLL_BLOCK_SIZE - 1) / CUDA_ROLL_BLOCK_SIZE;

    roll_f32_cuda<<<num_blocks, CUDA_ROLL_BLOCK_SIZE, 0, stream>>>(
        src0_d, dst_d, ne00, ne01, ne02, ne03, s0, s1, s2, s3);
}

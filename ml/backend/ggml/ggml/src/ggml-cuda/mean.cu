#include "hip/hip_runtime.h"
#include "mean.cuh"
#include "reduce_rows.cuh"

#ifdef GGML_CUDA_USE_CUB
#include <hipcub/hipcub.hpp>
using namespace hipcub;
#endif  // GGML_CUDA_USE_CUB

template <typename T> __global__ void divide_by_count(T * result, size_t count) {
    *result /= static_cast<T>(count);
}

void ggml_cuda_op_mean(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0   = dst->src[0];
    const float *       src0_d = (const float *) src0->data;
    float *             dst_d  = (float *) dst->data;
    hipStream_t        stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

// Special case for reducing vectors
#ifdef GGML_CUDA_USE_CUB
#ifdef USE_CUDA_GRAPH
    hipStreamCaptureStatus iscapturing;
    CUDA_CHECK(hipStreamIsCapturing(stream, &iscapturing));
#endif // USE_CUDA_GRAPH
    if ((nrows == 1) &&
#ifdef USE_CUDA_GRAPH
            // CUDA_GRAPHS_DISABLED
            ((ncols > 65536) &&
             ((ctx.cuda_graph->instance == nullptr) && (iscapturing == hipStreamCaptureStatusNone) ||
              ctx.cuda_graph->disable_due_to_gpu_arch || ctx.cuda_graph->disable_due_to_too_many_updates ||
              ctx.cuda_graph->disable_due_to_failed_graph_capture)) ||
        // CUDA_GRAPHS ENABLED
        ((ncols > 32768) &&
         !((ctx.cuda_graph->instance == nullptr) && (iscapturing == hipStreamCaptureStatusNone) ||
           ctx.cuda_graph->disable_due_to_gpu_arch || ctx.cuda_graph->disable_due_to_too_many_updates ||
           ctx.cuda_graph->disable_due_to_failed_graph_capture))) {
#else
        (ncols > 65536)) {
#endif // USE_CUDA_GRAPH
        // Single row - use device-wide reduction
        size_t           tmp_size = 0;
        ggml_cuda_pool & pool     = ctx.pool();

        DeviceReduce::Sum(nullptr, tmp_size, src0_d, dst_d, ncols, stream);

        ggml_cuda_pool_alloc<uint8_t> tmp_alloc(pool, tmp_size);
        DeviceReduce::Sum(tmp_alloc.ptr, tmp_size, src0_d, dst_d, ncols, stream);

        // Divide by ncols
        divide_by_count<float><<<1, 1, 0, stream>>>(dst_d, ncols);
        return;
    }
#endif // GGML_CUDA_USE_CUB

    const dim3 block_nums(nrows, 1, 1);

    const int id  = ggml_cuda_get_device();
    const int nsm = ggml_cuda_info().devices[id].nsm;
    if ((nrows / nsm) < 2) {
        const dim3 block_dims(512, 1, 1);
        reduce_rows_f32</*norm=*/true><<<block_nums, block_dims, 0, stream>>>(src0_d, dst_d, ncols);
    } else {
        const dim3 block_dims(ncols < 1024 ? 32 : 128, 1, 1);
        reduce_rows_f32</*norm=*/true><<<block_nums, block_dims, 0, stream>>>(src0_d, dst_d, ncols);
    }
}

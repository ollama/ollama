#include "hip/hip_runtime.h"
#include "mean.cuh"

void ggml_cuda_op_mean(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0   = dst->src[0];
    const float *       src0_d = (const float *) src0->data;
    float *             dst_d  = (float *) dst->data;
    hipStream_t        stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums(nrows, 1, 1);
    reduce_rows_f32</*norm*/ true><<<block_nums, block_dims, 0, stream>>>(src0_d, dst_d, ncols);
}

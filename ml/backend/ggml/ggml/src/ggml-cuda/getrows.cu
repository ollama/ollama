#include "hip/hip_runtime.h"
#include "getrows.cuh"
#include "dequantize.cuh"

template<int qk, int qr, dequantize_kernel_t dequantize_kernel, typename dst_t>
static __global__ void k_get_rows(
        const void * __restrict__ src0, const int32_t * __restrict__ src1, dst_t * __restrict__ dst,
        const int64_t ne00, /*const int64_t ne01, const int64_t ne02, const int64_t ne03,*/
        /*const int64_t ne10, const int64_t ne11,*/ const int64_t ne12, /*const int64_t ne13,*/
        /*const size_t s0,*/ const size_t s1, const size_t s2, const size_t s3,
        /*const size_t nb00,*/ const size_t nb01, const size_t nb02, const size_t nb03,
        const size_t s10, const size_t s11, const size_t s12/*, const size_t s13*/) {

    const int i00 = (blockIdx.x*blockDim.x + threadIdx.x)*2;
    const int i10 =  blockDim.y*blockIdx.y + threadIdx.y;
    const int i11 = (blockIdx.z*blockDim.z + threadIdx.z)/ne12;
    const int i12 = (blockIdx.z*blockDim.z + threadIdx.z)%ne12;

    if (i00 >= ne00) {
        return;
    }

    const int i01 = src1[i10*s10 + i11*s11 + i12*s12];

    dst_t * dst_row = dst + i10*s1 + i11*s2 + i12*s3;
    const void * src0_row = (const char *) src0 + i01*nb01 + i11*nb02 + i12*nb03;

    const int ib   =  i00/qk;      // block index
    const int iqs  = (i00%qk)/qr;  // quant index
    const int iybs = i00 - i00%qk; // dst block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    dfloat2 v;
    dequantize_kernel(src0_row, ib, iqs, v);

    dst_row[iybs + iqs + 0]        = v.x;
    dst_row[iybs + iqs + y_offset] = v.y;
}

template<typename src0_t, typename dst_t>
static __global__ void k_get_rows_float(
        const src0_t * __restrict__ src0, const int32_t * __restrict__ src1, dst_t * __restrict__ dst,
        const int64_t ne00, /*const int64_t ne01, const int64_t ne02, const int64_t ne03,*/
        /*const int64_t ne10, const int64_t ne11,*/ const int64_t ne12, /*const int64_t ne13,*/
        /*const size_t s0,*/ const size_t s1, const size_t s2, const size_t s3,
        /*const size_t nb00,*/ const size_t nb01, const size_t nb02, const size_t nb03,
        const size_t s10, const size_t s11, const size_t s12/*, const size_t s13*/) {

    const int i00 =  blockIdx.x*blockDim.x + threadIdx.x;
    const int i10 =  blockDim.y*blockIdx.y + threadIdx.y;
    const int i11 = (blockIdx.z*blockDim.z + threadIdx.z)/ne12;
    const int i12 = (blockIdx.z*blockDim.z + threadIdx.z)%ne12;

    if (i00 >= ne00) {
        return;
    }

    const int i01 = src1[i10*s10 + i11*s11 + i12*s12];

    dst_t * dst_row = dst + i10*s1 + i11*s2 + i12*s3;
    const src0_t * src0_row = (const src0_t *)((const char *) src0 + i01*nb01 + i11*nb02 + i12*nb03);

    dst_row[i00] = src0_row[i00];
}

template<typename grad_t, typename dst_t>
static __global__ void k_get_rows_back_float(
        const grad_t * __restrict__ grad, const int32_t * __restrict__ rows, dst_t * __restrict__ dst, const int64_t ncols, const int64_t nrows_grad) {
    const int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (col >= ncols) {
        return;
    }

    const int dst_row = blockIdx.y*blockDim.y + threadIdx.y;

    float sum = 0.0f;

    for (int64_t i = 0; i < nrows_grad; ++i) {
        if (rows[i] != dst_row) {
            continue;
        }
        sum += grad[i*ncols + col];
    }

    dst[dst_row*ncols + col] = sum;
}

template<int qk, int qr, dequantize_kernel_t dq>
static void get_rows_cuda(
        const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
        const void * src0_dd, const int32_t * src1_dd, float * dst_dd, hipStream_t stream) {

    GGML_TENSOR_BINARY_OP_LOCALS

    const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ne00 + 2*CUDA_GET_ROWS_BLOCK_SIZE - 1) / (2*CUDA_GET_ROWS_BLOCK_SIZE);
    const dim3 block_nums(block_num_x, ne10, ne11*ne12);

    // strides in elements
    //const size_t s0 = nb0 / ggml_element_size(dst);
    const size_t s1 = nb1 / ggml_element_size(dst);
    const size_t s2 = nb2 / ggml_element_size(dst);
    const size_t s3 = nb3 / ggml_element_size(dst);

    const size_t s10 = nb10 / ggml_element_size(src1);
    const size_t s11 = nb11 / ggml_element_size(src1);
    const size_t s12 = nb12 / ggml_element_size(src1);
    //const size_t s13 = nb13 / ggml_element_size(src1);

    GGML_ASSERT(ne00 % 2 == 0);

    k_get_rows<qk, qr, dq><<<block_nums, block_dims, 0, stream>>>(
        src0_dd, src1_dd, dst_dd,
        ne00, /*ne01, ne02, ne03,*/
        /*ne10, ne11,*/ ne12, /*ne13,*/
        /* s0,*/ s1, s2, s3,
        /* nb00,*/ nb01, nb02, nb03,
        s10, s11, s12/*, s13*/);

    GGML_UNUSED(dst);
}

template<typename src0_t>
static void get_rows_cuda_float(
        const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
        const src0_t * src0_dd, const int32_t * src1_dd, float * dst_dd, hipStream_t stream) {

    GGML_TENSOR_BINARY_OP_LOCALS

    GGML_ASSERT(ne13 == 1);

    const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ne00 + CUDA_GET_ROWS_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BLOCK_SIZE;
    const dim3 block_nums(block_num_x, ne10, ne11*ne12);

    // strides in elements
    //const size_t s0 = nb0 / ggml_element_size(dst);
    const size_t s1 = nb1 / ggml_element_size(dst);
    const size_t s2 = nb2 / ggml_element_size(dst);
    const size_t s3 = nb3 / ggml_element_size(dst);

    const size_t s10 = nb10 / ggml_element_size(src1);
    const size_t s11 = nb11 / ggml_element_size(src1);
    const size_t s12 = nb12 / ggml_element_size(src1);
    //const size_t s13 = nb13 / ggml_element_size(src1);

    k_get_rows_float<<<block_nums, block_dims, 0, stream>>>(
        src0_dd, src1_dd, dst_dd,
        ne00, /*ne01, ne02, ne03,*/
        /*ne10, ne11,*/ ne12, /*ne13,*/
        /* s0,*/ s1, s2, s3,
        /* nb00,*/ nb01, nb02, nb03,
        s10, s11, s12/*, s13*/);

    GGML_UNUSED(dst);
}

void ggml_cuda_op_get_rows(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    const void    * src0_d = (const void    *) src0->data;
    const int32_t * src1_d = (const int32_t *) src1->data;
    float         * dst_d  = (float         *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src1->type == GGML_TYPE_I32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    GGML_ASSERT(src0->nb[0] == ggml_type_size(src0->type));
    GGML_ASSERT(src1->nb[0] == ggml_type_size(src1->type));
    GGML_ASSERT(dst->nb[0]  == ggml_type_size(dst->type));

    switch (src0->type) {
        case GGML_TYPE_F16:
            get_rows_cuda_float(src0, src1, dst, (const half *) src0_d, src1_d, dst_d, stream);
            break;
        case GGML_TYPE_F32:
            get_rows_cuda_float(src0, src1, dst, (const float *) src0_d, src1_d, dst_d, stream);
            break;
        case GGML_TYPE_Q4_0:
            get_rows_cuda<QK4_0, QR4_0, dequantize_q4_0>(src0, src1, dst, src0_d, src1_d, dst_d, stream);
            break;
        case GGML_TYPE_Q4_1:
            get_rows_cuda<QK4_1, QR4_1, dequantize_q4_1>(src0, src1, dst, src0_d, src1_d, dst_d, stream);
            break;
        case GGML_TYPE_Q5_0:
            get_rows_cuda<QK5_0, QR5_0, dequantize_q5_0>(src0, src1, dst, src0_d, src1_d, dst_d, stream);
            break;
        case GGML_TYPE_Q5_1:
            get_rows_cuda<QK5_1, QR5_1, dequantize_q5_1>(src0, src1, dst, src0_d, src1_d, dst_d, stream);
            break;
        case GGML_TYPE_Q8_0:
            get_rows_cuda<QK8_0, QR8_0, dequantize_q8_0>(src0, src1, dst, src0_d, src1_d, dst_d, stream);
            break;
        default:
            // TODO: k-quants
            GGML_ABORT("%s: unsupported type: %s\n", __func__, ggml_type_name(src0->type));
            break;
    }
}

void ggml_cuda_op_get_rows_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // gradients of forward pass output
    const ggml_tensor * src1 = dst->src[1]; // src1 in forward pass

    GGML_TENSOR_BINARY_OP_LOCALS

    const float   * src0_d = (const float   *) src0->data;
    const int32_t * src1_d = (const int32_t *) src1->data;
    float         * dst_d  = (float         *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_I32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(dst));

    GGML_ASSERT(ne02*ne03 == 1);
    GGML_ASSERT(ne12*ne13 == 1);
    GGML_ASSERT(ne2*ne3 == 1);

    const dim3 block_dims(CUDA_GET_ROWS_BACK_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ne00 + CUDA_GET_ROWS_BACK_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BACK_BLOCK_SIZE;
    const dim3 block_nums(block_num_x, ne1, 1);

    k_get_rows_back_float<<<block_nums, block_dims, 0, stream>>>(src0_d, src1_d, dst_d, ne00, ne10);
}

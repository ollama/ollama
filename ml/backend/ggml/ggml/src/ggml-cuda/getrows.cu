#include "hip/hip_runtime.h"
#include "getrows.cuh"
#include "dequantize.cuh"

template<int qk, int qr, dequantize_kernel_t dequantize_kernel, typename dst_t>
static __global__ void k_get_rows(
            const void * src0, const int32_t * src1, dst_t * dst,
            int64_t ne00, /*int64_t ne01, int64_t ne02, int64_t ne03,*/
            /*int64_t ne10, int64_t ne11,*/ int64_t ne12, /*int64_t ne13,*/
            /*size_t s0,*/ size_t s1, size_t s2, size_t s3,
            /*size_t nb00,*/ size_t nb01, size_t nb02, size_t nb03,
            size_t s10, size_t s11, size_t s12/*, size_t s13*/) {

    const int i00 = (blockIdx.x*blockDim.x + threadIdx.x)*2;
    const int i10 = blockDim.y*blockIdx.y + threadIdx.y;
    const int i11 = (blockIdx.z*blockDim.z + threadIdx.z)/ne12;
    const int i12 = (blockIdx.z*blockDim.z + threadIdx.z)%ne12;

    if (i00 >= ne00) {
        return;
    }

    const int i01 = src1[i10*s10 + i11*s11 + i12*s12];

    dst_t * dst_row = dst + i10*s1 + i11*s2 + i12*s3;
    const void * src0_row = (const char *)src0 + i01*nb01 + i11*nb02 + i12*nb03;

    const int ib = i00/qk; // block index
    const int iqs = (i00%qk)/qr; // quant index
    const int iybs = i00 - i00%qk; // dst block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    dfloat2 v;
    dequantize_kernel(src0_row, ib, iqs, v);

    dst_row[iybs + iqs + 0]        = v.x;
    dst_row[iybs + iqs + y_offset] = v.y;
}

template<typename src0_t, typename dst_t>
static __global__ void k_get_rows_float(
            const src0_t * src0, const int32_t * src1, dst_t * dst,
            int64_t ne00, /*int64_t ne01, int64_t ne02, int64_t ne03,*/
            /*int64_t ne10, int64_t ne11,*/ int64_t ne12, /*int64_t ne13,*/
            /*size_t s0,*/ size_t s1, size_t s2, size_t s3,
            /*size_t nb00,*/ size_t nb01, size_t nb02, size_t nb03,
            size_t s10, size_t s11, size_t s12/*, size_t s13*/) {

    const int i00 = blockIdx.x*blockDim.x + threadIdx.x;
    const int i10 = blockDim.y*blockIdx.y + threadIdx.y;
    const int i11 = (blockIdx.z*blockDim.z + threadIdx.z)/ne12;
    const int i12 = (blockIdx.z*blockDim.z + threadIdx.z)%ne12;

    if (i00 >= ne00) {
        return;
    }

    const int i01 = src1[i10*s10 + i11*s11 + i12*s12];

    dst_t * dst_row = dst + i10*s1 + i11*s2 + i12*s3;
    const src0_t * src0_row = (const src0_t *)((const char *)src0 + i01*nb01 + i11*nb02 + i12*nb03);

    dst_row[i00] = src0_row[i00];
}

template<int qk, int qr, dequantize_kernel_t dq>
static void get_rows_cuda(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
                            const void * src0_dd, const int32_t * src1_dd, float * dst_dd, hipStream_t stream) {

    GGML_TENSOR_BINARY_OP_LOCALS

    const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ne00 + 2*CUDA_GET_ROWS_BLOCK_SIZE - 1) / (2*CUDA_GET_ROWS_BLOCK_SIZE);
    const dim3 block_nums(block_num_x, ne10, ne11*ne12);

    // strides in elements
    //const size_t s0 = nb0 / ggml_element_size(dst);
    const size_t s1 = nb1 / ggml_element_size(dst);
    const size_t s2 = nb2 / ggml_element_size(dst);
    const size_t s3 = nb3 / ggml_element_size(dst);

    const size_t s10 = nb10 / ggml_element_size(src1);
    const size_t s11 = nb11 / ggml_element_size(src1);
    const size_t s12 = nb12 / ggml_element_size(src1);
    //const size_t s13 = nb13 / ggml_element_size(src1);

    GGML_ASSERT(ne00 % 2 == 0);

    k_get_rows<qk, qr, dq><<<block_nums, block_dims, 0, stream>>>(
            src0_dd, src1_dd, dst_dd,
            ne00, /*ne01, ne02, ne03,*/
            /*ne10, ne11,*/ ne12, /*ne13,*/
            /* s0,*/ s1, s2, s3,
            /* nb00,*/ nb01, nb02, nb03,
            s10, s11, s12/*, s13*/);

    GGML_UNUSED(dst);
}

template<typename src0_t>
static void get_rows_cuda_float(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
                                const src0_t * src0_dd, const int32_t * src1_dd, float * dst_dd, hipStream_t stream) {

    GGML_TENSOR_BINARY_OP_LOCALS

    const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ne00 + CUDA_GET_ROWS_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BLOCK_SIZE;
    const dim3 block_nums(block_num_x, ne10, ne11*ne12);

    // strides in elements
    //const size_t s0 = nb0 / ggml_element_size(dst);
    const size_t s1 = nb1 / ggml_element_size(dst);
    const size_t s2 = nb2 / ggml_element_size(dst);
    const size_t s3 = nb3 / ggml_element_size(dst);

    const size_t s10 = nb10 / ggml_element_size(src1);
    const size_t s11 = nb11 / ggml_element_size(src1);
    const size_t s12 = nb12 / ggml_element_size(src1);
    //const size_t s13 = nb13 / ggml_element_size(src1);

    k_get_rows_float<<<block_nums, block_dims, 0, stream>>>(
            src0_dd, src1_dd, dst_dd,
            ne00, /*ne01, ne02, ne03,*/
            /*ne10, ne11,*/ ne12, /*ne13,*/
            /* s0,*/ s1, s2, s3,
            /* nb00,*/ nb01, nb02, nb03,
            s10, s11, s12/*, s13*/);

    GGML_UNUSED(dst);
}

void ggml_cuda_op_get_rows(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();


    GGML_ASSERT(src1->type == GGML_TYPE_I32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    GGML_ASSERT(src0->nb[0] == ggml_type_size(src0->type));
    GGML_ASSERT(src1->nb[0] == ggml_type_size(src1->type));
    GGML_ASSERT(dst->nb[0] == ggml_type_size(dst->type));

    const int32_t * src1_i32 = (const int32_t *) src1_d;

    switch (src0->type) {
        case GGML_TYPE_F16:
            get_rows_cuda_float(src0, src1, dst, (const half *)src0_d, src1_i32, dst_d, stream);
            break;
        case GGML_TYPE_F32:
            get_rows_cuda_float(src0, src1, dst, src0_d, src1_i32, dst_d, stream);
            break;
        case GGML_TYPE_Q4_0:
            get_rows_cuda<QK4_0, QR4_0, dequantize_q4_0>(src0, src1, dst, src0_d, src1_i32, dst_d, stream);
            break;
        case GGML_TYPE_Q4_1:
            get_rows_cuda<QK4_1, QR4_1, dequantize_q4_1>(src0, src1, dst, src0_d, src1_i32, dst_d, stream);
            break;
        case GGML_TYPE_Q5_0:
            get_rows_cuda<QK5_0, QR5_0, dequantize_q5_0>(src0, src1, dst, src0_d, src1_i32, dst_d, stream);
            break;
        case GGML_TYPE_Q5_1:
            get_rows_cuda<QK5_1, QR5_1, dequantize_q5_1>(src0, src1, dst, src0_d, src1_i32, dst_d, stream);
            break;
        case GGML_TYPE_Q8_0:
            get_rows_cuda<QK8_0, QR8_0, dequantize_q8_0>(src0, src1, dst, src0_d, src1_i32, dst_d, stream);
            break;
        default:
            // TODO: k-quants
            GGML_ABORT("%s: unsupported type: %s\n", __func__, ggml_type_name(src0->type));
            break;
    }
}

#include "hip/hip_runtime.h"
#include "getrows.cuh"
#include "dequantize.cuh"

template<int qk, int qr, dequantize_kernel_t dequantize_kernel, typename dst_t>
static __global__ void k_get_rows(
        const void * __restrict__ src0, const int32_t * __restrict__ src1, dst_t * __restrict__ dst,
        const int64_t ne00, /*const int64_t ne01, const int64_t ne02, const int64_t ne03,*/
        /*const int64_t ne10, const int64_t ne11,*/ const int64_t ne12, /*const int64_t ne13,*/
        /*const size_t s0,*/ const size_t s1, const size_t s2, const size_t s3,
        /*const size_t nb00,*/ const size_t nb01, const size_t nb02, const size_t nb03,
        const size_t s10, const size_t s11, const size_t s12/*, const size_t s13*/) {

    // The x and y dimensions of the grid are swapped because the maximum allowed grid size for x is higher.
    const int i00 = (blockIdx.y * blockDim.x + threadIdx.x)*2;
    const int i10 =  blockIdx.x;
    const int i11 =  blockIdx.z / ne12;
    const int i12 =  blockIdx.z % ne12;

    if (i00 >= ne00) {
        return;
    }

    const int i01 = src1[i10*s10 + i11*s11 + i12*s12];

    dst_t * dst_row = dst + i10*s1 + i11*s2 + i12*s3;
    const void * src0_row = (const char *) src0 + i01*nb01 + i11*nb02 + i12*nb03;

    const int ib   =  i00/qk;      // block index
    const int iqs  = (i00%qk)/qr;  // quant index
    const int iybs = i00 - i00%qk; // dst block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    dfloat2 v;
    dequantize_kernel(src0_row, ib, iqs, v);

    dst_row[iybs + iqs + 0]        = float(v.x);
    dst_row[iybs + iqs + y_offset] = float(v.y);
}

template<typename src0_t, typename dst_t>
static __global__ void k_get_rows_float(
        const src0_t * __restrict__ src0, const int32_t * __restrict__ src1, dst_t * __restrict__ dst,
        const int64_t ne00, /*const int64_t ne01, const int64_t ne02, const int64_t ne03,*/
        /*const int64_t ne10, const int64_t ne11,*/ const int64_t ne12, /*const int64_t ne13,*/
        /*const size_t s0,*/ const size_t s1, const size_t s2, const size_t s3,
        /*const size_t nb00,*/ const size_t nb01, const size_t nb02, const size_t nb03,
        const size_t s10, const size_t s11, const size_t s12/*, const size_t s13*/) {

    // The x and y dimensions of the grid are swapped because the maximum allowed grid size for x is higher.
    const int i00 = blockIdx.y * blockDim.x + threadIdx.x;
    const int i10 = blockIdx.x;
    const int i11 = blockIdx.z / ne12;
    const int i12 = blockIdx.z % ne12;

    if (i00 >= ne00) {
        return;
    }

    const int i01 = src1[i10*s10 + i11*s11 + i12*s12];

    dst_t * dst_row = dst + i10*s1 + i11*s2 + i12*s3;
    const src0_t * src0_row = (const src0_t *)((const char *) src0 + i01*nb01 + i11*nb02 + i12*nb03);

    dst_row[i00] = float(src0_row[i00]);
}

template<typename grad_t, typename dst_t>
static __global__ void k_get_rows_back_float(
        const grad_t * __restrict__ grad, const int32_t * __restrict__ rows, dst_t * __restrict__ dst, const int64_t ncols, const int64_t nrows_grad) {
    const int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (col >= ncols) {
        return;
    }

    const int dst_row = blockIdx.y*blockDim.y + threadIdx.y;

    float sum = 0.0f;

    for (int64_t i = 0; i < nrows_grad; ++i) {
        if (rows[i] != dst_row) {
            continue;
        }
        sum += grad[i*ncols + col];
    }

    dst[dst_row*ncols + col] = sum;
}

template<int qk, int qr, dequantize_kernel_t dq, typename dst_t>
static void get_rows_cuda_q(
        const void * src0_d, const int32_t * src1_d, dst_t * dst_d,
        const int64_t ne00, const size_t nb01, const size_t nb02, const size_t nb03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const size_t nb10, const size_t nb11, const size_t nb12,
        const size_t nb1, const size_t nb2, const size_t nb3,
        hipStream_t stream) {
    const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
    const int block_num_y = (ne00 + 2*CUDA_GET_ROWS_BLOCK_SIZE - 1) / (2*CUDA_GET_ROWS_BLOCK_SIZE);
    const dim3 block_nums(ne10, block_num_y, ne11*ne12);

    // strides in elements
    // const size_t s0 = nb0 / sizeof(dst_t);
    const size_t s1 = nb1 / sizeof(dst_t);
    const size_t s2 = nb2 / sizeof(dst_t);
    const size_t s3 = nb3 / sizeof(dst_t);

    const size_t s10 = nb10 / sizeof(int32_t);
    const size_t s11 = nb11 / sizeof(int32_t);
    const size_t s12 = nb12 / sizeof(int32_t);
    // const size_t s13 = nb13 / sizeof(int32_t);

    GGML_ASSERT(ne00 % 2 == 0);

    k_get_rows<qk, qr, dq><<<block_nums, block_dims, 0, stream>>>(
        src0_d, src1_d, dst_d,
        ne00, /*ne01, ne02, ne03,*/
        /*ne10, ne11,*/ ne12, /*ne13,*/
        /* s0,*/ s1, s2, s3,
        /* nb00,*/ nb01, nb02, nb03,
        s10, s11, s12/*, s13*/);
}

template<typename src0_t, typename dst_t>
static void get_rows_cuda_float(
        const src0_t * src0_d, const int32_t * src1_d, dst_t * dst_d,
        const int64_t ne00, const size_t nb01, const size_t nb02, const size_t nb03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const size_t nb10, const size_t nb11, const size_t nb12,
        const size_t nb1, const size_t nb2, const size_t nb3,
        hipStream_t stream) {
    const dim3 block_dims(CUDA_GET_ROWS_BLOCK_SIZE, 1, 1);
    const int block_num_y = (ne00 + CUDA_GET_ROWS_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BLOCK_SIZE;
    const dim3 block_nums(ne10, block_num_y, ne11*ne12);

    // strides in elements
    // const size_t s0 = nb0 / sizeof(dst_t);
    const size_t s1 = nb1 / sizeof(dst_t);
    const size_t s2 = nb2 / sizeof(dst_t);
    const size_t s3 = nb3 / sizeof(dst_t);

    const size_t s10 = nb10 / sizeof(int32_t);
    const size_t s11 = nb11 / sizeof(int32_t);
    const size_t s12 = nb12 / sizeof(int32_t);
    // const size_t s13 = nb13 / sizeof(int32_t);

    k_get_rows_float<<<block_nums, block_dims, 0, stream>>>(
        src0_d, src1_d, dst_d,
        ne00, /*ne01, ne02, ne03,*/
        /*ne10, ne11,*/ ne12, /*ne13,*/
        /* s0,*/ s1, s2, s3,
        /* nb00,*/ nb01, nb02, nb03,
        s10, s11, s12/*, s13*/);
}

template <typename dst_t>
static void ggml_cuda_get_rows_switch_src0_type(
        const void * src0_d, const ggml_type src0_type, const int32_t * src1_d, dst_t * dst_d,
        const int64_t ne00, const size_t nb01, const size_t nb02, const size_t nb03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const size_t nb10, const size_t nb11, const size_t nb12,
        const size_t nb1, const size_t nb2, const size_t nb3,
        hipStream_t stream) {
    switch (src0_type) {
        case GGML_TYPE_F16:
            get_rows_cuda_float((const half *) src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_F32:
            get_rows_cuda_float((const float *) src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_BF16:
            get_rows_cuda_float((const hip_bfloat16 *) src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_Q4_0:
            get_rows_cuda_q<QK4_0, QR4_0, dequantize_q4_0>(src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_Q4_1:
            get_rows_cuda_q<QK4_1, QR4_1, dequantize_q4_1>(src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_Q5_0:
            get_rows_cuda_q<QK5_0, QR5_0, dequantize_q5_0>(src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_Q5_1:
            get_rows_cuda_q<QK5_1, QR5_1, dequantize_q5_1>(src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_Q8_0:
            get_rows_cuda_q<QK8_0, QR8_0, dequantize_q8_0>(src0_d, src1_d, dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        default:
            // TODO: k-quants
            GGML_ABORT("%s: unsupported src0 type: %s\n", __func__, ggml_type_name(src0_type));
            break;
    }
}

void get_rows_cuda(
        const void * src0_d, ggml_type src0_type, const int32_t * src1_d, void * dst_d, ggml_type dst_type,
        int64_t ne00, size_t nb01, size_t nb02, size_t nb03,
        int64_t ne10, int64_t ne11, int64_t ne12, size_t nb10, size_t nb11, size_t nb12,
        size_t nb1, size_t nb2, size_t nb3,
        hipStream_t stream) {
    switch (dst_type) {
        case GGML_TYPE_F32:
            ggml_cuda_get_rows_switch_src0_type(src0_d, src0_type, src1_d, (float *) dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_F16:
            ggml_cuda_get_rows_switch_src0_type(src0_d, src0_type, src1_d, (half *) dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        case GGML_TYPE_BF16:
            ggml_cuda_get_rows_switch_src0_type(src0_d, src0_type, src1_d, (hip_bfloat16 *) dst_d,
                ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
            break;
        default:
            GGML_ABORT("%s: unsupported dst type: %s\n", __func__, ggml_type_name(dst_type));
            break;
    }
}

void ggml_cuda_op_get_rows(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    hipStream_t stream = ctx.stream();

    GGML_TENSOR_BINARY_OP_LOCALS

    GGML_ASSERT(src1->type == GGML_TYPE_I32);
    GGML_ASSERT(ne13 == 1);

    GGML_ASSERT(src0->nb[0] == ggml_type_size(src0->type));
    GGML_ASSERT(src1->nb[0] == ggml_type_size(src1->type));
    GGML_ASSERT(dst->nb[0]  == ggml_type_size(dst->type));

    get_rows_cuda(src0->data, src0->type, (const int32_t *) src1->data, dst->data, dst->type,
        ne00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb1, nb2, nb3, stream);
}

void ggml_cuda_op_get_rows_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // gradients of forward pass output
    const ggml_tensor * src1 = dst->src[1]; // src1 in forward pass

    GGML_TENSOR_BINARY_OP_LOCALS

    const float   * src0_d = (const float   *) src0->data;
    const int32_t * src1_d = (const int32_t *) src1->data;
    float         * dst_d  = (float         *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_I32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(dst));

    GGML_ASSERT(ne02*ne03 == 1);
    GGML_ASSERT(ne12*ne13 == 1);
    GGML_ASSERT(ne2*ne3 == 1);

    const dim3 block_dims(CUDA_GET_ROWS_BACK_BLOCK_SIZE, 1, 1);
    const int block_num_x = (ne00 + CUDA_GET_ROWS_BACK_BLOCK_SIZE - 1) / CUDA_GET_ROWS_BACK_BLOCK_SIZE;
    const dim3 block_nums(block_num_x, ne1, 1);

    k_get_rows_back_float<<<block_nums, block_dims, 0, stream>>>(src0_d, src1_d, dst_d, ne00, ne10);
}

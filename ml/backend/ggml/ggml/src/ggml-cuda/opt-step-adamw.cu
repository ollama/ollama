#include "hip/hip_runtime.h"
#include "ggml-impl.h"
#include "opt-step-adamw.cuh"

#include <cstdint>

static __global__ void opt_step_adamw_f32(
    float * __restrict__ x, const float * __restrict__ g, float * __restrict__ g_m, float * __restrict__ g_v,
    const float * __restrict__ pars, const int64_t k) {

    const int64_t i = (int64_t) blockIdx.x*blockDim.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    const float alpha  = pars[0];
    const float beta1  = pars[1];
    const float beta2  = pars[2];
    const float eps    = pars[3];
    const float wd     = pars[4];
    const float beta1h = pars[5];
    const float beta2h = pars[6];

    const float gi = g[i];
    const float gmi = g_m[i]*beta1 +    gi*(1.0f - beta1);
    const float gvi = g_v[i]*beta2 + gi*gi*(1.0f - beta2);

    g_m[i] = gmi;
    g_v[i] = gvi;

    const float mh =       gmi*beta1h;
    const float vh = sqrtf(gvi*beta2h) + eps;

    x[i] = x[i]*(1.0f - alpha*wd) - alpha*mh/vh;
}

static void opt_step_adamw_f32_cuda(
    float * x, const float * g, float * g_m, float * g_v, const float * pars, const int64_t k, hipStream_t stream) {

    const dim3 block_dims(CUDA_OPT_STEP_ADAMW_BLOCK_SIZE, 1, 1);
    const dim3 block_nums((k + CUDA_OPT_STEP_ADAMW_BLOCK_SIZE - 1) / CUDA_OPT_STEP_ADAMW_BLOCK_SIZE, 1, 1);
    opt_step_adamw_f32<<<block_nums, block_dims, 0, stream>>>(x, g, g_m, g_v, pars, k);
}

void ggml_cuda_opt_step_adamw(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0         = dst->src[0];
    const ggml_tensor * src0_grad    = dst->src[1];
    const ggml_tensor * src0_grad_m  = dst->src[2];
    const ggml_tensor * src0_grad_v  = dst->src[3];
    const ggml_tensor * adamw_params = dst->src[4];

    GGML_ASSERT(src0->type         == GGML_TYPE_F32);
    GGML_ASSERT(src0_grad->type    == GGML_TYPE_F32);
    GGML_ASSERT(src0_grad_m->type  == GGML_TYPE_F32);
    GGML_ASSERT(src0_grad_v->type  == GGML_TYPE_F32);
    GGML_ASSERT(adamw_params->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src0_grad));
    GGML_ASSERT(ggml_is_contiguous(src0_grad_m));
    GGML_ASSERT(ggml_is_contiguous(src0_grad_v));
    GGML_ASSERT(ggml_is_contiguous(adamw_params));
    GGML_ASSERT(ggml_are_same_shape(src0, src0_grad));
    GGML_ASSERT(ggml_are_same_shape(src0, src0_grad_m));
    GGML_ASSERT(ggml_are_same_shape(src0, src0_grad_v));
    GGML_ASSERT(ggml_nelements(adamw_params) == 7);

    float       * src0_d         = (float       *) src0->data;
    const float * src0_grad_d    = (const float *) src0_grad->data;
    float       * src0_grad_m_d  = (float       *) src0_grad_m->data;
    float       * src0_grad_v_d  = (float       *) src0_grad_v->data;
    const float * adamw_params_d = (const float *) adamw_params->data;

    hipStream_t stream = ctx.stream();

    const int64_t ne = ggml_nelements(src0);

    opt_step_adamw_f32_cuda(src0_d, src0_grad_d, src0_grad_m_d, src0_grad_v_d, adamw_params_d, ne, stream);
}

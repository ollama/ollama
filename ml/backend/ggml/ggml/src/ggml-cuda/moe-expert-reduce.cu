#include "hip/hip_runtime.h"
#include "moe-expert-reduce.cuh"

// This kernel is a fusion of the expert weight reduce, common in MoE models

template <int n_expert_used_template>
__global__ void moe_expert_reduce_cuda(const float * __restrict__ experts,
                                       const float * __restrict__ weights,
                                       float * __restrict__ dst,
                                       const int n_expert_used,
                                       const int n_cols) {
    const int row = blockIdx.x;
    const int col = blockIdx.y * blockDim.x + threadIdx.x;
    if (col >= n_cols) {
        return;
    }

    experts += row * n_cols * n_expert_used;
    weights += row * n_expert_used;
    dst += row * n_cols;

    float acc = 0.f;
    if constexpr (n_expert_used_template == 0) {
        for (int expert = 0; expert < n_expert_used; ++expert) {
            ggml_cuda_mad(acc, experts[col], weights[expert]);
            experts += n_cols;
        }
        dst[col] = acc;
    } else {
#pragma unroll
        for (int i = 0; i < n_expert_used_template; ++i) {
            ggml_cuda_mad(acc, experts[col], weights[i]);
            experts += n_cols;
        }
        dst[col] = acc;
    }
}

static void launch_moe_expert_reduce(ggml_backend_cuda_context & ctx,
                                     const float *               experts,
                                     const float *               weights,
                                     float *                     dst,
                                     const int                   n_expert_used,
                                     const int                   n_cols,
                                     const int                   n_rows) {
    const int block_size = 32;

    const int n_blocks_x = n_rows;
    const int n_blocks_y = (n_cols + block_size - 1) / block_size;

    dim3 block_dims(block_size);
    dim3 grid_dims(n_blocks_x, n_blocks_y);

    hipStream_t stream = ctx.stream();
    switch (n_expert_used) {
        case 1:
            moe_expert_reduce_cuda<1>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 2:
            moe_expert_reduce_cuda<2>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 4:
            moe_expert_reduce_cuda<4>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 6:
            moe_expert_reduce_cuda<6>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 8:
            moe_expert_reduce_cuda<8>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 16:
            moe_expert_reduce_cuda<16>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 32:
            moe_expert_reduce_cuda<32>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 64:
            moe_expert_reduce_cuda<64>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        case 128:
            moe_expert_reduce_cuda<128>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
        default:
            moe_expert_reduce_cuda<0>
                <<<grid_dims, block_dims, 0, stream>>>(experts, weights, dst, n_expert_used, n_cols);
            break;
    }
}

bool ggml_cuda_should_use_moe_expert_reduce(const ggml_cgraph * cgraph, int start_index, int end_index) {
    const ggml_tensor * mul = cgraph->nodes[start_index];

    if (mul->op != GGML_OP_MUL || !ggml_is_contiguous(mul->src[0]) || !ggml_is_contiguous(mul->src[1])) {
        return false;
    }

    int    current_node   = start_index + 1;
    size_t current_offset = 0;

    std::vector<const ggml_tensor *> view_nodes;
    //check if all are views of the expert in increasing order
    while (current_node < end_index && cgraph->nodes[current_node]->op == GGML_OP_VIEW) {
        const ggml_tensor * node = cgraph->nodes[current_node];
        if (node->view_src != mul) {
            return false;
        }
        if (node->view_offs < current_offset) {
            return false;
        }
        current_offset = node->view_offs;
        current_node++;
        view_nodes.push_back(node);
    }

    //check if all the adds are in increasing order
    const ggml_tensor * prev_add_src = view_nodes.empty() ? nullptr : view_nodes[0];
    int                 num_adds     = 0;
    int                 num_views    = view_nodes.size();
    while (current_node < end_index && cgraph->nodes[current_node]->op == GGML_OP_ADD) {
        const ggml_tensor * add_node = cgraph->nodes[current_node];

        bool is_first_op_ok  = num_views > num_adds ? add_node->src[0] == prev_add_src : false;
        bool is_second_op_ok = num_views > num_adds ? add_node->src[1] == view_nodes[num_adds + 1] : false;

        if (!is_first_op_ok || !is_second_op_ok) {
            return false;
        }
        prev_add_src = add_node;

        num_adds++;
        current_node++;
    }

    if (num_views != num_adds + 1) {
        return false;
    }

    return true;
}

void ggml_cuda_op_moe_expert_reduce(ggml_backend_cuda_context & ctx,
                                    const ggml_tensor *         experts,
                                    const ggml_tensor *         weights,
                                    ggml_tensor *               dst) {
    const int n_rows        = experts->ne[2];
    const int n_expert_used = experts->ne[1];
    const int n_cols        = experts->ne[0];

    GGML_ASSERT(experts->type == GGML_TYPE_F32);
    GGML_ASSERT(weights->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(experts));
    GGML_ASSERT(ggml_is_contiguous(weights));
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    const float * experts_d = (const float *) experts->data;
    const float * weights_d = (const float *) weights->data;
    float *       dst_d     = (float *) dst->data;

    launch_moe_expert_reduce(ctx, experts_d, weights_d, dst_d, n_expert_used, n_cols, n_rows);
}

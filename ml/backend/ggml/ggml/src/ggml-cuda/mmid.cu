#include "hip/hip_runtime.h"
#include "common.cuh"
#include "mmid.cuh"

// To reduce shared memory use, store "it" and "iex_used" with 22/10 bits each.
struct mm_ids_helper_store {
    uint32_t data;

    __device__ mm_ids_helper_store(const uint32_t it, const uint32_t iex_used) {
        data = (it & 0x003FFFFF) | (iex_used << 22);
    }

    __device__ uint32_t it() const {
        return data & 0x003FFFFF;
    }

    __device__ uint32_t iex_used() const {
        return data >> 22;
    }
};
static_assert(sizeof(mm_ids_helper_store) == 4, "unexpected size for mm_ids_helper_store");

// Helper function for mul_mat_id, converts ids to a more convenient format.
// ids_src1 describes how to permute the flattened column indices of src1 in order to get a compact src1 tensor sorted by expert.
// ids_dst describes the same mapping but for the dst tensor.
// The upper and lower bounds for the ith expert in the compact src1 tensor are stored in expert_bounds[i:i+1].
template <int n_expert_used_template>
__launch_bounds__(ggml_cuda_get_physical_warp_size(), 1)
static __global__ void mm_ids_helper(
        const int32_t * __restrict__ ids, int32_t * __restrict__ ids_src1, int32_t * __restrict__ ids_dst, int32_t * __restrict__ expert_bounds,
        const int n_tokens, const int n_expert_used_var, const int nchannels_y, const int si1, const int sis1) {
    constexpr int warp_size = ggml_cuda_get_physical_warp_size();
    const int n_expert_used = n_expert_used_template == 0 ? n_expert_used_var : n_expert_used_template;
    const int expert = blockIdx.x;

    extern __shared__ char data_mm_ids_helper[];
    mm_ids_helper_store * store = (mm_ids_helper_store *) data_mm_ids_helper;

    int nex_prev   = 0; // Number of columns for experts with a lower index.
    int it_compact = 0; // Running index for the compact slice of this expert.

    if constexpr (n_expert_used_template == 0) {
        // Generic implementation:
        for (int it = 0; it < n_tokens; ++it) {
            int iex_used = -1; // The index at which the expert is used, if any.
            for (int iex = threadIdx.x; iex < n_expert_used; iex += warp_size) {
                const int expert_used = ids[it*si1 + iex];
                nex_prev += expert_used < expert;
                if (expert_used == expert) {
                    iex_used = iex;
                }
            }

            if (iex_used != -1) {
                store[it_compact] = mm_ids_helper_store(it, iex_used);
            }

            if (warp_reduce_any<warp_size>(iex_used != -1)) {
                it_compact++;
            }
        }
    } else {
        // Implementation optimized for specific numbers of experts used:
        static_assert(n_expert_used == 6 || warp_size % n_expert_used == 0, "bad n_expert_used");
        const int neu_padded = n_expert_used == 6 ? 8 : n_expert_used; // Padded to next higher power of 2.
        for (int it0 = 0; it0 < n_tokens; it0 += warp_size/neu_padded) {
            const int it = it0 + threadIdx.x / neu_padded;

            const int iex = threadIdx.x % neu_padded; // The index at which the expert is used, if any.
            const int expert_used = (neu_padded == n_expert_used || iex < n_expert_used) && it < n_tokens ?
                ids[it*si1 + iex] : INT_MAX;
            const int iex_used = expert_used == expert ? iex : -1;
            nex_prev += expert_used < expert;

            // Whether the threads at this token position have used the expert:
            const int it_compact_add_self = warp_reduce_any<neu_padded>(iex_used != -1);

            // Do a scan over threads at lower token positions in warp to get the correct index for writing data:
            int it_compact_add_lower = 0;
#pragma unroll
            for (int offset = neu_padded; offset < warp_size; offset += neu_padded) {
                const int tmp = __shfl_up_sync(0xFFFFFFFF, it_compact_add_self, offset, warp_size);
                if (threadIdx.x >= static_cast<unsigned int>(offset)) {
                    it_compact_add_lower += tmp;
                }
            }

            if (iex_used != -1) {
                store[it_compact + it_compact_add_lower] = mm_ids_helper_store(it, iex_used);
            }

            // The thread with the highest index in the warp always has the sum over the whole warp, use it to increment all threads:
            it_compact += __shfl_sync(0xFFFFFFFF, it_compact_add_lower + it_compact_add_self, warp_size - 1, warp_size);
        }
    }
    nex_prev = warp_reduce_sum<warp_size>(nex_prev);

    for (int itc = threadIdx.x; itc < it_compact; itc += warp_size) {
        const mm_ids_helper_store store_it = store[itc];
        const int it       = store_it.it();
        const int iex_used = store_it.iex_used();
        ids_src1[nex_prev + itc] = it*sis1          + iex_used % nchannels_y;
        ids_dst [nex_prev + itc] = it*n_expert_used + iex_used;
    }

    if (threadIdx.x != 0) {
        return;
    }

    expert_bounds[expert] = nex_prev;

    if (expert < static_cast<int>(gridDim.x) - 1) {
        return;
    }

    expert_bounds[gridDim.x] = nex_prev + it_compact;
}

template <int n_expert_used_template>
static void launch_mm_ids_helper(
        const int32_t * __restrict__ ids, int32_t * __restrict__ ids_src1, int32_t * __restrict__ ids_dst, int32_t * __restrict__ expert_bounds,
        const int n_experts, const int n_tokens, const int n_expert_used_var, const int nchannels_y, const int si1, const int sis1, hipStream_t stream) {
    GGML_ASSERT(n_tokens          < (1 << 22) && "too few bits in mm_ids_helper_store");
    GGML_ASSERT(n_expert_used_var < (1 << 10) && "too few bits in mm_ids_helper_store");

    const int id = ggml_cuda_get_device();
    const int warp_size = ggml_cuda_info().devices[id].warp_size;
    const size_t smpbo = ggml_cuda_info().devices[id].smpbo;
    CUDA_SET_SHARED_MEMORY_LIMIT(mm_ids_helper<n_expert_used_template>, smpbo);

    const dim3 num_blocks(n_experts, 1, 1);
    const dim3 block_size(warp_size, 1, 1);
    const size_t nbytes_shared = n_tokens*sizeof(mm_ids_helper_store);
    GGML_ASSERT(nbytes_shared <= smpbo);
    mm_ids_helper<n_expert_used_template><<<num_blocks, block_size, nbytes_shared, stream>>>
        (ids, ids_src1, ids_dst, expert_bounds, n_tokens, n_expert_used_var, nchannels_y, si1, sis1);
}

void ggml_cuda_launch_mm_ids_helper(
        const int32_t * __restrict__ ids, int32_t * __restrict__ ids_src1, int32_t * __restrict__ ids_dst, int32_t * __restrict__ expert_bounds,
        const int n_experts, const int n_tokens, const int n_expert_used, const int nchannels_y, const int si1, const int sis1, hipStream_t stream) {
    switch (n_expert_used) {
        case  2:
            launch_mm_ids_helper< 2>(ids, ids_src1, ids_dst, expert_bounds, n_experts, n_tokens, n_expert_used, nchannels_y, si1, sis1, stream);
            break;
        case  4:
            launch_mm_ids_helper< 4>(ids, ids_src1, ids_dst, expert_bounds, n_experts, n_tokens, n_expert_used, nchannels_y, si1, sis1, stream);
            break;
        case  6:
            launch_mm_ids_helper< 6>(ids, ids_src1, ids_dst, expert_bounds, n_experts, n_tokens, n_expert_used, nchannels_y, si1, sis1, stream);
            break;
        case  8:
            launch_mm_ids_helper< 8>(ids, ids_src1, ids_dst, expert_bounds, n_experts, n_tokens, n_expert_used, nchannels_y, si1, sis1, stream);
            break;
        case 16:
            launch_mm_ids_helper<16>(ids, ids_src1, ids_dst, expert_bounds, n_experts, n_tokens, n_expert_used, nchannels_y, si1, sis1, stream);
            break;
        case 32:
            launch_mm_ids_helper<32>(ids, ids_src1, ids_dst, expert_bounds, n_experts, n_tokens, n_expert_used, nchannels_y, si1, sis1, stream);
            break;
        default:
            launch_mm_ids_helper< 0>(ids, ids_src1, ids_dst, expert_bounds, n_experts, n_tokens, n_expert_used, nchannels_y, si1, sis1, stream);
            break;
    }
}

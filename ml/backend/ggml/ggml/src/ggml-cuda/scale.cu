#include "hip/hip_runtime.h"
#include "scale.cuh"

static __global__ void scale_f32(const float * x, float * dst, const float scale, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = scale * x[i];
}

static void scale_f32_cuda(const float * x, float * dst, const float scale, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SCALE_BLOCK_SIZE - 1) / CUDA_SCALE_BLOCK_SIZE;
    scale_f32<<<num_blocks, CUDA_SCALE_BLOCK_SIZE, 0, stream>>>(x, dst, scale, k);
}

void ggml_cuda_op_scale(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float scale;
    memcpy(&scale, dst->op_params, sizeof(float));

    scale_f32_cuda(src0_d, dst_d, scale, ggml_nelements(src0), stream);
}

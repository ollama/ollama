#include "hip/hip_runtime.h"
#include "norm.cuh"
#include <cstdint>

template <int block_size>
static __global__ void norm_f32(
        const float * x, float * dst, const int ncols, const int64_t stride_row, const int64_t stride_channel,
        const int64_t stride_sample, const float eps) {
    const int nrows     = gridDim.x;
    const int nchannels = gridDim.y;

    const int row       = blockIdx.x;
    const int channel   = blockIdx.y;
    const int sample    = blockIdx.z;
    const int tid       = threadIdx.x;

    x   += sample*stride_sample + channel*stride_channel + row*stride_row;
    dst += ((sample*nchannels + channel)*nrows + row)*ncols;

    float2 mean_var = make_float2(0.0f, 0.0f);

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[col];
        mean_var.x += xi;
        mean_var.y += xi * xi;
    }

    // sum up partial sums
    mean_var = warp_reduce_sum(mean_var);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float2 s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = mean_var;
        }
        __syncthreads();
        mean_var = s_sum[lane_id];
        mean_var = warp_reduce_sum(mean_var);
    }

    const float mean = mean_var.x / ncols;
    const float var = mean_var.y / ncols - mean * mean;
    const float inv_std = rsqrtf(var + eps);

    for (int col = tid; col < ncols; col += block_size) {
        dst[col] = (x[col] - mean) * inv_std;
    }
}

template <int block_size>
static __global__ void group_norm_f32(const float * x, float * dst, const int group_size, const int ne_elements, const float eps) {
    // blockIdx.x: num_groups idx
    // threadIdx.x: block_size idx
    const int start =     blockIdx.x*group_size + threadIdx.x;
    const int end   = min(blockIdx.x*group_size + group_size,  ne_elements);

    float tmp = 0.0f; // partial sum for thread in warp

    for (int j = start; j < end; j += block_size) {
        tmp += x[j];
    }

    tmp = warp_reduce_sum(tmp);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float mean = tmp / group_size;
    tmp = 0.0f;

    for (int j = start; j < end; j += block_size) {
        const float xi = x[j] - mean;
        dst[j] = xi;
        tmp += xi * xi;
    }

    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __shared__ float s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float variance = tmp / group_size;
    const float scale = rsqrtf(variance + eps);
    for (int j = start; j < end; j += block_size) {
        dst[j] *= scale;
    }
}

template <int block_size>
static __global__ void rms_norm_f32(
        const float * x, float * dst, const int ncols, const int64_t stride_row, const int64_t stride_channel,
        const int64_t stride_sample, const float eps) {
    const int nrows     = gridDim.x;
    const int nchannels = gridDim.y;

    const int row       = blockIdx.x;
    const int channel   = blockIdx.y;
    const int sample    = blockIdx.z;
    const int tid       = threadIdx.x;

    x   += sample*stride_sample + channel*stride_channel + row*stride_row;
    dst += ((sample*nchannels + channel)*nrows + row)*ncols;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[col];
        tmp += xi * xi;
    }

    // sum up partial sums
    tmp = warp_reduce_sum(tmp);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float s_sum[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float mean = tmp / ncols;
    const float scale = rsqrtf(mean + eps);

    for (int col = tid; col < ncols; col += block_size) {
        dst[col] = scale * x[col];
    }
}

template <int block_size>
static __global__ void rms_norm_back_f32(
        const float * grad, const float * xf, float * dst, const int ncols, const float eps) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    grad += int64_t(row)*ncols;
    xf   += int64_t(row)*ncols;
    dst  += int64_t(row)*ncols;

    float sum_xx = 0.0f; // sum for squares of x, equivalent to forward pass
    float sum_xg = 0.0f; // sum for x * gradient, needed because RMS norm mixes inputs

    for (int col = tid; col < ncols; col += block_size) {
        const float xfi = xf[col];
        sum_xx += xfi * xfi;
        sum_xg += xfi * grad[col];
    }

    // sum up partial sums
    sum_xx = warp_reduce_sum(sum_xx);
    sum_xg = warp_reduce_sum(sum_xg);
    if constexpr (block_size > WARP_SIZE) {
        static_assert(block_size == 1024, "unexpected block_size");
        __shared__ float s_sum_xx[32];
        __shared__ float s_sum_xg[32];
        const int warp_id = threadIdx.x / WARP_SIZE;
        const int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum_xx[warp_id] = sum_xx;
            s_sum_xg[warp_id] = sum_xg;
        }
        __syncthreads();

        sum_xx = s_sum_xx[lane_id];
        sum_xx = warp_reduce_sum(sum_xx);

        sum_xg = s_sum_xg[lane_id];
        sum_xg = warp_reduce_sum(sum_xg);
    }

    const float mean_eps = sum_xx / ncols + eps;
    const float sum_eps  = sum_xx + ncols*eps;

    const float scale_grad = rsqrtf(mean_eps);
    const float scale_x    = -scale_grad * sum_xg/sum_eps;

    for (int col = tid; col < ncols; col += block_size) {
        dst[col] = scale_grad*grad[col] + scale_x*xf[col];
    }
}

static void norm_f32_cuda(
        const float * x, float * dst, const int ncols, const int nrows, const int nchannels, const int nsamples,
        const int64_t stride_row, const int64_t stride_channel, const int64_t stride_sample, const float eps, hipStream_t stream) {
    const dim3 blocks_num(nrows, nchannels, nsamples);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        norm_f32<WARP_SIZE><<<blocks_num, block_dims, 0, stream>>>(x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        norm_f32<1024><<<blocks_num, block_dims, 0, stream>>>(x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
}

static void group_norm_f32_cuda(
        const float * x, float * dst, const int num_groups, const float eps, const int group_size, const int ne_elements, hipStream_t stream) {
    if (group_size < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        group_norm_f32<WARP_SIZE><<<num_groups, block_dims, 0, stream>>>(x, dst, group_size, ne_elements, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        group_norm_f32<1024><<<num_groups, block_dims, 0, stream>>>(x, dst, group_size, ne_elements, eps);
    }
}

static void rms_norm_f32_cuda(
        const float * x, float * dst, const int ncols, const int nrows, const int nchannels, const int nsamples,
        const int64_t stride_row, const int64_t stride_channel, const int64_t stride_sample, const float eps, hipStream_t stream) {
    const dim3 blocks_num(nrows, nchannels, nsamples);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        rms_norm_f32<WARP_SIZE><<<blocks_num, block_dims, 0, stream>>>(x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        rms_norm_f32<1024><<<blocks_num, block_dims, 0, stream>>>(x, dst, ncols, stride_row, stride_channel, stride_sample, eps);
    }
}

static void rms_norm_back_f32_cuda(const float * grad, const float * xf, float * dst, const int ncols, const int nrows, const float eps, hipStream_t stream) {
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        rms_norm_back_f32<WARP_SIZE><<<nrows, block_dims, 0, stream>>>(grad, xf, dst, ncols, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        rms_norm_back_f32<1024><<<nrows, block_dims, 0, stream>>>(grad, xf, dst, ncols, eps);
    }
}

void ggml_cuda_op_norm(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *) src0->data;
    float * dst_d = (float *) dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_TENSOR_UNARY_OP_LOCALS;

    float eps;
    memcpy(&eps, dst->op_params, sizeof(float));
    GGML_ASSERT(eps >= 0.0f);

    const size_t ts0 = ggml_type_size(src0->type);
    GGML_ASSERT(nb00 == ts0);
    const int64_t s01 = nb01 / ts0;
    const int64_t s02 = nb02 / ts0;
    const int64_t s03 = nb03 / ts0;

    norm_f32_cuda(src0_d, dst_d, ne00, ne01, ne02, ne03, s01, s02, s03, eps, stream);
}

void ggml_cuda_op_group_norm(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    int num_groups = dst->op_params[0];

    float eps;
    memcpy(&eps, dst->op_params + 1, sizeof(float));
    GGML_ASSERT(eps >= 0.0f);

    int group_size = src0->ne[0] * src0->ne[1] * ((src0->ne[2] + num_groups - 1) / num_groups);
    group_norm_f32_cuda(src0_d, dst_d, num_groups * src0->ne[3], eps, group_size, ggml_nelements(src0), stream);
}

void ggml_cuda_op_rms_norm(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *) src0->data;
    float * dst_d = (float *) dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_TENSOR_UNARY_OP_LOCALS;

    float eps;
    memcpy(&eps, dst->op_params, sizeof(float));
    GGML_ASSERT(eps >= 0.0f);

    const size_t ts0 = ggml_type_size(src0->type);
    GGML_ASSERT(nb00 == ts0);
    const int64_t s01 = nb01 / ts0;
    const int64_t s02 = nb02 / ts0;
    const int64_t s03 = nb03 / ts0;

    rms_norm_f32_cuda(src0_d, dst_d, ne00, ne01, ne02, ne03, s01, s02, s03, eps, stream);
}

void ggml_cuda_op_rms_norm_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * grad  = dst->src[0]; // gradients
    const ggml_tensor * src0f = dst->src[1]; // src0 from forward pass

    const float * grad_d  = (const float *) grad->data;
    const float * src0f_d = (const float *) src0f->data;
    float       * dst_d   = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(grad));

    GGML_ASSERT( grad->type == GGML_TYPE_F32);
    GGML_ASSERT(src0f->type == GGML_TYPE_F32);
    GGML_ASSERT(  dst->type == GGML_TYPE_F32);

    const int64_t ne00 = src0f->ne[0];
    const int64_t nrows = ggml_nrows(src0f);

    float eps;
    memcpy(&eps, dst->op_params, sizeof(float));
    GGML_ASSERT(eps >= 0.0f);

    rms_norm_back_f32_cuda(grad_d, src0f_d, dst_d, ne00, nrows, eps, stream);
}

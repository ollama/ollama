#include "mmq.cuh"
#include "quantize.cuh"

#include <vector>

static void ggml_cuda_mul_mat_q_switch_type(ggml_backend_cuda_context & ctx, const mmq_args & args, hipStream_t stream) {
    switch (args.type_x) {
        case GGML_TYPE_Q4_0:
            mul_mat_q_case<GGML_TYPE_Q4_0>(ctx, args, stream);
            break;
        case GGML_TYPE_Q4_1:
            mul_mat_q_case<GGML_TYPE_Q4_1>(ctx, args, stream);
            break;
        case GGML_TYPE_Q5_0:
            mul_mat_q_case<GGML_TYPE_Q5_0>(ctx, args, stream);
            break;
        case GGML_TYPE_Q5_1:
            mul_mat_q_case<GGML_TYPE_Q5_1>(ctx, args, stream);
            break;
        case GGML_TYPE_Q8_0:
            mul_mat_q_case<GGML_TYPE_Q8_0>(ctx, args, stream);
            break;
        case GGML_TYPE_Q2_K:
            mul_mat_q_case<GGML_TYPE_Q2_K>(ctx, args, stream);
            break;
        case GGML_TYPE_Q3_K:
            mul_mat_q_case<GGML_TYPE_Q3_K>(ctx, args, stream);
            break;
        case GGML_TYPE_Q4_K:
            mul_mat_q_case<GGML_TYPE_Q4_K>(ctx, args, stream);
            break;
        case GGML_TYPE_Q5_K:
            mul_mat_q_case<GGML_TYPE_Q5_K>(ctx, args, stream);
            break;
        case GGML_TYPE_Q6_K:
            mul_mat_q_case<GGML_TYPE_Q6_K>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ2_XXS:
            mul_mat_q_case<GGML_TYPE_IQ2_XXS>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ2_XS:
            mul_mat_q_case<GGML_TYPE_IQ2_XS>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ2_S:
            mul_mat_q_case<GGML_TYPE_IQ2_S>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ3_XXS:
            mul_mat_q_case<GGML_TYPE_IQ3_XXS>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ3_S:
            mul_mat_q_case<GGML_TYPE_IQ3_S>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ1_S:
            mul_mat_q_case<GGML_TYPE_IQ1_S>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ4_XS:
            mul_mat_q_case<GGML_TYPE_IQ4_XS>(ctx, args, stream);
            break;
        case GGML_TYPE_IQ4_NL:
            mul_mat_q_case<GGML_TYPE_IQ4_NL>(ctx, args, stream);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

void ggml_cuda_mul_mat_q(
        ggml_backend_cuda_context & ctx, const ggml_tensor * src0, const ggml_tensor * src1, const ggml_tensor * ids, ggml_tensor * dst) {
    GGML_ASSERT(        src1->type == GGML_TYPE_F32);
    GGML_ASSERT(        dst->type  == GGML_TYPE_F32);
    GGML_ASSERT(!ids || ids->type  == GGML_TYPE_I32); // Optional, used for batched GGML_MUL_MAT_ID.

    GGML_TENSOR_BINARY_OP_LOCALS;

    hipStream_t stream = ctx.stream();
    const int cc = ggml_cuda_info().devices[ggml_cuda_get_device()].cc;

    const size_t ts_src0 = ggml_type_size(src0->type);
    const size_t ts_src1 = ggml_type_size(src1->type);
    const size_t ts_dst  = ggml_type_size(dst->type);

    GGML_ASSERT(        nb00       == ts_src0);
    GGML_ASSERT(        nb10       == ts_src1);
    GGML_ASSERT(        nb0        == ts_dst);
    GGML_ASSERT(!ids || ids->nb[0] == ggml_type_size(ids->type));

    const char  * src0_d = (const char  *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       *  dst_d = (float       *)  dst->data;

    const int64_t ne10_padded = GGML_PAD(ne10, MATRIX_ROW_PADDING);

    const int64_t s01 = src0->nb[1] / ts_src0;
    const int64_t s1  =  dst->nb[1] / ts_dst;
    const int64_t s02 = src0->nb[2] / ts_src0;
    const int64_t s2  =  dst->nb[2] / ts_dst;
    const int64_t s03 = src0->nb[3] / ts_src0;
    const int64_t s3  =  dst->nb[3] / ts_dst;

    const bool use_stream_k = GGML_CUDA_CC_IS_NVIDIA(cc) && ggml_cuda_highest_compiled_arch(cc) >= GGML_CUDA_CC_VOLTA;

    if (!ids) {
        const size_t nbytes_src1_q8_1 = ne13*ne12 * ne11*ne10_padded * sizeof(block_q8_1)/QK8_1 +
            get_mmq_x_max_host(cc)*sizeof(block_q8_1_mmq);
        ggml_cuda_pool_alloc<char> src1_q8_1(ctx.pool(), nbytes_src1_q8_1);

        {
            const int64_t s11 = src1->nb[1] / ts_src1;
            const int64_t s12 = src1->nb[2] / ts_src1;
            const int64_t s13 = src1->nb[3] / ts_src1;
            quantize_mmq_q8_1_cuda(src1_d, nullptr, src1_q8_1.get(), src0->type,
                ne10, s11, s12, s13, ne10_padded, ne11, ne12, ne13, stream);
        }

        const int64_t s12 = ne11*ne10_padded * sizeof(block_q8_1)/(QK8_1*sizeof(int));
        const int64_t s13 = ne12*s12;

        const mmq_args args = {
            src0_d, src0->type, (const int *) src1_q8_1.ptr, nullptr, nullptr, dst_d,
            ne00, ne01, ne1, s01, s1,
            ne02, ne12, s02, s12, s2,
            ne03, ne13, s03, s13, s3,
            use_stream_k};
        ggml_cuda_mul_mat_q_switch_type(ctx, args, stream);
        return;
    }

    GGML_ASSERT(ne13 == 1);
    GGML_ASSERT(nb12 % nb11 == 0);
    GGML_ASSERT(nb2  % nb1  == 0);

    const int64_t n_expert_used = ids->ne[0];
    const int64_t ne_get_rows = ne12 * n_expert_used;

    std::vector<char> ids_host(ggml_nbytes(ids));
    std::vector<int32_t> ids_src1_host;
    ids_src1_host.reserve(ne_get_rows);
    std::vector<int32_t> ids_dst_host;
    ids_dst_host.reserve(ne_get_rows);
    std::vector<int32_t> tokens_per_expert_host(ne02);
    std::vector<int32_t> expert_bounds_host(ne02 + 1);
    ggml_cuda_pool_alloc<int32_t> ids_buf_dev(ctx.pool());

    CUDA_CHECK(hipMemcpyAsync(ids_host.data(), ids->data, ggml_nbytes(ids), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int64_t i02 = 0; i02 < ne02; ++i02) { // expert matrices
        for (int64_t i12 = 0; i12 < ne12; ++i12) { // tokens
            for (int64_t iex = 0; iex < n_expert_used; ++iex) {
                const int32_t expert_to_use = *(const int32_t *)(ids_host.data() + i12*ids->nb[1] + iex*ids->nb[0]);
                assert(expert_to_use >= 0 && expert_to_use < ne02);
                if (expert_to_use == i02) {
                    ids_src1_host.push_back(i12*(nb12/nb11) + iex % ne11);
                    ids_dst_host.push_back(i12*ne1 + iex);
                    tokens_per_expert_host[i02]++;
                    break;
                }
            }
        }
    }

    int32_t cumsum = 0;
    for (int64_t i = 0; i < ne02; ++i) {
        expert_bounds_host[i] = cumsum;
        cumsum += tokens_per_expert_host[i];
    }
    expert_bounds_host[ne02] = cumsum;

    std::vector<int32_t> ids_buf_host;
    ids_buf_host.reserve(ids_src1_host.size() + ids_dst_host.size() + expert_bounds_host.size());
    ids_buf_host.insert(ids_buf_host.end(), ids_src1_host.begin(), ids_src1_host.end());
    ids_buf_host.insert(ids_buf_host.end(), ids_dst_host.begin(), ids_dst_host.end());
    ids_buf_host.insert(ids_buf_host.end(), expert_bounds_host.begin(), expert_bounds_host.end());
    ids_buf_dev.alloc(ids_buf_host.size() + get_mmq_x_max_host(cc)); // Expert bounds are padded on device.
    CUDA_CHECK(hipMemcpyAsync(ids_buf_dev.ptr, ids_buf_host.data(), ids_buf_host.size()*sizeof(int32_t), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    const int32_t * ids_src1_dev      = ids_buf_dev.ptr;
    const int32_t * ids_dst_dev       = ids_src1_dev + ids_src1_host.size();
    const int32_t * expert_bounds_dev = ids_dst_dev + ids_dst_host.size();

    const size_t nbytes_src1_q8_1 = ne12*n_expert_used*ne10_padded * sizeof(block_q8_1)/QK8_1 +
        get_mmq_x_max_host(cc)*sizeof(block_q8_1_mmq);
    ggml_cuda_pool_alloc<char> src1_q8_1(ctx.pool(), nbytes_src1_q8_1);

    const int64_t ne11_flat = ne12*n_expert_used;
    const int64_t ne12_flat = 1;
    const int64_t ne13_flat = 1;

    {
        const int64_t s11 = src1->nb[1] / ts_src1;
        const int64_t s12 = src1->nb[2] / ts_src1;
        const int64_t s13 = src1->nb[2] / ts_src1;
        quantize_mmq_q8_1_cuda(src1_d, ids_src1_dev, src1_q8_1.get(), src0->type,
            ne10, s11, s12, s13, ne10_padded, ne11_flat, ne12_flat, ne13_flat, stream);
    }

    const int64_t s12 = ne11*ne10_padded * sizeof(block_q8_1)/(QK8_1*sizeof(int));
    const int64_t s13 = ne12*s12;

    // Note that ne02 is used instead of ne12 because the number of y channels determines the z dimension of the CUDA grid.
    const mmq_args args = {
        src0_d, src0->type, (const int *) src1_q8_1.ptr, ids_dst_dev, expert_bounds_dev, dst_d,
        ne00, ne01, ne_get_rows, s01, s1,
        ne02, ne02, s02, s12, s2,
        ne03, ne13, s03, s13, s3,
        use_stream_k};

    ggml_cuda_mul_mat_q_switch_type(ctx, args, stream);
}

void ggml_cuda_op_mul_mat_q(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    GGML_ASSERT(ne10 % QK8_1 == 0);

    const int64_t ne0 = dst->ne[0];

    const int64_t row_diff = row_high - row_low;
    const int64_t stride01 = ne00 / ggml_blck_size(src0->type);

    const int id = ggml_cuda_get_device();
    const int cc = ggml_cuda_info().devices[id].cc;

    // the main device has a larger memory buffer to hold the results from all GPUs
    // nrows_dst == nrows of the matrix that the kernel writes into
    const int64_t nrows_dst = id == ctx.device ? ne0 : row_diff;

    // The stream-k decomposition is only faster for recent NVIDIA GPUs.
    // Also its fixup needs to allocate a temporary buffer in the memory pool.
    // There are multiple parallel CUDA streams for src1_ncols != ne11 which would introduce a race condition for this buffer.
    const bool use_stream_k = GGML_CUDA_CC_IS_NVIDIA(cc) &&
        ggml_cuda_highest_compiled_arch(cc) >= GGML_CUDA_CC_VOLTA && src1_ncols == ne11;
    const mmq_args args = {
        src0_dd_i, src0->type, (const int *) src1_ddq_i, nullptr, nullptr, dst_dd_i,
        ne00, row_diff, src1_ncols, stride01, nrows_dst,
        1, 1, 0, 0, 0,
        1, 1, 0, 0, 0,
        use_stream_k};

    ggml_cuda_mul_mat_q_switch_type(ctx, args, stream);

    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddf_i);
    GGML_UNUSED(src1_padded_row_size);
}

bool ggml_cuda_should_use_mmq(enum ggml_type type, int cc, int64_t ne11) {
#ifdef GGML_CUDA_FORCE_CUBLAS
    return false;
#endif // GGML_CUDA_FORCE_CUBLAS

    bool mmq_supported;

    switch (type) {
        case GGML_TYPE_Q4_0:
        case GGML_TYPE_Q4_1:
        case GGML_TYPE_Q5_0:
        case GGML_TYPE_Q5_1:
        case GGML_TYPE_Q8_0:
        case GGML_TYPE_Q2_K:
        case GGML_TYPE_Q3_K:
        case GGML_TYPE_Q4_K:
        case GGML_TYPE_Q5_K:
        case GGML_TYPE_Q6_K:
        case GGML_TYPE_IQ2_XXS:
        case GGML_TYPE_IQ2_XS:
        case GGML_TYPE_IQ2_S:
        case GGML_TYPE_IQ3_XXS:
        case GGML_TYPE_IQ3_S:
        case GGML_TYPE_IQ1_S:
        case GGML_TYPE_IQ4_XS:
        case GGML_TYPE_IQ4_NL:
            mmq_supported = true;
            break;
        default:
            mmq_supported = false;
            break;
    }

    if (!mmq_supported) {
        return false;
    }

    if (new_mma_available(cc)) {
        return true;
    }

    if (ggml_cuda_highest_compiled_arch(cc) < GGML_CUDA_CC_DP4A) {
        return false;
    }

#ifdef GGML_CUDA_FORCE_MMQ
    return true;
#endif //GGML_CUDA_FORCE_MMQ

    if (GGML_CUDA_CC_IS_NVIDIA(cc)) {
        return !fp16_mma_hardware_available(cc) || ne11 < MMQ_DP4A_MAX_BATCH_SIZE;
    }

    return (!GGML_CUDA_CC_IS_RDNA4(cc) && !GGML_CUDA_CC_IS_RDNA3(cc) && !GGML_CUDA_CC_IS_CDNA(cc)) || ne11 < MMQ_DP4A_MAX_BATCH_SIZE;
}

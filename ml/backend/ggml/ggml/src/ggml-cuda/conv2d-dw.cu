#include "hip/hip_runtime.h"
#include "conv2d-dw.cuh"

struct conv_params {
    int in_w, in_h;
    int out_w, out_h;
    int kernel_w, kernel_h;
    int stride_x, stride_y;
    int padding_x, padding_y;
    int dilation_x, dilation_y;
    int channels, batches;
};

struct kernel_bounds {
    int y_min, y_max;
    int x_min, x_max;
};

__device__ __forceinline__ kernel_bounds calculate_kernel_bounds(int out_x, int out_y, const conv_params & params) {
    kernel_bounds bounds;
    bounds.y_min = max(0, (params.padding_y - out_y * params.stride_y + params.dilation_y - 1) / params.dilation_y);
    bounds.y_max =
        min(params.kernel_h,
            (params.in_h + params.padding_y - out_y * params.stride_y + params.dilation_y - 1) / params.dilation_y);
    bounds.x_min = max(0, (params.padding_x - out_x * params.stride_x + params.dilation_x - 1) / params.dilation_x);
    bounds.x_max =
        min(params.kernel_w,
            (params.in_w + params.padding_x - out_x * params.stride_x + params.dilation_x - 1) / params.dilation_x);
    return bounds;
}

__device__ __forceinline__ int calculate_input_coord(int out_coord, int kern_coord, int stride, int dilation, int padding) {
    return out_coord * stride + kern_coord * dilation - padding;
}

struct whcn_layout {
    __device__ static int input_index(int n, int c, int y, int x, const conv_params & params) {
        return n * (params.channels * params.in_w * params.in_h) + c * params.in_w * params.in_h + y * params.in_w + x;
    }

    __device__ static int kernel_index(int c, int ky, int kx, const conv_params & params) {
        return c * params.kernel_h * params.kernel_w + ky * params.kernel_w + kx;
    }

    __device__ static int output_index(int n, int c, int y, int x, const conv_params & params) {
        return n * (params.channels * params.out_w * params.out_h) + c * params.out_w * params.out_h +
               y * params.out_w + x;
    }

    __device__ static void unpack_indices(int global_idx, const conv_params & params, int & n, int & c, int & out_y,
                                          int & out_x) {
        out_x = global_idx % params.out_w;
        out_y = (global_idx / params.out_w) % params.out_h;
        c     = (global_idx / (params.out_w * params.out_h)) % params.channels;
        n     = global_idx / (params.out_w * params.out_h * params.channels);
    }
};

struct cwhn_layout {
    __device__ static int input_index(int n, int c, int y, int x, const conv_params & params) {
        return n * (params.channels * params.in_w * params.in_h) + (y * params.in_w + x) * params.channels + c;
    }

    __device__ static int kernel_index(int c, int ky, int kx, const conv_params & params) {
        return (ky * params.kernel_w + kx) * params.channels + c;
    }

    __device__ static int output_index(int n, int c, int y, int x, const conv_params & params) {
        return n * (params.channels * params.out_w * params.out_h) + y * (params.out_w * params.channels) +
               x * params.channels + c;
    }

    __device__ static void unpack_indices(int global_idx, const conv_params & params, int & n, int & c, int & out_y,
                                          int & out_x) {
        c     = global_idx % params.channels;
        out_x = (global_idx / params.channels) % params.out_w;
        out_y = (global_idx / (params.channels * params.out_w)) % params.out_h;
        n     = global_idx / (params.channels * params.out_w * params.out_h);
    }
};

template <typename T, typename Layout>
__global__ void conv2d_dw_kernel(const T * __restrict__ input, const T * __restrict__ kernel, T * __restrict__ output,
                                 const int in_w, const int in_h, const int out_w, const int out_h,
                                 const int kernel_w, const int kernel_h, const int stride_x, const int stride_y,
                                 const int padding_x, const int padding_y, const int dilation_x, const int dilation_y,
                                 const int channels, const int batches) {
    const int global_idx     = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_elements = batches * channels * out_h * out_w;

    if (global_idx >= total_elements) {
        return;
    }

    conv_params params = { in_w,     in_h,      out_w,     out_h,      kernel_w,   kernel_h, stride_x,
                           stride_y, padding_x, padding_y, dilation_x, dilation_y, channels, batches };

    int batch_idx, channel_idx, out_y_idx, out_x_idx;
    Layout::unpack_indices(global_idx, params, batch_idx, channel_idx, out_y_idx, out_x_idx);

    T accumulator = 0;
    kernel_bounds bounds = calculate_kernel_bounds(out_x_idx, out_y_idx, params);

    for (int kern_y = bounds.y_min; kern_y < bounds.y_max; ++kern_y) {
        int in_y_idx = calculate_input_coord(out_y_idx, kern_y, params.stride_y, params.dilation_y, params.padding_y);

        for (int kern_x = bounds.x_min; kern_x < bounds.x_max; ++kern_x) {
            int in_x_idx = calculate_input_coord(out_x_idx, kern_x, params.stride_x, params.dilation_x, params.padding_x);

            const T input_val  = input[Layout::input_index(batch_idx, channel_idx, in_y_idx, in_x_idx, params)];
            const T kernel_val = kernel[Layout::kernel_index(channel_idx, kern_y, kern_x, params)];

            accumulator += input_val * kernel_val;
        }
    }

    output[Layout::output_index(batch_idx, channel_idx, out_y_idx, out_x_idx, params)] = accumulator;
}

void ggml_cuda_op_conv2d_dw(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * kernel = dst->src[0];
    const ggml_tensor * input  = dst->src[1];

    GGML_ASSERT(kernel->type == GGML_TYPE_F32 && input->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32);
    const float * w_d = (const float *) kernel->data;
    const float * x_d = (const float *) input->data;
    float *       y_d = (float *) dst->data;

    const int32_t * p          = (const int32_t *) dst->op_params;
    const int       stride_x   = p[0];
    const int       stride_y   = p[1];
    const int       padding_x  = p[2];
    const int       padding_y  = p[3];
    const int       dilation_x = p[4];
    const int       dilation_y = p[5];

    const int in_w     = input->ne[0];
    const int in_h     = input->ne[1];
    const int kernel_w = kernel->ne[0];
    const int kernel_h = kernel->ne[1];
    const int out_w    = dst->ne[0];
    const int out_h    = dst->ne[1];
    const int channels = dst->ne[2];
    const int batches  = dst->ne[3];

    hipStream_t st = ctx.stream();

    const int total  = batches * channels * out_h * out_w;
    const int blocks = (total + CUDA_CONV2D_DW_BLOCK_SIZE - 1) / CUDA_CONV2D_DW_BLOCK_SIZE;

    if (ggml_is_contiguous(input)) {
        conv2d_dw_kernel<float, whcn_layout><<<blocks, CUDA_CONV2D_DW_BLOCK_SIZE, 0, st>>>(
            x_d, w_d, y_d, in_w, in_h, out_w, out_h, kernel_w, kernel_h, stride_x, stride_y, padding_x, padding_y,
            dilation_x, dilation_y, channels, batches);
    } else if (ggml_is_contiguous_channels(input)) {
        conv2d_dw_kernel<float, cwhn_layout><<<blocks, CUDA_CONV2D_DW_BLOCK_SIZE, 0, st>>>(
            x_d, w_d, y_d, in_w, in_h, out_w, out_h, kernel_w, kernel_h, stride_x, stride_y, padding_x, padding_y,
            dilation_x, dilation_y, channels, batches);
    } else {
        GGML_ABORT("Unsupported memory layout for conv_2d_dw");
    }
}

#include "hip/hip_runtime.h"
#include "quantize.cuh"
#include <cstdint>

static __global__ void quantize_q8_1(const float * __restrict__ x, void * __restrict__ vy, const int64_t kx, const int64_t kx0_padded) {
    const int64_t ix0 = (int64_t)blockDim.x*blockIdx.x + threadIdx.x;

    if (ix0 >= kx0_padded) {
        return;
    }

    const int64_t ix1 = blockIdx.y;

    const int64_t i_padded = ix1*kx0_padded + ix0;

    block_q8_1 * y = (block_q8_1 *) vy;

    const int64_t ib = i_padded / QK8_1; // block index
    const int64_t iqs = i_padded % QK8_1; // quant index

    const float xi = ix0 < kx ? x[ix1*kx + ix0] : 0.0f;
    float amax = fabsf(xi);
    float sum = xi;

    amax = warp_reduce_max(amax);
    sum = warp_reduce_sum(sum);

    const float d = amax / 127;
    const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    y[ib].qs[iqs] = q;

    if (iqs > 0) {
        return;
    }

    reinterpret_cast<half&>(y[ib].ds.x) = d;
    reinterpret_cast<half&>(y[ib].ds.y) = sum;
}

template <mmq_q8_1_ds_layout ds_layout>
static __global__ void quantize_mmq_q8_1(
    const float * __restrict__ x, void * __restrict__ vy, const int64_t kx0, const int64_t kx1, const int64_t kx0_padded) {

    constexpr int vals_per_scale = ds_layout == MMQ_Q8_1_DS_LAYOUT_D2S6 ? 64 : 32;
    constexpr int vals_per_sum   = ds_layout == MMQ_Q8_1_DS_LAYOUT_D2S6 ? 16 : 32;

    const int64_t ix0 = ((int64_t)blockDim.x*blockIdx.x + threadIdx.x)*4;

    if (ix0 >= kx0_padded) {
        return;
    }

    const float4 * x4 = (const float4 *) x;

    const int64_t ix1 = kx1*blockIdx.z + blockIdx.y;

    block_q8_1_mmq * y = (block_q8_1_mmq *) vy;

    const int64_t ib0 = blockIdx.z*((int64_t)gridDim.y*gridDim.x*blockDim.x/QK8_1); // first block of channel
    const int64_t ib  = ib0 + (ix0 / (4*QK8_1))*kx1 + blockIdx.y;                   // block index in channel
    const int64_t iqs = ix0 % (4*QK8_1);                                            // quant index in block

    // Load 4 floats per thread and calculate max. abs. value between them:
    const float4 xi = ix0 < kx0 ? x4[(ix1*kx0 + ix0)/4] : make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float amax = fabsf(xi.x);
    amax = fmaxf(amax, fabsf(xi.y));
    amax = fmaxf(amax, fabsf(xi.z));
    amax = fmaxf(amax, fabsf(xi.w));

    // Exchange max. abs. value between vals_per_scale/4 threads.
#pragma unroll
    for (int offset = vals_per_scale/8; offset > 0; offset >>= 1) {
        amax = fmaxf(amax, __shfl_xor_sync(0xFFFFFFFF, amax, offset, WARP_SIZE));
    }

    float sum;
    if (ds_layout != MMQ_Q8_1_DS_LAYOUT_D4) {
        sum = xi.x + xi.y + xi.z + xi.w;

        // Exchange calculate sum across vals_per_sum/4 threads.
#pragma unroll
        for (int offset = vals_per_sum/8; offset > 0; offset >>= 1) {
            sum += __shfl_xor_sync(0xFFFFFFFF, sum, offset, WARP_SIZE);
        }
    }

    const float d_inv = 127.0f / amax;
    char4 q;
    q.x = roundf(xi.x*d_inv);
    q.y = roundf(xi.y*d_inv);
    q.z = roundf(xi.z*d_inv);
    q.w = roundf(xi.w*d_inv);

    // Write back 4 int8 values as a single 32 bit value for better memroy bandwidth:
    char4 * yqs4 = (char4 *) y[ib].qs;
    yqs4[iqs/4] = q;

    if (ds_layout == MMQ_Q8_1_DS_LAYOUT_D2S6) {
        if (iqs % 16 != 0 || iqs >= 96) {
            return;
        }

        y[ib].d2s6[2 + iqs/16] = sum;

        if (iqs % 64 != 0) {
            return;
        }

        const float d = 1.0f / d_inv;

        y[ib].d2s6[iqs/64] = d;

        return;
    }

    if (iqs % 32 != 0) {
        return;
    }

    const float d = 1.0f / d_inv;

    if (ds_layout == MMQ_Q8_1_DS_LAYOUT_DS4) {
        y[ib].ds4[iqs/32] = make_half2(d, sum);
    } else {
        y[ib].d4[iqs/32]  = d;
    }
}

void quantize_row_q8_1_cuda(
    const float * x, void * vy, const int64_t kx0, const int64_t kx1, const int64_t channels,
    const int64_t kx0_padded, const ggml_type type_x, hipStream_t stream) {

    GGML_ASSERT(kx0_padded % QK8_1 == 0);

    const int64_t block_num_x = (kx0_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
    const dim3 num_blocks(block_num_x, kx1*channels, 1);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx0_padded);

    GGML_UNUSED(type_x);
}

void quantize_mmq_q8_1_cuda(
    const float * x, void * vy, const int64_t kx0, const int64_t kx1, const int64_t channels,
    const int64_t kx0_padded, const ggml_type type_x, hipStream_t stream) {

    GGML_ASSERT(kx0_padded % (4*QK8_1) == 0);

    const int64_t block_num_x = (kx0_padded + 4*CUDA_QUANTIZE_BLOCK_SIZE_MMQ - 1) / (4*CUDA_QUANTIZE_BLOCK_SIZE_MMQ);
    const dim3 num_blocks(block_num_x, kx1, channels);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE_MMQ, 1, 1);
    switch (mmq_get_q8_1_ds_layout(type_x)) {
        case MMQ_Q8_1_DS_LAYOUT_D4:
            quantize_mmq_q8_1<MMQ_Q8_1_DS_LAYOUT_D4>
                <<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx1, kx0_padded);
            break;
        case MMQ_Q8_1_DS_LAYOUT_DS4:
            quantize_mmq_q8_1<MMQ_Q8_1_DS_LAYOUT_DS4>
                <<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx1, kx0_padded);
            break;
        case MMQ_Q8_1_DS_LAYOUT_D2S6:
            quantize_mmq_q8_1<MMQ_Q8_1_DS_LAYOUT_D2S6>
                <<<num_blocks, block_size, 0, stream>>>(x, vy, kx0, kx1, kx0_padded);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

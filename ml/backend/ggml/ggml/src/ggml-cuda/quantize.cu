#include "hip/hip_runtime.h"
#include "quantize.cuh"
#include <cstdint>

static __global__ void quantize_q8_1(
        const float * __restrict__ x, void * __restrict__ vy,
        const int64_t ne00, const int64_t s01, const int64_t s02, const int64_t s03,
        const int64_t ne0, const int ne1, const int ne2) {
    const int64_t i0 = (int64_t)blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= ne0) {
        return;
    }

    const int64_t i1 = blockIdx.y;
    const int64_t i2 = blockIdx.z % ne2;
    const int64_t i3 = blockIdx.z / ne2;

    const int64_t & i00 = i0;
    const int64_t & i01 = i1;
    const int64_t & i02 = i2;
    const int64_t & i03 = i3;

    const int64_t i_cont = ((i3*ne2 + i2) * ne1 + i1) * ne0 + i0;

    block_q8_1 * y = (block_q8_1 *) vy;

    const int64_t ib  = i_cont / QK8_1; // block index
    const int64_t iqs = i_cont % QK8_1; // quant index

    const float xi = i0 < ne00 ? x[i03*s03 + i02*s02 + i01*s01 + i00] : 0.0f;
    float amax = fabsf(xi);
    float sum = xi;

    amax = warp_reduce_max(amax);
    sum  = warp_reduce_sum(sum);

    const float  d = amax / 127;
    const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    y[ib].qs[iqs] = q;

    if (iqs > 0) {
        return;
    }

    reinterpret_cast<half&>(y[ib].ds.x) = d;
    reinterpret_cast<half&>(y[ib].ds.y) = sum;
}

template <mmq_q8_1_ds_layout ds_layout>
static __global__ void quantize_mmq_q8_1(
        const float * __restrict__ x, const int32_t * __restrict__ ids, void * __restrict__ vy,
        const int64_t ne00, const int64_t s01, const int64_t s02, const int64_t s03,
        const int64_t ne0, const int ne1, const int ne2) {

    constexpr int vals_per_scale = ds_layout == MMQ_Q8_1_DS_LAYOUT_D2S6 ? 64 : 32;
    constexpr int vals_per_sum   = ds_layout == MMQ_Q8_1_DS_LAYOUT_D2S6 ? 16 : 32;

    const int64_t i0 = ((int64_t)blockDim.x*blockIdx.x + threadIdx.x)*4;

    if (i0 >= ne0) {
        return;
    }

    const int64_t i1 = blockIdx.y;
    const int64_t i2 = blockIdx.z % ne2;
    const int64_t i3 = blockIdx.z / ne2;

    const int64_t i00 = i0;
    const int64_t i01 = ids ? ids[i1] : i1;
    const int64_t i02 = i2;
    const int64_t i03 = i3;

    const float4 * x4 = (const float4 *) x;

    block_q8_1_mmq * y = (block_q8_1_mmq *) vy;

    const int64_t ib0 = blockIdx.z*((int64_t)gridDim.y*gridDim.x*blockDim.x/QK8_1); // first block of channel
    const int64_t ib  = ib0 + (i0 / (4*QK8_1))*ne1 + blockIdx.y;                    // block index in channel
    const int64_t iqs = i0 % (4*QK8_1);                                             // quant index in block

    // Load 4 floats per thread and calculate max. abs. value between them:
    const float4 xi = i0 < ne00 ? x4[(i03*s03 + i02*s02 + i01*s01 + i00)/4] : make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float amax = fabsf(xi.x);
    amax = fmaxf(amax, fabsf(xi.y));
    amax = fmaxf(amax, fabsf(xi.z));
    amax = fmaxf(amax, fabsf(xi.w));

    // Exchange max. abs. value between vals_per_scale/4 threads.
#pragma unroll
    for (int offset = vals_per_scale/8; offset > 0; offset >>= 1) {
        amax = fmaxf(amax, __shfl_xor_sync(0xFFFFFFFF, amax, offset, WARP_SIZE));
    }

    float sum;
    if (ds_layout != MMQ_Q8_1_DS_LAYOUT_D4) {
        sum = xi.x + xi.y + xi.z + xi.w;

        // Calculate sums across vals_per_sum/4 threads.
#pragma unroll
        for (int offset = vals_per_sum/8; offset > 0; offset >>= 1) {
            sum += __shfl_xor_sync(0xFFFFFFFF, sum, offset, WARP_SIZE);
        }
    }

    const float d_inv = 127.0f / amax;
    char4 q;
    q.x = roundf(xi.x*d_inv);
    q.y = roundf(xi.y*d_inv);
    q.z = roundf(xi.z*d_inv);
    q.w = roundf(xi.w*d_inv);

    // Write back 4 int8 values as a single 32 bit value for better memroy bandwidth:
    char4 * yqs4 = (char4 *) y[ib].qs;
    yqs4[iqs/4] = q;

    if (ds_layout == MMQ_Q8_1_DS_LAYOUT_D2S6) {
        if (iqs % 16 != 0 || iqs >= 96) {
            return;
        }

        y[ib].d2s6[2 + iqs/16] = sum;

        if (iqs % 64 != 0) {
            return;
        }

        const float d = 1.0f / d_inv;

        y[ib].d2s6[iqs/64] = d;

        return;
    }

    if (iqs % 32 != 0) {
        return;
    }

    const float d = 1.0f / d_inv;

    if (ds_layout == MMQ_Q8_1_DS_LAYOUT_DS4) {
        y[ib].ds4[iqs/32] = make_half2(d, sum);
    } else {
        y[ib].d4[iqs/32]  = d;
    }
}

void quantize_row_q8_1_cuda(
        const float * x, const int32_t * ids, void * vy, const ggml_type type_src0,
        const int64_t ne00, const int64_t s01, const int64_t s02, const int64_t s03,
        const int64_t ne0, const int64_t ne1, const int64_t ne2, const int64_t ne3, hipStream_t stream) {
    GGML_ASSERT(!ids);
    GGML_ASSERT(ne0 % QK8_1 == 0);

    const int64_t block_num_x = (ne0 + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
    const dim3 num_blocks(block_num_x, ne1, ne2*ne3);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(x, vy, ne00, s01, s02, s03, ne0, ne1, ne2);
    GGML_UNUSED(type_src0);
}

void quantize_mmq_q8_1_cuda(
        const float * x, const int32_t * ids, void * vy, const ggml_type type_src0,
        const int64_t ne00, const int64_t s01, const int64_t s02, const int64_t s03,
        const int64_t ne0, const int64_t ne1, const int64_t ne2, const int64_t ne3, hipStream_t stream) {
    GGML_ASSERT(ne0 % (4*QK8_1) == 0);

    const int64_t block_num_x = (ne0 + 4*CUDA_QUANTIZE_BLOCK_SIZE_MMQ - 1) / (4*CUDA_QUANTIZE_BLOCK_SIZE_MMQ);
    const dim3 num_blocks(block_num_x, ne1, ne2*ne3);
    const dim3 block_size(CUDA_QUANTIZE_BLOCK_SIZE_MMQ, 1, 1);
    switch (mmq_get_q8_1_ds_layout(type_src0)) {
        case MMQ_Q8_1_DS_LAYOUT_D4:
            quantize_mmq_q8_1<MMQ_Q8_1_DS_LAYOUT_D4>
                <<<num_blocks, block_size, 0, stream>>>(x, ids, vy, ne00, s01, s02, s03, ne0, ne1, ne2);
            break;
        case MMQ_Q8_1_DS_LAYOUT_DS4:
            quantize_mmq_q8_1<MMQ_Q8_1_DS_LAYOUT_DS4>
                <<<num_blocks, block_size, 0, stream>>>(x, ids, vy, ne00, s01, s02, s03, ne0, ne1, ne2);
            break;
        case MMQ_Q8_1_DS_LAYOUT_D2S6:
            quantize_mmq_q8_1<MMQ_Q8_1_DS_LAYOUT_D2S6>
                <<<num_blocks, block_size, 0, stream>>>(x, ids, vy, ne00, s01, s02, s03, ne0, ne1, ne2);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

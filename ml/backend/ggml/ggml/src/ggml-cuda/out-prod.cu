#include "out-prod.cuh"

#include <cstdint>

void ggml_cuda_out_prod(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_TENSOR_BINARY_OP_LOCALS

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    GGML_ASSERT(ne01 == ne11);
    GGML_ASSERT(ne0 == ne00);
    GGML_ASSERT(ne1 == ne10);

    GGML_ASSERT(ne2 % src0->ne[2] == 0);
    GGML_ASSERT(ne3 % src0->ne[3] == 0);

    GGML_ASSERT(ne2 == src1->ne[2]);
    GGML_ASSERT(ne3 == src1->ne[3]);

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       *  dst_d = (float       *)  dst->data;

    hipStream_t   stream = ctx.stream();
    hipblasHandle_t handle = ctx.cublas_handle();

    const float alpha = 1.0f;
    const float beta = 0.0f;

    CUBLAS_CHECK(hipblasSetStream(handle, stream));

    const int64_t lda = nb01 / sizeof(float);
    const int64_t ldc = nb1  / sizeof(float);

    const bool src1_T = ggml_is_transposed(src1);
    const hipblasOperation_t src1_cublas_op =  src1_T ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    const int64_t           ldb            = (src1_T ?        nb10 :        nb11) /  sizeof(float);
    GGML_ASSERT(                             (src1_T ?        nb11 :        nb10) == sizeof(float));

    // data strides in dimensions 2/3
    const size_t s02 = nb02 / sizeof(float);
    const size_t s03 = nb03 / sizeof(float);
    const size_t s12 = nb12 / sizeof(float);
    const size_t s13 = nb13 / sizeof(float);
    const size_t s2  = nb2  / sizeof(float);
    const size_t s3  = nb3  / sizeof(float);

    // dps == dst per src0, used for group query attention
    const int64_t dps2 = ne2 / ne02;
    const int64_t dps3 = ne3 / ne03;

    // TODO batched matrix multiplication
    for (int64_t i3 = 0; i3 < ne3; ++i3) {
        for (int64_t i2 = 0; i2 < ne2; ++i2) {
            CUBLAS_CHECK(
                hipblasSgemm(handle, HIPBLAS_OP_N, src1_cublas_op,
                        ne0, ne1, ne01,
                        &alpha, src0_d + (i3/dps3)*s03 + (i2/dps2)*s02, lda,
                                src1_d +  i3      *s13 +  i2      *s12, ldb,
                        &beta,  dst_d  +  i3      *s3  +  i2      *s2,  ldc));
        }
    }
}

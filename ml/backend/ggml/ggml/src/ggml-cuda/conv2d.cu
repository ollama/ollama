#include "hip/hip_runtime.h"
#include "conv2d.cuh"
#include "convert.cuh"

struct conv_params {
    const int64_t IW, IH;
    const int64_t OW, OH;
    const int64_t KW, KH;
    const int64_t ST_X, ST_Y;
    const int64_t PD_X, PD_Y;
    const int64_t DL_X, DL_Y;
    const int64_t IC, OC;
    const int64_t B;
    const int64_t TOTAL;
};

struct kernel_bounds {
    int64_t y_min, y_max;
    int64_t x_min, x_max;
};

__device__ __forceinline__ int64_t max64(int64_t a, int64_t b) {
    return (a > b) ? a : b;
}

__device__ __forceinline__ int64_t min64(int64_t a, int64_t b) {
    return (a < b) ? a : b;
}

__device__ __forceinline__ kernel_bounds calculate_kernel_bounds(int64_t out_x, int64_t out_y, const conv_params & P) {
    kernel_bounds bounds;
    bounds.y_min = max64(0, (P.PD_Y - out_y * P.ST_Y + P.DL_Y - 1) / P.DL_Y);
    bounds.y_max = min64(P.KH, (P.IH + P.PD_Y - out_y * P.ST_Y + P.DL_Y - 1) / P.DL_Y);
    bounds.x_min = max64(0, (P.PD_X - out_x * P.ST_X + P.DL_X - 1) / P.DL_X);
    bounds.x_max = min64(P.KW, (P.IW + P.PD_X - out_x * P.ST_X + P.DL_X - 1) / P.DL_X);
    return bounds;
}

__device__ __forceinline__ int calculate_input_coord(int64_t out_coord,
                                                     int64_t kern_coord,
                                                     int64_t stride,
                                                     int64_t dilation,
                                                     int64_t padding) {
    return out_coord * stride + kern_coord * dilation - padding;
}

struct whcn_layout {
    __device__ static int64_t input_index(int64_t n, int64_t c, int64_t y, int64_t x, const conv_params & P) {
        return n * (P.IC * P.IW * P.IH) + c * P.IW * P.IH + y * P.IW + x;
    }

    __device__ static int64_t kernel_index(int64_t c_out, int64_t c_in, int64_t ky, int64_t kx, const conv_params & P) {
        return c_out * (P.IC * P.KH * P.KW) + c_in * (P.KH * P.KW) + ky * P.KW + kx;
    }

    __device__ static int64_t output_index(int64_t n, int64_t c, int64_t y, int64_t x, const conv_params & P) {
        return n * (P.OC * P.OW * P.OH) + c * P.OW * P.OH + y * P.OW + x;
    }

    __device__ static void unpack_indices(int64_t             global_idx,
                                          const conv_params & P,
                                          int64_t &           n,
                                          int64_t &           c,
                                          int64_t &           out_y,
                                          int64_t &           out_x) {
        out_x = global_idx % P.OW;
        out_y = (global_idx / P.OW) % P.OH;
        c     = (global_idx / (P.OW * P.OH)) % P.OC;
        n     = global_idx / (P.OW * P.OH * P.OC);
    }
};

template <typename T, typename Layout>
static __global__ void conv2d_kernel(const float * __restrict__ input,
                                     const T * __restrict__ kernel,
                                     float * __restrict__ output,
                                     const conv_params P) {
    const int64_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_idx >= P.TOTAL) {
        return;
    }

    int64_t n, c_out, out_y, out_x;
    Layout::unpack_indices(global_idx, P, n, c_out, out_y, out_x);

    float acc = 0.0f;

    for (int64_t c_in = 0; c_in < P.IC; ++c_in) {
        kernel_bounds bounds = calculate_kernel_bounds(out_x, out_y, P);

        for (int64_t ky = bounds.y_min; ky < bounds.y_max; ++ky) {
            const int64_t in_y = calculate_input_coord(out_y, ky, P.ST_Y, P.DL_Y, P.PD_Y);

            for (int64_t kx = bounds.x_min; kx < bounds.x_max; ++kx) {
                const int64_t in_x = calculate_input_coord(out_x, kx, P.ST_X, P.DL_X, P.PD_X);

                const float input_val = input[Layout::input_index(n, c_in, in_y, in_x, P)];
                const T kernel_val = kernel[Layout::kernel_index(c_out, c_in, ky, kx, P)];
                acc += (input_val * ggml_cuda_cast<float>(kernel_val));
            }
        }
    }

    // [N, OC, OH, OW]
    output[Layout::output_index(n, c_out, out_y, out_x, P)] = acc;
}

template <typename T>
static void conv2d_cuda(const float * X_D, const T * K_D, float * Y_D, const conv_params P, hipStream_t st) {
    const int blocks = (P.TOTAL + CUDA_CONV2D_BLOCK_SIZE - 1) / CUDA_CONV2D_BLOCK_SIZE;
    conv2d_kernel<T, whcn_layout><<<blocks, CUDA_CONV2D_BLOCK_SIZE, 0, st>>>(X_D, K_D, Y_D, P);
}

static void conv2d_cuda_f16(const float * X_D, const half * K_D, float * Y_D, const conv_params P, hipStream_t st) {
    conv2d_cuda<half>(X_D, K_D, Y_D, P, st);
}

static void conv2d_cuda_f32(const float * X_D, const float * K_D, float * Y_D, const conv_params P, hipStream_t st) {
    conv2d_cuda<float>(X_D, K_D, Y_D, P, st);
}

void ggml_cuda_op_conv2d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * kernel = dst->src[0];
    const ggml_tensor * input  = dst->src[1];
    float *             K_D    = (float *) kernel->data;
    const float *       X_D    = (const float *) input->data;
    float *             Y_D    = (float *) dst->data;

    GGML_ASSERT(ggml_is_contiguous(kernel));
    GGML_ASSERT(kernel->type == GGML_TYPE_F16 || kernel->type == GGML_TYPE_F32);

    // same number of input channels
    GGML_ASSERT(input->ne[2] == kernel->ne[2]);

    hipStream_t st = ctx.stream();

    const int32_t * p    = (const int32_t *) dst->op_params;
    const int       ST_X = p[0];  // stride_x
    const int       ST_Y = p[1];  // stride_y
    const int       PD_X = p[2];  // padding_x
    const int       PD_Y = p[3];  // padding_y
    const int       DL_X = p[4];  // dilation_x
    const int       DL_Y = p[5];  // dilation_y

    // No cwhn
    GGML_ASSERT(p[6] == false);

    const int IW = input->ne[0];   // input_w
    const int IH = input->ne[1];   // input_h
    const int OW = dst->ne[0];     // output_w
    const int OH = dst->ne[1];     // output_h
    const int KW = kernel->ne[0];  // kernel_w
    const int KH = kernel->ne[1];  // kernel_h
    const int IC = input->ne[2];   // input_channels
    const int OC = kernel->ne[3];  // ouptut_chanles
    const int B  = input->ne[3];   // n_batches

    const int64_t total  = B * OC * OH * OW;
    conv_params   params = { IW, IH, OW, OH, KW, KH, ST_X, ST_Y, PD_X, PD_Y, DL_X, DL_Y, IC, OC, B, total };

    if (kernel->type == GGML_TYPE_F16) {
        conv2d_cuda_f16(X_D, (half *) K_D, Y_D, params, st);
    } else {
        conv2d_cuda_f32(X_D, K_D, Y_D, params, st);
    }
}

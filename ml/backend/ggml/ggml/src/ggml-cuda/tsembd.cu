#include "hip/hip_runtime.h"
#include "tsembd.cuh"

static __global__ void timestep_embedding_f32(const float * timesteps, float * dst, const int nb1, const int dim, const int max_period) {
    // blockIDx.y: idx of timesteps->ne[0]
    // blockIDx.x: idx of ((dim + 1) / 2) / BLOCK_SIZE
    int i = blockIdx.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    float * embed_data = (float *)((char *)dst +  i*nb1);

    if (dim % 2 != 0 && j == ((dim + 1) / 2)) {
        embed_data[dim] = 0.f;
    }

    int half = dim / 2;
    if (j >= half) {
        return;
    }

    float timestep = timesteps[i];
    float freq = (float)expf(-logf(max_period) * j / half);
    float arg = timestep * freq;
    embed_data[j] = cosf(arg);
    embed_data[j + half] = sinf(arg);
}

static void timestep_embedding_f32_cuda(const float * x, float * dst, const int ne00, const int nb1,
                                        const int dim, const int max_period, hipStream_t stream) {
    int half_ceil = (dim + 1) / 2;
    int num_blocks = (half_ceil + CUDA_TIMESTEP_EMBEDDING_BLOCK_SIZE - 1) / CUDA_TIMESTEP_EMBEDDING_BLOCK_SIZE;
    dim3 gridDim(num_blocks, ne00, 1);
    timestep_embedding_f32<<<gridDim, CUDA_TIMESTEP_EMBEDDING_BLOCK_SIZE, 0, stream>>>(x, dst, nb1, dim, max_period);
}

void ggml_cuda_op_timestep_embedding(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    const int dim = dst->op_params[0];
    const int max_period = dst->op_params[1];

    timestep_embedding_f32_cuda(src0_d, dst_d, src0->ne[0], dst->nb[1], dim, max_period, stream);
}

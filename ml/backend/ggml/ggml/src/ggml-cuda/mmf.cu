#include "ggml.h"
#include "mmf.cuh"
#include "mmid.cuh"


void ggml_cuda_mul_mat_f(ggml_backend_cuda_context & ctx, const ggml_tensor * src0, const ggml_tensor * src1, const ggml_tensor * ids, ggml_tensor * dst) {
    GGML_ASSERT(        src1->type == GGML_TYPE_F32);
    GGML_ASSERT(!ids ||  ids->type == GGML_TYPE_I32);
    GGML_ASSERT(         dst->type == GGML_TYPE_F32);


    GGML_TENSOR_BINARY_OP_LOCALS;

    const size_t ts_src0 = ggml_type_size(src0->type);
    const size_t ts_src1 = ggml_type_size(src1->type);
    const size_t ts_dst  = ggml_type_size(dst->type);

    GGML_ASSERT(ne13 == ne3);

    GGML_ASSERT(        nb00       == ts_src0);
    GGML_ASSERT(        nb10       == ts_src1);
    GGML_ASSERT(!ids || ids->nb[0] == ggml_type_size(ids->type));
    GGML_ASSERT(        nb0        == ts_dst);

    const float   * src1_d =       (const float   *) src1->data;
    const int32_t *  ids_d = ids ? (const int32_t *)  ids->data : nullptr;
    float         *  dst_d =       (float         *)  dst->data;

    const int64_t s01 = src0->nb[1] / ts_src0;
    const int64_t s11 = src1->nb[1] / ts_src1;
    const int64_t s1  =  dst->nb[1] / ts_dst;
    const int64_t s02 = src0->nb[2] / ts_src0;
    const int64_t s12 = src1->nb[2] / ts_src1;
    const int64_t s2  =  dst->nb[2] / ts_dst;
    const int64_t s03 = src0->nb[3] / ts_src0;
    const int64_t s13 = src1->nb[3] / ts_src1;
    const int64_t s3  =  dst->nb[3] / ts_dst;

    const int64_t ids_s0 = ids ? ids->nb[0] / ggml_type_size(ids->type) : 0;
    const int64_t ids_s1 = ids ? ids->nb[1] / ggml_type_size(ids->type) : 0;

    mmf_ids_data ids_info{};
    mmf_ids_data * ids_info_ptr = nullptr;
    ggml_cuda_pool_alloc<int32_t> ids_src_compact_dev;
    ggml_cuda_pool_alloc<int32_t> ids_dst_compact_dev;
    ggml_cuda_pool_alloc<int32_t> expert_bounds_dev;

    // For MUL_MAT_ID the memory layout is different than for MUL_MAT:
    const int64_t ncols_dst          = ids ? ne2  : ne1;
    const int64_t nchannels_dst      = ids ? ne1 : ne2;

    const int64_t stride_col_dst     = ids ? s2   : s1;
    const int64_t stride_col_y       = ids ? s12  : s11;
    const int64_t stride_channel_dst = ids ? s1 : s2;

    int64_t stride_channel_y         = ids ? s11  : s12;
    int64_t nchannels_y              = ids ? ne11 : ne12;

    //mul_mat_id: handle broadcast
    if (ids && nchannels_y == 1) {
        stride_channel_y = 0;
        nchannels_y      = ids->ne[0];
    }

    if (ids && ncols_dst > 16) {
        const int64_t n_expert_used = ids->ne[0];
        const int64_t n_experts     = ne02;
        const int64_t n_tokens      = ne12;
        const int64_t ne_get_rows   = n_tokens * n_expert_used;

        ids_src_compact_dev.alloc(ctx.pool(), ne_get_rows);
        ids_dst_compact_dev.alloc(ctx.pool(), ne_get_rows);
        expert_bounds_dev.alloc(ctx.pool(), n_experts + 1);

        const int si1  = static_cast<int>(ids_s1);
        const int sis1 = static_cast<int>(src1->nb[2] / src1->nb[1]);

        GGML_ASSERT(sis1 > 0);

        ggml_cuda_launch_mm_ids_helper(ids_d, ids_src_compact_dev.get(), ids_dst_compact_dev.get(), expert_bounds_dev.get(),
            static_cast<int>(n_experts), static_cast<int>(n_tokens), static_cast<int>(n_expert_used), static_cast<int>(ne11), si1, sis1, ctx.stream());
        CUDA_CHECK(hipGetLastError());

        ids_info.ids_src_compact   = ids_src_compact_dev.get();
        ids_info.ids_dst_compact   = ids_dst_compact_dev.get();
        ids_info.expert_bounds_dev = expert_bounds_dev.get();
        ids_info.n_experts         = static_cast<int>(n_experts);
        ids_info.sis1              = sis1;
        ids_info_ptr = &ids_info;
    }

    switch (src0->type) {
        case GGML_TYPE_F32: {
            const float * src0_d = (const float *) src0->data;
            constexpr int vals_per_T = 1;
            mul_mat_f_switch_cols_per_block(
                src0_d, src1_d, ids_d, dst_d, ne00/vals_per_T, ne01, ncols_dst, s01/vals_per_T, stride_col_y/vals_per_T, stride_col_dst,
                ids_s0, ids_s1, ne02, nchannels_y, nchannels_dst, s02/vals_per_T, stride_channel_y, stride_channel_dst,
                ne03, ne3, s03/vals_per_T, s13, s3, ctx.stream(), ids_info_ptr);
        } break;
        case GGML_TYPE_F16: {
            const half2 * src0_d = (const half2 *) src0->data;
            constexpr int vals_per_T = 2;
            mul_mat_f_switch_cols_per_block(
                src0_d, src1_d, ids_d, dst_d, ne00/vals_per_T, ne01, ncols_dst, s01/vals_per_T, stride_col_y/vals_per_T, stride_col_dst,
                ids_s0, ids_s1, ne02, nchannels_y, nchannels_dst, s02/vals_per_T, stride_channel_y, stride_channel_dst,
                ne03, ne3, s03/vals_per_T, s13, s3, ctx.stream(), ids_info_ptr);
        } break;
        case GGML_TYPE_BF16: {
            const nv_bfloat162 * src0_d = (const nv_bfloat162 *) src0->data;
            constexpr int vals_per_T = 2;
            mul_mat_f_switch_cols_per_block(
                src0_d, src1_d, ids_d, dst_d, ne00/vals_per_T, ne01, ncols_dst, s01/vals_per_T, stride_col_y/vals_per_T, stride_col_dst,
                ids_s0, ids_s1, ne02, nchannels_y, nchannels_dst, s02/vals_per_T, stride_channel_y, stride_channel_dst,
                ne03, ne3, s03/vals_per_T, s13, s3, ctx.stream(), ids_info_ptr);
        } break;
        default:
            GGML_ABORT("unsupported type: %s", ggml_type_name(src0->type));
    }
}

bool ggml_cuda_should_use_mmf(enum ggml_type type, int cc, int warp_size, const int64_t * src0_ne, const int src1_ncols, bool mul_mat_id) {

    if (ggml_is_quantized(type)) {
        return false;
    }

    if (src0_ne[0] % (warp_size * (4/ggml_type_size(type))) != 0) {
        return false;
    }
    if (src0_ne[1] % MMF_ROWS_PER_BLOCK != 0) {
        return false;
    }

    if (mul_mat_id) {
        if (src0_ne[1] <= 1024 && src1_ncols > 512) {
            return false;
        } else if(src0_ne[1] > 1024 && src1_ncols > 128) {
            return false;
        }
    } else {
        if (src1_ncols > 16) {
            return false;
        }
    }

    switch (type) {
        case GGML_TYPE_F32:
            return ampere_mma_available(cc);
        case GGML_TYPE_F16:
            return volta_mma_available(cc) || turing_mma_available(cc);
        case GGML_TYPE_BF16:
            return ampere_mma_available(cc);
        default:
            return false;
    }
}

#include "hip/hip_runtime.h"
#include "ggml.h"
#include "common.cuh"
#include "mma.cuh"
#include "mmf.cuh"

using namespace ggml_cuda_mma;

#define MMF_ROWS_PER_BLOCK 32

template <typename T, int rows_per_block, int cols_per_block, int nwarps>
__launch_bounds__(ggml_cuda_get_physical_warp_size()*nwarps, 1)
static __global__ void mul_mat_f(
        const T * __restrict__ x, const float * __restrict__ y, const int32_t * __restrict__ ids, float * __restrict__ dst,
        const int ncols, const int nchannels_y, const int stride_row, const int stride_col_y, const int stride_col_dst,
        const int channel_ratio, const int stride_channel_x, const int stride_channel_y, const int stride_channel_dst,
        const int sample_ratio, const int stride_sample_x, const int stride_sample_y, const int stride_sample_dst) {
#if !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA)
    typedef tile<16, 8, T>     tile_A;
    typedef tile< 8, 8, T>     tile_B;
    typedef tile<16, 8, float> tile_C;

    constexpr int warp_size = ggml_cuda_get_physical_warp_size();
    constexpr int tile_k_padded = warp_size + 4;
    constexpr int ntA = rows_per_block / tile_A::I;
    constexpr int ntB = (cols_per_block + tile_B::I - 1) / tile_B::I;

    const int row0        = blockIdx.x * rows_per_block;
    const int channel_dst = blockIdx.y;
    const int channel_x   = channel_dst / channel_ratio;
    const int channel_y   = channel_dst;
    const int sample_dst  = blockIdx.z;
    const int sample_x    = sample_dst / sample_ratio;
    const int sample_y    = sample_dst;

    x   += int64_t(sample_x)  *stride_sample_x   + channel_x  *stride_channel_x   + row0*stride_row ;
    y   += int64_t(sample_y)  *stride_sample_y   + channel_y  *stride_channel_y;
    dst += int64_t(sample_dst)*stride_sample_dst + channel_dst*stride_channel_dst;

    const float2 * y2 = (const float2 *) y;

    extern __shared__ char data_mmv[];

    tile_C C[ntA][ntB];

    T * tile_xy = (T *) data_mmv + threadIdx.y*(tile_A::I * tile_k_padded);

    for (int col = threadIdx.y*warp_size + threadIdx.x; col < ncols; col += nwarps*warp_size) {
        tile_A A[ntA][warp_size / tile_A::J];
#pragma unroll
        for (int itA = 0; itA < ntA; ++itA) {
#pragma unroll
            for (int i = 0; i < tile_A::I; ++i) {
                tile_xy[i*tile_k_padded + threadIdx.x] = x[(itA*tile_A::I + i)*stride_row  + col];
            }
#pragma unroll
            for (int k0 = 0; k0 < warp_size; k0 += tile_A::J) {
                load_ldmatrix(A[itA][k0/tile_A::J], tile_xy + k0, tile_k_padded);
            }
        }

#pragma unroll
        for (int itB = 0; itB < ntB; ++itB) {
            if constexpr (std::is_same_v<T, float>) {
#pragma unroll
                for (int j0 = 0; j0 < tile_B::I; ++j0) {
                    const int j = j0 + itB*tile_B::I;

                    tile_xy[j0*tile_k_padded + threadIdx.x] = j < cols_per_block ? y[j*stride_col_y + col] : 0.0f;
                }
            } else if constexpr (std::is_same_v<T, half2> || std::is_same_v<T, nv_bfloat162>) {
#pragma unroll
                for (int j0 = 0; j0 < tile_B::I; ++j0) {
                    const int j = j0 + itB*tile_B::I;

                    const float2 tmp = j < cols_per_block ? y2[j*stride_col_y + col] : make_float2(0.0f, 0.0f);
                    tile_xy[j0*tile_k_padded + threadIdx.x] = {tmp.x, tmp.y};
                }
            } else {
                static_assert(std::is_same_v<T, void>, "unsupported type");
            }
#pragma unroll
            for (int k0 = 0; k0 < warp_size; k0 += tile_B::J) {
                tile_B B;
                load_ldmatrix(B, tile_xy + k0, tile_k_padded);
#pragma unroll
                for (int itA = 0; itA < ntA; ++itA) {
                    mma(C[itA][itB], A[itA][k0/tile_B::J], B);
                }
            }
        }
    }

    float * buf_iw = (float *) data_mmv;
    constexpr int kiw = nwarps*rows_per_block + 4;

    if (nwarps > 1) {
        __syncthreads();
    }
#pragma unroll
    for (int itB = 0; itB < ntB; ++itB) {
#pragma unroll
        for (int itA = 0; itA < ntA; ++itA) {
#pragma unroll
            for (int l = 0; l < tile_C::ne; ++l) {
                const int i = threadIdx.y*rows_per_block + itA*tile_C::I + tile_C::get_i(l);
                const int j = itB*tile_C::J + tile_C::get_j(l);
                buf_iw[j*kiw + i] = C[itA][itB].x[l];
            }
        }
    }

    if (nwarps > 1) {
        __syncthreads();
    }

#pragma unroll
    for (int j0 = 0; j0 < cols_per_block; j0 += nwarps) {
        const int j = j0 + threadIdx.y;

        if (j0 + nwarps > cols_per_block && j >= cols_per_block) {
            return;
        }

        float sum = 0.0f;
        static_assert(rows_per_block == warp_size, "need loop/check");
#pragma unroll
        for (int i0 = 0; i0 < nwarps*rows_per_block; i0 += rows_per_block) {
            const int i = i0 + threadIdx.x;

            sum += buf_iw[j*kiw + i];
        }
        dst[j*stride_col_dst + row0 + threadIdx.x] = sum;
    }
#else
    NO_DEVICE_CODE;
    GGML_UNUSED(x); GGML_UNUSED(y); GGML_UNUSED(ids); GGML_UNUSED(dst);
    GGML_UNUSED(ncols); GGML_UNUSED(nchannels_y); GGML_UNUSED(stride_row); GGML_UNUSED(stride_col_y); GGML_UNUSED(stride_col_dst);
    GGML_UNUSED(channel_ratio); GGML_UNUSED(stride_channel_x); GGML_UNUSED(stride_channel_y); GGML_UNUSED(stride_channel_dst);
    GGML_UNUSED(sample_ratio); GGML_UNUSED(stride_sample_x); GGML_UNUSED(stride_sample_y); GGML_UNUSED(stride_sample_dst);
#endif // !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA)
}

template <typename T, int cols_per_block>
static void mul_mat_f_cuda(
        const T * x, const float * y, const int32_t * ids, float * dst,
        const int64_t ncols_x, const int64_t nrows_x,
        const int64_t stride_row, const int64_t stride_col_y, const int64_t stride_col_dst,
        const int64_t nchannels_x, const int64_t nchannels_y, const int64_t nchannels_dst,
        const int64_t stride_channel_x, const int64_t stride_channel_y, const int64_t stride_channel_dst, const int64_t nsamples_x,
        const int64_t nsamples_dst, const int64_t stride_sample_x, const int64_t stride_sample_y, const int64_t stride_sample_dst,
        hipStream_t stream) {
    typedef tile<16, 8, T>     tile_A;
    typedef tile< 8, 8, T>     tile_B;
    typedef tile<16, 8, float> tile_C;

    GGML_ASSERT(!ids && "mul_mat_id not implemented");

    GGML_ASSERT(ncols_x      % 2 == 0);
    GGML_ASSERT(stride_row   % 2 == 0);
    GGML_ASSERT(stride_col_y % 2 == 0);
    GGML_ASSERT(ids || nchannels_dst % nchannels_x == 0);
    GGML_ASSERT(       nsamples_dst  % nsamples_x  == 0);
    const int64_t channel_ratio = nchannels_dst / nchannels_x;
    const int64_t sample_ratio  = nsamples_dst  / nsamples_x;

    const int device = ggml_cuda_get_device();
    const int warp_size = ggml_cuda_info().devices[device].warp_size;

    int64_t nwarps_best     = 1;
    int64_t niter_best      = (ncols_x + warp_size*2 - 1) / (warp_size*2);
    int64_t max_block_size  = 256;
    for (int64_t nwarps = 2; nwarps <= max_block_size/warp_size; nwarps++) {
        const int64_t niter = (ncols_x + nwarps*warp_size*2 - 1) / (nwarps*warp_size*2);
        if (niter < niter_best) {
            niter_best  = niter;
            nwarps_best = nwarps;
        }
    }

    constexpr int rows_per_block = MMF_ROWS_PER_BLOCK;
    const int nbytes_shared_iter = nwarps_best * tile_A::I * (warp_size + 4) * 4;
    const int nbytes_shared_combine = GGML_PAD(cols_per_block, tile_B::I) * (nwarps_best*rows_per_block + 4) * 4;
    const int nbytes_shared = std::max(nbytes_shared_iter, nbytes_shared_combine);
    const dim3 block_nums(nrows_x/rows_per_block, nchannels_dst, nsamples_dst);
    const dim3 block_dims(warp_size, nwarps_best, 1);
    switch (nwarps_best) {
        case 1: {
            mul_mat_f<T, rows_per_block, cols_per_block, 1><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        case 2: {
            mul_mat_f<T, rows_per_block, cols_per_block, 2><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        case 3: {
            mul_mat_f<T, rows_per_block, cols_per_block, 3><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        case 4: {
            mul_mat_f<T, rows_per_block, cols_per_block, 4><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        case 5: {
            mul_mat_f<T, rows_per_block, cols_per_block, 5><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        case 6: {
            mul_mat_f<T, rows_per_block, cols_per_block, 6><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        case 7: {
            mul_mat_f<T, rows_per_block, cols_per_block, 7><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        case 8: {
            mul_mat_f<T, rows_per_block, cols_per_block, 8><<<block_nums, block_dims, nbytes_shared, stream>>>
                (x, y, ids, dst, ncols_x, nchannels_y, stride_row, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
        } break;
        default: {
            GGML_ABORT("fatal error");
        } break;
    }
}

template <typename T>
static void mul_mat_f_switch_cols_per_block(
        const T * x, const float * y, const int32_t * ids, float * dst,
        const int64_t ncols_x, const int64_t nrows_x, const int64_t ncols_dst,
        const int64_t stride_row, const int64_t stride_col_y, const int64_t stride_col_dst,
        const int64_t nchannels_x, const int64_t nchannels_y, const int64_t nchannels_dst,
        const int64_t stride_channel_x, const int64_t stride_channel_y, const int64_t stride_channel_dst, const int64_t nsamples_x,
        const int64_t nsamples_dst, const int64_t stride_sample_x, const int64_t stride_sample_y, const int64_t stride_sample_dst,
        hipStream_t stream) {
    switch (ncols_dst) {
        case  1: {
            mul_mat_f_cuda<T,  1>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  2: {
            mul_mat_f_cuda<T,  2>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  3: {
            mul_mat_f_cuda<T,  3>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  4: {
            mul_mat_f_cuda<T,  4>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  5: {
            mul_mat_f_cuda<T,  5>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  6: {
            mul_mat_f_cuda<T,  6>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  7: {
            mul_mat_f_cuda<T,  7>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  8: {
            mul_mat_f_cuda<T,  8>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case  9: {
            mul_mat_f_cuda<T,  9>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case 10: {
            mul_mat_f_cuda<T, 10>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case 11: {
            mul_mat_f_cuda<T, 11>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case 12: {
            mul_mat_f_cuda<T, 12>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case 13: {
            mul_mat_f_cuda<T, 13>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case 14: {
            mul_mat_f_cuda<T, 14>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case 15: {
            mul_mat_f_cuda<T, 15>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        case 16: {
            mul_mat_f_cuda<T, 16>(x, y, ids, dst, ncols_x, nrows_x, stride_row, stride_col_y, stride_col_dst,
                nchannels_x, nchannels_y,  nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                nsamples_x,                nsamples_dst,  stride_sample_x,  stride_sample_y,  stride_sample_dst,  stream);
        } break;
        default: {
            GGML_ABORT("fatal error");
        } break;
    }
}

void ggml_cuda_mul_mat_f(ggml_backend_cuda_context & ctx, const ggml_tensor * src0, const ggml_tensor * src1, const ggml_tensor * ids, ggml_tensor * dst) {
    GGML_ASSERT(        src1->type == GGML_TYPE_F32);
    GGML_ASSERT(!ids ||  ids->type == GGML_TYPE_I32);
    GGML_ASSERT(         dst->type == GGML_TYPE_F32);

    GGML_TENSOR_BINARY_OP_LOCALS;

    const size_t ts_src0 = ggml_type_size(src0->type);
    const size_t ts_src1 = ggml_type_size(src1->type);
    const size_t ts_dst  = ggml_type_size(dst->type);

    GGML_ASSERT(ne13 == ne3);

    GGML_ASSERT(        nb00       == ts_src0);
    GGML_ASSERT(        nb10       == ts_src1);
    GGML_ASSERT(!ids || ids->nb[0] == ggml_type_size(ids->type));
    GGML_ASSERT(        nb0        == ts_dst);

    const int cc = ggml_cuda_info().devices[ggml_cuda_get_device()].cc;
    const enum ggml_prec prec = fast_fp16_available(cc) ? ggml_prec(dst->op_params[0]) : GGML_PREC_F32;

    const float   * src1_d =       (const float   *) src1->data;
    const int32_t *  ids_d = ids ? (const int32_t *)  ids->data : nullptr;
    float         *  dst_d =       (float         *)  dst->data;

    const int64_t s01 = src0->nb[1] / ts_src0;
    const int64_t s11 = src1->nb[1] / ts_src1;
    const int64_t s1  =  dst->nb[1] / ts_dst;
    const int64_t s02 = src0->nb[2] / ts_src0;
    const int64_t s12 = src1->nb[2] / ts_src1;
    const int64_t s2  =  dst->nb[2] / ts_dst;
    const int64_t s03 = src0->nb[3] / ts_src0;
    const int64_t s13 = src1->nb[3] / ts_src1;
    const int64_t s3  =  dst->nb[3] / ts_dst;

    // For MUL_MAT_ID the memory layout is different than for MUL_MAT:
    const int64_t ncols_dst          = ids ? ne2  : ne1;
    const int64_t nchannels_y        = ids ? ne11 : ne12;
    const int64_t nchannels_dst      = ids ? ne1  : ne2;
    const int64_t stride_channel_dst = ids ? s1   : s2;
    const int64_t stride_channel_y   = ids ? s11  : s12;

    GGML_ASSERT(!ids || ncols_dst == 1);

    switch (src0->type) {
        case GGML_TYPE_F32: {
            const float * src0_d = (const float *) src0->data;
            constexpr int vals_per_T = 1;
            mul_mat_f_switch_cols_per_block(
                src0_d, src1_d, ids_d, dst_d, ne00/vals_per_T, ne01, ncols_dst, s01/vals_per_T, s11/vals_per_T, s1,
                ne02, nchannels_y, nchannels_dst, s02/vals_per_T, stride_channel_y, stride_channel_dst,
                ne03,              ne3,           s03/vals_per_T, s13,              s3,                 ctx.stream());
        } break;
        case GGML_TYPE_F16: {
            const half2 * src0_d = (const half2 *) src0->data;
            constexpr int vals_per_T = 2;
            mul_mat_f_switch_cols_per_block(
                src0_d, src1_d, ids_d, dst_d, ne00/vals_per_T, ne01, ncols_dst, s01/vals_per_T, s11/vals_per_T, s1,
                ne02, nchannels_y, nchannels_dst, s02/vals_per_T, stride_channel_y, stride_channel_dst,
                ne03,              ne3,           s03/vals_per_T, s13,              s3,                 ctx.stream());
        } break;
        case GGML_TYPE_BF16: {
            const nv_bfloat162 * src0_d = (const nv_bfloat162 *) src0->data;
            constexpr int vals_per_T = 2;
            mul_mat_f_switch_cols_per_block(
                src0_d, src1_d, ids_d, dst_d, ne00/vals_per_T, ne01, ncols_dst, s01/vals_per_T, s11/vals_per_T, s1,
                ne02, nchannels_y, nchannels_dst, s02/vals_per_T, stride_channel_y, stride_channel_dst,
                ne03,              ne3,           s03/vals_per_T, s13,              s3,                 ctx.stream());
        } break;
        default:
            GGML_ABORT("unsupported type: %s", ggml_type_name(src0->type));
    }
}

bool ggml_cuda_should_use_mmf(enum ggml_type type, int cc, int warp_size, const int64_t * src0_ne, int64_t ne11) {
    if (src0_ne[0] % (warp_size * (4/ggml_type_size(type))) != 0) {
        return false;
    }
    if (src0_ne[1] % MMF_ROWS_PER_BLOCK != 0) {
        return false;
    }
    if (ne11 > 16) {
        return false;
    }
    switch (type) {
        case GGML_TYPE_F32:
            return ampere_mma_available(cc);
        case GGML_TYPE_F16:
            return turing_mma_available(cc);
        case GGML_TYPE_BF16:
            return ampere_mma_available(cc);
        default:
            return false;
    }
}

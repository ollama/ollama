#include "hip/hip_runtime.h"
#include "pad_reflect_1d.cuh"

static __global__ __launch_bounds__(CUDA_PAD_REFLECT_1D_BLOCK_SIZE, 1) void
    pad_reflect_1d_kernel_f32(
        const void * __restrict__ src0,
        void * __restrict__       dst,
        const int64_t             ne0,
        const int64_t             ne00,
        const uint3               ne01,
        const int64_t             ne02,
        const int64_t             ne03,
        const int64_t             nb00,
        const int64_t             nb01,
        const int64_t             nb02,
        const int64_t             nb03,
        const int64_t             nb0,
        const int64_t             nb1,
        const int64_t             nb2,
        const int64_t             nb3,
        const int                 p0,
        const int                 p1) {
    const int64_t i3 = blockIdx.z;
    const int64_t i2 = blockIdx.y;

    const uint2   div_mod_packed = fast_div_modulo(blockIdx.x, ne01);
    const int64_t tile1          = div_mod_packed.y;  // i1
    const int64_t tile0          = div_mod_packed.x;  // nth i0 tile
    const int64_t i1             = tile1;
    const int64_t i0             = threadIdx.x + tile0 * blockDim.x;

    // ne01.z is original value of unpacked ne01 (see init_fastdiv_values in common.cuh)
    if (i0 >= ne0 || i1 >= ne01.z || i2 >= ne02 || i3 >= ne03) {
        return;
    }

    const char * src0_ptr = (const char *) src0 + i3 * nb03 + i2 * nb02 + i1 * nb01;
    char *       dst_ptr  = (char *) dst + i3 * nb3 + i2 * nb2 + i1 * nb1;

    const int64_t rel_i0 = i0 - p0;  // relative i0 in src0
    int64_t src_idx;

    if (rel_i0 < 0) {
        // Left padding - reflect
        src_idx = -rel_i0;
    } else if (rel_i0 < ne00) {
        // Middle - copy
        src_idx = rel_i0;
    } else {
        // Right padding - reflect
        src_idx = 2 * ne00 - 2 - rel_i0;
    }
    const float value               = *(const float *) (src0_ptr + src_idx * nb00);
    *(float *) (dst_ptr + i0 * nb0) = value;

    GGML_UNUSED(p1);
}

void ggml_cuda_op_pad_reflect_1d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0   = dst->src[0];
    hipStream_t        stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    const int32_t * opts = (const int32_t *) dst->op_params;
    const int       p0   = opts[0];
    const int       p1   = opts[1];

    const int64_t ne00        = src0->ne[0];
    const int64_t ne01        = src0->ne[1];
    const uint3   ne01_packed = init_fastdiv_values(ne01);
    const int64_t ne02        = src0->ne[2];
    const int64_t ne03        = src0->ne[3];

    const int64_t ne0 = dst->ne[0];

    // sanity: padded length matches
    GGML_ASSERT(ne0 == ne00 + p0 + p1);

    constexpr int64_t bx     = CUDA_PAD_REFLECT_1D_BLOCK_SIZE;  // threads per block (x)
    const int64_t     tiles0 = (ne0 + bx - 1) / bx;             // number of tiles along i0
    // grid.x covers i1 and all tiles of i0: [ne01 * tiles0]
    // grid.y covers i2: [ne02]
    // grid.z covers i3: [ne03]
    const dim3        grid_dims((unsigned) (ne01 * tiles0), (unsigned) ne02, (unsigned) ne03);
    const dim3        block_dims((unsigned) bx, 1, 1);

    pad_reflect_1d_kernel_f32<<<grid_dims, block_dims, 0, stream>>>(
        src0->data, dst->data, ne0, ne00, ne01_packed, ne02, ne03, src0->nb[0], src0->nb[1], src0->nb[2], src0->nb[3],
        dst->nb[0], dst->nb[1], dst->nb[2], dst->nb[3], p0, p1);
}

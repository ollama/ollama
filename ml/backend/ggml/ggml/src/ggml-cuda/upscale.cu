#include "hip/hip_runtime.h"
#include "upscale.cuh"

static __global__ void upscale_f32(const float * x, float * dst,
        const int nb00, const int nb01, const int nb02, const int nb03,
        const int ne10, const int ne11, const int ne12, const int ne13,
        const float sf0, const float sf1, const float sf2, const float sf3) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= ne10 * ne11 * ne12 * ne13) {
        return;
    }

    int i10 = index % ne10;
    int i11 = (index / ne10) % ne11;
    int i12 = (index / (ne10 * ne11)) % ne12;
    int i13 = (index / (ne10 * ne11 * ne12)) % ne13;

    int i00 = i10 / sf0;
    int i01 = i11 / sf1;
    int i02 = i12 / sf2;
    int i03 = i13 / sf3;

    dst[index] = *( (const float *)((const char *)x + i03 * nb03 + i02 * nb02 + i01 * nb01 + i00 * nb00) );
}

static __global__ void upscale_f32_bilinear(const float * x, float * dst,
        const int nb00, const int nb01, const int nb02, const int nb03,
        const int ne00_src, const int ne01_src,
        const int ne10_dst, const int ne11_dst, const int ne12_dst, const int ne13_dst,
        const float sf0, const float sf1, const float sf2, const float sf3,
        const float pixel_offset) {
    const int64_t index              = threadIdx.x + blockIdx.x * blockDim.x;
    const int64_t dst_total_elements = ne10_dst * ne11_dst * ne12_dst * ne13_dst;

    if (index >= dst_total_elements) {
        return;
    }

    const int i10_dst = index % ne10_dst;
    const int i11_dst = (index / ne10_dst) % ne11_dst;
    const int i12_dst = (index / (ne10_dst * ne11_dst)) % ne12_dst;
    const int i13_dst = index / (ne10_dst * ne11_dst * ne12_dst);

    const int i02_src = (int)(i12_dst / sf2);
    const int i03_src = (int)(i13_dst / sf3);

    const float y_src_f = ((float)i11_dst + pixel_offset) / sf1 - pixel_offset;
    int y0_src    = (int)floorf(y_src_f);
    int y1_src    = y0_src + 1;

    y0_src = max(0, min(y0_src, ne01_src - 1));
    y1_src = max(0, min(y1_src, ne01_src - 1));

    float dy = y_src_f - (float)y0_src;
    dy       = max(0.0f, min(dy, 1.0f));

    float x_src_f = ((float)i10_dst + pixel_offset) / sf0 - pixel_offset;
    int x0_src    = (int)floorf(x_src_f);
    int x1_src    = x0_src + 1;

    x0_src = max(0, min(x0_src, ne00_src - 1));
    x1_src = max(0, min(x1_src, ne00_src - 1));

    float dx = x_src_f - (float)x0_src;
    dx = max(0.0f, min(dx, 1.0f));

    const float * p_a = (const float *)((const char *)x + (int64_t)x0_src * nb00 + (int64_t)y0_src * nb01 + (int64_t)i02_src * nb02 + (int64_t)i03_src * nb03);
    const float * p_b = (const float *)((const char *)x + (int64_t)x1_src * nb00 + (int64_t)y0_src * nb01 + (int64_t)i02_src * nb02 + (int64_t)i03_src * nb03);
    const float * p_c = (const float *)((const char *)x + (int64_t)x0_src * nb00 + (int64_t)y1_src * nb01 + (int64_t)i02_src * nb02 + (int64_t)i03_src * nb03);
    const float * p_d = (const float *)((const char *)x + (int64_t)x1_src * nb00 + (int64_t)y1_src * nb01 + (int64_t)i02_src * nb02 + (int64_t)i03_src * nb03);

    const float val_a = *p_a;
    const float val_b = *p_b;
    const float val_c = *p_c;
    const float val_d = *p_d;

    float result = val_a * (1.0f - dx) * (1.0f - dy) +
                   val_b * dx * (1.0f - dy) +
                   val_c * (1.0f - dx) * dy +
                   val_d * dx * dy;

    dst[index] = result;
}

static void upscale_f32_cuda(const float * x, float * dst,
        const int nb00, const int nb01, const int nb02, const int nb03,
        const int ne10, const int ne11, const int ne12, const int ne13,
        const float sf0, const float sf1, const float sf2, const float sf3,
        hipStream_t stream) {
    const int64_t dst_size   = ne10 * ne11 * ne12 * ne13;
    const int64_t num_blocks = (dst_size + CUDA_UPSCALE_BLOCK_SIZE - 1) / CUDA_UPSCALE_BLOCK_SIZE;

    upscale_f32<<<num_blocks, CUDA_UPSCALE_BLOCK_SIZE,0,stream>>>(x, dst, nb00, nb01, nb02, nb03, ne10, ne11, ne12, ne13, sf0, sf1, sf2, sf3);
}

static void upscale_f32_bilinear_cuda(const float * x, float * dst,
        const int nb00, const int nb01, const int nb02, const int nb03,
        const int ne00_src, const int ne01_src,
        const int ne10_dst, const int ne11_dst, const int ne12_dst, const int ne13_dst,
        const float sf0, const float sf1, const float sf2, const float sf3,
        const float pixel_offset, hipStream_t stream) {
    const int64_t dst_size   = ne10_dst * ne11_dst * ne12_dst * ne13_dst;
    const int64_t num_blocks = (dst_size + CUDA_UPSCALE_BLOCK_SIZE - 1) / CUDA_UPSCALE_BLOCK_SIZE;

    upscale_f32_bilinear<<<num_blocks, CUDA_UPSCALE_BLOCK_SIZE,0,stream>>>(x, dst, nb00, nb01, nb02, nb03, ne00_src, ne01_src, ne10_dst, ne11_dst, ne12_dst, ne13_dst, sf0, sf1, sf2, sf3, pixel_offset);
}

void ggml_cuda_op_upscale(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int mode_flags = dst->op_params[0];
    const ggml_scale_mode mode = (ggml_scale_mode)(mode_flags & 0xFF);

    float sf0 = (float)dst->ne[0]/src0->ne[0];
    float sf1 = (float)dst->ne[1]/src0->ne[1];
    float sf2 = (float)dst->ne[2]/src0->ne[2];
    const float sf3 = (float)dst->ne[3]/src0->ne[3];

    if (mode == GGML_SCALE_MODE_NEAREST) {
        upscale_f32_cuda(src0_d, dst_d, src0->nb[0], src0->nb[1], src0->nb[2], src0->nb[3], dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3], sf0, sf1, sf2, sf3, stream);
    } else if (mode == GGML_SCALE_MODE_BILINEAR) {
        float pixel_offset = 0.5f;
        if (mode_flags & GGML_SCALE_FLAG_ALIGN_CORNERS) {
            sf0          = (float)(dst->ne[0] - 1) / (src0->ne[0] - 1);
            sf1          = (float)(dst->ne[1] - 1) / (src0->ne[1] - 1);
            pixel_offset = 0.0f;
        }
        upscale_f32_bilinear_cuda(src0_d, dst_d, src0->nb[0], src0->nb[1], src0->nb[2], src0->nb[3],
                                 src0->ne[0], src0->ne[1], dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3],
                                 sf0, sf1, sf2, sf3, pixel_offset, stream);
    }
}

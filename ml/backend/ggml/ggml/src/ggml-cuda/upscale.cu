#include "hip/hip_runtime.h"
#include "upscale.cuh"

static __global__ void upscale_f32(const float * x, float * dst,
        const int nb00, const int nb01, const int nb02, const int nb03,
        const int ne10, const int ne11, const int ne12, const int ne13,
        const float sf0, const float sf1, const float sf2, const float sf3) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= ne10 * ne11 * ne12 * ne13) {
        return;
    }

    int i10 = index % ne10;
    int i11 = (index / ne10) % ne11;
    int i12 = (index / (ne10 * ne11)) % ne12;
    int i13 = (index / (ne10 * ne11 * ne12)) % ne13;

    int i00 = i10 / sf0;
    int i01 = i11 / sf1;
    int i02 = i12 / sf2;
    int i03 = i13 / sf3;

    dst[index] = *(float *)((char *)x + i03 * nb03 + i02 * nb02 + i01 * nb01 + i00 * nb00);
}

static void upscale_f32_cuda(const float * x, float * dst,
        const int nb00, const int nb01, const int nb02, const int nb03,
        const int ne10, const int ne11, const int ne12, const int ne13,
        const float sf0, const float sf1, const float sf2, const float sf3,
        hipStream_t stream) {
    int dst_size = ne10 * ne11 * ne12 * ne13;
    int num_blocks = (dst_size + CUDA_UPSCALE_BLOCK_SIZE - 1) / CUDA_UPSCALE_BLOCK_SIZE;

    upscale_f32<<<num_blocks, CUDA_UPSCALE_BLOCK_SIZE,0,stream>>>(x, dst, nb00, nb01, nb02, nb03, ne10, ne11, ne12, ne13, sf0, sf1, sf2, sf3);
}

void ggml_cuda_op_upscale(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const float sf0 = (float)dst->ne[0]/src0->ne[0];
    const float sf1 = (float)dst->ne[1]/src0->ne[1];
    const float sf2 = (float)dst->ne[2]/src0->ne[2];
    const float sf3 = (float)dst->ne[3]/src0->ne[3];

    upscale_f32_cuda(src0_d, dst_d, src0->nb[0], src0->nb[1], src0->nb[2], src0->nb[3], dst->ne[0], dst->ne[1], dst->ne[2], dst->ne[3], sf0, sf1, sf2, sf3, stream);
}

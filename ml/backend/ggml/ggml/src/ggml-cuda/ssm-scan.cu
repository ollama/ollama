#include "hip/hip_runtime.h"
#include "ssm-scan.cuh"

template <size_t splitD, size_t N>
__global__ void __launch_bounds__(splitD, 2)
    ssm_scan_f32(const float * __restrict__ src0, const float * __restrict__ src1, const float * __restrict__ src2,
                 const float * __restrict__ src3, const float * __restrict__ src4, const float * __restrict__ src5,
                 const int src0_nb1, const int src0_nb2, const int src1_nb0, const int src1_nb1, const int src1_nb2,
                 const int src1_nb3, const int src2_nb0, const int src2_nb1, const int src2_nb2, const int src3_nb1,
                 const int src4_nb1, const int src4_nb2, const int src5_nb1, const int src5_nb2,
                 float * __restrict__ dst, const int64_t L) {
    GGML_UNUSED(src1_nb0);
    GGML_UNUSED(src2_nb0);
    const int bidx = blockIdx.x;  // split along B
    const int bidy = blockIdx.y;  // split along D
    const int tid  = threadIdx.x;
    const int wid  = tid / 32;
    const int wtid = tid % 32;

    extern __shared__ float smem[];
    const int               stride_sA  = N + 1;
    const int               stride_ss0 = N + 1;
    float *                 smem_A     = smem;
    float *                 smem_s0    = smem_A + splitD * stride_sA;

    const float * s0_block = (const float *) ((const char *) src0 + bidx * src0_nb2 + bidy * splitD * src0_nb1);
    const float * x_block  = (const float *) ((const char *) src1 + (bidx * src1_nb2) + bidy * splitD * sizeof(float));
    const float * dt_block = (const float *) ((const char *) src2 + (bidx * src2_nb2) + bidy * splitD * sizeof(float));
    const float * A_block  = (const float *) ((const char *) src3 + bidy * splitD * src3_nb1);
    const float * B_block  = (const float *) ((const char *) src4 + (bidx * src4_nb2));
    const float * C_block  = (const float *) ((const char *) src5 + (bidx * src5_nb2));
    float *       y_block  = (float *) ((char *) dst + (bidx * src1_nb2) + bidy * splitD * sizeof(float));
    float *       s_block  = (float *) ((char *) dst + src1_nb3 + bidx * src0_nb2 + bidy * splitD * src0_nb1);

    const int stride_s0 = src0_nb1 / sizeof(float);
    const int stride_x  = src1_nb1 / sizeof(float);
    const int stride_dt = src2_nb1 / sizeof(float);
    const int stride_A  = src3_nb1 / sizeof(float);
    const int stride_B  = src4_nb1 / sizeof(float);
    const int stride_C  = src5_nb1 / sizeof(float);
    const int stride_s  = stride_s0;
    const int stride_y  = stride_x;

    // can N not be 16? for example 32?
    if (N == 16) {
#pragma unroll
        for (size_t i = 0; i < splitD / 4; i += 2) {
            float value = A_block[(wid * warpSize + i) * stride_A + wtid];
            // todo: bank conflict
            // I am always confused with how to use the swizzling method to solve
            // bank conflit. Hoping somebody can tell me.
            smem_A[(wid * warpSize + i) * stride_sA + wtid + ((wtid / 16) > 0 ? 1 : 0)] = value;
        }
#pragma unroll
        for (size_t i = 0; i < splitD / 4; i += 2) {
            float value = s0_block[(wid * warpSize + i) * stride_s0 + wtid];
            smem_s0[(wid * warpSize + i) * stride_ss0 + wtid + ((wtid / 16) > 0 ? 1 : 0)] = value;
        }
    }

    __syncthreads();

    for (int64_t i = 0; i < L; i++) {
        float dt_soft_plus = dt_block[i * stride_dt + tid];
        if (dt_soft_plus <= 20.0f) {
            dt_soft_plus = log1pf(exp(dt_soft_plus));
        }
        float x_dt = x_block[i * stride_x + tid] * dt_soft_plus;
        float sumf = 0.0f;
#pragma unroll
        for (size_t j = 0; j < N; j++) {
            float state = (smem_s0[tid * stride_ss0 + j] * expf(dt_soft_plus * smem_A[tid * stride_sA + j])) +
                          (B_block[i * stride_B + j] * x_dt);
            sumf += state * C_block[i * stride_C + j];
            if (i == L - 1) {
                s_block[tid * stride_s + j] = state;
            } else {
                smem_s0[tid * stride_ss0 + j] = state;
            }
        }
        __syncthreads();
        y_block[i * stride_y + tid] = sumf;
    }
}

static void ssm_scan_f32_cuda(const float * src0, const float * src1, const float * src2, const float * src3,
                              const float * src4, const float * src5, const int src0_nb1, const int src0_nb2,
                              const int src1_nb0, const int src1_nb1, const int src1_nb2, const int src1_nb3,
                              const int src2_nb0, const int src2_nb1, const int src2_nb2, const int src3_nb1,
                              const int src4_nb1, const int src4_nb2, const int src5_nb1, const int src5_nb2,
                              float * dst, const int64_t N, const int64_t D, const int64_t L, const int64_t B,
                              hipStream_t stream) {
    const int threads = 128;
    // todo: consider D cannot be divided,does this situation exist?
    GGML_ASSERT(D % threads == 0);
    const dim3 blocks(B, (D + threads - 1) / threads, 1);
    const int  smem_size = (threads * (N + 1) * 2) * sizeof(float);
    if (N == 16) {
        ssm_scan_f32<128, 16><<<blocks, threads, smem_size, stream>>>(
            src0, src1, src2, src3, src4, src5, src0_nb1, src0_nb2, src1_nb0, src1_nb1, src1_nb2, src1_nb3, src2_nb0,
            src2_nb1, src2_nb2, src3_nb1, src4_nb1, src4_nb2, src5_nb1, src5_nb2, dst, L);
    } else {
        GGML_ABORT("doesn't support N!=16.");
    }
}

void ggml_cuda_op_ssm_scan(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0];  // s
    const struct ggml_tensor * src1 = dst->src[1];  // x
    const struct ggml_tensor * src2 = dst->src[2];  // dt
    const struct ggml_tensor * src3 = dst->src[3];  // A
    const struct ggml_tensor * src4 = dst->src[4];  // B
    const struct ggml_tensor * src5 = dst->src[5];  // C

    //   const int64_t d_state = src0->ne[0];
    //   const int64_t d_inner = src0->ne[1];
    //   const int64_t l = src1->ne[1];
    //   const int64_t b = src0->ne[2];

    const int64_t nc  = src0->ne[0];  // d_state
    const int64_t nr  = src0->ne[1];  // d_inner
    const int64_t n_t = src1->ne[1];  // number of tokens per sequence
    const int64_t n_s = src0->ne[2];  // number of sequences in the batch

    GGML_ASSERT(ggml_nelements(src1) + ggml_nelements(src0) == ggml_nelements(dst));
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src2->nb[0] == sizeof(float));
    GGML_ASSERT(src3->nb[0] == sizeof(float));
    GGML_ASSERT(src4->nb[0] == sizeof(float));
    GGML_ASSERT(src5->nb[0] == sizeof(float));
    // required for the dot product between s and C
    GGML_ASSERT(src0->nb[1] == src0->ne[0] * sizeof(float));
    // required for per-sequence offsets for states
    GGML_ASSERT(src0->nb[2] == src0->ne[0] * src0->ne[1] * sizeof(float));
    // required to get correct offset for state destination (i.e. src1->nb[3])
    GGML_ASSERT(src1->nb[3] == src1->ne[0] * src1->ne[1] * src1->ne[2] * sizeof(float));

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    const float * src2_d = (const float *) src2->data;
    const float * src3_d = (const float *) src3->data;
    const float * src4_d = (const float *) src4->data;
    const float * src5_d = (const float *) src5->data;
    float *       dst_d  = (float *) dst->data;
    hipStream_t  stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    ssm_scan_f32_cuda(src0_d, src1_d, src2_d, src3_d, src4_d, src5_d, src0->nb[1], src0->nb[2], src1->nb[0],
                      src1->nb[1], src1->nb[2], src1->nb[3], src2->nb[0], src2->nb[1], src2->nb[2], src3->nb[1],
                      src4->nb[1], src4->nb[2], src5->nb[1], src5->nb[2], dst_d, nc, nr, n_t, n_s, stream);
}

#include "hip/hip_runtime.h"
#include "ssm-scan.cuh"

template <size_t splitD, size_t N>
__global__ void __launch_bounds__(splitD, 2)
    ssm_scan_f32(const float * __restrict__ src0, const float * __restrict__ src1, const float * __restrict__ src2,
                 const float * __restrict__ src3, const float * __restrict__ src4, const float * __restrict__ src5,
                 const int32_t * __restrict__ src6, float * __restrict__ dst,
                 const int src0_nb2, const int src0_nb3, const int src1_nb2, const int src1_nb3,
                 const int src2_nb1, const int src2_nb2, const int src3_nb1,
                 const int src4_nb2, const int src4_nb3, const int src5_nb2, const int src5_nb3,
                 const int64_t s_off, const int64_t d_inner, const int64_t L) {

    constexpr int warp_size = ggml_cuda_get_physical_warp_size();
    const int bidx = blockIdx.x;  // split along B (sequences)
    const int bidy = blockIdx.y;  // split along D (d_inner)
    const int tid  = threadIdx.x;
    const int wid  = tid / 32;
    const int wtid = tid % 32;

    extern __shared__ float smem[];
    const int               stride_sA  = N + 1;
    const int               stride_ss0 = N + 1;
    float *                 smem_A     = smem;
    float *                 smem_s0    = smem_A + splitD * stride_sA;

    const float * s0_block = (const float *) ((const char *) src0 + src6[bidx] * src0_nb3 + bidy * splitD * src0_nb2);
    const float * x_block  = (const float *) ((const char *) src1 + (bidx * src1_nb3) + bidy * splitD * sizeof(float));
    const float * dt_block = (const float *) ((const char *) src2 + (bidx * src2_nb2) + bidy * splitD * sizeof(float));
    const float * A_block  = (const float *) ((const char *) src3 + bidy * splitD * src3_nb1);
    const float * B_block  = (const float *) ((const char *) src4 + (bidx * src4_nb3));
    const float * C_block  = (const float *) ((const char *) src5 + (bidx * src5_nb3));
    float *       y_block  = (float *) ((char *) dst + (bidx * d_inner * L * sizeof(float)) + bidy * splitD * sizeof(float));
    float *       s_block  = (float *) ((char *) dst + s_off + bidx * src0_nb3 + bidy * splitD * src0_nb2);

    const int stride_s0 = src0_nb2 / sizeof(float);
    const int stride_x  = src1_nb2 / sizeof(float);
    const int stride_dt = src2_nb1 / sizeof(float);
    const int stride_A  = src3_nb1 / sizeof(float);
    const int stride_B  = src4_nb2 / sizeof(float);
    const int stride_C  = src5_nb2 / sizeof(float);
    const int stride_s  = stride_s0;
    const int stride_y  = d_inner;

    // can N not be 16? for example 32?
    if (N == 16) {
#pragma unroll
        for (size_t i = 0; i < splitD / 4; i += 2) {
            float value = A_block[(wid * warp_size + i) * stride_A + wtid];
            // todo: bank conflict
            // I am always confused with how to use the swizzling method to solve
            // bank conflit. Hoping somebody can tell me.
            smem_A[(wid * warp_size + i) * stride_sA + wtid + ((wtid / 16) > 0 ? 1 : 0)] = value;
        }
#pragma unroll
        for (size_t i = 0; i < splitD / 4; i += 2) {
            float value = s0_block[(wid * warp_size + i) * stride_s0 + wtid];
            smem_s0[(wid * warp_size + i) * stride_ss0 + wtid + ((wtid / 16) > 0 ? 1 : 0)] = value;
        }
    }

    __syncthreads();

    for (int64_t i = 0; i < L; i++) {
        float dt_soft_plus = dt_block[i * stride_dt + tid];
        if (dt_soft_plus <= 20.0f) {
            dt_soft_plus = log1pf(exp(dt_soft_plus));
        }
        float x_dt = x_block[i * stride_x + tid] * dt_soft_plus;
        float sumf = 0.0f;
#pragma unroll
        for (size_t j = 0; j < N; j++) {
            float state = (smem_s0[tid * stride_ss0 + j] * expf(dt_soft_plus * smem_A[tid * stride_sA + j])) +
                          (B_block[i * stride_B + j] * x_dt);
            sumf += state * C_block[i * stride_C + j];
            if (i == L - 1) {
                s_block[tid * stride_s + j] = state;
            } else {
                smem_s0[tid * stride_ss0 + j] = state;
            }
        }
        __syncthreads();
        y_block[i * stride_y + tid] = sumf;
    }
}

// assumes as many threads as d_state
template <int splitH, int d_state>
__global__ void __launch_bounds__(d_state, 1)
    ssm_scan_f32_group(
        const float * __restrict__ src0, const float * __restrict__ src1, const float * __restrict__ src2,
        const float * __restrict__ src3, const float * __restrict__ src4, const float * __restrict__ src5,
        const int32_t * __restrict__ src6, float * __restrict__ dst,
        const int src0_nb2, const int src0_nb3, const int src1_nb2, const int src1_nb3,
        const int src2_nb1, const int src2_nb2, const int src3_nb1,
        const int src4_nb2, const int src4_nb3, const int src5_nb2, const int src5_nb3,
        const int64_t s_off, const int64_t n_head, const int64_t d_head, const int64_t n_group, const int64_t n_tok) {

    const int head_idx = (blockIdx.x * splitH) / d_head;
    const int head_off = ((blockIdx.x * splitH) % d_head) * sizeof(float);
    const int seq_idx = blockIdx.y;

    const int group_off = (head_idx & (n_group - 1)) * d_state * sizeof(float);

    const float * s0_block = (const float *) ((const char *) src0 + src6[seq_idx] * src0_nb3 + head_idx * src0_nb2 + head_off * d_state);
    const float * x_block  = (const float *) ((const char *) src1 + (seq_idx * src1_nb3) + blockIdx.x * splitH * sizeof(float));
    const float * dt_block = (const float *) ((const char *) src2 + (seq_idx * src2_nb2) + head_idx * sizeof(float));
    const float * A_block  = (const float *) ((const char *) src3 + head_idx * src3_nb1);
    const float * B_block  = (const float *) ((const char *) src4 + (seq_idx * src4_nb3) + (group_off));
    const float * C_block  = (const float *) ((const char *) src5 + (seq_idx * src5_nb3) + (group_off));
    float *       y_block  = dst + (seq_idx * n_tok * n_head * d_head) + blockIdx.x * splitH;
    float *       s_block  = (float *) ((char *) dst + s_off + seq_idx * src0_nb3 + head_idx * src0_nb2 + head_off * d_state);

    // strides across n_seq_tokens
    const int stride_x  = src1_nb2 / sizeof(float);
    const int stride_dt = src2_nb1 / sizeof(float);
    const int stride_B  = src4_nb2 / sizeof(float);
    const int stride_C  = src5_nb2 / sizeof(float);
    const int stride_y  = n_head * d_head;

    float state[splitH];
    // for the parallel accumulation
    __shared__ float stateC[splitH * d_state];

#pragma unroll
    for (int j = 0; j < splitH; j++) {
        state[j] = s0_block[j * d_state + threadIdx.x];
    }

    for (int64_t i = 0; i < n_tok; i++) {
        // TODO: only calculate dA and dt_soft_plus once per head instead of every splitH head elements
        // TODO: only calculate B and C once per head group
        // NOTE: dt_soft_plus, dA and x_dt have the same value across threads here.
        float dt_soft_plus = dt_block[i * stride_dt];
        if (dt_soft_plus <= 20.0f) {
            dt_soft_plus = log1pf(expf(dt_soft_plus));
        }
        const float dA = expf(dt_soft_plus * A_block[0]);
        const float B = B_block[i * stride_B + threadIdx.x];
        const float C = C_block[i * stride_C + threadIdx.x];

        // across d_head
#pragma unroll
        for (int j = 0; j < splitH; j++) {
            const float x_dt = x_block[i * stride_x + j] * dt_soft_plus;

            state[j] = (state[j] * dA) + (B * x_dt);

            stateC[j * d_state + threadIdx.x] = state[j] * C;
        }

        __syncthreads();

        // parallel accumulation for stateC
        // TODO: simplify
        {
            static_assert((d_state & -d_state) == d_state, "the state size has to be a power of 2");
            static_assert((splitH & -splitH) == splitH, "splitH has to be a power of 2");

            // reduce until w matches the warp size
            // TODO: does this work even when the physical warp size is 64?
#pragma unroll
            for (int w = d_state; w > WARP_SIZE; w >>= 1) {
                // (assuming there are d_state threads)
#pragma unroll
                for (int j = 0; j < ((w >> 1) * splitH + d_state - 1) / d_state; j++) {
                    // TODO: check for bank conflicts
                    const int k = (threadIdx.x % (w >> 1)) + (d_state * (threadIdx.x / (w >> 1))) + j * d_state * (d_state / (w >> 1));
                    stateC[k] += stateC[k + (w >> 1)];

                }
                __syncthreads();
            }

            static_assert(splitH >= d_state / WARP_SIZE);

#pragma unroll
            for (int j = 0; j < splitH / (d_state / WARP_SIZE); j++) {
                float y = stateC[(threadIdx.x % WARP_SIZE) + d_state * (threadIdx.x / WARP_SIZE) + j * d_state * (d_state / WARP_SIZE)];
                y = warp_reduce_sum(y);

                // store the above accumulations
                if (threadIdx.x % WARP_SIZE == 0) {
                    const int k = threadIdx.x / WARP_SIZE + j * (d_state / WARP_SIZE);
                    y_block[i * stride_y + k] = y;
                }
            }
        }
    }

    // write back the state
#pragma unroll
    for (int j = 0; j < splitH; j++) {
        s_block[j * d_state + threadIdx.x] = state[j];
    }
}

static void ssm_scan_f32_cuda(const float * src0, const float * src1, const float * src2, const float * src3,
                              const float * src4, const float * src5, const int32_t * src6, float * dst,
                              const int src0_nb2, const int src0_nb3, const int src1_nb2, const int src1_nb3, const int src2_nb1,
                              const int src2_nb2, const int src3_nb1, const int src4_nb2, const int src4_nb3, const int src5_nb2,
                              const int src5_nb3, const int64_t s_off, const int64_t d_state, const int64_t head_dim,
                              const int64_t n_head, const int64_t n_group, const int64_t n_tok, const int64_t n_seq,
                              hipStream_t stream) {
    // NOTE: if you change conditions here, be sure to update the corresponding supports_op condition!
    if (src3_nb1 == sizeof(float)) {
        // Mamba-2
        if (d_state == 128) {
            const int threads = 128;
            GGML_ASSERT(d_state % threads == 0);
            // NOTE: can be any power of two between 4 and 64
            const int splitH = 16;
            GGML_ASSERT(head_dim % splitH == 0);
            const dim3 blocks((n_head * head_dim + (splitH - 1)) / splitH, n_seq, 1);
            ssm_scan_f32_group<16, 128><<<blocks, threads, 0, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                    src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2, src3_nb1,
                    src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, head_dim, n_group, n_tok);
        } else if (d_state == 256) { // Falcon-H1
            const int threads = 256;
            // NOTE: can be any power of two between 8 and 64
            const int splitH = 16;
            GGML_ASSERT(head_dim % splitH == 0);
            const dim3 blocks((n_head * head_dim + (splitH - 1)) / splitH, n_seq, 1);
            ssm_scan_f32_group<16, 256><<<blocks, threads, 0, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                    src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2, src3_nb1,
                    src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, head_dim, n_group, n_tok);
        } else {
            GGML_ABORT("doesn't support d_state!=(128 or 256).");
        }
    } else {
        const int threads = 128;
        // Mamba-1
        GGML_ASSERT(n_head % threads == 0);
        GGML_ASSERT(head_dim == 1);
        GGML_ASSERT(n_group == 1);
        const dim3 blocks(n_seq, (n_head + threads - 1) / threads, 1);
        const int  smem_size = (threads * (d_state + 1) * 2) * sizeof(float);
        if (d_state == 16) {
            ssm_scan_f32<128, 16><<<blocks, threads, smem_size, stream>>>(
                src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
        } else {
            GGML_ABORT("doesn't support d_state!=16.");
        }
    }
}

void ggml_cuda_op_ssm_scan(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0];  // s
    const struct ggml_tensor * src1 = dst->src[1];  // x
    const struct ggml_tensor * src2 = dst->src[2];  // dt
    const struct ggml_tensor * src3 = dst->src[3];  // A
    const struct ggml_tensor * src4 = dst->src[4];  // B
    const struct ggml_tensor * src5 = dst->src[5];  // C
    const struct ggml_tensor * src6 = dst->src[6];  // ids

    const int64_t nc  = src0->ne[0];  // d_state
    const int64_t nr  = src0->ne[1];  // head_dim or 1
    const int64_t nh  = src1->ne[1];  // n_head
    const int64_t ng  = src4->ne[1];  // n_group
    const int64_t n_t = src1->ne[2];  // number of tokens per sequence
    const int64_t n_s = src1->ne[3];  // number of sequences in the batch

    const int64_t s_off = ggml_nelements(src1) * sizeof(float);

    GGML_ASSERT(ggml_nelements(src1) + nc*nr*nh*n_s == ggml_nelements(dst));
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src2->nb[0] == sizeof(float));
    GGML_ASSERT(src3->nb[0] == sizeof(float));
    GGML_ASSERT(src4->nb[0] == sizeof(float));
    GGML_ASSERT(src5->nb[0] == sizeof(float));
    GGML_ASSERT(src6->nb[0] == sizeof(int32_t));

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    const float * src2_d = (const float *) src2->data;
    const float * src3_d = (const float *) src3->data;
    const float * src4_d = (const float *) src4->data;
    const float * src5_d = (const float *) src5->data;
    const int32_t * src6_d = (const int32_t *) src6->data;
    float *       dst_d  = (float *) dst->data;
    hipStream_t  stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src6->type == GGML_TYPE_I32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    ssm_scan_f32_cuda(src0_d, src1_d, src2_d, src3_d, src4_d, src5_d, src6_d, dst_d,
                      src0->nb[2], src0->nb[3], src1->nb[2], src1->nb[3], src2->nb[1], src2->nb[2],
                      src3->nb[1], src4->nb[2], src4->nb[3], src5->nb[2], src5->nb[3],
                      s_off, nc, nr, nh, ng, n_t, n_s, stream);
}

#include "hip/hip_runtime.h"
#if !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11070
#define USE_CUB
#endif // !defined(GGML_USE_HIP) && !defined(GGML_USE_MUSA) && CUDART_VERSION >= 11070

#ifdef USE_CUB
#include <hipcub/hipcub.hpp>
using namespace hipcub;
#endif // USE_CUB

#include "ssm-scan.cuh"

// We would like to keep pragma unroll for cases where L_template is not 0,
// so we suppress the clang transformation warning.
#ifdef __clang__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wpass-failed"
#endif // __clang__
template <size_t splitD, size_t N, size_t L_template>
__global__ void __launch_bounds__(splitD, 1)
    ssm_scan_f32(const float *__restrict__ src0, const float *__restrict__ src1, const float *__restrict__ src2,
                 const float *__restrict__ src3, const float *__restrict__ src4, const float *__restrict__ src5,
                 const int32_t * __restrict__ src6, float * __restrict__ dst,
                 const int src0_nb2, const int src0_nb3, const int src1_nb2, const int src1_nb3,
                 const int src2_nb1, const int src2_nb2, const int src3_nb1,
                 const int src4_nb2, const int src4_nb3, const int src5_nb2, const int src5_nb3,
                 const int64_t s_off, const int64_t d_inner, const int64_t L_param)
{
    const size_t L = L_template == 0 ? L_param : L_template;
    const float *s0_block = (const float *)((const char *)src0 + src6[blockIdx.x] * src0_nb3 + blockIdx.y * splitD * src0_nb2);
    const float *x_block = (const float *)((const char *)src1 + (blockIdx.x * src1_nb3) + blockIdx.y * splitD * sizeof(float));
    const float *dt_block = (const float *)((const char *)src2 + (blockIdx.x * src2_nb2) + blockIdx.y * splitD * sizeof(float));
    const float *A_block = (const float *)((const char *)src3 + blockIdx.y * splitD * src3_nb1);
    const float *B_block = (const float *)((const char *)src4 + (blockIdx.x * src4_nb3));
    const float *C_block = (const float *)((const char *)src5 + (blockIdx.x * src5_nb3));
    float *y_block = (float *)((char *)dst + (blockIdx.x * d_inner * L * sizeof(float)) + blockIdx.y * splitD * sizeof(float));
    float *s_block = (float *)((char *)dst + s_off + blockIdx.x * src0_nb3 + blockIdx.y * splitD * src0_nb2);

    const int stride_x = src1_nb2 / sizeof(float);
    const int stride_dt = src2_nb1 / sizeof(float);
    const int stride_B = src4_nb2 / sizeof(float);
    const int stride_C = src5_nb2 / sizeof(float);
    const int stride_y = d_inner;

    float regA[N];
    float regs0[N];

    __shared__ float smemB[N];
    __shared__ float smemC[N];

#ifdef USE_CUB
    using BlockLoad = hipcub::BlockLoad<float, splitD, N, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockStore = hipcub::BlockStore<float, splitD, N, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;

    union CubTempStorage {
        typename BlockLoad::TempStorage load_temp;
        typename BlockStore::TempStorage store_temp;
    };
    __shared__ CubTempStorage cub_temp_storage;

    BlockLoad(cub_temp_storage.load_temp).Load(A_block, regA);
    BlockLoad(cub_temp_storage.load_temp).Load(s0_block, regs0);
#else
    const int stride_s0 = src0_nb2 / sizeof(float);
    const int stride_A = src3_nb1 / sizeof(float);
#pragma unroll
    for (size_t n = 0; n < N; ++n)
    {
        regA[n] = A_block[threadIdx.x * stride_A + n];
        regs0[n] = s0_block[threadIdx.x * stride_s0 + n];
    }
#endif

#pragma unroll
    for (size_t i = 0; i < L; i++)
    {
        if (threadIdx.x < N)
        {
            smemB[threadIdx.x] = B_block[i * stride_B + threadIdx.x];
            smemC[threadIdx.x] = C_block[i * stride_C + threadIdx.x];
        }
        __syncthreads();

        float dt_soft_plus = dt_block[i * stride_dt + threadIdx.x];
        if (dt_soft_plus <= 20.0f)
        {
            dt_soft_plus = log1pf(expf(dt_soft_plus));
        }
        float x_dt = x_block[i * stride_x + threadIdx.x] * dt_soft_plus;

        float sumf = 0.0f;
#pragma unroll
        for (size_t n = 0; n < N; n++)
        {
            float state = regs0[n] * expf(dt_soft_plus * regA[n]) + smemB[n] * x_dt;
            sumf += state * smemC[n];
            regs0[n] = state;
        }
        y_block[i * stride_y + threadIdx.x] = sumf;
    }

#ifdef USE_CUB
    BlockStore(cub_temp_storage.store_temp).Store(s_block, regs0);
#else
    const int stride_s = stride_s0;
#pragma unroll
    for (size_t n = 0; n < N; ++n)
    {
        s_block[threadIdx.x * stride_s + n] = regs0[n];
    }
#endif
}
#ifdef __clang__
#pragma clang diagnostic pop
#endif // __clang__

// assumes as many threads as d_state
template <int splitH, int d_state>
__global__ void __launch_bounds__(d_state, 1)
    ssm_scan_f32_group(
        const float * __restrict__ src0, const float * __restrict__ src1, const float * __restrict__ src2,
        const float * __restrict__ src3, const float * __restrict__ src4, const float * __restrict__ src5,
        const int32_t * __restrict__ src6, float * __restrict__ dst,
        const int src0_nb2, const int src0_nb3, const int src1_nb2, const int src1_nb3,
        const int src2_nb1, const int src2_nb2, const int src3_nb1,
        const int src4_nb2, const int src4_nb3, const int src5_nb2, const int src5_nb3,
        const int64_t s_off, const int64_t n_head, const int64_t d_head, const int64_t n_group, const int64_t n_tok) {

    const int head_idx = (blockIdx.x * splitH) / d_head;
    const int head_off = ((blockIdx.x * splitH) % d_head) * sizeof(float);
    const int seq_idx = blockIdx.y;

    const int group_off = (head_idx / (n_head / n_group)) * d_state * sizeof(float);

    const float * s0_block = (const float *) ((const char *) src0 + src6[seq_idx] * src0_nb3 + head_idx * src0_nb2 + head_off * d_state);
    const float * x_block  = (const float *) ((const char *) src1 + (seq_idx * src1_nb3) + blockIdx.x * splitH * sizeof(float));
    const float * dt_block = (const float *) ((const char *) src2 + (seq_idx * src2_nb2) + head_idx * sizeof(float));
    const float * A_block  = (const float *) ((const char *) src3 + head_idx * src3_nb1);
    const float * B_block  = (const float *) ((const char *) src4 + (seq_idx * src4_nb3) + (group_off));
    const float * C_block  = (const float *) ((const char *) src5 + (seq_idx * src5_nb3) + (group_off));
    float *       y_block  = dst + (seq_idx * n_tok * n_head * d_head) + blockIdx.x * splitH;
    float *       s_block  = (float *) ((char *) dst + s_off + seq_idx * src0_nb3 + head_idx * src0_nb2 + head_off * d_state);

    // strides across n_seq_tokens
    const int stride_x  = src1_nb2 / sizeof(float);
    const int stride_dt = src2_nb1 / sizeof(float);
    const int stride_B  = src4_nb2 / sizeof(float);
    const int stride_C  = src5_nb2 / sizeof(float);
    const int stride_y  = n_head * d_head;

    float state[splitH];
    // for the parallel accumulation
    __shared__ float stateC[splitH * d_state];

#pragma unroll
    for (int j = 0; j < splitH; j++) {
        state[j] = s0_block[j * d_state + threadIdx.x];
    }

    for (int64_t i = 0; i < n_tok; i++) {
        // TODO: only calculate dA and dt_soft_plus once per head instead of every splitH head elements
        // TODO: only calculate B and C once per head group
        // NOTE: dt_soft_plus, dA and x_dt have the same value across threads here.
        float dt_soft_plus = dt_block[i * stride_dt];
        if (dt_soft_plus <= 20.0f) {
            dt_soft_plus = log1pf(expf(dt_soft_plus));
        }
        const float dA = expf(dt_soft_plus * A_block[0]);
        const float B = B_block[i * stride_B + threadIdx.x];
        const float C = C_block[i * stride_C + threadIdx.x];

        // across d_head
#pragma unroll
        for (int j = 0; j < splitH; j++) {
            const float x_dt = x_block[i * stride_x + j] * dt_soft_plus;

            state[j] = (state[j] * dA) + (B * x_dt);

            stateC[j * d_state + threadIdx.x] = state[j] * C;
        }

        __syncthreads();

        // parallel accumulation for stateC
        // TODO: simplify
        {
            static_assert((d_state & -d_state) == d_state, "the state size has to be a power of 2");
            static_assert((splitH & -splitH) == splitH, "splitH has to be a power of 2");

            // reduce until w matches the warp size
            // TODO: does this work even when the physical warp size is 64?
#pragma unroll
            for (int w = d_state; w > WARP_SIZE; w >>= 1) {
                // (assuming there are d_state threads)
#pragma unroll
                for (int j = 0; j < ((w >> 1) * splitH + d_state - 1) / d_state; j++) {
                    // TODO: check for bank conflicts
                    const int k = (threadIdx.x % (w >> 1)) + (d_state * (threadIdx.x / (w >> 1))) + j * d_state * (d_state / (w >> 1));
                    stateC[k] += stateC[k + (w >> 1)];

                }
                __syncthreads();
            }

            static_assert(splitH >= d_state / WARP_SIZE);

#pragma unroll
            for (int j = 0; j < splitH / (d_state / WARP_SIZE); j++) {
                float y = stateC[(threadIdx.x % WARP_SIZE) + d_state * (threadIdx.x / WARP_SIZE) + j * d_state * (d_state / WARP_SIZE)];
                y = warp_reduce_sum(y);

                // store the above accumulations
                if (threadIdx.x % WARP_SIZE == 0) {
                    const int k = threadIdx.x / WARP_SIZE + j * (d_state / WARP_SIZE);
                    y_block[i * stride_y + k] = y;
                }
            }
        }
    }

    // write back the state
#pragma unroll
    for (int j = 0; j < splitH; j++) {
        s_block[j * d_state + threadIdx.x] = state[j];
    }
}

static void ssm_scan_f32_cuda(const float * src0, const float * src1, const float * src2, const float * src3,
                              const float * src4, const float * src5, const int32_t * src6, float * dst,
                              const int src0_nb2, const int src0_nb3, const int src1_nb2, const int src1_nb3, const int src2_nb1,
                              const int src2_nb2, const int src3_nb1, const int src4_nb2, const int src4_nb3, const int src5_nb2,
                              const int src5_nb3, const int64_t s_off, const int64_t d_state, const int64_t head_dim,
                              const int64_t n_head, const int64_t n_group, const int64_t n_tok, const int64_t n_seq,
                              hipStream_t stream) {
    const int threads = 128;
    // NOTE: if you change conditions here, be sure to update the corresponding supports_op condition!
    if (src3_nb1 == sizeof(float)) {
        // Mamba-2
        if (d_state == 128) {
            GGML_ASSERT(d_state % threads == 0);
            // NOTE: can be any power of two between 4 and 64
            const int splitH = 16;
            GGML_ASSERT(head_dim % splitH == 0);
            const dim3 blocks((n_head * head_dim + (splitH - 1)) / splitH, n_seq, 1);
            ssm_scan_f32_group<16, 128><<<blocks, threads, 0, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                    src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2, src3_nb1,
                    src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, head_dim, n_group, n_tok);
        } else if (d_state == 256) { // Falcon-H1
            const int threads = 256;
            // NOTE: can be any power of two between 8 and 64
            const int splitH = 16;
            GGML_ASSERT(head_dim % splitH == 0);
            const dim3 blocks((n_head * head_dim + (splitH - 1)) / splitH, n_seq, 1);
            ssm_scan_f32_group<16, 256><<<blocks, threads, 0, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                    src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2, src3_nb1,
                    src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, head_dim, n_group, n_tok);
        } else {
            GGML_ABORT("doesn't support d_state!=(128 or 256).");
        }
    } else {
        // Mamba-1
        GGML_ASSERT(n_head % threads == 0);
        GGML_ASSERT(head_dim == 1);
        GGML_ASSERT(n_group == 1);
        const dim3 blocks(n_seq, (n_head + threads - 1) / threads, 1);
        const int  smem_size = (threads * (d_state + 1) * 2) * sizeof(float);
        if (d_state == 16) {
            switch (n_tok)
            {
            case 1:
                ssm_scan_f32<threads, 16, 1><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            case 2:
                ssm_scan_f32<threads, 16, 2><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            case 3:
                ssm_scan_f32<threads, 16, 3><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            case 4:
                ssm_scan_f32<threads, 16, 4><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            case 5:
                ssm_scan_f32<threads, 16, 5><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            case 6:
                ssm_scan_f32<threads, 16, 6><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            case 7:
                ssm_scan_f32<threads, 16, 7><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            case 8:
                ssm_scan_f32<threads, 16, 8><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            default:
                ssm_scan_f32<threads, 16, 0><<<blocks, threads, smem_size, stream>>>(
                    src0, src1, src2, src3, src4, src5, src6, dst,
                src0_nb2, src0_nb3, src1_nb2, src1_nb3, src2_nb1, src2_nb2,
                src3_nb1, src4_nb2, src4_nb3, src5_nb2, src5_nb3, s_off, n_head, n_tok);
                break;
            }
        } else {
            GGML_ABORT("doesn't support d_state!=16.");
        }
    }
}

void ggml_cuda_op_ssm_scan(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0];  // s
    const struct ggml_tensor * src1 = dst->src[1];  // x
    const struct ggml_tensor * src2 = dst->src[2];  // dt
    const struct ggml_tensor * src3 = dst->src[3];  // A
    const struct ggml_tensor * src4 = dst->src[4];  // B
    const struct ggml_tensor * src5 = dst->src[5];  // C
    const struct ggml_tensor * src6 = dst->src[6];  // ids

    const int64_t nc  = src0->ne[0];  // d_state
    const int64_t nr  = src0->ne[1];  // head_dim or 1
    const int64_t nh  = src1->ne[1];  // n_head
    const int64_t ng  = src4->ne[1];  // n_group
    const int64_t n_t = src1->ne[2];  // number of tokens per sequence
    const int64_t n_s = src1->ne[3];  // number of sequences in the batch

    const int64_t s_off = ggml_nelements(src1) * sizeof(float);

    GGML_ASSERT(ggml_nelements(src1) + nc*nr*nh*n_s == ggml_nelements(dst));
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src2->nb[0] == sizeof(float));
    GGML_ASSERT(src3->nb[0] == sizeof(float));
    GGML_ASSERT(src4->nb[0] == sizeof(float));
    GGML_ASSERT(src5->nb[0] == sizeof(float));
    GGML_ASSERT(src6->nb[0] == sizeof(int32_t));

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    const float * src2_d = (const float *) src2->data;
    const float * src3_d = (const float *) src3->data;
    const float * src4_d = (const float *) src4->data;
    const float * src5_d = (const float *) src5->data;
    const int32_t * src6_d = (const int32_t *) src6->data;
    float *       dst_d  = (float *) dst->data;
    hipStream_t  stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src6->type == GGML_TYPE_I32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    ssm_scan_f32_cuda(src0_d, src1_d, src2_d, src3_d, src4_d, src5_d, src6_d, dst_d,
                      src0->nb[2], src0->nb[3], src1->nb[2], src1->nb[3], src2->nb[1], src2->nb[2],
                      src3->nb[1], src4->nb[2], src4->nb[3], src5->nb[2], src5->nb[3],
                      s_off, nc, nr, nh, ng, n_t, n_s, stream);
}

#include "hip/hip_runtime.h"
#include "mmvq.cuh"
#include "vecdotq.cuh"

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs);

static constexpr __device__ vec_dot_q_cuda_t get_vec_dot_q_cuda(ggml_type type) {
    return type == GGML_TYPE_Q4_0 ? vec_dot_q4_0_q8_1 :
        type == GGML_TYPE_Q4_1 ? vec_dot_q4_1_q8_1 :
        type == GGML_TYPE_Q5_0 ? vec_dot_q5_0_q8_1 :
        type == GGML_TYPE_Q5_1 ? vec_dot_q5_1_q8_1 :
        type == GGML_TYPE_Q8_0 ? vec_dot_q8_0_q8_1 :
        type == GGML_TYPE_Q2_K ? vec_dot_q2_K_q8_1 :
        type == GGML_TYPE_Q3_K ? vec_dot_q3_K_q8_1 :
        type == GGML_TYPE_Q4_K ? vec_dot_q4_K_q8_1 :
        type == GGML_TYPE_Q5_K ? vec_dot_q5_K_q8_1 :
        type == GGML_TYPE_Q6_K ? vec_dot_q6_K_q8_1 :
        type == GGML_TYPE_IQ2_XXS ? vec_dot_iq2_xxs_q8_1 :
        type == GGML_TYPE_IQ2_XS ? vec_dot_iq2_xs_q8_1 :
        type == GGML_TYPE_IQ2_S ? vec_dot_iq2_s_q8_1 :
        type == GGML_TYPE_IQ3_XXS ? vec_dot_iq3_xxs_q8_1 :
        type == GGML_TYPE_IQ1_S ? vec_dot_iq1_s_q8_1 :
        type == GGML_TYPE_IQ1_M ? vec_dot_iq1_m_q8_1 :
        type == GGML_TYPE_IQ4_NL ? vec_dot_iq4_nl_q8_1 :
        type == GGML_TYPE_IQ4_XS ? vec_dot_iq4_xs_q8_1 :
        type == GGML_TYPE_IQ3_S ? vec_dot_iq3_s_q8_1 :
        nullptr;
}

static constexpr __device__ int get_vdr_mmvq(ggml_type type) {
    return type == GGML_TYPE_Q4_0 ? VDR_Q4_0_Q8_1_MMVQ :
        type == GGML_TYPE_Q4_1    ? VDR_Q4_1_Q8_1_MMVQ :
        type == GGML_TYPE_Q5_0    ? VDR_Q5_0_Q8_1_MMVQ :
        type == GGML_TYPE_Q5_1    ? VDR_Q5_1_Q8_1_MMVQ :
        type == GGML_TYPE_Q8_0    ? VDR_Q8_0_Q8_1_MMVQ :
        type == GGML_TYPE_Q2_K    ? VDR_Q2_K_Q8_1_MMVQ :
        type == GGML_TYPE_Q3_K    ? VDR_Q3_K_Q8_1_MMVQ :
        type == GGML_TYPE_Q4_K    ? VDR_Q4_K_Q8_1_MMVQ :
        type == GGML_TYPE_Q5_K    ? VDR_Q5_K_Q8_1_MMVQ :
        type == GGML_TYPE_Q6_K    ? VDR_Q6_K_Q8_1_MMVQ :
        type == GGML_TYPE_IQ2_XXS ? VDR_IQ2_XXS_Q8_1_MMVQ :
        type == GGML_TYPE_IQ2_XS  ? VDR_IQ2_XS_Q8_1_MMVQ :
        type == GGML_TYPE_IQ2_S   ? VDR_IQ2_S_Q8_1_MMVQ :
        type == GGML_TYPE_IQ3_XXS ? VDR_IQ3_XXS_Q8_1_MMVQ :
        type == GGML_TYPE_IQ3_S   ? VDR_IQ3_S_Q8_1_MMVQ :
        type == GGML_TYPE_IQ4_NL  ? VDR_IQ4_NL_Q8_1_MMVQ :
        type == GGML_TYPE_IQ4_XS  ? VDR_IQ4_XS_Q8_1_MMVQ :
        1;
}

template <ggml_type type, int ncols_y>
#if !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__((ncols_y <= 4 ? 4 : 2)*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))
static __global__ void mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;
    constexpr int vdr = get_vdr_mmvq(type);

    constexpr vec_dot_q_cuda_t vec_dot_q_cuda = get_vec_dot_q_cuda(type);

#if defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int nwarps              = 1;
    constexpr int rows_per_cuda_block = 1;
#else
    constexpr int nwarps              = ncols_y <= 4 ? 4 : 2;
    constexpr int rows_per_cuda_block = ncols_y == 1 ? 1 : 2;
#endif // defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp[j][i] += vec_dot_q_cuda(vx, &y[j*blocks_per_col_y + kby], (row0 + i)*blocks_per_row_x + kbx, kqs);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            dst[j*nrows_dst + row0 + threadIdx.x] = tmp[j][threadIdx.x];
        }
    }
}

template <ggml_type type>
static void mul_mat_vec_q_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    GGML_ASSERT(ncols_x % ggml_blck_size(type) == 0);
    GGML_ASSERT(ncols_y <= MMVQ_MAX_BATCH_SIZE);

    int id = ggml_cuda_get_device();

    int64_t nwarps = 1;
    int64_t rows_per_cuda_block = 1;

    if (ggml_cuda_info().devices[id].cc < GGML_CUDA_CC_CDNA || ggml_cuda_info().devices[id].cc == GGML_CUDA_CC_RDNA1) { // NVIDIA and AMD older than RDNA2 but not CDNA
        switch(ncols_y) {
            case 1:
                nwarps = 4;
                rows_per_cuda_block = 1;
                break;
            case 2:
            case 3:
            case 4:
                nwarps = 4;
                rows_per_cuda_block = 2;
                break;
            case 5:
            case 6:
            case 7:
            case 8:
                nwarps = 2;
                rows_per_cuda_block = 2;
                break;
            default:
                GGML_ABORT("fatal error");
                break;
        }
    }
    const int64_t nblocks = (nrows_x + rows_per_cuda_block - 1) / rows_per_cuda_block;
    const dim3 block_nums(nblocks, 1, 1);
    const dim3 block_dims(WARP_SIZE, nwarps, 1);

    switch (ncols_y) {
        case 1:
            mul_mat_vec_q<type, 1><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 2:
            mul_mat_vec_q<type, 2><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 3:
            mul_mat_vec_q<type, 3><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 4:
            mul_mat_vec_q<type, 4><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 5:
            mul_mat_vec_q<type, 5><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 6:
            mul_mat_vec_q<type, 6><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 7:
            mul_mat_vec_q<type, 7><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 8:
            mul_mat_vec_q<type, 8><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

static void mul_mat_vec_q4_0_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q4_0>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q4_1_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q4_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q5_0_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q5_0>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q5_1_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q5_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q8_0_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q8_0>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q2_K_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q2_K>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q3_K_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q3_K>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q4_K_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q4_K>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q5_K_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q5_K>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_q6_K_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_Q6_K>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq2_xxs_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ2_XXS>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq2_xs_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ2_XS>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq2_s_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ2_S>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq3_xxs_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ3_XXS>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq1_s_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ1_S>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq1_m_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ1_M>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq4_nl_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ4_NL>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq4_xs_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ4_XS>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

static void mul_mat_vec_iq3_s_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    mul_mat_vec_q_cuda<GGML_TYPE_IQ3_S>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void ggml_cuda_op_mul_mat_vec_q(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t row_diff = row_high - row_low;

    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);

    const int64_t ne0 = dst->ne[0];

    int id = ggml_cuda_get_device();

    // the main device has a larger memory buffer to hold the results from all GPUs
    // nrows_dst == nrows of the matrix that the kernel writes into
    const int64_t nrows_dst = id == ctx.device ? ne0 : row_diff;

    switch (src0->type) {
        case GGML_TYPE_Q4_0:
            mul_mat_vec_q4_0_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q4_1:
            mul_mat_vec_q4_1_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q5_0:
            mul_mat_vec_q5_0_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q5_1:
            mul_mat_vec_q5_1_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q8_0:
            mul_mat_vec_q8_0_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q2_K:
            mul_mat_vec_q2_K_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q3_K:
            mul_mat_vec_q3_K_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q4_K:
            mul_mat_vec_q4_K_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q5_K:
            mul_mat_vec_q5_K_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_Q6_K:
            mul_mat_vec_q6_K_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ2_XXS:
            mul_mat_vec_iq2_xxs_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ2_XS:
            mul_mat_vec_iq2_xs_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ2_S:
            mul_mat_vec_iq2_s_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ3_XXS:
            mul_mat_vec_iq3_xxs_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ1_S:
            mul_mat_vec_iq1_s_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ1_M:
            mul_mat_vec_iq1_m_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ4_NL:
            mul_mat_vec_iq4_nl_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ4_XS:
            mul_mat_vec_iq4_xs_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        case GGML_TYPE_IQ3_S:
            mul_mat_vec_iq3_s_q8_1_cuda(src0_dd_i, src1_ddq_i, dst_dd_i, ne00, row_diff, src1_padded_row_size, src1_ncols, nrows_dst, stream);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }

    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddf_i);
    GGML_UNUSED(src1_ncols);
    GGML_UNUSED(src1_padded_row_size);
}

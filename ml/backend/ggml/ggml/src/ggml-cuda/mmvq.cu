#include "hip/hip_runtime.h"
#include "mmvq.cuh"
#include "quantize.cuh"
#include "vecdotq.cuh"

#include <cstdint>

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs);

static constexpr __device__ vec_dot_q_cuda_t get_vec_dot_q_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:    return vec_dot_q4_0_q8_1;
        case GGML_TYPE_Q4_1:    return vec_dot_q4_1_q8_1;
        case GGML_TYPE_Q5_0:    return vec_dot_q5_0_q8_1;
        case GGML_TYPE_Q5_1:    return vec_dot_q5_1_q8_1;
        case GGML_TYPE_Q8_0:    return vec_dot_q8_0_q8_1;
        case GGML_TYPE_Q2_K:    return vec_dot_q2_K_q8_1;
        case GGML_TYPE_Q3_K:    return vec_dot_q3_K_q8_1;
        case GGML_TYPE_Q4_K:    return vec_dot_q4_K_q8_1;
        case GGML_TYPE_Q5_K:    return vec_dot_q5_K_q8_1;
        case GGML_TYPE_Q6_K:    return vec_dot_q6_K_q8_1;
        case GGML_TYPE_IQ2_XXS: return vec_dot_iq2_xxs_q8_1;
        case GGML_TYPE_IQ2_XS:  return vec_dot_iq2_xs_q8_1;
        case GGML_TYPE_IQ2_S:   return vec_dot_iq2_s_q8_1;
        case GGML_TYPE_IQ3_XXS: return vec_dot_iq3_xxs_q8_1;
        case GGML_TYPE_IQ1_S:   return vec_dot_iq1_s_q8_1;
        case GGML_TYPE_IQ1_M:   return vec_dot_iq1_m_q8_1;
        case GGML_TYPE_IQ4_NL:  return vec_dot_iq4_nl_q8_1;
        case GGML_TYPE_IQ4_XS:  return vec_dot_iq4_xs_q8_1;
        case GGML_TYPE_IQ3_S:   return vec_dot_iq3_s_q8_1;
        default:                return nullptr;
    }
}

static constexpr __device__ int get_vdr_mmvq(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:    return VDR_Q4_0_Q8_1_MMVQ;
        case GGML_TYPE_Q4_1:    return VDR_Q4_1_Q8_1_MMVQ;
        case GGML_TYPE_Q5_0:    return VDR_Q5_0_Q8_1_MMVQ;
        case GGML_TYPE_Q5_1:    return VDR_Q5_1_Q8_1_MMVQ;
        case GGML_TYPE_Q8_0:    return VDR_Q8_0_Q8_1_MMVQ;
        case GGML_TYPE_Q2_K:    return VDR_Q2_K_Q8_1_MMVQ;
        case GGML_TYPE_Q3_K:    return VDR_Q3_K_Q8_1_MMVQ;
        case GGML_TYPE_Q4_K:    return VDR_Q4_K_Q8_1_MMVQ;
        case GGML_TYPE_Q5_K:    return VDR_Q5_K_Q8_1_MMVQ;
        case GGML_TYPE_Q6_K:    return VDR_Q6_K_Q8_1_MMVQ;
        case GGML_TYPE_IQ2_XXS: return VDR_IQ2_XXS_Q8_1_MMVQ;
        case GGML_TYPE_IQ2_XS:  return VDR_IQ2_XS_Q8_1_MMVQ;
        case GGML_TYPE_IQ2_S:   return VDR_IQ2_S_Q8_1_MMVQ;
        case GGML_TYPE_IQ3_XXS: return VDR_IQ3_XXS_Q8_1_MMVQ;
        case GGML_TYPE_IQ3_S:   return VDR_IQ3_S_Q8_1_MMVQ;
        case GGML_TYPE_IQ4_NL:  return VDR_IQ4_NL_Q8_1_MMVQ;
        case GGML_TYPE_IQ4_XS:  return VDR_IQ4_XS_Q8_1_MMVQ;
        default:                return 1;
    }
}

enum mmvq_parameter_table_id {
    MMVQ_PARAMETERS_GENERIC = 0,
    MMVQ_PARAMETERS_GCN,
    MMVQ_PARAMETERS_RDNA2
};

static constexpr __device__ mmvq_parameter_table_id get_device_table_id() {
#if defined(RDNA2) || defined(RDNA3) || defined(RDNA4)
    return MMVQ_PARAMETERS_RDNA2;
#elif defined(GCN) || defined(CDNA)
    return MMVQ_PARAMETERS_GCN;
#else
    return MMVQ_PARAMETERS_GENERIC;
#endif
}

static __host__ mmvq_parameter_table_id get_device_table_id(int cc) {
    if (GGML_CUDA_CC_IS_RDNA2(cc) || GGML_CUDA_CC_IS_RDNA3(cc) || GGML_CUDA_CC_IS_RDNA4(cc)) {
        return MMVQ_PARAMETERS_RDNA2;
    }
    if (GGML_CUDA_CC_IS_GCN(cc) || GGML_CUDA_CC_IS_CDNA(cc)) {
        return MMVQ_PARAMETERS_GCN;
    }
    return MMVQ_PARAMETERS_GENERIC;
}

static constexpr __host__ __device__ int calc_nwarps(int ncols_dst,  mmvq_parameter_table_id table_id) {
    if (table_id == MMVQ_PARAMETERS_GENERIC) {
        switch (ncols_dst) {
            case 1:
            case 2:
            case 3:
            case 4:
                return 4;
            case 5:
            case 6:
            case 7:
            case 8:
                return 2;
            default:
                return 1;
        }
    } else if (table_id == MMVQ_PARAMETERS_GCN) {
        switch (ncols_dst) {
            case 1:
            case 2:
            case 3:
            case 4:
                return 2;
            case 5:
            case 6:
            case 7:
            case 8:
            default:
                return 1;
        }
    }
    return 1;
}

static constexpr __host__ __device__ int calc_rows_per_block(int ncols_dst, int table_id) {
    if (table_id == MMVQ_PARAMETERS_GENERIC || table_id == MMVQ_PARAMETERS_GCN) {
        switch (ncols_dst) {
            case 1:
                return 1;
            case 2:
            case 3:
            case 4:
            case 5:
            case 6:
            case 7:
            case 8:
                return 2;
            default:
                return 1;
        }
    }
    return 1;
}

template <ggml_type type, int ncols_dst>
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__(calc_nwarps(ncols_dst, get_device_table_id())*ggml_cuda_get_physical_warp_size(), 1)
static __global__ void mul_mat_vec_q(
        const void * __restrict__ vx, const void * __restrict__ vy, const int32_t * __restrict__ ids, float * __restrict__ dst,
        const int ncols_x, const int nchannels_y, const int stride_row_x, const int stride_col_y, const int stride_col_dst,
        const int channel_ratio, const int stride_channel_x, const int stride_channel_y, const int stride_channel_dst,
        const int sample_ratio, const int stride_sample_x, const int stride_sample_y, const int stride_sample_dst) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;
    constexpr int vdr = get_vdr_mmvq(type);
    constexpr mmvq_parameter_table_id table_id = get_device_table_id();
    constexpr int nwarps = calc_nwarps(ncols_dst, table_id);
    constexpr int rows_per_cuda_block = calc_rows_per_block(ncols_dst, table_id);
    constexpr int warp_size = ggml_cuda_get_physical_warp_size();

    constexpr vec_dot_q_cuda_t vec_dot_q_cuda = get_vec_dot_q_cuda(type);

    const     int tid = warp_size*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    constexpr int blocks_per_iter = vdr * nwarps*warp_size / qi;

    // The MUL_MAT_ID code path with ids != nullptr is only implemented for ncols_dst == 1.
    const int channel_dst = blockIdx.y;
    const int channel_x   = ncols_dst == 1 && ids ? ids[channel_dst]          : channel_dst / channel_ratio;
    const int channel_y   = ncols_dst == 1 && ids ? channel_dst % nchannels_y : channel_dst;
    const int sample_dst  = blockIdx.z;
    const int sample_x    = sample_dst / sample_ratio;
    const int sample_y    = sample_dst;

    // partial sum for each thread
    float tmp[ncols_dst][rows_per_cuda_block] = {{0.0f}};

    const block_q8_1 * y = ((const block_q8_1 *) vy) + sample_y*stride_sample_y + channel_y*stride_channel_y;
    const int kbx_offset = sample_x*stride_sample_x + channel_x*stride_channel_x + row0*stride_row_x;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_dst; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp[j][i] += vec_dot_q_cuda(
                    vx, &y[j*stride_col_y + kby], kbx_offset + i*stride_row_x + kbx, kqs);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_dst][rows_per_cuda_block][warp_size];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_dst; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    dst += sample_dst*stride_sample_dst + channel_dst*stride_channel_dst + row0;

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_dst; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum<warp_size>(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + int(threadIdx.x) < stride_col_dst)) {
            dst[j*stride_col_dst + threadIdx.x] = tmp[j][threadIdx.x];
        }
    }
}

static std::pair<dim3, dim3> calc_launch_params(
        const int ncols_dst, const int nrows_x, const int nchannels_y, const int nsamples_y,
        const int warp_size, const mmvq_parameter_table_id table_id) {
    const int64_t nblocks = (nrows_x + calc_rows_per_block(ncols_dst, table_id) - 1) / calc_rows_per_block(ncols_dst, table_id);
    const dim3 block_nums(nblocks, nchannels_y, nsamples_y);
    const dim3 block_dims(warp_size, calc_nwarps(ncols_dst, table_id), 1);
    return {block_nums, block_dims};
}

template <ggml_type type>
static void mul_mat_vec_q_switch_ncols_dst(
        const void * vx, const void * vy, const int32_t * ids, float * dst,
        const int ncols_x, const int nrows_x, const int ncols_dst,
        const int stride_row_x, const int stride_col_y, const int stride_col_dst,
        const int nchannels_x, const int nchannels_y, const int nchannels_dst,
        const int stride_channel_x, const int stride_channel_y, const int stride_channel_dst,
        const int nsamples_x, const int nsamples_dst, const int stride_sample_x, const int stride_sample_y, const int stride_sample_dst,
        hipStream_t stream) {

    GGML_ASSERT(ncols_x % ggml_blck_size(type) == 0);
    GGML_ASSERT(ncols_dst <= MMVQ_MAX_BATCH_SIZE);

    const int channel_ratio = nchannels_dst / nchannels_x;
    const int sample_ratio  = nsamples_dst  / nsamples_x;

    const int device = ggml_cuda_get_device();
    const int warp_size = ggml_cuda_info().devices[device].warp_size;
    const mmvq_parameter_table_id table_id = get_device_table_id(ggml_cuda_info().devices[device].cc);

    GGML_ASSERT(!ids || ncols_dst == 1);
    switch (ncols_dst) {
        case 1:
        {
            constexpr int c_ncols_dst = 1;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        case 2:
        {
            constexpr int c_ncols_dst = 2;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        case 3:
        {
            constexpr int c_ncols_dst = 3;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        case 4:
        {
            constexpr int c_ncols_dst = 4;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        case 5:
        {
            constexpr int c_ncols_dst = 5;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        case 6:
        {
            constexpr int c_ncols_dst = 6;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        case 7:
        {
            constexpr int c_ncols_dst = 7;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        case 8:
        {
            constexpr int c_ncols_dst = 8;
            std::pair<dim3, dim3> dims = calc_launch_params(c_ncols_dst, nrows_x, nchannels_dst, nsamples_dst, warp_size, table_id);
            mul_mat_vec_q<type, c_ncols_dst><<<dims.first, dims.second, 0, stream>>>
                (vx, vy, ids, dst, ncols_x, nchannels_y, stride_row_x, stride_col_y, stride_col_dst,
                 channel_ratio, stride_channel_x, stride_channel_y, stride_channel_dst,
                 sample_ratio, stride_sample_x, stride_sample_y, stride_sample_dst);
            break;
        }
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

static void mul_mat_vec_q_switch_type(
        const void * vx, const ggml_type type_x, const void * vy, const int32_t * ids, float * dst,
        const int ncols_x, const int nrows_x, const int ncols_dst,
        const int stride_row_x, const int stride_col_y, const int stride_col_dst,
        const int nchannels_x, const int nchannels_y, const int nchannels_dst,
        const int stride_channel_x, const int stride_channel_y, const int stride_channel_dst,
        const int nsamples_x, const int nsamples_dst, const int stride_sample_x, const int stride_sample_y, const int stride_sample_dst,
        hipStream_t stream) {
    switch (type_x) {
        case GGML_TYPE_Q4_0:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q4_0>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q4_1:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q4_1>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q5_0:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q5_0>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q5_1:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q5_1>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q8_0:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q8_0>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q2_K:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q2_K>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q3_K:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q3_K>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q4_K:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q4_K>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q5_K:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q5_K>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_Q6_K:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_Q6_K>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ2_XXS:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ2_XXS>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ2_XS:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ2_XS>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ2_S:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ2_S>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ3_XXS:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ3_XXS>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ1_S:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ1_S>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ1_M:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ1_M>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ4_NL:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ4_NL>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ4_XS:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ4_XS>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        case GGML_TYPE_IQ3_S:
            mul_mat_vec_q_switch_ncols_dst<GGML_TYPE_IQ3_S>
                (vx, vy, ids, dst, ncols_x, nrows_x, ncols_dst, stride_row_x, stride_col_y, stride_col_dst,
                 nchannels_x, nchannels_y, nchannels_dst, stride_channel_x, stride_channel_y, stride_channel_dst,
                 nsamples_x, nsamples_dst, stride_sample_x, stride_sample_y, stride_sample_dst,
                 stream);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}

void ggml_cuda_mul_mat_vec_q(
        ggml_backend_cuda_context & ctx, const ggml_tensor * src0, const ggml_tensor * src1, const ggml_tensor * ids, ggml_tensor * dst) {
    GGML_ASSERT(        src1->type == GGML_TYPE_F32);
    GGML_ASSERT(        dst->type  == GGML_TYPE_F32);
    GGML_ASSERT(!ids || ids->type  == GGML_TYPE_I32); // Optional, used for batched GGML_MUL_MAT_ID.

    GGML_TENSOR_BINARY_OP_LOCALS;

    hipStream_t stream = ctx.stream();

    const size_t ts_src0 = ggml_type_size(src0->type);
    const size_t ts_src1 = ggml_type_size(src1->type);
    const size_t ts_dst  = ggml_type_size(dst->type);

    GGML_ASSERT(        nb00       == ts_src0);
    GGML_ASSERT(        nb10       == ts_src1);
    GGML_ASSERT(        nb0        == ts_dst);
    GGML_ASSERT(!ids || ids->nb[0] == ggml_type_size(ids->type));

    GGML_ASSERT(!ids || ne12 == 1); // Implementation is only correct for batch size 1.

    const float   * src1_d =       (const float   *) src1->data;
    const int32_t *  ids_d = ids ? (const int32_t *)  ids->data : nullptr;
    float         *  dst_d =       (float         *)  dst->data;

    const int64_t ne10_padded = GGML_PAD(ne10, MATRIX_ROW_PADDING);
    ggml_cuda_pool_alloc<char> src1_q8_1(ctx.pool(), ne13*ne12 * ne11*ne10_padded * sizeof(block_q8_1)/QK8_1);
    {
        const int64_t s11 = src1->nb[1] / ts_src1;
        const int64_t s12 = src1->nb[2] / ts_src1;
        const int64_t s13 = src1->nb[3] / ts_src1;
        quantize_row_q8_1_cuda(src1_d, nullptr, src1_q8_1.get(), src0->type, ne10, s11, s12, s13, ne10_padded, ne11, ne12, ne13, stream);
    }

    const int64_t s01 = src0->nb[1] / ts_src0;
    const int64_t s11 = ne10_padded / QK8_1;
    const int64_t s1  =  dst->nb[1] / ts_dst;
    const int64_t s02 = src0->nb[2] / ts_src0;
    const int64_t s2  =  dst->nb[2] / ts_dst;
    const int64_t s03 = src0->nb[3] / ts_src0;
    const int64_t s3  =  dst->nb[3] / ts_dst;

    const int64_t s12 = ne11*s11;
    const int64_t s13 = ne12*s12;

    // For MUL_MAT_ID the memory layout is different than for MUL_MAT:
    const int64_t ncols_dst          = ids ? ne2  : ne1;
    const int64_t nchannels_y        = ids ? ne11 : ne12;
    const int64_t nchannels_dst      = ids ? ne1  : ne2;
    const int64_t stride_col_dst     = ids ? s2   : s1;
    const int64_t stride_col_y       = ids ? s12  : s11;
    const int64_t stride_channel_dst = ids ? s1   : s2;
    const int64_t stride_channel_y   = ids ? s11  : s12;

    mul_mat_vec_q_switch_type(
        src0->data, src0->type, src1_q8_1.get(), ids_d, dst_d, ne00,
        ne01,              ncols_dst,     s01, stride_col_y,     stride_col_dst,
        ne02, nchannels_y, nchannels_dst, s02, stride_channel_y, stride_channel_dst,
        ne03,              ne3,           s03, s13,              s3,                 stream);
}

void ggml_cuda_op_mul_mat_vec_q(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t row_diff = row_high - row_low;

    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);

    const int64_t ne0 = dst->ne[0];

    int id = ggml_cuda_get_device();

    // the main device has a larger memory buffer to hold the results from all GPUs
    // nrows_dst == nrows of the matrix that the kernel writes into
    const int64_t nrows_dst = id == ctx.device ? ne0 : row_diff;

    const int stride_row_x = ne00 / ggml_blck_size(src0->type);
    const int stride_col_y = src1_padded_row_size / QK8_1;

    mul_mat_vec_q_switch_type(
        src0_dd_i, src0->type, src1_ddq_i, nullptr, dst_dd_i, ne00, row_diff, src1_ncols, stride_row_x, stride_col_y, nrows_dst,
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, stream);

    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddf_i);
    GGML_UNUSED(src1_ncols);
    GGML_UNUSED(src1_padded_row_size);
}

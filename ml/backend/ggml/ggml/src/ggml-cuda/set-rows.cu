#include "hip/hip_runtime.h"
#include "set-rows.cuh"

typedef void (*set_rows_kernel_t)(const char * src, char * dst);

template<typename src_t, typename dst_t>
__device__ void set_rows_1(const src_t * src_f, dst_t * dst_f) {
    GGML_UNUSED(src_f);
    GGML_UNUSED(dst_f);
}

template<>
__device__ __forceinline__ void set_rows_1<float, half>(const float * src_f, half * dst_h) {
    *dst_h = __float2half(*src_f);
}

template<>
__device__ __forceinline__ void set_rows_1<float, hip_bfloat16>(const float * src_f, hip_bfloat16 * dst_b) {
    *dst_b = *src_f;
}

template<>
__device__ __forceinline__ void set_rows_1<float, float>(const float * src_f, float * dst_f) {
    *dst_f = *src_f;
}

template<typename src_t, typename dst_t>
static __global__ void k_set_rows(
        const src_t * __restrict__ src0, const int64_t * __restrict__ src1, dst_t * __restrict__ dst,
        const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const int64_t s01, const int64_t s02, const int64_t s03,
        const int64_t s10, const int64_t s11, const int64_t s12,
        const int64_t s1, const int64_t s2, const int64_t s3) {

    const int64_t i = int64_t(blockDim.x) * blockIdx.x + threadIdx.x;
    const int64_t ne_total = ne00 * ne01 * ne02 * ne03;

    if (i >= ne_total) {
        return;
    }

    const int64_t i03 = i / (ne00 * ne01 * ne02);
    const int64_t i02 = (i - i03 * ne00 * ne01 * ne02) / (ne00 * ne01);
    const int64_t i01 = (i - i03 * ne00 * ne01 * ne02 - i02 * ne00 * ne01) / ne00;
    const int64_t i00 = i - i03 * ne00 * ne01 * ne02 - i02 * ne00 * ne01 - i01 * ne00;

    const int64_t i12 = i03 % ne12;
    const int64_t i11 = i02 % ne11;
    const int64_t i10 = i01;

    const int64_t dst_row = *(src1 + i10*s10 + i11*s11 + i12*s12);

    const src_t * src0_row = src0 + i01*s01 + i02*s02 + i03*s03;
    dst_t * dst_row_ptr    = dst + dst_row*s1 + i02*s2 + i03*s3;

    const src_t* src_elem = src0_row + i00;
    dst_t* dst_elem = dst_row_ptr + i00;
    set_rows_1(src_elem, dst_elem);

    GGML_UNUSED(ne10);
    GGML_UNUSED(ne13);
}

template<typename src_t, typename dst_t>
static void set_rows_cuda(
        const src_t * src0_d, const int64_t * src1_d, dst_t * dst_d,
        const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const size_t nb01, const size_t nb02, const size_t nb03,
        const size_t nb10, const size_t nb11, const size_t nb12,
        const size_t nb1, const size_t nb2, const size_t nb3,
        hipStream_t stream) {

    const int64_t ne_total = ne00 * ne01 * ne02 * ne03;
    const int num_blocks = (ne_total + CUDA_SET_ROWS_BLOCK_SIZE - 1) / CUDA_SET_ROWS_BLOCK_SIZE;
    const dim3 block_size(CUDA_SET_ROWS_BLOCK_SIZE);
    const dim3 grid_size(num_blocks);


    const int64_t s01 = nb01/sizeof(src_t);
    const int64_t s02 = nb02/sizeof(src_t);
    const int64_t s03 = nb03/sizeof(src_t);
    const int64_t s10 = nb10/sizeof(int64_t);
    const int64_t s11 = nb11/sizeof(int64_t);
    const int64_t s12 = nb12/sizeof(int64_t);
    const int64_t s1  = nb1/sizeof(dst_t);
    const int64_t s2  = nb2/sizeof(dst_t);
    const int64_t s3  = nb3/sizeof(dst_t);

    if (ne_total > 0) {
        k_set_rows<<<grid_size, block_size, 0, stream>>>(
            src0_d, src1_d, dst_d,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            s01, s02, s03,
            s10, s11, s12,
            s1, s2, s3);
    }
}


void ggml_cuda_op_set_rows(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_I64);

    GGML_TENSOR_BINARY_OP_LOCALS

    const float * src0_d   = (const float *)src0->data;
    const int64_t * src1_d = (const int64_t *)src1->data;

    hipStream_t stream = ctx.stream();



    if (dst->type == GGML_TYPE_F32) {
        set_rows_cuda(
            src0_d, src1_d, (float*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_F16) {
        set_rows_cuda(
            src0_d, src1_d, (half*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_BF16) {
        set_rows_cuda(
            src0_d, src1_d, (hip_bfloat16*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else {
        GGML_ABORT("unsupported type");
    }
}

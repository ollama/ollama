#include "hip/hip_runtime.h"
#include "set-rows.cuh"
#include "cpy-utils.cuh"

typedef void (*set_rows_kernel_t)(const char * src, char * dst);

template<typename src_t, typename dst_t>
__device__ __forceinline__ void set_rows_1(const src_t * src_f, dst_t * dst_f) {
    convert_flt(src_f, dst_f);
}

// Generic quantized set_rows kernel template
template<typename block_type, int qk, void (*quantize_func)(const float*, block_type*)>
static __global__ void k_set_rows_quant(
        const float * __restrict__ src0, const int64_t * __restrict__ src1, block_type * __restrict__ dst,
        const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const int64_t s01, const int64_t s02, const int64_t s03,
        const int64_t s10, const int64_t s11, const int64_t s12,
        const int64_t s1, const int64_t s2, const int64_t s3) {

    const int64_t i = int64_t(blockDim.x) * blockIdx.x + threadIdx.x;
    const int64_t ne_total = (ne00 * ne01 * ne02 * ne03) / qk;

    if (i >= ne_total) {
        return;
    }

    const int64_t i_base = i * qk;
    const int64_t i03 = i_base / (ne00 * ne01 * ne02);
    const int64_t i02 = (i_base - i03 * ne00 * ne01 * ne02) / (ne00 * ne01);
    const int64_t i01 = (i_base - i03 * ne00 * ne01 * ne02 - i02 * ne00 * ne01) / ne00;
    const int64_t i00 = i_base - i03 * ne00 * ne01 * ne02 - i02 * ne00 * ne01 - i01 * ne00;

    const int64_t i12 = i03 % ne12;
    const int64_t i11 = i02 % ne11;
    const int64_t i10 = i01;

    const int64_t dst_row = *(src1 + i10*s10 + i11*s11 + i12*s12);

    const float * src0_row = src0 + i01*s01 + i02*s02 + i03*s03;
    block_type * dst_row_ptr = dst + (dst_row*s1 + i02*s2 + i03*s3) / sizeof(block_type);

    const float * src_block = src0_row + i00;
    block_type * dst_block = dst_row_ptr + i00 / qk;

    quantize_func(src_block, dst_block);

    GGML_UNUSED(ne10);
    GGML_UNUSED(ne13);
}

// Template dispatch function for quantized set_rows
template<typename block_type, int qk, void (*quantize_func)(const float*, block_type*)>
static void set_rows_cuda_quant(
        const float * src0_d, const int64_t * src1_d, block_type * dst_d,
        const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const size_t nb01, const size_t nb02, const size_t nb03,
        const size_t nb10, const size_t nb11, const size_t nb12,
        const size_t nb1, const size_t nb2, const size_t nb3,
        hipStream_t stream) {

    GGML_ASSERT(ne00 % qk == 0);
    const int64_t ne_total = (ne00 * ne01 * ne02 * ne03) / qk;
    const int num_blocks = (ne_total + CUDA_SET_ROWS_BLOCK_SIZE - 1) / CUDA_SET_ROWS_BLOCK_SIZE;
    const dim3 block_size(CUDA_SET_ROWS_BLOCK_SIZE);
    const dim3 grid_size(num_blocks);

    const int64_t s01 = nb01/sizeof(float);
    const int64_t s02 = nb02/sizeof(float);
    const int64_t s03 = nb03/sizeof(float);
    const int64_t s10 = nb10/sizeof(int64_t);
    const int64_t s11 = nb11/sizeof(int64_t);
    const int64_t s12 = nb12/sizeof(int64_t);
    const int64_t s1  = nb1;
    const int64_t s2  = nb2;
    const int64_t s3  = nb3;

    if (ne_total > 0) {
        k_set_rows_quant<block_type, qk, quantize_func><<<grid_size, block_size, 0, stream>>>(
            src0_d, src1_d, dst_d,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            s01, s02, s03,
            s10, s11, s12,
            s1, s2, s3);
    }
}

template<typename src_t, typename dst_t>
static __global__ void k_set_rows(
        const src_t * __restrict__ src0, const int64_t * __restrict__ src1, dst_t * __restrict__ dst,
        const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const int64_t s01, const int64_t s02, const int64_t s03,
        const int64_t s10, const int64_t s11, const int64_t s12,
        const int64_t s1, const int64_t s2, const int64_t s3) {

    const int64_t i = int64_t(blockDim.x) * blockIdx.x + threadIdx.x;
    const int64_t ne_total = ne00 * ne01 * ne02 * ne03;

    if (i >= ne_total) {
        return;
    }

    const int64_t i03 = i / (ne00 * ne01 * ne02);
    const int64_t i02 = (i - i03 * ne00 * ne01 * ne02) / (ne00 * ne01);
    const int64_t i01 = (i - i03 * ne00 * ne01 * ne02 - i02 * ne00 * ne01) / ne00;
    const int64_t i00 = i - i03 * ne00 * ne01 * ne02 - i02 * ne00 * ne01 - i01 * ne00;

    const int64_t i12 = i03 % ne12;
    const int64_t i11 = i02 % ne11;
    const int64_t i10 = i01;

    const int64_t dst_row = *(src1 + i10*s10 + i11*s11 + i12*s12);

    const src_t * src0_row = src0 + i01*s01 + i02*s02 + i03*s03;
    dst_t * dst_row_ptr    = dst + dst_row*s1 + i02*s2 + i03*s3;

    const src_t* src_elem = src0_row + i00;
    dst_t* dst_elem = dst_row_ptr + i00;
    set_rows_1(src_elem, dst_elem);

    GGML_UNUSED(ne10);
    GGML_UNUSED(ne13);
}

template<typename src_t, typename dst_t>
static void set_rows_cuda(
        const src_t * src0_d, const int64_t * src1_d, dst_t * dst_d,
        const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const size_t nb01, const size_t nb02, const size_t nb03,
        const size_t nb10, const size_t nb11, const size_t nb12,
        const size_t nb1, const size_t nb2, const size_t nb3,
        hipStream_t stream) {

    const int64_t ne_total = ne00 * ne01 * ne02 * ne03;
    const int num_blocks = (ne_total + CUDA_SET_ROWS_BLOCK_SIZE - 1) / CUDA_SET_ROWS_BLOCK_SIZE;
    const dim3 block_size(CUDA_SET_ROWS_BLOCK_SIZE);
    const dim3 grid_size(num_blocks);


    const int64_t s01 = nb01/sizeof(src_t);
    const int64_t s02 = nb02/sizeof(src_t);
    const int64_t s03 = nb03/sizeof(src_t);
    const int64_t s10 = nb10/sizeof(int64_t);
    const int64_t s11 = nb11/sizeof(int64_t);
    const int64_t s12 = nb12/sizeof(int64_t);
    const int64_t s1  = nb1/sizeof(dst_t);
    const int64_t s2  = nb2/sizeof(dst_t);
    const int64_t s3  = nb3/sizeof(dst_t);

    if (ne_total > 0) {
        k_set_rows<<<grid_size, block_size, 0, stream>>>(
            src0_d, src1_d, dst_d,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            s01, s02, s03,
            s10, s11, s12,
            s1, s2, s3);
    }
}


void ggml_cuda_op_set_rows(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_I64);

    GGML_TENSOR_BINARY_OP_LOCALS

    const float * src0_d   = (const float *)src0->data;
    const int64_t * src1_d = (const int64_t *)src1->data;

    hipStream_t stream = ctx.stream();



    if (dst->type == GGML_TYPE_F32) {
        set_rows_cuda(
            src0_d, src1_d, (float*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_F16) {
        set_rows_cuda(
            src0_d, src1_d, (half*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_BF16) {
        set_rows_cuda(
            src0_d, src1_d, (hip_bfloat16*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_Q4_0) {
        set_rows_cuda_quant<block_q4_0, QK4_0, quantize_f32_q4_0_block>(
            src0_d, src1_d, (block_q4_0*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_Q4_1) {
        set_rows_cuda_quant<block_q4_1, QK4_1, quantize_f32_q4_1_block>(
            src0_d, src1_d, (block_q4_1*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_Q5_0) {
        set_rows_cuda_quant<block_q5_0, QK5_0, quantize_f32_q5_0_block>(
            src0_d, src1_d, (block_q5_0*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_Q5_1) {
        set_rows_cuda_quant<block_q5_1, QK5_1, quantize_f32_q5_1_block>(
            src0_d, src1_d, (block_q5_1*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_Q8_0) {
        set_rows_cuda_quant<block_q8_0, QK8_0, quantize_f32_q8_0_block>(
            src0_d, src1_d, (block_q8_0*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else if (dst->type == GGML_TYPE_IQ4_NL) {
        set_rows_cuda_quant<block_iq4_nl, QK4_NL, quantize_f32_iq4_nl_block>(
            src0_d, src1_d, (block_iq4_nl*)dst->data,
            ne00, ne01, ne02, ne03,
            ne10, ne11, ne12, ne13,
            nb01, nb02, nb03,
            nb10, nb11, nb12,
            nb1, nb2, nb3,
            stream
        );
    } else {
        GGML_ABORT("unsupported type %s", ggml_type_name(dst->type));
    }
}

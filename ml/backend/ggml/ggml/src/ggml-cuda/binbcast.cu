#include "hip/hip_runtime.h"
#include "binbcast.cuh"
#include <cstdint>
#include <utility>

static __device__ __forceinline__ float op_repeat(const float a, const float b) {
    return b;
    GGML_UNUSED(a);
}

static __device__ __forceinline__ float op_add(const float a, const float b) {
    return a + b;
}

static __device__ __forceinline__ float op_sub(const float a, const float b) {
    return a - b;
}

static __device__ __forceinline__ float op_mul(const float a, const float b) {
    return a * b;
}

static __device__ __forceinline__ float op_div(const float a, const float b) {
    return a / b;
}

template <float (*bin_op)(const float, const float),
          typename src0_t,
          typename src1_t,
          typename dst_t,
          typename... src1_ptrs>
static __global__ void k_bin_bcast(const src0_t *         src0,
                                   const src1_t *         src1,
                                   dst_t *                dst,
                                   const int              ne0,
                                   const int              ne1,
                                   const int              ne2,
                                   const uint3            ne3,
                                   const uint3            ne10,
                                   const uint3            ne11,
                                   const uint3            ne12,
                                   const uint3            ne13,
                                   /*int s0, */ const int s1,
                                   const int              s2,
                                   const int              s3,
                                   /*int s00,*/ const int s01,
                                   const int              s02,
                                   const int              s03,
                                   /*int s10,*/ const int s11,
                                   const int              s12,
                                   const int              s13,
                                   src1_ptrs... src1s) {
    const uint32_t i0s = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t i1  = (blockDim.y * blockIdx.y + threadIdx.y);
    const uint32_t i2  = fastdiv((blockDim.z * blockIdx.z + threadIdx.z), ne3);
    const uint32_t i3  = (blockDim.z * blockIdx.z + threadIdx.z) - (i2 * ne3.z);

    if (i0s >= (uint32_t)ne0 || i1 >= (uint32_t)ne1 || i2 >= (uint32_t)ne2 || i3 >= ne3.z) {
        return;
    }

    const uint32_t i11 = fastmodulo(i1, ne11);
    const uint32_t i12 = fastmodulo(i2, ne12);
    const uint32_t i13 = fastmodulo(i3, ne13);

    const size_t i_src0 =  i3*s03 +  i2*s02 +  i1*s01;
    const size_t i_src1 = i13*s13 + i12*s12 + i11*s11;
    const size_t i_dst  =  i3*s3  +  i2*s2  +  i1*s1;

    const src0_t * src0_row = src0 ? (src0 + i_src0) : nullptr;
    dst_t * dst_row = dst + i_dst;

    for (int i0 = i0s; i0 < ne0; i0 += blockDim.x * gridDim.x) {
        const uint32_t i10 = fastmodulo(i0, ne10);

        float result = src0_row ? (float) src0_row[i0] : 0.0f;
        if constexpr (sizeof...(src1_ptrs) > 0) {
            result = (..., (result = bin_op(result, (float)src1s[i_src1 + i10])));
        } else {
            result = bin_op(result, (float)src1[i_src1 + i10]);
        }

        dst_row[i0] = (dst_t) result;
    }
}

template <float (*bin_op)(const float, const float),
          typename src0_t,
          typename src1_t,
          typename dst_t,
          typename... src1_ptrs>
static __global__ void k_bin_bcast_unravel(const src0_t *         src0,
                                           const src1_t *         src1,
                                           dst_t *                dst,
                                           const uint3            ne0,
                                           const uint3            ne1,
                                           const uint3            ne2,
                                           const uint32_t         ne3,
                                           const uint3            prod_012,
                                           const uint3            prod_01,
                                           const uint3            ne10,
                                           const uint3            ne11,
                                           const uint3            ne12,
                                           const uint3            ne13,
                                           /*int s0, */ const int s1,
                                           const int              s2,
                                           const int              s3,
                                           /*int s00,*/ const int s01,
                                           const int              s02,
                                           const int              s03,
                                           /*int s10,*/ const int s11,
                                           const int              s12,
                                           const int              s13,
                                           src1_ptrs... src1s) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    const uint32_t i3 = fastdiv(i, prod_012);
    const uint32_t i2 = fastdiv(i - i3 * prod_012.z, prod_01);
    const uint32_t i1 = fastdiv(i - i3 * prod_012.z - i2 * prod_01.z, ne0);
    const uint32_t i0 = i - i3 * prod_012.z - i2 * prod_01.z - i1 * ne0.z;

    if (i0 >= ne0.z || i1 >= ne1.z || i2 >= ne2.z || i3 >= ne3) {
        return;
    }

    const int i11 = fastmodulo(i1, ne11);
    const int i12 = fastmodulo(i2, ne12);
    const int i13 = fastmodulo(i3, ne13);

    const size_t i_src0 =  i3*s03 +  i2*s02 +  i1*s01;
    const size_t i_src1 = i13*s13 + i12*s12 + i11*s11;
    const size_t i_dst  =  i3*s3  +  i2*s2  +  i1*s1;

    const src0_t * src0_row = src0 ? (src0 + i_src0) : nullptr;
    dst_t * dst_row = dst + i_dst;

    const int i10 = fastmodulo(i0, ne10);

    float result = src0_row ? (float) src0_row[i0] : 0.0f;
    if constexpr (sizeof...(src1_ptrs) > 0) {
        result = (..., (result = bin_op(result, (float)src1s[i_src1 + i10])));
    } else {
        result = bin_op(result, (float)src1[i_src1 + i10]);
    }

    dst_row[i0] = (dst_t) result;
}

template <float (*bin_op)(const float, const float), typename src0_t, typename src1_t, typename dst_t, size_t... I>
static void launch_bin_bcast_pack(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
                                  const src0_t * src0_dd, const src1_t * src1_dd, dst_t * dst_dd,
                                  hipStream_t stream, std::index_sequence<I...>) {
    GGML_TENSOR_BINARY_OP_LOCALS

    int nr0 = ne10 / ne0;
    int nr1 = ne11 / ne1;
    int nr2 = ne12 / ne2;
    int nr3 = ne13 / ne3;

    int nr[4] = { nr0, nr1, nr2, nr3 };

    int64_t cne[]  = { ne0, ne1, ne2, ne3 };
    int64_t cne0[] = { ne00, ne01, ne02, ne03 };
    int64_t cne1[] = { ne10, ne11, ne12, ne13 };

    size_t cnb[]  = { nb0, nb1, nb2, nb3 };
    size_t cnb0[] = { nb00, nb01, nb02, nb03 };
    size_t cnb1[] = { nb10, nb11, nb12, nb13 };

    auto collapse = [](int64_t cne[]) {
        cne[0] *= cne[1];
        cne[1] = cne[2];
        cne[2] = cne[3];
        cne[3] = 1;
    };

    auto collapse_nb = [](size_t cnb[], const int64_t cne[]) {
        cnb[1] *= cne[1];
        cnb[2] *= cne[2];
        cnb[3] *= cne[3];
    };

    if (ggml_is_contiguous(src0) && ggml_is_contiguous(src1) && ggml_is_contiguous(dst)) {
        for (int i = 0; i < 4; i++) {
            if (nr[i] != 1) {
                break;
            }
            if (i > 0) {
                collapse_nb(cnb, cne);
                collapse_nb(cnb0, cne0);
                collapse_nb(cnb1, cne1);
                collapse(cne);
                collapse(cne0);
                collapse(cne1);
            }
        }
    }

    {
        int64_t ne0 = cne[0];
        int64_t ne1 = cne[1];
        int64_t ne2 = cne[2];
        int64_t ne3 = cne[3];

        //int64_t ne00 = cne0[0]; GGML_UNUSED(ne00);
        //int64_t ne01 = cne0[1]; GGML_UNUSED(ne01);
        //int64_t ne02 = cne0[2]; GGML_UNUSED(ne02);
        //int64_t ne03 = cne0[3]; GGML_UNUSED(ne03);

        size_t nb0 = cnb[0];
        size_t nb1 = cnb[1];
        size_t nb2 = cnb[2];
        size_t nb3 = cnb[3];

        size_t nb00 = cnb0[0];
        size_t nb01 = cnb0[1];
        size_t nb02 = cnb0[2];
        size_t nb03 = cnb0[3];

        size_t nb10 = cnb1[0];
        size_t nb11 = cnb1[1];
        size_t nb12 = cnb1[2];
        size_t nb13 = cnb1[3];

        size_t s0 = nb0 / sizeof(dst_t);
        size_t s1 = nb1 / sizeof(dst_t);
        size_t s2 = nb2 / sizeof(dst_t);
        size_t s3 = nb3 / sizeof(dst_t);

        size_t s10 = nb10 / sizeof(src1_t);
        size_t s11 = nb11 / sizeof(src1_t);
        size_t s12 = nb12 / sizeof(src1_t);
        size_t s13 = nb13 / sizeof(src1_t);

        size_t s00 = nb00 / sizeof(src0_t);
        size_t s01 = nb01 / sizeof(src0_t);
        size_t s02 = nb02 / sizeof(src0_t);
        size_t s03 = nb03 / sizeof(src0_t);

        GGML_ASSERT(nb0 % sizeof(dst_t) == 0);
        GGML_ASSERT(nb1 % sizeof(dst_t) == 0);
        GGML_ASSERT(nb2 % sizeof(dst_t) == 0);
        GGML_ASSERT(nb3 % sizeof(dst_t) == 0);

        GGML_ASSERT(nb00 % sizeof(src0_t) == 0);
        GGML_ASSERT(nb01 % sizeof(src0_t) == 0);
        GGML_ASSERT(nb02 % sizeof(src0_t) == 0);
        GGML_ASSERT(nb03 % sizeof(src0_t) == 0);

        GGML_ASSERT(nb10 % sizeof(src1_t) == 0);
        GGML_ASSERT(nb11 % sizeof(src1_t) == 0);
        GGML_ASSERT(nb12 % sizeof(src1_t) == 0);
        GGML_ASSERT(nb13 % sizeof(src1_t) == 0);

        GGML_ASSERT(s0 == 1);
        GGML_ASSERT(s00 == 1);
        GGML_ASSERT(s10 == 1);

        const int block_size = 128;

        int64_t hne0 = std::max(ne0 / 2LL, 1LL);

        dim3 block_dims;
        block_dims.x = std::min<unsigned int>(hne0, block_size);
        block_dims.y = std::min<unsigned int>(ne1, block_size / block_dims.x);
        block_dims.z = std::min(std::min<unsigned int>(ne2 * ne3, block_size / block_dims.x / block_dims.y), 64U);

        dim3 block_nums((hne0 + block_dims.x - 1) / block_dims.x, (ne1 + block_dims.y - 1) / block_dims.y,
                        (ne2 * ne3 + block_dims.z - 1) / block_dims.z);

        const uint3 ne10 = init_fastdiv_values((uint32_t) cne1[0]);
        const uint3 ne11 = init_fastdiv_values((uint32_t) cne1[1]);
        const uint3 ne12 = init_fastdiv_values((uint32_t) cne1[2]);
        const uint3 ne13 = init_fastdiv_values((uint32_t) cne1[3]);

        if (block_nums.z > 65535) {
            int         block_num  = (ne0 * ne1 * ne2 * ne3 + block_size - 1) / block_size;
            const uint3 prod_012    = init_fastdiv_values((uint32_t) (ne0 * ne1 * ne2));
            const uint3 prod_01     = init_fastdiv_values((uint32_t) (ne0 * ne1));
            const uint3 ne0_fastdiv = init_fastdiv_values((uint32_t) ne0);
            const uint3 ne1_fastdiv = init_fastdiv_values((uint32_t) ne1);
            const uint3 ne2_fastdiv = init_fastdiv_values((uint32_t) ne2);

            if constexpr (sizeof...(I) > 0) {
                k_bin_bcast_unravel<bin_op, src0_t, src1_t, dst_t><<<block_num, block_size, 0, stream>>>(
                    src0_dd, src1_dd, dst_dd, ne0_fastdiv, ne1_fastdiv, ne2_fastdiv, ne3, prod_012, prod_01, ne10, ne11,
                    ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00,*/ s01, s02, s03,
                    /* s10,*/ s11, s12, s13, (const src1_t *) dst->src[I + 1]->data...);
            } else {
                k_bin_bcast_unravel<bin_op, src0_t, src1_t, dst_t>
                    <<<block_num, block_size, 0, stream>>>(src0_dd, src1_dd, dst_dd, ne0_fastdiv, ne1_fastdiv,
                                                           ne2_fastdiv, ne3, prod_012, prod_01, ne10, ne11, ne12, ne13,
                                                           /* s0, */ s1, s2, s3,
                                                           /* s00,*/ s01, s02, s03,
                                                           /* s10,*/ s11, s12, s13);
            }
        } else {
            const uint3 ne3_fastdiv = init_fastdiv_values((uint32_t) ne3);
            if constexpr (sizeof...(I) > 0) {
                k_bin_bcast<bin_op, src0_t, src1_t, dst_t><<<block_nums, block_dims, 0, stream>>>(
                    src0_dd, src1_dd, dst_dd, ne0, ne1, ne2, ne3_fastdiv, ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00,*/ s01, s02, s03,
                    /* s10,*/ s11, s12, s13, (const src1_t *) dst->src[I + 1]->data...);
            } else {
                k_bin_bcast<bin_op, src0_t, src1_t, dst_t><<<block_nums, block_dims, 0, stream>>>(
                    src0_dd, src1_dd, dst_dd, ne0, ne1, ne2, ne3_fastdiv, ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00,*/ s01, s02, s03,
                    /* s10,*/ s11, s12, s13);
            }
        }
    }
}

template <typename T>
static __global__ void k_repeat_back(
    const T * __restrict__ src, T * __restrict__ dst, const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
    const size_t s00, const size_t s01, const size_t s02, const size_t s03,
    const int64_t ne0, const int64_t ne1, const int64_t ne2, const int64_t ne3) {

    const int64_t tid0  = int64_t(blockIdx.x)*blockDim.x + threadIdx.x;
    const int64_t tid1  = int64_t(blockIdx.y)*blockDim.y + threadIdx.y;
    const int64_t tid23 = int64_t(blockIdx.z)*blockDim.z + threadIdx.z;
    const int64_t tid2  = tid23 % ne2;
    const int64_t tid3  = tid23 / ne2;

    if (tid0 >= ne0) {
        return;
    }

    T sum = 0;
    for (int64_t i3 = tid3; i3 < ne03; i3 += ne3) {
        for (int64_t i2 = tid2; i2 < ne02; i2 += ne2) {
            for (int64_t i1 = tid1; i1 < ne01; i1 += ne1) {
                for (int64_t i0 = tid0; i0 < ne00; i0 += ne0) {
                    sum += src[i3*s03 + i2*s02 + i1*s01 + i0*s00];
                }
            }
        }
    }
    dst[tid3*ne2*ne1*ne0 + tid2*ne1*ne0 + tid1*ne0 + tid0] = sum;
}

template <float (*bin_op)(const float, const float), int n_fuse = 1>
struct bin_bcast_cuda {
    template<typename src0_t, typename src1_t, typename dst_t>
    void operator()(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst,
            const src0_t * src0_dd, const src1_t * src1_dd, dst_t * dst_dd,
            hipStream_t stream) {
        launch_bin_bcast_pack<bin_op, src0_t, src1_t, dst_t>(
            src0, src1, dst, src0_dd, src1_dd, dst_dd, stream, std::make_index_sequence<n_fuse>{});
    }
};

template <typename T>
static void repeat_back_cuda(
    const T * src, T * dst, const int64_t ne00, const int64_t ne01, const int64_t ne02, const int64_t ne03,
    const size_t s00, const size_t s01, const size_t s02, const size_t s03,
    const int64_t ne0, const int64_t ne1, const int64_t ne2, const int64_t ne3, hipStream_t stream) {

    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums((ne0 + WARP_SIZE - 1) / WARP_SIZE, ne1, ne2*ne3);
    k_repeat_back<T><<<block_nums, block_dims, 0, stream>>>
        (src, dst, ne00, ne01, ne02, ne03, s00, s01, s02, s03, ne0, ne1, ne2, ne3);
}

template<class op>
static void ggml_cuda_op_bin_bcast(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    const void * src0_dd, const void * src1_dd, void * dst_dd, hipStream_t stream) {

    GGML_ASSERT(src1->type == GGML_TYPE_F32 || src1->type == GGML_TYPE_F16);

    if (src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32) {
        op()(src0, src1, dst, (const float *)src0_dd, (const float *)src1_dd, (float *)dst_dd, stream);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F16) {
        op()(src0, src1, dst, (const half *) src0_dd, (const half *)src1_dd, (half *) dst_dd, stream);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F16) {
        op()(src0, src1, dst, (const half *) src0_dd, (const float *)src1_dd, (half *) dst_dd, stream);
    } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F32) {
        op()(src0, src1, dst, (const half *) src0_dd, (const float *)src1_dd, (float *)dst_dd, stream);
    } else {
        fprintf(stderr, "%s: unsupported types: dst: %s, src0: %s, src1: %s\n", __func__,
            ggml_type_name(dst->type), ggml_type_name(src0->type), ggml_type_name(src1->type));
        GGML_ABORT("fatal error");
    }
}

void ggml_cuda_op_repeat(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_repeat, 0>>(dst, dst->src[0], dst, nullptr, dst->src[0]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_add>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_sub(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_sub>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_mul(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_mul>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_div(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_div>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

template <float (*op)(const float, const float), int n_fuse>
static void ggml_cuda_op_fused_binbcast_impl(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    hipStream_t stream = ctx.stream();

    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    if (src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32) {
        launch_bin_bcast_pack<op, float, float, float>(src0, src1, dst,
            (const float *) src0->data, (const float *) src1->data, (float *) dst->data,
            stream, std::make_index_sequence<n_fuse>{});
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F16) {
        launch_bin_bcast_pack<op, half, half, half>(src0, src1, dst,
            (const half *) src0->data, (const half *) src1->data, (half *) dst->data,
            stream, std::make_index_sequence<n_fuse>{});
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F16) {
        launch_bin_bcast_pack<op, half, float, half>(src0, src1, dst,
            (const half *) src0->data, (const float *) src1->data, (half *) dst->data,
            stream, std::make_index_sequence<n_fuse>{});
    } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F32) {
        launch_bin_bcast_pack<op, half, float, float>(src0, src1, dst,
            (const half *) src0->data, (const float *) src1->data, (float *) dst->data,
            stream, std::make_index_sequence<n_fuse>{});
    } else {
        fprintf(stderr,
                "%s: unsupported types for fusion: dst: %s, src0: %s, src1: %s\n",
                __func__, ggml_type_name(dst->type), ggml_type_name(src0->type), ggml_type_name(src1->type));
        GGML_ABORT("fatal error");
    }
}


void ggml_cuda_op_fused_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst, int n_fuse) {
    GGML_ASSERT(2 <= n_fuse && n_fuse <= 8);

    switch (n_fuse) {
        case 2:
            ggml_cuda_op_fused_binbcast_impl<op_add, 2>(ctx, dst);
            break;
        case 3:
            ggml_cuda_op_fused_binbcast_impl<op_add, 3>(ctx, dst);
            break;
        case 4:
            ggml_cuda_op_fused_binbcast_impl<op_add, 4>(ctx, dst);
            break;
        case 5:
            ggml_cuda_op_fused_binbcast_impl<op_add, 5>(ctx, dst);
            break;
        case 6:
            ggml_cuda_op_fused_binbcast_impl<op_add, 6>(ctx, dst);
            break;
        case 7:
            ggml_cuda_op_fused_binbcast_impl<op_add, 7>(ctx, dst);
            break;
        case 8:
            ggml_cuda_op_fused_binbcast_impl<op_add, 8>(ctx, dst);
            break;
        default:
            GGML_ASSERT(false && "Unsupported n_fuse value");
    }
}

void ggml_cuda_op_repeat_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];

    GGML_ASSERT(src0->type == dst->type);
    GGML_ASSERT(ggml_is_contiguous(dst));
    GGML_ASSERT(ggml_can_repeat(dst, src0));

    hipStream_t stream = ctx.stream();

    GGML_TENSOR_UNARY_OP_LOCALS;

    GGML_ASSERT(ne2*ne3 <= (1 << 15));

    const size_t ts = ggml_type_size(src0->type);
    const size_t s00 = nb00 / ts;
    const size_t s01 = nb01 / ts;
    const size_t s02 = nb02 / ts;
    const size_t s03 = nb03 / ts;

    switch (dst->type) {
        case GGML_TYPE_F32: {
            const float * src0_d = (const float *) src0->data;
            float       * dst_d  = (float       *) dst->data;
            repeat_back_cuda(src0_d, dst_d, ne00, ne01, ne02, ne03, s00, s01, s02, s03, ne0, ne1, ne2, ne3, stream);
        } break;
        default: {
            GGML_ASSERT(false);
        } break;
    }
}

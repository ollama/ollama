#include "hip/hip_runtime.h"
#include "binbcast.cuh"
#include <cstdint>

static __device__ __forceinline__ float op_repeat(const float a, const float b) {
    return b;
    GGML_UNUSED(a);
}

static __device__ __forceinline__ float op_add(const float a, const float b) {
    return a + b;
}

static __device__ __forceinline__ float op_sub(const float a, const float b) {
    return a - b;
}

static __device__ __forceinline__ float op_mul(const float a, const float b) {
    return a * b;
}

static __device__ __forceinline__ float op_div(const float a, const float b) {
    return a / b;
}

template<float (*bin_op)(const float, const float), typename src0_t, typename src1_t, typename dst_t>
static __global__ void k_bin_bcast(const src0_t * src0, const src1_t * src1, dst_t * dst,
        int ne0, int ne1, int ne2, int ne3,
        int ne10, int ne11, int ne12, int ne13,
        /*int s0, */ int s1,  int s2,  int s3,
        /*int s00,*/ int s01, int s02, int s03,
        /*int s10,*/ int s11, int s12, int s13) {
    const int i0s = blockDim.x*blockIdx.x + threadIdx.x;
    const int i1 = (blockDim.y*blockIdx.y + threadIdx.y);
    const int i2 = (blockDim.z*blockIdx.z + threadIdx.z) / ne3;
    const int i3 = (blockDim.z*blockIdx.z + threadIdx.z) % ne3;

    if (i0s >= ne0 || i1 >= ne1 || i2 >= ne2 || i3 >= ne3) {
        return;
    }

    const int i11 = i1 % ne11;
    const int i12 = i2 % ne12;
    const int i13 = i3 % ne13;

    const size_t i_src0 =  i3*s03 +  i2*s02 +  i1*s01;
    const size_t i_src1 = i13*s13 + i12*s12 + i11*s11;
    const size_t i_dst  =  i3*s3  +  i2*s2  +  i1*s1;

    const src0_t * src0_row = src0 + i_src0;
    const src1_t * src1_row = src1 + i_src1;
    dst_t * dst_row = dst + i_dst;

    for (int i0 = i0s; i0 < ne0; i0 += blockDim.x*gridDim.x) {
        const int i10 = i0 % ne10;
        dst_row[i0] = (dst_t)bin_op(src0 ? (float)src0_row[i0] : 0.0f, (float)src1_row[i10]);
    }
}

template<float (*bin_op)(const float, const float), typename src0_t, typename src1_t, typename dst_t>
static __global__ void k_bin_bcast_unravel(const src0_t * src0, const src1_t * src1, dst_t * dst,
        int ne0, int ne1, int ne2, int ne3,
        int ne10, int ne11, int ne12, int ne13,
        /*int s0, */ int s1,  int s2,  int s3,
        /*int s00,*/ int s01, int s02, int s03,
        /*int s10,*/ int s11, int s12, int s13) {

    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    const int i3 = i/(ne2*ne1*ne0);
    const int i2 = (i/(ne1*ne0)) % ne2;
    const int i1 = (i/ne0) % ne1;
    const int i0 = i % ne0;

    if (i0 >= ne0 || i1 >= ne1 || i2 >= ne2 || i3 >= ne3) {
        return;
    }

    const int i11 = i1 % ne11;
    const int i12 = i2 % ne12;
    const int i13 = i3 % ne13;

    const size_t i_src0 =  i3*s03 +  i2*s02 +  i1*s01;
    const size_t i_src1 = i13*s13 + i12*s12 + i11*s11;
    const size_t i_dst  =  i3*s3  +  i2*s2  +  i1*s1;

    const src0_t * src0_row = src0 + i_src0;
    const src1_t * src1_row = src1 + i_src1;
    dst_t * dst_row = dst + i_dst;

    const int i10 = i0 % ne10;
    dst_row[i0] = (dst_t)bin_op(src0 ? (float)src0_row[i0] : 0.0f, (float)src1_row[i10]);
}

template <typename T>
static __global__ void k_repeat_back(
    const T * __restrict__ src, T * __restrict__ dst, const int64_t ne00, const int64_t ne01, const int64_t ne02,
    const int64_t ne0, const int64_t ne1, const int64_t ne2) {

    const int64_t tid0 = (int64_t) blockIdx.x*blockDim.x + threadIdx.x;
    const int64_t tid1 = (int64_t) blockIdx.y*blockDim.y + threadIdx.y;
    const int64_t tid2 = (int64_t) blockIdx.z*blockDim.z + threadIdx.z;

    if (tid0 >= ne0) {
        return;
    }

    T sum = 0;
    for (int64_t i2 = tid2; i2 < ne02; i2 += ne2) {
        for (int64_t i1 = tid1; i1 < ne01; i1 += ne1) {
            for (int64_t i0 = tid0; i0 < ne00; i0 += ne0) {
                sum += src[i2*ne01*ne00 + i1*ne00 + i0];
            }
        }
    }
    dst[tid2*ne1*ne0 + tid1*ne0 + tid0] = sum;
}

template<float (*bin_op)(const float, const float)>
struct bin_bcast_cuda {
    template<typename src0_t, typename src1_t, typename dst_t>
    void operator()(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst,
            const src0_t * src0_dd, const src1_t * src1_dd, dst_t * dst_dd,
            hipStream_t stream) {

        GGML_TENSOR_BINARY_OP_LOCALS

        int nr0 = ne10/ne0;
        int nr1 = ne11/ne1;
        int nr2 = ne12/ne2;
        int nr3 = ne13/ne3;

        int nr[4] = { nr0, nr1, nr2, nr3 };

        // collapse dimensions until first broadcast dimension
        int64_t cne[] = {ne0, ne1, ne2, ne3};
        int64_t cne0[] = {ne00, ne01, ne02, ne03};
        int64_t cne1[] = {ne10, ne11, ne12, ne13};

        size_t cnb[] = {nb0, nb1, nb2, nb3};
        size_t cnb0[] = {nb00, nb01, nb02, nb03};
        size_t cnb1[] = {nb10, nb11, nb12, nb13};

        auto collapse = [](int64_t cne[]) {
            cne[0] *= cne[1];
            cne[1] = cne[2];
            cne[2] = cne[3];
            cne[3] = 1;
        };

        auto collapse_nb = [](size_t cnb[], const int64_t cne[]) {
            cnb[1] *= cne[1];
            cnb[2] *= cne[2];
            cnb[3] *= cne[3];
        };

        if (ggml_is_contiguous(src0) && ggml_is_contiguous(src1) && ggml_is_contiguous(dst)) {
            for (int i = 0; i < 4; i++) {
                if (nr[i] != 1) {
                    break;
                }
                if (i > 0) {
                    collapse_nb(cnb, cne);
                    collapse_nb(cnb0, cne0);
                    collapse_nb(cnb1, cne1);
                    collapse(cne);
                    collapse(cne0);
                    collapse(cne1);
                }
            }
        }

        {
            int64_t ne0 = cne[0];
            int64_t ne1 = cne[1];
            int64_t ne2 = cne[2];
            int64_t ne3 = cne[3];

            //int64_t ne00 = cne0[0]; GGML_UNUSED(ne00);
            //int64_t ne01 = cne0[1]; GGML_UNUSED(ne01);
            //int64_t ne02 = cne0[2]; GGML_UNUSED(ne02);
            //int64_t ne03 = cne0[3]; GGML_UNUSED(ne03);

            int64_t ne10 = cne1[0];
            int64_t ne11 = cne1[1];
            int64_t ne12 = cne1[2];
            int64_t ne13 = cne1[3];

            size_t nb0 = cnb[0];
            size_t nb1 = cnb[1];
            size_t nb2 = cnb[2];
            size_t nb3 = cnb[3];

            size_t nb00 = cnb0[0];
            size_t nb01 = cnb0[1];
            size_t nb02 = cnb0[2];
            size_t nb03 = cnb0[3];

            size_t nb10 = cnb1[0];
            size_t nb11 = cnb1[1];
            size_t nb12 = cnb1[2];
            size_t nb13 = cnb1[3];

            size_t s0 = nb0 / sizeof(dst_t);
            size_t s1 = nb1 / sizeof(dst_t);
            size_t s2 = nb2 / sizeof(dst_t);
            size_t s3 = nb3 / sizeof(dst_t);

            size_t s10 = nb10 / sizeof(src1_t);
            size_t s11 = nb11 / sizeof(src1_t);
            size_t s12 = nb12 / sizeof(src1_t);
            size_t s13 = nb13 / sizeof(src1_t);

            size_t s00 = nb00 / sizeof(src0_t);
            size_t s01 = nb01 / sizeof(src0_t);
            size_t s02 = nb02 / sizeof(src0_t);
            size_t s03 = nb03 / sizeof(src0_t);

            GGML_ASSERT(nb0 % sizeof(dst_t) == 0);
            GGML_ASSERT(nb1 % sizeof(dst_t) == 0);
            GGML_ASSERT(nb2 % sizeof(dst_t) == 0);
            GGML_ASSERT(nb3 % sizeof(dst_t) == 0);

            GGML_ASSERT(nb00 % sizeof(src0_t) == 0);
            GGML_ASSERT(nb01 % sizeof(src0_t) == 0);
            GGML_ASSERT(nb02 % sizeof(src0_t) == 0);
            GGML_ASSERT(nb03 % sizeof(src0_t) == 0);

            GGML_ASSERT(nb10 % sizeof(src1_t) == 0);
            GGML_ASSERT(nb11 % sizeof(src1_t) == 0);
            GGML_ASSERT(nb12 % sizeof(src1_t) == 0);
            GGML_ASSERT(nb13 % sizeof(src1_t) == 0);

            GGML_ASSERT(s0 == 1);
            GGML_ASSERT(s00 == 1);
            GGML_ASSERT(s10 == 1);

            const int block_size = 128;

            int64_t hne0 = std::max(ne0/2LL, 1LL);

            dim3 block_dims;
            block_dims.x = std::min<unsigned int>(hne0, block_size);
            block_dims.y = std::min<unsigned int>(ne1, block_size / block_dims.x);
            block_dims.z = std::min(std::min<unsigned int>(ne2*ne3, block_size / block_dims.x / block_dims.y), 64U);

            dim3 block_nums(
                (hne0 + block_dims.x - 1) / block_dims.x,
                (ne1 + block_dims.y - 1) / block_dims.y,
                (ne2*ne3 + block_dims.z - 1) / block_dims.z
            );

            if (block_nums.z > 65535) {
                // this is the maximum number of blocks in z dimension, fallback to 1D grid kernel
                int block_num = (ne0*ne1*ne2*ne3 + block_size - 1) / block_size;
                k_bin_bcast_unravel<bin_op><<<block_num, block_size, 0, stream>>>(
                    src0_dd, src1_dd, dst_dd,
                    ne0, ne1, ne2, ne3,
                    ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00, */ s01, s02, s03,
                    /* s10, */ s11, s12, s13);
            } else {
                k_bin_bcast<bin_op><<<block_nums, block_dims, 0, stream>>>(
                    src0_dd, src1_dd, dst_dd,
                    ne0, ne1, ne2, ne3,
                    ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00, */ s01, s02, s03,
                    /* s10, */ s11, s12, s13);
            }
        }
    }
};

template <typename T>
static void repeat_back_cuda(
    const T * src, T * dst, const int64_t ne00, const int64_t ne01, const int64_t ne02,
    const int64_t ne0, const int64_t ne1, const int64_t ne2, hipStream_t stream) {

    const dim3 block_dims(WARP_SIZE, 1, 1);
    const dim3 block_nums((ne0 + WARP_SIZE - 1) / WARP_SIZE, ne1, ne2);
    k_repeat_back<T><<<block_nums, block_dims, 0, stream>>>(src, dst, ne00, ne01, ne02, ne0, ne1, ne2);
}

template<class op>
static void ggml_cuda_op_bin_bcast(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    const void * src0_dd, const void * src1_dd, void * dst_dd, hipStream_t stream) {

    GGML_ASSERT(src1->type == GGML_TYPE_F32);

    if (src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32) {
        op()(src0, src1, dst, (const float *)src0_dd, (const float *)src1_dd, (float *)dst_dd, stream);
    } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F16) {
        op()(src0, src1, dst, (const half *) src0_dd, (const float *)src1_dd, (half *) dst_dd, stream);
    } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F32) {
        op()(src0, src1, dst, (const half *) src0_dd, (const float *)src1_dd, (float *)dst_dd, stream);
    } else {
        fprintf(stderr, "%s: unsupported types: dst: %s, src0: %s, src1: %s\n", __func__,
            ggml_type_name(dst->type), ggml_type_name(src0->type), ggml_type_name(src1->type));
        GGML_ABORT("fatal error");
    }
}

void ggml_cuda_op_repeat(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_repeat>>(dst, dst->src[0], dst, nullptr, dst->src[0]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_add>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_sub(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_sub>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_mul(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_mul>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_div(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_div>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_repeat_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];

    GGML_ASSERT(src0->type == dst->type);
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(dst));
    GGML_ASSERT(ggml_can_repeat(dst, src0));

    hipStream_t stream = ctx.stream();

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    GGML_ASSERT(src0->ne[3] == 1);

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];
    const int64_t ne2 = dst->ne[2];
    GGML_ASSERT(dst->ne[3] == 1);

    switch (dst->type) {
        case GGML_TYPE_F32: {
            const float * src0_d = (const float *) src0->data;
            float       * dst_d  = (float       *) dst->data;
            repeat_back_cuda<float>(src0_d, dst_d, ne00, ne01, ne02, ne0, ne1, ne2, stream);
        } break;
        default: {
            GGML_ASSERT(false);
        } break;
    }
}

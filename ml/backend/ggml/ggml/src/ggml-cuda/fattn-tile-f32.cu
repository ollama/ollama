#include "hip/hip_runtime.h"
#include "common.cuh"
#include "fattn-common.cuh"
#include "fattn-tile-f32.cuh"

#define FATTN_KQ_STRIDE_TILE_F32 32

template<int D, int ncols, int nwarps, int parallel_blocks, bool use_logit_softcap> // D == head size
#if !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(nwarps*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIP) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_tile_ext_f32(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const uint32_t n_head_log2,
        const float logit_softcap,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int nb21,
        const int nb22,
        const int nb23,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#ifndef FLASH_ATTN_AVAILABLE
    NO_DEVICE_CODE;
    return;
#endif // FLASH_ATTN_AVAILABLE
    // Skip unused kernel variants for faster compilation:
    if (use_logit_softcap && !(D == 128 || D == 256)) {
        NO_DEVICE_CODE;
        return;
    }

    // In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    const int ic0 = (blockIdx.x / parallel_blocks) * ncols; // Index of the Q/QKV column to work on.
    const int ip  =  blockIdx.x % parallel_blocks; // Index in group of blocks running for the same column in parallel.

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.
    const float2 * Q_f2  = (const float2 *) (Q    + nb02* blockIdx.y              + nb01*ic0);
    const half2  * K_h2  = (const half2  *) (K    + nb12*(blockIdx.y / gqa_ratio));
    const half2  * V_h2  = (const half2  *) (V    + nb12*(blockIdx.y / gqa_ratio)); // K and V have same shape
    const half   * maskh = (const half   *)  mask + ne11*ic0;

    const int stride_KV2 = nb11 / sizeof(half2);

    const float slope = get_alibi_slope(max_bias, blockIdx.y, n_head_log2, m0, m1);

    static_assert(D % (2*WARP_SIZE) == 0, "D not divisible by 2*WARP_SIZE == 64.");

    __shared__ float KQ[ncols*FATTN_KQ_STRIDE_TILE_F32];

    __shared__ float KV_tmp[FATTN_KQ_STRIDE_TILE_F32][D + 1]; // Pad D to avoid memory bank conflicts.
    float2 * KV_tmp2 = (float2 *) KV_tmp;

    float kqmax[ncols/nwarps];
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        kqmax[j0/nwarps] = -FLT_MAX/2.0f;
    }
    float kqsum[ncols/nwarps] = {0.0f};

    float2 VKQ[ncols/nwarps][(D/2)/WARP_SIZE] = {{{0.0f, 0.0f}}};

    // Convert Q to half2 and store in registers:
    __shared__ float Q_f[ncols][D];
#pragma unroll
    for (int j0 = 0; j0 < ncols; j0 += nwarps) {
        const int j = j0 + threadIdx.y;

#pragma unroll
        for (int i0 = 0; i0 < D; i0 += 2*WARP_SIZE) {
            float2 tmp = ic0 + j < ne01 ? Q_f2[j*(nb01/sizeof(float2)) + i0/2 + threadIdx.x] : make_float2(0.0f, 0.0f);
            Q_f[j][i0 + 0*WARP_SIZE + threadIdx.x] = tmp.x * scale;
            Q_f[j][i0 + 1*WARP_SIZE + threadIdx.x] = tmp.y * scale;
        }
    }

    __syncthreads();

    const int k_start = parallel_blocks == 1 ? 0 : ip*FATTN_KQ_STRIDE_TILE_F32;
    for (int k_VKQ_0 = k_start; k_VKQ_0 < ne11; k_VKQ_0 += parallel_blocks*FATTN_KQ_STRIDE_TILE_F32) {
        // Calculate KQ tile and keep track of new maximum KQ values:

        float kqmax_new[ncols/nwarps];
#pragma unroll
        for (int j = 0; j < ncols/nwarps; ++j) {
            kqmax_new[j] = kqmax[j];
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += nwarps) {
            const int i_KQ = i_KQ_0 + threadIdx.y;

#pragma unroll
            for (int k_KQ_0 = 0; k_KQ_0 < D; k_KQ_0 += 2*WARP_SIZE) {
                const half2 tmp = K_h2[(k_VKQ_0 + i_KQ)*stride_KV2 + k_KQ_0/2 + threadIdx.x];
                KV_tmp[i_KQ][k_KQ_0 + 0*WARP_SIZE + threadIdx.x] =  __low2float(tmp);
                KV_tmp[i_KQ][k_KQ_0 + 1*WARP_SIZE + threadIdx.x] = __high2float(tmp);
            }
        }

        __syncthreads();

        float sum[FATTN_KQ_STRIDE_TILE_F32/WARP_SIZE][ncols/nwarps] = {{0.0f}};

#pragma unroll
        for (int k_KQ = 0; k_KQ < D; ++k_KQ) {
            float K_k[FATTN_KQ_STRIDE_TILE_F32/WARP_SIZE];
            float Q_k[ncols/nwarps];

#pragma unroll
            for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += WARP_SIZE) {
                const int i_KQ = i_KQ_0 + threadIdx.x;

                K_k[i_KQ_0/WARP_SIZE] = KV_tmp[i_KQ][k_KQ];
            }
#pragma unroll
            for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                const int j_KQ = j_KQ_0 + threadIdx.y;

                Q_k[j_KQ_0/nwarps] = Q_f[j_KQ][k_KQ];
            }

#pragma unroll
            for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += WARP_SIZE) {
#pragma unroll
                for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                    sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps] += K_k[i_KQ_0/WARP_SIZE] * Q_k[j_KQ_0/nwarps];
                }
            }
        }

#pragma unroll
        for (int i_KQ_0 = 0; i_KQ_0 < FATTN_KQ_STRIDE_TILE_F32; i_KQ_0 += WARP_SIZE) {
            const int i_KQ = i_KQ_0 + threadIdx.x;

#pragma unroll
            for (int j_KQ_0 = 0; j_KQ_0 < ncols; j_KQ_0 += nwarps) {
                const int j_KQ = j_KQ_0 + threadIdx.y;

                if (use_logit_softcap) {
                    sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps] = logit_softcap * tanhf(sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps]);
                }

                sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps] += mask ? slope*__half2float(maskh[j_KQ*ne11 + k_VKQ_0 + i_KQ]) : 0.0f;

                kqmax_new[j_KQ_0/nwarps] = fmaxf(kqmax_new[j_KQ_0/nwarps], sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps]);

                KQ[j_KQ*FATTN_KQ_STRIDE_TILE_F32 + i_KQ] = sum[i_KQ_0/WARP_SIZE][j_KQ_0/nwarps];
            }
        }

        __syncthreads();

#pragma unroll
        for (int j0 = 0; j0 < ncols; j0 += nwarps) {
            const int j = j0 + threadIdx.y;

            kqmax_new[j0/nwarps] = warp_reduce_max(kqmax_new[j0/nwarps]);
            const float KQ_max_scale = expf(kqmax[j0/nwarps] - kqmax_new[j0/nwarps]);
            kqmax[j0/nwarps] = kqmax_new[j0/nwarps];

            float kqsum_add = 0.0f;
#pragma unroll
            for (int i0 = 0; i0 < FATTN_KQ_STRIDE_TILE_F32; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                const float diff = KQ[j*FATTN_KQ_STRIDE_TILE_F32 + i] - kqmax[j0/nwarps];
                const float val = expf(diff);
                kqsum_add += val;
                KQ[j*FATTN_KQ_STRIDE_TILE_F32 + i] = val;
            }
            kqsum[j0/nwarps] = kqsum[j0/nwarps]*KQ_max_scale + kqsum_add;

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                VKQ[j0/nwarps][i0/WARP_SIZE].x *= KQ_max_scale;
                VKQ[j0/nwarps][i0/WARP_SIZE].y *= KQ_max_scale;
            }
        }

        __syncthreads();

#pragma unroll
        for (int k0 = 0; k0 < FATTN_KQ_STRIDE_TILE_F32; k0 += nwarps) {
            const int k = k0 + threadIdx.y;

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                KV_tmp2[k*(D/2) + i].x =  __low2float(V_h2[(k_VKQ_0 + k)*stride_KV2 + i]);
                KV_tmp2[k*(D/2) + i].y = __high2float(V_h2[(k_VKQ_0 + k)*stride_KV2 + i]);
            }
        }

        __syncthreads();

#pragma unroll
        for (int k = 0; k < FATTN_KQ_STRIDE_TILE_F32; ++k) {
            float2 V_k[(D/2)/WARP_SIZE];
            float  KQ_k[ncols/nwarps];

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
                const int i = i0 + threadIdx.x;

                V_k[i0/WARP_SIZE] = KV_tmp2[k*(D/2) + i];
            }
#pragma unroll
            for (int j0 = 0; j0 < ncols; j0 += nwarps) {
                const int j = j0 + threadIdx.y;

                KQ_k[j0/nwarps] = KQ[j*FATTN_KQ_STRIDE_TILE_F32 + k];
            }

#pragma unroll
            for (int i0 = 0; i0 < D/2; i0 += WARP_SIZE) {
#pragma unroll
                for (int j0 = 0; j0 < ncols; j0 += nwarps) {
                    VKQ[j0/nwarps][i0/WARP_SIZE].x += V_k[i0/WARP_SIZE].x*KQ_k[j0/nwarps];
                    VKQ[j0/nwarps][i0/WARP_SIZE].y += V_k[i0/WARP_SIZE].y*KQ_k[j0/nwarps];
                }
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int j_VKQ_0 = 0; j_VKQ_0 < ncols; j_VKQ_0 += nwarps) {
        const int j_VKQ = j_VKQ_0 + threadIdx.y;

        if (ic0 + j_VKQ >= ne01) {
            return;
        }

        float kqsum_j = kqsum[j_VKQ_0/nwarps];
        kqsum_j = warp_reduce_sum(kqsum_j);

#pragma unroll
        for (int i00 = 0; i00 < D; i00 += 2*WARP_SIZE) {
            const int i0 = i00 + 2*threadIdx.x;

            float2 dst_val = VKQ[j_VKQ_0/nwarps][i0/(2*WARP_SIZE)];
            if (parallel_blocks == 1) {
                dst_val.x /= kqsum_j;
                dst_val.y /= kqsum_j;
            }
            const int j_dst = (ic0 + j_VKQ)*parallel_blocks + ip;
            dst[j_dst*D*gridDim.y + D*blockIdx.y + i0 + 0] = dst_val.x;
            dst[j_dst*D*gridDim.y + D*blockIdx.y + i0 + 1] = dst_val.y;
        }

        if (parallel_blocks != 1 && threadIdx.x == 0) {
            dst_meta[(ic0 + j_VKQ)*gridDim.y*parallel_blocks + blockIdx.y*parallel_blocks + ip] = make_float2(kqmax[j_VKQ_0/nwarps], kqsum_j);
        }
    }
}

template <int cols_per_block, int parallel_blocks, bool use_logit_softcap>
void launch_fattn_tile_f32_64_128(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];
    switch (Q->ne[0]) {
        case  64: {
            constexpr int      D = 64;
            constexpr int nwarps = 8;
            fattn_kernel_t fattn_kernel = flash_attn_tile_ext_f32<D, cols_per_block, nwarps, parallel_blocks, use_logit_softcap>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block, true, true);
        } break;
        case 128: {
            constexpr int      D = 128;
            constexpr int nwarps = 8;
            fattn_kernel_t fattn_kernel = flash_attn_tile_ext_f32<D, cols_per_block, nwarps, parallel_blocks, use_logit_softcap>;
            launch_fattn<D, parallel_blocks>(ctx, dst, fattn_kernel, nwarps, cols_per_block, true, true);
        } break;
        default: {
            GGML_ABORT("FlashAttention without tensor cores only supports head sizes 64 and 128.");
        } break;
    }
}

void ggml_cuda_flash_attn_ext_tile_f32(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV = dst;
    const ggml_tensor * Q = dst->src[0];

    float logit_softcap;
    memcpy(&logit_softcap, (const float *) KQV->op_params + 2, sizeof(float));

    if (Q->ne[1] <= 16) {
        constexpr int cols_per_block = 16;
        constexpr int parallel_blocks = 4;
        if (logit_softcap == 0.0f) {
            constexpr bool use_logit_softcap = false;
            launch_fattn_tile_f32_64_128<cols_per_block, parallel_blocks, use_logit_softcap>(ctx, dst);
        } else {
            constexpr bool use_logit_softcap = true;
            launch_fattn_tile_f32_64_128<cols_per_block, parallel_blocks, use_logit_softcap>(ctx, dst);
        }
        return;
    }

    if (Q->ne[1] <= 32) {
        constexpr int cols_per_block = 32;
        constexpr int parallel_blocks = 4;
        if (logit_softcap == 0.0f) {
            constexpr bool use_logit_softcap = false;
            launch_fattn_tile_f32_64_128<cols_per_block, parallel_blocks, use_logit_softcap>(ctx, dst);
        } else {
            constexpr bool use_logit_softcap = true;
            launch_fattn_tile_f32_64_128<cols_per_block, parallel_blocks, use_logit_softcap>(ctx, dst);
        }
        return;
    }

    constexpr int cols_per_block = 32;
    constexpr int parallel_blocks = 1;
    if (logit_softcap == 0.0f) {
        constexpr bool use_logit_softcap = false;
        launch_fattn_tile_f32_64_128<cols_per_block, parallel_blocks, use_logit_softcap>(ctx, dst);
    } else {
        constexpr bool use_logit_softcap = true;
        launch_fattn_tile_f32_64_128<cols_per_block, parallel_blocks, use_logit_softcap>(ctx, dst);
    }
}

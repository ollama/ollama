#include "hip/hip_runtime.h"
#include "argsort.cuh"

#ifdef GGML_CUDA_USE_CUB
#    include <hipcub/hipcub.hpp>
using namespace hipcub;
#endif  // GGML_CUDA_USE_CUB

static __global__ void init_indices(int * indices, const int ncols, const int nrows) {
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y;

    if (col < ncols && row < nrows) {
        indices[row * ncols + col] = col;
    }
}

static __global__ void init_offsets(int * offsets, const int ncols, const int nrows) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx <= nrows) {
        offsets[idx] = idx * ncols;
    }
}

#ifdef GGML_CUDA_USE_CUB
static void argsort_f32_i32_cuda_cub(ggml_cuda_pool & pool,
                                     const float *    x,
                                     int *            dst,
                                     const int        ncols,
                                     const int        nrows,
                                     ggml_sort_order  order,
                                     hipStream_t     stream) {
    ggml_cuda_pool_alloc<int>   temp_indices_alloc(pool, ncols * nrows);
    ggml_cuda_pool_alloc<float> temp_keys_alloc(pool, ncols * nrows);
    ggml_cuda_pool_alloc<int>   offsets_alloc(pool, nrows + 1);

    int *   temp_indices = temp_indices_alloc.get();
    float * temp_keys    = temp_keys_alloc.get();
    int *   d_offsets    = offsets_alloc.get();

    static const int block_size = 256;
    const dim3 grid_size((ncols + block_size - 1) / block_size, nrows);
    init_indices<<<grid_size, block_size, 0, stream>>>(temp_indices, ncols, nrows);

    const dim3 offset_grid((nrows + block_size - 1) / block_size);
    init_offsets<<<offset_grid, block_size, 0, stream>>>(d_offsets, ncols, nrows);

    hipMemcpyAsync(temp_keys, x, ncols * nrows * sizeof(float), hipMemcpyDeviceToDevice, stream);

    size_t temp_storage_bytes = 0;

    if (order == GGML_SORT_ORDER_ASC) {
        DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_bytes, temp_keys, temp_keys,  // keys (in-place)
                                            temp_indices, dst,                                  // values (indices)
                                            ncols * nrows, nrows,                            // num items, num segments
                                            d_offsets, d_offsets + 1, 0, sizeof(float) * 8,  // all bits
                                            stream);
    } else {
        DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_bytes, temp_keys, temp_keys, temp_indices,
                                                      dst, ncols * nrows, nrows, d_offsets, d_offsets + 1, 0,
                                                      sizeof(float) * 8, stream);
    }

    ggml_cuda_pool_alloc<uint8_t> temp_storage_alloc(pool, temp_storage_bytes);
    void *                        d_temp_storage = temp_storage_alloc.get();

    if (order == GGML_SORT_ORDER_ASC) {
        DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, temp_keys, temp_keys, temp_indices, dst,
                                            ncols * nrows, nrows, d_offsets, d_offsets + 1, 0, sizeof(float) * 8,
                                            stream);
    } else {
        DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, temp_keys, temp_keys,
                                                      temp_indices, dst, ncols * nrows, nrows, d_offsets, d_offsets + 1,
                                                      0, sizeof(float) * 8, stream);
    }
}
#endif  // GGML_CUDA_USE_CUB

// Bitonic sort implementation
template<typename T>
static inline __device__ void ggml_cuda_swap(T & a, T & b) {
    T tmp = a;
    a = b;
    b = tmp;
}

template<ggml_sort_order order>
static __global__ void k_argsort_f32_i32(const float * x, int * dst, const int ncols, int ncols_pad) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.x;

    if (col >= ncols_pad) {
        return;
    }

    const float * x_row = x + row * ncols;
    extern __shared__ int dst_row[];

    // initialize indices
    dst_row[col] = col;

    __syncthreads();

    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (dst_row[col] >= ncols ||
                        (dst_row[ixj] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] > x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] < x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                } else {
                    if (dst_row[ixj] >= ncols ||
                        (dst_row[col] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] < x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] > x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }

    // copy the result to dst without the padding
    if (col < ncols) {
        dst[row * ncols + col] = dst_row[col];
    }
}

static int next_power_of_2(int x) {
    int n = 1;
    while (n < x) {
        n *= 2;
    }
    return n;
}

static void argsort_f32_i32_cuda_bitonic(const float *   x,
                                         int *           dst,
                                         const int       ncols,
                                         const int       nrows,
                                         ggml_sort_order order,
                                         hipStream_t    stream) {
    // bitonic sort requires ncols to be power of 2
    const int ncols_pad = next_power_of_2(ncols);

    const dim3 block_dims(ncols_pad, 1, 1);
    const dim3 block_nums(nrows, 1, 1);
    const size_t shared_mem = ncols_pad * sizeof(int);

    // FIXME: this limit could be raised by ~2-4x on Ampere or newer
    GGML_ASSERT(shared_mem <= ggml_cuda_info().devices[ggml_cuda_get_device()].smpb);

    if (order == GGML_SORT_ORDER_ASC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_ASC>
            <<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else if (order == GGML_SORT_ORDER_DESC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_DESC>
            <<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else {
        GGML_ABORT("fatal error");
    }
}


template<ggml_sort_order order>
static __global__ void k_argsort_i32_i32(const int32_t * x, int * dst, const int ncols, const int ncols_pad) {
    extern __shared__ int shared_mem[];
    int * indices = shared_mem;

    const int tid = threadIdx.x;
    const int row = blockIdx.y;

    // Initialize all indices, handling the case where threads < ncols_pad
    for (int i = tid; i < ncols_pad; i += blockDim.x) {
        indices[i] = i < ncols ? i : 0; // Use 0 for padding indices
    }
    __syncthreads();

    // Bitonic sort
    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k/2; j > 0; j /= 2) {
            for (int i = tid; i < ncols_pad; i += blockDim.x) {
                const int ij = i ^ j;
                if (ij > i) {
                    // Only compare values within the actual data range
                    if (i < ncols && ij < ncols) {
                        if ((i & k) == 0) {
                            if (order == GGML_SORT_ORDER_ASC) {
                                if (x[row * ncols + indices[i]] > x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            } else {
                                if (x[row * ncols + indices[i]] < x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            }
                        } else {
                            if (order == GGML_SORT_ORDER_ASC) {
                                if (x[row * ncols + indices[i]] < x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            } else {
                                if (x[row * ncols + indices[i]] > x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            }
                        }
                    }
                }
            }
            __syncthreads();
        }
    }

    // Write sorted indices to output, only threads handling valid data
    for (int i = tid; i < ncols; i += blockDim.x) {
        dst[row * ncols + i] = indices[i];
    }
}

static void argsort_i32_i32_cuda(const int32_t * x, int * dst, const int ncols, const int nrows, ggml_sort_order order, hipStream_t stream) {
    // Bitonic sort requires ncols to be power of 2
    const int ncols_pad = next_power_of_2(ncols);

    // Ensure thread count doesn't exceed maximum (typically 1024)
    const int max_threads = 1024;  // This is the typical max for most GPUs
    const int threads_per_block = ncols_pad > max_threads ? max_threads : ncols_pad;

    const dim3 block_dims(threads_per_block, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    const size_t shared_mem = ncols_pad * sizeof(int);

    // Check if shared memory size is within limits
    const size_t max_shared_mem = ggml_cuda_info().devices[ggml_cuda_get_device()].smpb;

    // Instead of logging an error, use GGML_ASSERT with a descriptive message
    GGML_ASSERT(shared_mem <= max_shared_mem && "argsort: required shared memory exceeds device limit");

    // Launch kernels with the updated thread configuration
    if (order == GGML_SORT_ORDER_ASC) {
        k_argsort_i32_i32<GGML_SORT_ORDER_ASC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else if (order == GGML_SORT_ORDER_DESC) {
        k_argsort_i32_i32<GGML_SORT_ORDER_DESC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else {
        GGML_ABORT("fatal error");
    }
}


void ggml_cuda_op_argsort(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_I32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    enum ggml_sort_order order = (enum ggml_sort_order) dst->op_params[0];

    if (src0->type == GGML_TYPE_I32) {
        argsort_i32_i32_cuda((const int32_t *)src0_d, (int *)dst_d, ncols, nrows, order, stream);
    } else {
#ifdef GGML_CUDA_USE_CUB
        const int    ncols_pad      = next_power_of_2(ncols);
        const size_t shared_mem     = ncols_pad * sizeof(int);
        const size_t max_shared_mem = ggml_cuda_info().devices[ggml_cuda_get_device()].smpb;

        if (shared_mem > max_shared_mem || ncols > 1024) {
            ggml_cuda_pool & pool = ctx.pool();
            argsort_f32_i32_cuda_cub(pool, src0_d, (int *) dst_d, ncols, nrows, order, stream);
        } else {
            argsort_f32_i32_cuda_bitonic(src0_d, (int *) dst_d, ncols, nrows, order, stream);
        }
#else
        argsort_f32_i32_cuda_bitonic(src0_d, (int *) dst_d, ncols, nrows, order, stream);
#endif
    }
}

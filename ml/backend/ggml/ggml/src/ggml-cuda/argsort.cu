#include "hip/hip_runtime.h"
#include "argsort.cuh"

template<typename T>
static inline __device__ void ggml_cuda_swap(T & a, T & b) {
    T tmp = a;
    a = b;
    b = tmp;
}

template<ggml_sort_order order>
static __global__ void k_argsort_f32_i32(const float * x, int * dst, const int ncols, int ncols_pad) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.y;

    if (col >= ncols_pad) {
        return;
    }

    const float * x_row = x + row * ncols;
    extern __shared__ int dst_row[];

    // initialize indices
    dst_row[col] = col;

    __syncthreads();

    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (dst_row[col] >= ncols ||
                        (dst_row[ixj] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] > x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] < x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                } else {
                    if (dst_row[ixj] >= ncols ||
                        (dst_row[col] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] < x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] > x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }

    // copy the result to dst without the padding
    if (col < ncols) {
        dst[row * ncols + col] = dst_row[col];
    }
}

static int next_power_of_2(int x) {
    int n = 1;
    while (n < x) {
        n *= 2;
    }
    return n;
}

static void argsort_f32_i32_cuda(const float * x, int * dst, const int ncols, const int nrows, ggml_sort_order order, hipStream_t stream) {
    // bitonic sort requires ncols to be power of 2
    const int ncols_pad = next_power_of_2(ncols);

    const dim3 block_dims(ncols_pad, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    const size_t shared_mem = ncols_pad * sizeof(int);

    // FIXME: this limit could be raised by ~2-4x on Ampere or newer
    GGML_ASSERT(shared_mem <= ggml_cuda_info().devices[ggml_cuda_get_device()].smpb);

    if (order == GGML_SORT_ORDER_ASC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_ASC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else if (order == GGML_SORT_ORDER_DESC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_DESC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else {
        GGML_ABORT("fatal error");
    }
}

void ggml_cuda_op_argsort(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    enum ggml_sort_order order = (enum ggml_sort_order) dst->op_params[0];

    argsort_f32_i32_cuda(src0_d, (int *)dst_d, ncols, nrows, order, stream);
}

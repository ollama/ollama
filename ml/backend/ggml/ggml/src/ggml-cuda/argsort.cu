#include "hip/hip_runtime.h"
#include "argsort.cuh"

template<typename T>
static inline __device__ void ggml_cuda_swap(T & a, T & b) {
    T tmp = a;
    a = b;
    b = tmp;
}

template<ggml_sort_order order>
static __global__ void k_argsort_f32_i32(const float * x, int * dst, const int ncols, int ncols_pad) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.y;

    if (col >= ncols_pad) {
        return;
    }

    const float * x_row = x + row * ncols;
    extern __shared__ int dst_row[];

    // initialize indices
    dst_row[col] = col;

    __syncthreads();

    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (dst_row[col] >= ncols ||
                        (dst_row[ixj] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] > x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] < x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                } else {
                    if (dst_row[ixj] >= ncols ||
                        (dst_row[col] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] < x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] > x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }

    // copy the result to dst without the padding
    if (col < ncols) {
        dst[row * ncols + col] = dst_row[col];
    }
}

static int next_power_of_2(int x) {
    int n = 1;
    while (n < x) {
        n *= 2;
    }
    return n;
}

static void argsort_f32_i32_cuda(const float * x, int * dst, const int ncols, const int nrows, ggml_sort_order order, hipStream_t stream) {
    // bitonic sort requires ncols to be power of 2
    const int ncols_pad = next_power_of_2(ncols);

    const dim3 block_dims(ncols_pad, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    const size_t shared_mem = ncols_pad * sizeof(int);

    // FIXME: this limit could be raised by ~2-4x on Ampere or newer
    GGML_ASSERT(shared_mem <= ggml_cuda_info().devices[ggml_cuda_get_device()].smpb);

    if (order == GGML_SORT_ORDER_ASC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_ASC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else if (order == GGML_SORT_ORDER_DESC) {
        k_argsort_f32_i32<GGML_SORT_ORDER_DESC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else {
        GGML_ABORT("fatal error");
    }
}


template<ggml_sort_order order>
static __global__ void k_argsort_i32_i32(const int32_t * x, int * dst, const int ncols, const int ncols_pad) {
    extern __shared__ int shared_mem[];
    int * indices = shared_mem;

    const int tid = threadIdx.x;
    const int row = blockIdx.y;

    // Initialize all indices, handling the case where threads < ncols_pad
    for (int i = tid; i < ncols_pad; i += blockDim.x) {
        indices[i] = i < ncols ? i : 0; // Use 0 for padding indices
    }
    __syncthreads();

    // Bitonic sort
    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k/2; j > 0; j /= 2) {
            for (int i = tid; i < ncols_pad; i += blockDim.x) {
                const int ij = i ^ j;
                if (ij > i) {
                    // Only compare values within the actual data range
                    if (i < ncols && ij < ncols) {
                        if ((i & k) == 0) {
                            if (order == GGML_SORT_ORDER_ASC) {
                                if (x[row * ncols + indices[i]] > x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            } else {
                                if (x[row * ncols + indices[i]] < x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            }
                        } else {
                            if (order == GGML_SORT_ORDER_ASC) {
                                if (x[row * ncols + indices[i]] < x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            } else {
                                if (x[row * ncols + indices[i]] > x[row * ncols + indices[ij]]) {
                                    int tmp = indices[i];
                                    indices[i] = indices[ij];
                                    indices[ij] = tmp;
                                }
                            }
                        }
                    }
                }
            }
            __syncthreads();
        }
    }

    // Write sorted indices to output, only threads handling valid data
    for (int i = tid; i < ncols; i += blockDim.x) {
        dst[row * ncols + i] = indices[i];
    }
}

static void argsort_i32_i32_cuda(const int32_t * x, int * dst, const int ncols, const int nrows, ggml_sort_order order, hipStream_t stream) {
    // Bitonic sort requires ncols to be power of 2
    const int ncols_pad = next_power_of_2(ncols);

    // Ensure thread count doesn't exceed maximum (typically 1024)
    const int max_threads = 1024;  // This is the typical max for most GPUs
    const int threads_per_block = ncols_pad > max_threads ? max_threads : ncols_pad;

    const dim3 block_dims(threads_per_block, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    const size_t shared_mem = ncols_pad * sizeof(int);

    // Check if shared memory size is within limits
    const size_t max_shared_mem = ggml_cuda_info().devices[ggml_cuda_get_device()].smpb;

    // Instead of logging an error, use GGML_ASSERT with a descriptive message
    GGML_ASSERT(shared_mem <= max_shared_mem && "argsort: required shared memory exceeds device limit");

    // Launch kernels with the updated thread configuration
    if (order == GGML_SORT_ORDER_ASC) {
        k_argsort_i32_i32<GGML_SORT_ORDER_ASC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else if (order == GGML_SORT_ORDER_DESC) {
        k_argsort_i32_i32<GGML_SORT_ORDER_DESC><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad);
    } else {
        GGML_ABORT("fatal error");
    }
}


void ggml_cuda_op_argsort(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_I32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    enum ggml_sort_order order = (enum ggml_sort_order) dst->op_params[0];

    if (src0->type == GGML_TYPE_I32) {
        argsort_i32_i32_cuda((const int32_t *)src0_d, (int *)dst_d, ncols, nrows, order, stream);
    } else {
        argsort_f32_i32_cuda(src0_d, (int *)dst_d, ncols, nrows, order, stream);
    }
}

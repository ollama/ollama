#include "hip/hip_runtime.h"
#include "ggml-cuda/common.cuh"
#include "ggml.h"
#include "topk-moe.cuh"

#include <initializer_list>

/*
    This kernel does the following:
    1. softmax over the logits per token [n_experts, n_tokens]
    2. argmax reduce over the top-k (n_experts_used) logits
    3. write weights + ids to global memory
    4. optionally normalize the weights

    It is intended as fusion of softmax->top-k->get_rows pipeline for MoE models
*/
template <int n_experts, bool with_norm>
__launch_bounds__(4 * WARP_SIZE, 1) __global__ void topk_moe_cuda(const float * logits,
                                                                  float *       weights,
                                                                  int32_t *     ids,
                                                                  const int     n_rows,
                                                                  const int     n_expert_used) {
    const int row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= n_rows) {
        return;
    }

    logits += n_experts * row;
    weights += n_expert_used * row;
    ids += n_experts * row;

    constexpr int experts_per_thread = (n_experts > WARP_SIZE) ? n_experts / WARP_SIZE : 1;

    float logits_r[experts_per_thread];

#pragma unroll
    for (int i = 0; i < n_experts; i += WARP_SIZE) {
        const int expert        = i + threadIdx.x;
        logits_r[i / WARP_SIZE] = n_experts % WARP_SIZE == 0 || expert < n_experts ? logits[expert] : -INFINITY;
    }

    float max_val = logits_r[0];

#pragma unroll
    for (int i = 1; i < experts_per_thread; i++) {
        const float val = logits_r[i];
        max_val         = max(val, max_val);
    }

    max_val = warp_reduce_max(max_val);

    float wt[experts_per_thread];
    float tmp = 0.f;

#pragma unroll
    for (int i = 0; i < experts_per_thread; i++) {
        const float val = logits_r[i];
        wt[i]           = expf(val - max_val);
        tmp += wt[i];
    }

    tmp = warp_reduce_sum(tmp);

    const float inv_sum = 1.0f / tmp;

#pragma unroll
    for (int i = 0; i < experts_per_thread; i++) {
        wt[i] = wt[i] * inv_sum;
    }

    //at this point, each thread holds a portion of softmax,
    //we do the argmax reduce over n_expert_used, each time marking
    //the expert weight as -inf to exclude from the next iteration

    float wt_sum = 0.f;

    extern __shared__ float data_topk_shared[];
    float *                 wt_shared_ptr = data_topk_shared + threadIdx.y * n_expert_used;

    for (int k = 0; k < n_expert_used; k++) {
        float max_val    = wt[0];
        int   max_expert = threadIdx.x;

#pragma unroll
        for (int i = 1; i < experts_per_thread; i++) {
            const int expert = threadIdx.x + i * WARP_SIZE;
            if ((n_experts % WARP_SIZE == 0 || expert < n_experts) && wt[i] > max_val) {
                max_val    = wt[i];
                max_expert = expert;
            }
        }

#pragma unroll
        for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2) {
            const float val    = __shfl_xor_sync(0xFFFFFFFF, max_val, mask, WARP_SIZE);
            const int   expert = __shfl_xor_sync(0xFFFFFFFF, max_expert, mask, WARP_SIZE);
            if (val > max_val || (val == max_val && expert < max_expert)) {
                max_val    = val;
                max_expert = expert;
            }
        }

        if ((max_expert & (WARP_SIZE - 1)) == threadIdx.x) {
            wt[max_expert / WARP_SIZE] = -INFINITY;

            wt_shared_ptr[k] = max_val;
            ids[k]           = max_expert;
            if constexpr (with_norm) {
                wt_sum += max_val;
            }
        }
    }

    if constexpr (with_norm) {
        wt_sum              = warp_reduce_sum(wt_sum);
        const float inv_sum = 1.0f / wt_sum;

        for (int i = threadIdx.x; i < n_expert_used; i += WARP_SIZE) {
            wt_shared_ptr[i] = wt_shared_ptr[i] * inv_sum;
        }
    }

    for (int i = threadIdx.x; i < n_expert_used; i += WARP_SIZE) {
        weights[i] = wt_shared_ptr[i];
    }
}

template <bool with_norm>
static void launch_topk_moe_cuda(ggml_backend_cuda_context & ctx,
                                 const float *               logits,
                                 float *                     weights,
                                 int32_t *                   ids,
                                 const int                   n_rows,
                                 const int                   n_expert,
                                 const int                   n_expert_used) {
    const int    rows_per_block = 4;
    dim3         grid_dims((n_rows + rows_per_block - 1) / rows_per_block, 1, 1);
    dim3         block_dims(WARP_SIZE, rows_per_block, 1);
    hipStream_t stream = ctx.stream();

    const int nbytes_shared = n_expert_used * rows_per_block * sizeof(float);

    switch (n_expert) {
        case 1:
            topk_moe_cuda<1, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 2:
            topk_moe_cuda<2, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 4:
            topk_moe_cuda<4, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 8:
            topk_moe_cuda<8, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 16:
            topk_moe_cuda<16, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 32:
            topk_moe_cuda<32, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 64:
            topk_moe_cuda<64, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 128:
            topk_moe_cuda<128, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 256:
            topk_moe_cuda<256, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        case 512:
            topk_moe_cuda<512, with_norm>
                <<<grid_dims, block_dims, nbytes_shared, stream>>>(logits, weights, ids, n_rows, n_expert_used);
            break;
        default:
            GGML_ASSERT(false && "fatal error");
            break;
    }
}

void ggml_cuda_op_topk_moe(ggml_backend_cuda_context & ctx,
                           const ggml_tensor *         logits,
                           ggml_tensor *               weights,
                           ggml_tensor *               ids,
                           const bool                  with_norm) {
    GGML_ASSERT(logits->type == GGML_TYPE_F32);
    GGML_ASSERT(weights->type == GGML_TYPE_F32);
    GGML_ASSERT(ids->type == GGML_TYPE_I32);

    const int n_experts = logits->ne[0];
    const int n_rows    = logits->ne[1];

    const float * logits_d  = (const float *) logits->src[0]->data;
    float *       weights_d = (float *) weights->data;
    int32_t *     ids_d     = (int32_t *) ids->data;

    GGML_ASSERT(ids->nb[1] / ggml_type_size(ids->type) == (size_t) n_experts);

    const int n_expert_used = weights->ne[1];

    if (with_norm) {
        launch_topk_moe_cuda<true>(ctx, logits_d, weights_d, ids_d, n_rows, n_experts, n_expert_used);
    } else {
        launch_topk_moe_cuda<false>(ctx, logits_d, weights_d, ids_d, n_rows, n_experts, n_expert_used);
    }
}

bool ggml_cuda_should_use_topk_moe(const ggml_tensor * softmax, const ggml_tensor * weights) {
    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale, (const float *) softmax->op_params + 0, sizeof(float));
    memcpy(&max_bias, (const float *) softmax->op_params + 1, sizeof(float));

    if (!ggml_is_contiguous(softmax->src[0]) || !ggml_is_contiguous(weights)) {
        return false;
    }

    if (scale != 1.0f || max_bias != 0.0f) {
        return false;
    }

    // don't fuse when masks or sinks are present
    if (softmax->src[1] || softmax->src[2]) {
        return false;
    }

    const int n_expert = softmax->ne[0];
    // n_expert must be a power of 2
    if ((n_expert & (n_expert - 1)) != 0 || n_expert > 512) {
        return false;
    }

    return true;
}

std::initializer_list<enum ggml_op> ggml_cuda_topk_moe_ops(bool norm) {
    static std::initializer_list<enum ggml_op> norm_ops = { GGML_OP_SOFT_MAX, GGML_OP_RESHAPE,  GGML_OP_ARGSORT,
                                                            GGML_OP_VIEW,     GGML_OP_GET_ROWS, GGML_OP_RESHAPE,
                                                            GGML_OP_SUM_ROWS, GGML_OP_DIV,      GGML_OP_RESHAPE };

    static std::initializer_list<enum ggml_op> no_norm_ops = { GGML_OP_SOFT_MAX, GGML_OP_RESHAPE, GGML_OP_ARGSORT,
                                                               GGML_OP_VIEW, GGML_OP_GET_ROWS };

    if (norm) {
        return norm_ops;
    }
    return no_norm_ops;
}

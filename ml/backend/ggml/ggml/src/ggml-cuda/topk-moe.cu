#include "hip/hip_runtime.h"
#include "ggml-cuda/common.cuh"
#include "ggml.h"
#include "topk-moe.cuh"

#include <cmath>
#include <initializer_list>

// Warp-local softmax used for both the pre-top-k logits and the post-top-k delayed path.
template <int experts_per_thread, bool use_limit>
__device__ void softmax_warp_inplace(float (&vals)[experts_per_thread], const int limit, const int lane) {
    float max_val = -INFINITY;

#pragma unroll
    for (int i = 0; i < experts_per_thread; i++) {
        const int  idx    = lane + i * WARP_SIZE;
        const bool active = !use_limit || (idx < limit);
        if (active) {
            max_val = max(max_val, vals[i]);
        }
    }

    max_val = warp_reduce_max(max_val);

    float sum = 0.f;

#pragma unroll
    for (int i = 0; i < experts_per_thread; i++) {
        const int  idx    = lane + i * WARP_SIZE;
        const bool active = !use_limit || (idx < limit);
        if (active) {
            const float val = expf(vals[i] - max_val);
            vals[i]         = val;
            sum += val;
        } else {
            vals[i] = 0.f;
        }
    }

    sum = warp_reduce_sum(sum);

    const float inv_sum = 1.0f / sum;

#pragma unroll
    for (int i = 0; i < experts_per_thread; i++) {
        const int  idx    = lane + i * WARP_SIZE;
        const bool active = !use_limit || (idx < limit);
        if (active) {
            vals[i] *= inv_sum;
        }
    }
}

/*
    This kernel does the following:
    1. optionally softmax over the logits per token [n_experts, n_tokens]
    2. argmax reduce over the top-k (n_experts_used) logits
    3. write weights + ids to global memory
    4. optionally normalize the weights or apply softmax over the selected logits

    It is intended as fusion of softmax->top-k->get_rows pipeline for MoE models
*/
template <int n_experts, bool with_norm, bool delayed_softmax = false>
__launch_bounds__(4 * WARP_SIZE, 1) __global__ void topk_moe_cuda(const float * logits,
                                                                  float *       weights,
                                                                  int32_t *     ids,
                                                                  const int     n_rows,
                                                                  const int     n_expert_used,
                                                                  const float   clamp_val) {
    const int row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= n_rows) {
        return;
    }

    logits += n_experts * row;
    weights += n_expert_used * row;
    ids += n_experts * row;

    constexpr int experts_per_thread = (n_experts > WARP_SIZE) ? n_experts / WARP_SIZE : 1;

    float wt[experts_per_thread];

#pragma unroll
    for (int i = 0; i < n_experts; i += WARP_SIZE) {
        const int expert  = i + threadIdx.x;
        wt[i / WARP_SIZE] = (n_experts % WARP_SIZE == 0 || expert < n_experts) ? logits[expert] : -INFINITY;
    }

    if constexpr (!delayed_softmax) {
        softmax_warp_inplace<experts_per_thread, false>(wt, n_experts, threadIdx.x);
    }

    //at this point, each thread holds either a portion of the softmax distribution
    //or the raw logits. We do the argmax reduce over n_expert_used, each time marking
    //the expert weight as -inf to exclude from the next iteration

    float wt_sum = 0.f;

    float output_weights[experts_per_thread];

#pragma unroll
    for (int i = 0; i < experts_per_thread; i++) {
        output_weights[i] = 0.f;
    }

    for (int k = 0; k < n_expert_used; k++) {
        float max_val    = wt[0];
        int   max_expert = threadIdx.x;

#pragma unroll
        for (int i = 1; i < experts_per_thread; i++) {
            const int expert = threadIdx.x + i * WARP_SIZE;
            if ((n_experts % WARP_SIZE == 0 || expert < n_experts) && wt[i] > max_val) {
                max_val    = wt[i];
                max_expert = expert;
            }
        }

#pragma unroll
        for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2) {
            const float val    = __shfl_xor_sync(0xFFFFFFFF, max_val, mask, WARP_SIZE);
            const int   expert = __shfl_xor_sync(0xFFFFFFFF, max_expert, mask, WARP_SIZE);
            if (val > max_val || (val == max_val && expert < max_expert)) {
                max_val    = val;
                max_expert = expert;
            }
        }

        if ((k & (WARP_SIZE - 1)) == threadIdx.x) {
            output_weights[k / WARP_SIZE] = max_val;
        }

        if ((max_expert & (WARP_SIZE - 1)) == threadIdx.x) {
            wt[max_expert / WARP_SIZE] = -INFINITY;

            ids[k] = max_expert;
            if constexpr (with_norm) {
                wt_sum += max_val;
            }
        }
    }

    if constexpr (with_norm) {
        wt_sum              = warp_reduce_sum(wt_sum);
        wt_sum              = max(wt_sum, clamp_val);
        const float inv_sum = 1.0f / wt_sum;

        for (int i = 0; i < experts_per_thread; i++) {
            output_weights[i] *= inv_sum;
        }
    }

    if constexpr (delayed_softmax) {
        softmax_warp_inplace<experts_per_thread, true>(output_weights, n_expert_used, threadIdx.x);
    }

#pragma unroll
    for (int i = 0; i < experts_per_thread; i++) {
        const int idx = i * WARP_SIZE + threadIdx.x;
        if (idx < n_expert_used) {
            weights[idx] = output_weights[i];
        }
    }

    if (!with_norm) {
        GGML_UNUSED(clamp_val);
    }
}

template <bool with_norm, bool delayed_softmax = false>
static void launch_topk_moe_cuda(ggml_backend_cuda_context & ctx,
                                 const float *               logits,
                                 float *                     weights,
                                 int32_t *                   ids,
                                 const int                   n_rows,
                                 const int                   n_expert,
                                 const int                   n_expert_used,
                                 const float                 clamp_val) {
    static_assert(!(with_norm && delayed_softmax), "delayed softmax is not supported with weight normalization");
    const int    rows_per_block = 4;
    dim3         grid_dims((n_rows + rows_per_block - 1) / rows_per_block, 1, 1);
    dim3         block_dims(WARP_SIZE, rows_per_block, 1);
    hipStream_t stream = ctx.stream();

    switch (n_expert) {
        case 1:
            topk_moe_cuda<1, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 2:
            topk_moe_cuda<2, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 4:
            topk_moe_cuda<4, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 8:
            topk_moe_cuda<8, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 16:
            topk_moe_cuda<16, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 32:
            topk_moe_cuda<32, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 64:
            topk_moe_cuda<64, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 128:
            topk_moe_cuda<128, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 256:
            topk_moe_cuda<256, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        case 512:
            topk_moe_cuda<512, with_norm, delayed_softmax>
                <<<grid_dims, block_dims, 0, stream>>>(logits, weights, ids, n_rows, n_expert_used, clamp_val);
            break;
        default:
            GGML_ASSERT(false && "fatal error");
            break;
    }
}

void ggml_cuda_op_topk_moe(ggml_backend_cuda_context & ctx,
                           const ggml_tensor *         logits,
                           ggml_tensor *               weights,
                           ggml_tensor *               ids,
                           const bool                  with_norm,
                           const bool                  delayed_softmax,
                           ggml_tensor *               clamp) {
    GGML_ASSERT(logits->type == GGML_TYPE_F32);
    GGML_ASSERT(weights->type == GGML_TYPE_F32);
    GGML_ASSERT(ids->type == GGML_TYPE_I32);

    const int n_experts = logits->ne[0];
    const int n_rows    = logits->ne[1];

    const float * logits_d  = (const float *) logits->data;
    float *       weights_d = (float *) weights->data;
    int32_t *     ids_d     = (int32_t *) ids->data;

    GGML_ASSERT(ids->nb[1] / ggml_type_size(ids->type) == (size_t) n_experts);

    const int n_expert_used = weights->ne[1];

    float clamp_val = -INFINITY;
    if (with_norm) {
        if (clamp) {
            clamp_val = ggml_get_op_params_f32(clamp, 0);
        }
        launch_topk_moe_cuda<true>(ctx, logits_d, weights_d, ids_d, n_rows, n_experts, n_expert_used, clamp_val);
    } else {
        GGML_ASSERT(clamp == nullptr);
        if (delayed_softmax) {
            launch_topk_moe_cuda<false, true>(ctx, logits_d, weights_d, ids_d, n_rows, n_experts, n_expert_used,
                                              clamp_val);
        } else {
            launch_topk_moe_cuda<false, false>(ctx, logits_d, weights_d, ids_d, n_rows, n_experts, n_expert_used,
                                               clamp_val);
        }
    }
}

bool ggml_cuda_should_use_topk_moe(const ggml_tensor * softmax, const ggml_tensor * weights, const ggml_tensor * clamp) {
    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale, (const float *) softmax->op_params + 0, sizeof(float));
    memcpy(&max_bias, (const float *) softmax->op_params + 1, sizeof(float));

    if (!ggml_is_contiguous(softmax->src[0]) || !ggml_is_contiguous(weights)) {
        return false;
    }

    if (scale != 1.0f || max_bias != 0.0f) {
        return false;
    }

    // don't fuse when masks or sinks are present
    if (softmax->src[1] || softmax->src[2]) {
        return false;
    }

    const int n_expert = softmax->ne[0];
    // n_expert must be a power of 2
    if ((n_expert & (n_expert - 1)) != 0 || n_expert > 512) {
        return false;
    }

    if (clamp) {
        if (clamp->op != GGML_OP_CLAMP) {
            return false;
        }
        float max_val = ggml_get_op_params_f32(clamp, 1);

        if (max_val != INFINITY) {
            return false;
        }
    }


    return true;
}

std::initializer_list<enum ggml_op> ggml_cuda_topk_moe_ops(bool norm, bool delayed_softmax) {
    static std::initializer_list<enum ggml_op> norm_ops = { GGML_OP_SOFT_MAX, GGML_OP_RESHAPE,  GGML_OP_ARGSORT,
                                                            GGML_OP_VIEW,     GGML_OP_GET_ROWS, GGML_OP_RESHAPE,
                                                            GGML_OP_SUM_ROWS, GGML_OP_CLAMP,    GGML_OP_DIV,
                                                            GGML_OP_RESHAPE };

    static std::initializer_list<enum ggml_op> no_norm_ops = { GGML_OP_SOFT_MAX, GGML_OP_RESHAPE, GGML_OP_ARGSORT,
                                                               GGML_OP_VIEW, GGML_OP_GET_ROWS };

    static std::initializer_list<enum ggml_op> delayed_softmax_ops = { GGML_OP_ARGSORT,  GGML_OP_VIEW,
                                                                       GGML_OP_GET_ROWS, GGML_OP_RESHAPE,
                                                                       GGML_OP_SOFT_MAX, GGML_OP_RESHAPE };

    GGML_ASSERT(!norm || !delayed_softmax);

    if (delayed_softmax) {
        return delayed_softmax_ops;
    }

    if (norm) {
        return norm_ops;
    }

    return no_norm_ops;
}

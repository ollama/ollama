#include "hip/hip_runtime.h"
#include "common.cuh"
#include "gla.cuh"

template<int HEAD_SIZE>
static __global__ void gated_linear_attn_f32(const int B, const int T, const int C, const int H, const float scale,
     const float * k, const float * v, const float * r, const float * td, const float * s, float * dst) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int head_size = HEAD_SIZE;
    const int batch_i = bid / H;
    const int head_i = bid % H;
    const int state_size = C * head_size;
    const int n_seq_tokens = T / B;

    float state[head_size];
    __shared__ float _k[head_size], _r[head_size], _td[head_size];

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        state[i] = s[batch_i * state_size + head_i * head_size * head_size + i * head_size + tid];
    }

    for (int t = batch_i * n_seq_tokens * C + head_i * head_size + tid; t < (batch_i + 1) * n_seq_tokens * C + head_i * head_size + tid; t += C) {
        __syncthreads();
        _k[tid] = k[t];
        _r[tid] = r[t];
        _td[tid] = td[t];
        __syncthreads();

        const float _v = v[t];
        float y = 0;
        for (int j = 0; j < head_size; j += 4) {
            const float4 & k = (float4 &)(_k[j]);
            const float4 & r = (float4 &)(_r[j]);
            const float4 & td = (float4 &)(_td[j]);
            float4 & s = (float4 &)(state[j]);
            float4 kv;

            kv.x = k.x * _v;
            kv.y = k.y * _v;
            kv.z = k.z * _v;
            kv.w = k.w * _v;

            s.x = s.x * td.x + kv.x;
            s.y = s.y * td.y + kv.y;
            s.z = s.z * td.z + kv.z;
            s.w = s.w * td.w + kv.w;

            y += r.x * s.x;
            y += r.y * s.y;
            y += r.z * s.z;
            y += r.w * s.w;
        }
        dst[t] = y * scale;
    }

    #pragma unroll
    for (int i = 0; i < head_size; i++) {
        dst[T * C + batch_i * state_size + head_i * head_size * head_size + i * head_size + tid] = state[i];
    }
}

void ggml_cuda_op_gated_linear_attn(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const float * k_d  = (const float *)dst->src[0]->data;
    const float * v_d  = (const float *)dst->src[1]->data;
    const float * r_d  = (const float *)dst->src[2]->data;
    const float * td_d = (const float *)dst->src[3]->data;
    const float * s_d  = (const float *)dst->src[4]->data;

    const int64_t B = dst->src[4]->ne[1];
    const int64_t T = dst->src[0]->ne[2];
    const int64_t C = dst->ne[0];
    const int64_t H = dst->src[0]->ne[1];

    float scale;
    memcpy(&scale, (float*)dst->op_params, sizeof(float));

    float * dst_d = (float *)dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(dst->src[4]->type == GGML_TYPE_F32);
    GGML_ASSERT(C % H == 0);
    GGML_ASSERT(C / H == 64 || C / H == 128);


    if (C / H == 64) {
        gated_linear_attn_f32<64><<<B * H, C / H, 0, stream>>>(B, T, C, H, scale, k_d, v_d, r_d, td_d, s_d, dst_d);
    } else {
        gated_linear_attn_f32<128><<<B * H, C / H, 0, stream>>>(B, T, C, H, scale, k_d, v_d, r_d, td_d, s_d, dst_d);
    }
}

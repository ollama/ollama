#include "hip/hip_runtime.h"
#include "clamp.cuh"

static __global__ void clamp_f32(const float * x, float * dst, const float min, const float max, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = x[i] < min ? min : (x[i] > max ? max : x[i]);
}

static void clamp_f32_cuda(const float * x, float * dst, const float min, const float max, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_CLAMP_BLOCK_SIZE - 1) / CUDA_CLAMP_BLOCK_SIZE;
    clamp_f32<<<num_blocks, CUDA_CLAMP_BLOCK_SIZE, 0, stream>>>(x, dst, min, max, k);
}


void ggml_cuda_op_clamp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float min;
    float max;
    memcpy(&min, dst->op_params, sizeof(float));
    memcpy(&max, (float *) dst->op_params + 1, sizeof(float));

    clamp_f32_cuda(src0_d, dst_d, min, max, ggml_nelements(src0), stream);
}

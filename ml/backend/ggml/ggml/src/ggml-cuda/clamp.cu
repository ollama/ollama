#include "hip/hip_runtime.h"
#include "clamp.cuh"

static __device__ __forceinline__ float op_clamp(float x, float min, float max) {
    return fminf(fmaxf(x, min), max);
}

template <class T>
static __global__ void op_clamp_kernel(const T * x, T * dst, const T min, const T max, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op_clamp((float)x[i], (float)min, (float)max);
}

template <class T>
static void clamp_cuda(const T * x, T * dst, const T min, const T max, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_CLAMP_BLOCK_SIZE - 1) / CUDA_CLAMP_BLOCK_SIZE;
    op_clamp_kernel<<<num_blocks, CUDA_CLAMP_BLOCK_SIZE, 0, stream>>>(x, dst, min, max, k);
}


void ggml_cuda_op_clamp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    float min;
    float max;
    memcpy(&min, dst->op_params, sizeof(float));
    memcpy(&max, (float *) dst->op_params + 1, sizeof(float));

    if (src0->type == GGML_TYPE_F16) {
        clamp_cuda((const half *)src0_d, (half *)dst_d, (half)min, (half)max, ggml_nelements(src0), stream);
    } else {
        clamp_cuda((const float *)src0_d, (float *)dst_d, (float)min, (float)max, ggml_nelements(src0), stream);
    }
}

#include "hip/hip_runtime.h"
#include "common.cuh"
#include "count-equal.cuh"

#include <cstdint>

template <typename T>
static __global__ void count_equal(const T * __restrict__ x, const T * __restrict__ y, int64_t * __restrict__ dst, const int64_t dk, const int64_t k) {
    const int64_t i0 = (int64_t) blockIdx.x*dk;
    const int64_t i1 = min(i0 + dk, k);

    int nequal = 0;

    for (int64_t i = i0 + threadIdx.x; i < i1; i += WARP_SIZE) {
        const T xi = x[i];
        const T yi = y[i];
        nequal += xi == yi;
    }

    nequal = warp_reduce_sum(nequal);

    if (threadIdx.x != 0) {
        return;
    }

    atomicAdd((int *) dst, nequal);
}

void ggml_cuda_count_equal(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_ASSERT(src0->type == src1->type);
    GGML_ASSERT( dst->type == GGML_TYPE_I64);

    GGML_ASSERT(ggml_are_same_shape(src0, src1));
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(ggml_is_contiguous(dst));

    int64_t * dst_d  = (int64_t *) dst->data;

    hipStream_t stream = ctx.stream();
    const int nsm = ggml_cuda_info().devices[ggml_cuda_get_device()].nsm;

    const int64_t ne = ggml_nelements(src0);
    GGML_ASSERT(ne < (1 << 30) && "atomicAdd implementation only supports int");
    const int64_t dne = GGML_PAD((ne + 4*nsm - 1) / (4*nsm), CUDA_COUNT_EQUAL_CHUNK_SIZE);

    CUDA_CHECK(hipMemsetAsync(dst_d, 0, ggml_nbytes(dst), stream));

    const dim3 blocks_dim(WARP_SIZE, 1, 1);
    const dim3 blocks_num(std::min((int64_t)4*nsm, (ne + CUDA_COUNT_EQUAL_CHUNK_SIZE - 1)/CUDA_COUNT_EQUAL_CHUNK_SIZE), 1, 1);

    switch (src0->type) {
        case GGML_TYPE_I32: {
            const int * src0_d = (const int *) src0->data;
            const int * src1_d = (const int *) src1->data;
            count_equal<<<blocks_num, blocks_dim, 0, stream>>>(src0_d, src1_d, dst_d, dne, ne);
        } break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

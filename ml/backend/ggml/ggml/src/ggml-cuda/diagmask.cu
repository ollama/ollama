#include "hip/hip_runtime.h"
#include "diagmask.cuh"

static __global__ void diag_mask_inf_f32(const float * x, float * dst, const int ncols, const int rows_per_channel, const int n_past) {
    const int col = blockDim.y*blockIdx.y + threadIdx.y;
    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    if (col >= ncols) {
        return;
    }

    const int i = row*ncols + col;
    //dst[i] = col > (n_past + row % rows_per_channel) ? -INFINITY : x[i];
    //dst[i] = x[i] - (col > n_past + row % rows_per_channel) * INT_MAX; // equivalent within rounding error but slightly faster on GPU
    dst[i] = x[i] - (col > n_past + row % rows_per_channel) * FLT_MAX;
}

static void diag_mask_inf_f32_cuda(const float * x, float * dst, const int ncols_x, const int nrows_x, const int rows_per_channel, const int n_past, hipStream_t stream) {
    const dim3 block_dims(1, CUDA_DIAG_MASK_INF_BLOCK_SIZE, 1);
    const int block_num_x = (ncols_x + CUDA_DIAG_MASK_INF_BLOCK_SIZE - 1) / CUDA_DIAG_MASK_INF_BLOCK_SIZE;
    const dim3 block_nums(nrows_x, block_num_x, 1);
    diag_mask_inf_f32<<<block_nums, block_dims, 0, stream>>>(x, dst, ncols_x, rows_per_channel, n_past);
}

void ggml_cuda_op_diag_mask_inf(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int nrows0 = ggml_nrows(src0);

    const int n_past = ((int32_t *) dst->op_params)[0];

    diag_mask_inf_f32_cuda(src0_d, dst_d, ne00, nrows0, ne01, n_past, stream);
}

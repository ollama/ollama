#include "hip/hip_runtime.h"
#include "im2col.cuh"

template <typename T>
static  __global__ void im2col_kernel(
        const float * x, T * dst, int64_t batch_offset,
        int64_t offset_delta, int64_t IC, int64_t IW, int64_t IH, int64_t OH, int64_t OW, int64_t KW, int64_t KH, int64_t pelements, int64_t CHW,
        int s0, int s1, int p0, int p1, int d0, int d1) {
    const int64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= pelements) {
        return;
    }

    const int64_t  ksize = OW * (KH > 1 ? KW : 1);
    const int64_t  kx = i / ksize;
    const int64_t  kd = kx * ksize;
    const int64_t  ky = (i - kd) / OW;
    const int64_t  ix = i % OW;

    const int64_t  oh = blockIdx.y;
    const int64_t  batch = blockIdx.z / IC;
    const int64_t  ic = blockIdx.z % IC;

    const int64_t iiw = ix * s0 + kx * d0 - p0;
    const int64_t iih = oh * s1 + ky * d1 - p1;

    const int64_t offset_dst =
        ((batch * OH + oh) * OW + ix) * CHW +
        (ic * (KW * KH) + ky * KW + kx);

    if (iih < 0 || iih >= IH || iiw < 0 || iiw >= IW) {
        dst[offset_dst] = 0.0f;
    } else {
        const int64_t offset_src = ic * offset_delta + batch * batch_offset;
        dst[offset_dst] = x[offset_src + iih * IW + iiw];
    }
}

template <typename T>
static void im2col_cuda(const float * x, T* dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t batch, int64_t batch_offset, int64_t offset_delta,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {
    const int parallel_elements = OW * KW * KH;
    const int num_blocks = (parallel_elements + CUDA_IM2COL_BLOCK_SIZE - 1) / CUDA_IM2COL_BLOCK_SIZE;
    dim3 block_nums(num_blocks, OH, batch * IC);
    im2col_kernel<<<block_nums, CUDA_IM2COL_BLOCK_SIZE, 0, stream>>>(x, dst, batch_offset, offset_delta, IC, IW, IH, OH, OW, KW, KH, parallel_elements, (IC * KH * KW), s0, s1, p0, p1, d0, d1);
}

static void im2col_cuda_f16(const float * x, half * dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t batch, int64_t batch_offset, int64_t offset_delta,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {

    im2col_cuda<half>(x, dst, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, offset_delta, s0, s1, p0, p1, d0, d1, stream);
}

static void im2col_cuda_f32(const float * x, float * dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t batch, int64_t batch_offset, int64_t offset_delta,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {

    im2col_cuda<float>(x, dst, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, offset_delta, s0, s1, p0, p1, d0, d1, stream);
}

void ggml_cuda_op_im2col(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const float * src1_d = (const float *)src1->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F16 || dst->type == GGML_TYPE_F32);

    const int32_t s0 = ((const int32_t*)(dst->op_params))[0];
    const int32_t s1 = ((const int32_t*)(dst->op_params))[1];
    const int32_t p0 = ((const int32_t*)(dst->op_params))[2];
    const int32_t p1 = ((const int32_t*)(dst->op_params))[3];
    const int32_t d0 = ((const int32_t*)(dst->op_params))[4];
    const int32_t d1 = ((const int32_t*)(dst->op_params))[5];

    const bool is_2D = ((const int32_t*)(dst->op_params))[6] == 1;

    const int64_t IC = src1->ne[is_2D ? 2 : 1];
    const int64_t IH = is_2D ? src1->ne[1] : 1;
    const int64_t IW =         src1->ne[0];

    const int64_t KH = is_2D ? src0->ne[1] : 1;
    const int64_t KW =         src0->ne[0];

    const int64_t OH = is_2D ? dst->ne[2] : 1;
    const int64_t OW =         dst->ne[1];

    const size_t  delta_offset = src1->nb[is_2D ? 2 : 1] / 4; // nb is byte offset, src is type float32
    const int64_t batch        = src1->ne[is_2D ? 3 : 2];
    const size_t  batch_offset = src1->nb[is_2D ? 3 : 2] / 4; // nb is byte offset, src is type float32

    if(dst->type == GGML_TYPE_F16) {
        im2col_cuda_f16(src1_d, (half *) dst_d, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, delta_offset, s0, s1, p0, p1, d0, d1, stream);
    } else {
        im2col_cuda_f32(src1_d, (float *) dst_d, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, delta_offset, s0, s1, p0, p1, d0, d1, stream);
    }
}

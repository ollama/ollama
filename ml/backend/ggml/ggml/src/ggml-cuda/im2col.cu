#include "hip/hip_runtime.h"
#include "im2col.cuh"

#define MAX_GRIDDIM_Z 65535

template <typename T>
static  __global__ void im2col_kernel(
        const float * x, T * dst,
        int64_t IC, int64_t IW, int64_t IH, int64_t OH, int64_t OW, int64_t KW, int64_t KH,
        int64_t IC_IH_IW, int64_t IH_IW, int64_t N_OH, int64_t KH_KW, int64_t IC_KH_KW,
        int s0, int s1, int p0, int p1, int d0, int d1) {
    const int64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= IC_KH_KW) {
        return;
    }

    const int64_t iic = i / (KH_KW);
    const int64_t rem = i - iic * KH_KW;
    const int64_t ikh = rem / KW;
    const int64_t ikw = rem - ikh * KW;

    const int64_t  iow = blockIdx.y;
    for (int64_t iz = blockIdx.z; iz < N_OH; iz+=MAX_GRIDDIM_Z) {
        const int64_t  in = iz / OH;
        const int64_t  ioh = iz - in * OH;

        const int64_t iiw = iow * s0 + ikw * d0 - p0;
        const int64_t iih = ioh * s1 + ikh * d1 - p1;

        const int64_t offset_dst =
            ((in * OH + ioh) * OW + iow) * IC_KH_KW + iic * KH_KW + ikh * KW + ikw;

        if (iih < 0 || iih >= IH || iiw < 0 || iiw >= IW) {
            dst[offset_dst] = 0.0f;
        } else {
            const int64_t offset_src = iic * IC_IH_IW + in * IH_IW;
            dst[offset_dst] = x[offset_src + iih * IW + iiw];
        }
    }

    GGML_UNUSED(IC);
    GGML_UNUSED(KH);
}

// im2col: [N, IC, IH, IW] => [N, OH, OW, IC*KH*KW]
template <typename T>
static void im2col_cuda(const float * x, T* dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t N, int64_t IC_IH_IW, int64_t IH_IW,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {
    const int64_t IC_KH_KW = IC * KH * KW;
    const int64_t num_blocks = (IC_KH_KW + CUDA_IM2COL_BLOCK_SIZE - 1) / CUDA_IM2COL_BLOCK_SIZE;
    const int64_t N_OH = N * OH;
    const int64_t KH_KW = KW*KH;
    dim3 block_nums(num_blocks, OW, MIN(N_OH, MAX_GRIDDIM_Z));
    im2col_kernel<<<block_nums, MIN(IC_KH_KW, CUDA_IM2COL_BLOCK_SIZE) , 0, stream>>>(x, dst, IC, IW, IH, OH, OW, KW, KH,
                                                                                     IC_IH_IW, IH_IW, N_OH, KH_KW, IC_KH_KW,
                                                                                     s0, s1, p0, p1, d0, d1);
}

static void im2col_cuda_f16(const float * x, half * dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t N, int64_t IC_IH_IW, int64_t IH_IW,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {

    im2col_cuda<half>(x, dst, IW, IH, OW, OH, KW, KH, IC, N, IC_IH_IW, IH_IW, s0, s1, p0, p1, d0, d1, stream);
}

static void im2col_cuda_f32(const float * x, float * dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t N, int64_t IC_IH_IW, int64_t IH_IW,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {

    im2col_cuda<float>(x, dst, IW, IH, OW, OH, KW, KH, IC, N, IC_IH_IW, IH_IW, s0, s1, p0, p1, d0, d1, stream);
}

void ggml_cuda_op_im2col(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const float * src1_d = (const float *)src1->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F16 || dst->type == GGML_TYPE_F32);

    const int32_t s0 = ((const int32_t*)(dst->op_params))[0];
    const int32_t s1 = ((const int32_t*)(dst->op_params))[1];
    const int32_t p0 = ((const int32_t*)(dst->op_params))[2];
    const int32_t p1 = ((const int32_t*)(dst->op_params))[3];
    const int32_t d0 = ((const int32_t*)(dst->op_params))[4];
    const int32_t d1 = ((const int32_t*)(dst->op_params))[5];

    const bool is_2D = ((const int32_t*)(dst->op_params))[6] == 1;

    const int64_t IC = src1->ne[is_2D ? 2 : 1];
    const int64_t IH = is_2D ? src1->ne[1] : 1;
    const int64_t IW =         src1->ne[0];

    const int64_t KH = is_2D ? src0->ne[1] : 1;
    const int64_t KW =         src0->ne[0];

    const int64_t OH = is_2D ? dst->ne[2] : 1;
    const int64_t OW =         dst->ne[1];

    const int64_t IC_IH_IW = src1->nb[is_2D ? 2 : 1] / 4; // nb is byte offset, src is type float32
    const int64_t N        = src1->ne[is_2D ? 3 : 2];
    const int64_t IH_IW    = src1->nb[is_2D ? 3 : 2] / 4; // nb is byte offset, src is type float32

    if(dst->type == GGML_TYPE_F16) {
        im2col_cuda_f16(src1_d, (half *) dst_d, IW, IH, OW, OH, KW, KH, IC, N, IC_IH_IW, IH_IW, s0, s1, p0, p1, d0, d1, stream);
    } else {
        im2col_cuda_f32(src1_d, (float *) dst_d, IW, IH, OW, OH, KW, KH, IC, N, IC_IH_IW, IH_IW, s0, s1, p0, p1, d0, d1, stream);
    }
}
